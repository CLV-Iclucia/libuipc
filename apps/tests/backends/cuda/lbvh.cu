#include "hip/hip_runtime.h"
#include <muda/ext/eigen/eigen_core_cxx20.h>  // to use Eigen in CUDA
#include <app/test_common.h>
#include <app/asset_dir.h>
#include <uipc/common/type_define.h>
#include <Eigen/Geometry>
#include <muda/buffer/device_buffer.h>
#include <muda/cub/device/device_reduce.h>
#include <muda/cub/device/device_radix_sort.h>
#include <uipc/geometry.h>
#include <uipc/common/enumerate.h>
#include <muda/atomic.h>
#include <muda/cub/device/device_scan.h>
#include <cuda/atomic>
#include <muda/ext/eigen/atomic.h>
#include <cub/util_ptx.cuh>
#include <cuda/atomic>

namespace uipc::test::backend::cuda
{
namespace detail
{
    MUDA_DEVICE MUDA_INLINE int common_upper_bits(const unsigned int lhs,
                                                  const unsigned int rhs) noexcept
    {
        return ::__clz(lhs ^ rhs);
    }
    MUDA_DEVICE MUDA_INLINE int common_upper_bits(const unsigned long long int lhs,
                                                  const unsigned long long int rhs) noexcept
    {
        return ::__clzll(lhs ^ rhs);
    }

    MUDA_GENERIC MUDA_INLINE std::uint32_t expand_bits(std::uint32_t v) noexcept
    {
        v = (v * 0x00010001u) & 0xFF0000FFu;
        v = (v * 0x00000101u) & 0x0F00F00Fu;
        v = (v * 0x00000011u) & 0xC30C30C3u;
        v = (v * 0x00000005u) & 0x49249249u;
        return v;
    }

    MUDA_GENERIC MUDA_INLINE std::uint32_t morton_code(Vector3 xyz) noexcept
    {
        xyz = xyz.cwiseMin(1.0).cwiseMax(0.0);
        const std::uint32_t xx =
            expand_bits(static_cast<std::uint32_t>(xyz.x() * 1024.0));
        const std::uint32_t yy =
            expand_bits(static_cast<std::uint32_t>(xyz.y() * 1024.0));
        const std::uint32_t zz =
            expand_bits(static_cast<std::uint32_t>(xyz.z() * 1024.0));
        return xx * 4 + yy * 2 + zz;
    }

    struct LBVHNode
    {
        uint32_t parent_idx = 0xFFFFFFFF;  // parent node
        uint32_t left_idx   = 0xFFFFFFFF;  // index of left  child node
        uint32_t right_idx  = 0xFFFFFFFF;  // index of right child node
        uint32_t object_idx = 0xFFFFFFFF;  // == 0xFFFFFFFF if internal node.
    };

    struct LBVHMortonIndex
    {
        MUDA_GENERIC LBVHMortonIndex(uint32_t m, uint32_t idx) noexcept
        {
            morton = m;
            morton <<= 32;
            morton |= idx;
        }

        MUDA_GENERIC LBVHMortonIndex() noexcept = default;

        uint64_t morton = 0;
    };

    MUDA_GENERIC bool operator==(const LBVHMortonIndex& lhs, const LBVHMortonIndex& rhs) noexcept
    {
        return lhs.morton == rhs.morton;
        //&&lhs.idx == rhs.idx;
    }

    MUDA_DEVICE uint2 determine_range(muda::Dense1D<LBVHMortonIndex> node_code,
                                      const uint32_t                 num_leaves,
                                      uint32_t                       idx)
    {
        if(idx == 0)
        {
            return make_uint2(0, num_leaves - 1);
        }

        // determine direction of the range
        const auto self_code = node_code(idx);
        const int  L_delta =
            common_upper_bits(self_code.morton, node_code(idx - 1).morton);
        const int R_delta =
            common_upper_bits(self_code.morton, node_code(idx + 1).morton);
        const int d = (R_delta > L_delta) ? 1 : -1;

        // Compute upper bound for the length of the range

        const int delta_min = thrust::min(L_delta, R_delta);
        int       l_max     = 2;
        int       delta     = -1;
        int       i_tmp     = idx + d * l_max;
        if(0 <= i_tmp && i_tmp < num_leaves)
        {
            delta = common_upper_bits(self_code.morton, node_code(i_tmp).morton);
        }
        while(delta > delta_min)
        {
            l_max <<= 1;
            i_tmp = idx + d * l_max;
            delta = -1;
            if(0 <= i_tmp && i_tmp < num_leaves)
            {
                delta = common_upper_bits(self_code.morton, node_code(i_tmp).morton);
            }
        }

        // Find the other end by binary search
        int l = 0;
        int t = l_max >> 1;
        while(t > 0)
        {
            i_tmp = idx + (l + t) * d;
            delta = -1;
            if(0 <= i_tmp && i_tmp < num_leaves)
            {
                delta = common_upper_bits(self_code.morton, node_code(i_tmp).morton);
            }
            if(delta > delta_min)
            {
                l += t;
            }
            t >>= 1;
        }
        uint32_t jdx = idx + l * d;
        if(d < 0)
        {
            thrust::swap(idx, jdx);  // make it sure that idx < jdx
        }
        return make_uint2(idx, jdx);
    }


    MUDA_DEVICE uint32_t find_split(muda::Dense1D<LBVHMortonIndex> node_code,
                                    const uint32_t                 num_leaves,
                                    const uint32_t                 first,
                                    const uint32_t last) noexcept
    {
        const auto first_code = node_code(first);
        const auto last_code  = node_code(last);
        if(first_code == last_code)
        {
            return (first + last) >> 1;
        }
        const int delta_node = common_upper_bits(first_code.morton, last_code.morton);

        // binary search...
        int split  = first;
        int stride = last - first;
        do
        {
            stride           = (stride + 1) >> 1;
            const int middle = split + stride;
            if(middle < last)
            {
                const int delta =
                    common_upper_bits(first_code.morton, node_code(middle).morton);
                if(delta > delta_node)
                {
                    split = middle;
                }
            }
        } while(stride > 1);

        return split;
    }
}  // namespace detail

class LBVH;

template <bool IsConst>
class LBVHViewerT : muda::ViewerBase<IsConst>
{
    MUDA_VIEWER_COMMON_NAME(LBVHViewerT);

    using Base = muda::ViewerBase<IsConst>;
    template <typename U>
    using auto_const_t = typename Base::template auto_const_t<U>;

    friend class LBVH;
    using Node = detail::LBVHNode;


  public:
    using ConstViewer    = LBVHViewerT<true>;
    using NonConstViewer = LBVHViewerT<false>;
    using ThisViewer     = LBVHViewerT<IsConst>;
    using AABB           = Eigen::AlignedBox<Float, 3>;


    struct DefaultQueryCallback
    {
        MUDA_GENERIC void operator()(uint32_t obj_idx) const noexcept {}
    };

    MUDA_GENERIC LBVHViewerT(const uint32_t      num_nodes,
                             const uint32_t      num_objects,
                             auto_const_t<Node>* nodes,
                             auto_const_t<AABB>* aabbs)
        : m_num_nodes(num_nodes)
        , m_num_objects(num_objects)
        , m_nodes(nodes, num_nodes)
        , m_aabbs(aabbs, num_nodes)
    {
        m_aabbs.copy_name(*this);
        m_nodes.copy_name(*this);
    }

    MUDA_GENERIC auto as_const() const noexcept
    {
        return ConstViewer{m_num_nodes, m_num_objects, m_nodes.data(), m_aabbs.data()};
    }

    MUDA_GENERIC operator ConstViewer() const noexcept { return as_const(); }

    MUDA_GENERIC auto num_nodes() const noexcept { return m_num_nodes; }
    MUDA_GENERIC auto num_objects() const noexcept { return m_num_objects; }

    /**
     * @brief query AABBs that intersect with the given point q.
     * 
     * @param q query point
     * @param callback callback function that is called when an AABB is found (may be called multiple times)
     * 
     * @return the number of found AABBs
     */
    template <uint32_t StackNum = 64, std::invocable<uint32_t> CallbackF = DefaultQueryCallback>
    MUDA_GENERIC uint32_t query(const Vector3& q,
                                CallbackF callback = DefaultQueryCallback{}) const noexcept
    {
        uint32_t stack[StackNum];
        return this->query(q, stack, StackNum, callback);
    }

    template <std::invocable<uint32_t> CallbackF = DefaultQueryCallback>
    MUDA_GENERIC uint32_t query(const Vector3& q,
                                uint32_t*      stack,
                                uint32_t       stack_num,
                                CallbackF callback = DefaultQueryCallback{}) const noexcept
    {
        return this->query(
            q,
            [](const AABB& aabb, const Vector3& q) { return aabb.contains(q); },
            stack,
            stack_num,
            callback);
    }

    /**
     * @brief query AABBs that intersect with the given AABB q.
     * 
     * @param q query AABB
     * @param callback callback function that is called when an AABB is found (may be called multiple times)
     * 
     * @return the number of found AABBs
     */
    template <uint32_t StackNum = 64, std::invocable<uint32_t> CallbackF = DefaultQueryCallback>
    MUDA_GENERIC uint32_t query(const AABB& aabb,
                                CallbackF callback = DefaultQueryCallback{}) const noexcept
    {
        uint32_t stack[StackNum];
        return this->query(aabb, stack, StackNum, callback);
    }

    template <std::invocable<uint32_t> CallbackF = DefaultQueryCallback>
    MUDA_GENERIC uint32_t query(const AABB& aabb,
                                uint32_t*   stack,
                                uint32_t    stack_num,
                                CallbackF callback = DefaultQueryCallback{}) const noexcept
    {
        return this->query(
            aabb, [](const AABB& A, const AABB& B) { return A.intersects(B); }, stack, stack_num, callback);
    }


    /**
     * @brief check if the stack overflow occurs during the query.
     */
    bool stack_overflow() const noexcept { return m_stack_overflow; }

  private:
    uint32_t m_num_nodes;    // (# of internal node) + (# of leaves), 2N+1
    uint32_t m_num_objects;  // (# of leaves), the same as the number of objects

    muda::Dense1DBase<IsConst, AABB> m_aabbs;
    muda::Dense1DBase<IsConst, Node> m_nodes;

    MUDA_INLINE MUDA_GENERIC void check_index(const uint32_t idx) const noexcept
    {
        MUDA_KERNEL_ASSERT(idx < m_num_objects,
                           "BVHViewer[%s:%s]: index out of range, idx=%u, num_objects=%u",
                           this->name(),
                           this->kernel_name(),
                           idx,
                           m_num_objects);
    }

    MUDA_INLINE MUDA_GENERIC void stack_overflow_warning(uint32_t num_found,
                                                         uint32_t stack_num) const noexcept
    {
        if constexpr(muda::RUNTIME_CHECK_ON)
        {
            MUDA_KERNEL_WARN_WITH_LOCATION("BVHViewer[%s:%s]: stack overflow, num_found=%u, stack_num=%u, the return value may be invalid, enlarge the stack please.",
                                           this->name(),
                                           this->kernel_name(),
                                           num_found,
                                           stack_num);
        }
    }

    mutable bool m_stack_overflow = false;

    template <typename QueryType, typename IntersectF, typename CallbackF>
    MUDA_GENERIC uint32_t query(const QueryType& Q,
                                IntersectF       Intersect,
                                uint32_t*        stack,
                                uint32_t         stack_num,
                                CallbackF        Callback) const noexcept
    {
        uint32_t* stack_ptr = stack;
        uint32_t* stack_end = stack + stack_num;
        *stack_ptr++        = 0;  // root node is always 0

        if(m_num_objects == 1)
        {
            if(Intersect(m_aabbs(0), Q))
            {
                Callback(m_nodes(0).object_idx);
                return 1;
            }
        }

        uint32_t num_found = 0;
        do
        {
            const uint32_t node  = *--stack_ptr;
            const uint32_t L_idx = m_nodes(node).left_idx;
            const uint32_t R_idx = m_nodes(node).right_idx;

            if(Intersect(m_aabbs(L_idx), Q))
            {
                const auto obj_idx = m_nodes(L_idx).object_idx;
                if(obj_idx != 0xFFFFFFFF)
                {
                    Callback(obj_idx);
                    ++num_found;
                }
                else  // the node is not a leaf.
                {
                    *stack_ptr++ = L_idx;
                }
            }
            if(Intersect(m_aabbs(R_idx), Q))
            {
                const auto obj_idx = m_nodes(R_idx).object_idx;
                if(obj_idx != 0xFFFFFFFF)
                {
                    Callback(obj_idx);
                    ++num_found;
                }
                else  // the node is not a leaf.
                {
                    *stack_ptr++ = R_idx;
                }
            }
            if(stack_ptr >= stack_end)
            {
                stack_overflow_warning(num_found, stack_num);
                break;
            }
        } while(stack < stack_ptr);
        return num_found;
    }
};

using LBVHViewer  = LBVHViewerT<false>;
using CLBVHViewer = LBVHViewerT<true>;

class LBVH
{
    using Node = detail::LBVHNode;

  public:
    using AABB        = Eigen::AlignedBox<Float, 3>;
    using MortonIndex = detail::LBVHMortonIndex;

    // now we only use default stream
    void build(muda::CBufferView<AABB> aabbs, muda::Stream& s = muda::Stream::Default())
    {
        using namespace muda;

        if(aabbs.size() == 0)
            return;

        const uint32_t num_objects        = aabbs.size();
        const uint32_t num_internal_nodes = num_objects - 1;
        const uint32_t leaf_start         = num_internal_nodes;
        const uint32_t num_nodes          = num_objects * 2 - 1;

        AABB default_aabb;
        resize(s, m_aabbs, num_nodes);
        BufferLaunch(s).fill(m_aabbs.view(), default_aabb);

        resize(s, m_sorted_mortons, num_objects);
        resize(s, m_sorted_mortons, num_objects);

        resize(s, m_indices, num_objects);
        resize(s, m_new_to_old, num_objects);

        resize(s, m_mortons, num_objects);
        resize(s, m_morton_idx, num_objects);

        Node default_node;
        m_nodes.resize(num_nodes, default_node);
        resize(s, m_nodes, num_nodes);

        resize(s, m_flags, num_internal_nodes);
        BufferLaunch(s).fill(m_flags.view(), 0);

        // 1) get max aabb
        DeviceReduce(s).Reduce(
            aabbs.data(),
            m_max_aabb.data(),
            aabbs.size(),
            [] HIPCUB_RUNTIME_FUNCTION(const AABB& a, const AABB& b) -> AABB
            { return a.merged(b); },
            default_aabb);

        //AABB max_aabb;
        //max_aabb = m_max_aabb;

        //std::cout << "max_aabb=" << max_aabb.min().transpose() << ", "
        //          << max_aabb.max().transpose() << std::endl;

        // 2) calculate morton code
        on(s)
            .next<ParallelFor>()
            .kernel_name("LBVH::MortonCode")
            .apply(num_objects,
                   [max_aabb = m_max_aabb.viewer().name("max_aabb"),
                    aabbs    = aabbs.viewer().name("filled_aabbs"),
                    mortons = m_mortons.viewer().name("mortons")] __device__(int i) mutable
                   {
                       Vector3 p = aabbs(i).center();
                       p -= max_aabb->min();
                       p.array() /= max_aabb->sizes().array();
                       mortons(i) = detail::morton_code(p);
                   });

        // 3) sort morton code
        on(s)
            .next<ParallelFor>()
            .kernel_name("LBVH::Iota")
            .apply(m_indices.size(),
                   [indices = m_indices.viewer()] __device__(int i) mutable
                   { indices(i) = i; });

        // 4) sort morton code
        DeviceRadixSort(s).SortPairs(m_mortons.data(),
                                     m_sorted_mortons.data(),
                                     m_indices.data(),
                                     m_new_to_old.data(),
                                     num_objects);

        // 5) expand morton code to 64bit, the last 32bit is the index
        on(s)
            .next<ParallelFor>()
            .kernel_name("LBVH::ExpandMorton")
            .apply(m_mortons.size(),
                   [morton64s = m_morton_idx.viewer().name("morton64s"),
                    mortons   = m_sorted_mortons.viewer().name("mortons"),
                    indices = m_new_to_old.viewer().name("indices")] __device__(int i) mutable
                   {
                       MortonIndex morton{mortons(i), indices(i)};
                       morton64s(i) = morton;
                   });

        // 6) setup leaf nodes
        auto leaf_aabbs = m_aabbs.view(leaf_start);  // offset = leaf_start
        auto leaf_nodes = m_nodes.view(leaf_start);  // offset = leaf_start
        on(s)
            .next<ParallelFor>()
            .kernel_name("LBVH::SetupLeafNodes")
            .apply(num_objects,
                   [leaf_nodes = leaf_nodes.viewer().name("leaf_nodes"),
                    indices    = m_new_to_old.viewer().name("indices"),
                    aabbs      = aabbs.viewer().name("aabbs"),
                    sorted_aabbs = leaf_aabbs.viewer().name("sorted_aabbs")] __device__(int i) mutable
                   {
                       Node node;
                       node.parent_idx = 0xFFFFFFFF;
                       node.left_idx   = 0xFFFFFFFF;
                       node.right_idx  = 0xFFFFFFFF;
                       node.object_idx = indices(i);
                       leaf_nodes(i)   = node;
                       sorted_aabbs(i) = aabbs(node.object_idx);
                   });

        // 7) construct internal nodes
        on(s)
            .next<ParallelFor>()
            .kernel_name("LBVH::ConstructInternalNodes")
            .apply(num_internal_nodes,
                   [nodes      = m_nodes.viewer().name("nodes"),
                    morton_idx = m_morton_idx.viewer().name("morton_idx"),
                    num_objects] __device__(int idx) mutable
                   {
                       nodes(idx).object_idx = 0xFFFFFFFF;  //  internal nodes

                       const uint2 ij =
                           detail::determine_range(morton_idx, num_objects, idx);
                       const int gamma =
                           detail::find_split(morton_idx, num_objects, ij.x, ij.y);

                       nodes(idx).left_idx  = gamma;
                       nodes(idx).right_idx = gamma + 1;
                       if(thrust::min(ij.x, ij.y) == gamma)
                       {
                           nodes(idx).left_idx += num_objects - 1;
                       }
                       if(thrust::max(ij.x, ij.y) == gamma + 1)
                       {
                           nodes(idx).right_idx += num_objects - 1;
                       }
                       nodes(nodes(idx).left_idx).parent_idx  = idx;
                       nodes(nodes(idx).right_idx).parent_idx = idx;
                   });

        // 8) calculate the AABB of internal nodes
        auto internal_aabbs = m_aabbs.view(0, num_internal_nodes);
        on(s)
            .next<ParallelFor>()
            .kernel_name("LBVH::CalculateInternalAABB")
            .apply(num_objects,
                   [nodes = m_nodes.cviewer().name("nodes"),
                    aabbs = m_aabbs.viewer().name("aabbs"),
                    flags = m_flags.viewer().name("flags"),
                    leaf_start] __device__(int I) mutable
                   {
                       auto leaf_idx = I + leaf_start;
                       auto parent   = nodes(leaf_idx).parent_idx;


                       while(parent != 0xFFFFFFFF)  // means idx == 0
                       {
                           const int old = muda::atomic_add(&flags(parent), 1);
                           if(old == 0)
                           {
                               // this is the first thread entered here.
                               // wait the other thread from the other child node.
                               return;
                           }
                           MUDA_KERNEL_ASSERT(old == 1, "old=%d", old);
                           //here, the flag has already been 1. it means that this
                           //thread is the 2nd thread. merge AABB of both childlen.

                           // the memory fence is necessary to disable reordering of the memory access.
                           // we need to ensure that this thread can get the updated value of AABB.
                           ::cuda::atomic_thread_fence(::cuda::memory_order_acquire,
                                                       ::cuda::thread_scope_system);

                           const auto lidx = nodes(parent).left_idx;
                           const auto ridx = nodes(parent).right_idx;
                           auto&      lbox = aabbs(lidx);
                           auto&      rbox = aabbs(ridx);

                           // to avoid cache coherency problem, we must use atomic operation.
                           auto atomic_fetch = [](AABB& aabb) -> AABB
                           {
                               Vector3 zero  = Vector3::Zero();
                               AABB    aabb_ = aabb;

                               // without atomic_thread_fence, this loop may be infinite.
                               while(aabb_.isEmpty())
                               {
                                   Vector3 min_ = eigen::atomic_add(aabb.min(), zero);
                                   Vector3 max_ = eigen::atomic_add(aabb.max(), zero);
                                   aabb_ = AABB{min_, max_};
                               };

                               return aabb_;
                           };

                           auto L = atomic_fetch(lbox);
                           auto R = atomic_fetch(rbox);

                           aabbs(parent) = L.merged(R);

                           // look the next parent...
                           parent = nodes(parent).parent_idx;
                       }
                   });
    }

    auto viewer() noexcept
    {
        return LBVHViewer{(uint32_t)m_nodes.size(),
                          (uint32_t)m_mortons.size(),
                          m_nodes.data(),
                          m_aabbs.data()};
    }

    auto viewer() const noexcept
    {
        return CLBVHViewer{(uint32_t)m_nodes.size(),
                           (uint32_t)m_mortons.size(),
                           m_nodes.data(),
                           m_aabbs.data()};
    }
    template <typename T>
    void resize(muda::Stream& s, muda::DeviceBuffer<T>& V, size_t size)
    {
        if(size > V.capacity())
            muda::BufferLaunch(s).reserve(V, size * m_resize_factor);
        muda::BufferLaunch(s).resize(V, size);
    }

    muda::DeviceBuffer<AABB>        m_aabbs;
    muda::DeviceBuffer<uint32_t>    m_mortons;
    muda::DeviceBuffer<uint32_t>    m_sorted_mortons;
    muda::DeviceBuffer<uint32_t>    m_indices;
    muda::DeviceBuffer<uint32_t>    m_new_to_old;
    muda::DeviceBuffer<MortonIndex> m_morton_idx;
    muda::DeviceBuffer<int>         m_flags;
    muda::DeviceBuffer<Node>        m_nodes;
    muda::DeviceVar<AABB>           m_max_aabb;

    Float m_resize_factor = 1.5;
};
}  // namespace uipc::test::backend::cuda


using namespace uipc;
using namespace uipc::test::backend::cuda;
using namespace uipc::geometry;
using namespace muda;


void tree_consistency_test(const DeviceBuffer<detail::LBVHNode>& d_a,
                           const DeviceBuffer<detail::LBVHNode>& d_b,
                           const DeviceBuffer<LBVH::AABB>&       d_a_AABB,
                           const DeviceBuffer<LBVH::AABB>&       d_b_AABB)
{
    std::vector<detail::LBVHNode> a;
    d_a.copy_to(a);

    std::vector<detail::LBVHNode> b;
    d_b.copy_to(b);

    std::vector<LBVH::AABB> a_AABB;
    d_a_AABB.copy_to(a_AABB);

    std::vector<LBVH::AABB> b_AABB;
    d_b_AABB.copy_to(b_AABB);

    {
        auto it = std::mismatch(a.begin(),
                                a.end(),
                                b.begin(),
                                b.end(),
                                [](const auto& lhs, const auto& rhs)
                                {
                                    return lhs.parent_idx == rhs.parent_idx
                                           && lhs.left_idx == rhs.left_idx
                                           && lhs.right_idx == rhs.right_idx
                                           && lhs.object_idx == rhs.object_idx;
                                });

        REQUIRE(it.first == a.end());
        REQUIRE(it.second == b.end());

        if(it.first != a.end() || it.second != b.end())
        {
            std::cout << "tree inconsistency detected:" << std::endl;

            for(int i = 0; i < a.size(); ++i)
            {
                auto f_node = a[i];
                auto s_node = b[i];

                std::cout
                    << "node=" << i << "parent=(" << f_node.parent_idx << ", "
                    << s_node.parent_idx << ")"
                    << "left=(" << f_node.left_idx << ", " << s_node.left_idx << ")"
                    << "right=(" << f_node.right_idx << ", " << s_node.right_idx << ")"
                    << "obj=(" << f_node.object_idx << ", " << s_node.object_idx
                    << ")" << std::endl;
            }
        }
    }

    {
        auto it = std::mismatch(a_AABB.begin(),
                                a_AABB.end(),
                                b_AABB.begin(),
                                b_AABB.end(),
                                [](const auto& lhs, const auto& rhs) {
                                    return lhs.min() == rhs.min()
                                           && lhs.max() == rhs.max();
                                });

        CHECK(it.first == a_AABB.end());
        CHECK(it.second == b_AABB.end());

        while(it.first != a_AABB.end() || it.second != b_AABB.end())
        {
            std::cout << "AABB inconsistency detected: id="
                      << std::distance(a_AABB.begin(), it.first) << std::endl;

            auto f_aabb = *it.first;
            auto s_aabb = *it.second;

            std::cout << "aabb=(" << f_aabb.min().transpose() << ", "
                      << f_aabb.max().transpose() << ")\n"
                      << "aabb=(" << s_aabb.min().transpose() << ", "
                      << s_aabb.max().transpose() << ")" << std::endl;

            it = std::mismatch(++it.first,
                               a_AABB.end(),
                               ++it.second,
                               b_AABB.end(),
                               [](const auto& lhs, const auto& rhs) {
                                   return lhs.min() == rhs.min()
                                          && lhs.max() == rhs.max();
                               });
        }
    }
}


std::vector<Vector2i> lbvh_cp(span<const LBVH::AABB> aabbs)
{
    DeviceBuffer<LBVH::AABB> d_aabbs(aabbs.size());
    d_aabbs.view().copy_from(aabbs.data());

    // enlarge the aabbs by a 0.1 * diagonal length
    ParallelFor()
        .kernel_name("LBVHTest::Enlarge")
        .apply(aabbs.size(),
               [aabbs = d_aabbs.viewer().name("aabbs")] __device__(int i) mutable
               {
                   auto aabb = aabbs(i);
                   auto diag = aabb.sizes().norm();
                   //aabb.min().array() -= 0.1 * diag;
                   //aabb.max().array() += 0.1 * diag;
               });


    LBVH lbvh;
    lbvh.build(d_aabbs);

    DeviceBuffer<IndexT> counts(aabbs.size() + 1ull);
    DeviceBuffer<IndexT> offsets(aabbs.size() + 1ull);

    //for(int i = 0; i < aabbs.size(); ++i)
    //{
    //    auto aabb = aabbs[i];
    //    std::cout << "[" << aabb.min().transpose() << "],"
    //              << "[" << aabb.max().transpose() << "]" << std::endl;
    //}

    ParallelFor()
        .kernel_name("LBVHTest::Query")
        .apply(aabbs.size(),
               [lbvh  = lbvh.viewer().name("lbvh"),
                aabbs = d_aabbs.viewer().name("aabbs"),
                counts = counts.viewer().name("counts")] __device__(int i) mutable
               {
                   auto N = aabbs.total_size();

                   auto aabb  = aabbs(i);
                   auto count = 0;
                   lbvh.query(aabb,
                              [&](uint32_t id)
                              {
                                  if(id > i)
                                      count++;
                              });
                   counts(i) = count;

                   if(i == 0)
                   {
                       counts(N) = 0;
                   }
               });

    DeviceScan().ExclusiveSum(counts.data(), offsets.data(), counts.size());
    IndexT total;
    offsets.view(aabbs.size()).copy_to(&total);

    DeviceBuffer<Vector2i> pairs(total);


    ParallelFor()
        .kernel_name("LBVHTest::Query")
        .apply(aabbs.size(),
               [lbvh    = lbvh.viewer().name("lbvh"),
                aabbs   = d_aabbs.viewer().name("aabbs"),
                counts  = counts.viewer().name("counts"),
                offsets = offsets.viewer().name("offsets"),
                pairs = pairs.viewer().name("pairs")] __device__(int i) mutable
               {
                   auto N = aabbs.total_size();

                   auto aabb   = aabbs(i);
                   auto count  = counts(i);
                   auto offset = offsets(i);

                   auto pair = pairs.subview(offset, count);
                   int  j    = 0;
                   lbvh.query(aabb,
                              [&](uint32_t id)
                              {
                                  if(id > i)
                                      pair(j++) = Vector2i(i, id);
                              });
                   MUDA_ASSERT(j == count, "j = %d, count=%d", j, count);
               });

    DeviceBuffer<detail::LBVHNode> nodes_1 = lbvh.m_nodes;
    DeviceBuffer<LBVH::AABB>       aabbs_1 = lbvh.m_aabbs;

    lbvh.build(d_aabbs);  // build again, the internal nodes should be the same.
    DeviceBuffer<detail::LBVHNode> nodes_2 = lbvh.m_nodes;
    DeviceBuffer<LBVH::AABB>       aabbs_2 = lbvh.m_aabbs;

    tree_consistency_test(nodes_1, nodes_2, aabbs_1, aabbs_2);

    LBVH lbvh2;
    lbvh2.build(d_aabbs);

    DeviceBuffer<detail::LBVHNode> nodes_3 = lbvh2.m_nodes;
    DeviceBuffer<LBVH::AABB>       aabbs_3 = lbvh2.m_aabbs;

    //tree_consistency_test(nodes_1, nodes_3, aabbs_1, aabbs_3);

    std::vector<Vector2i> pairs_host;
    pairs.copy_to(pairs_host);

    //std::vector<LBVH::AABB> aabbs_host;
    //lbvh.m_aabbs.copy_to(aabbs_host);
    //for(auto&& [i, aabb] : enumerate(aabbs_host))
    //{
    //    std::cout << "[" << aabb.min().transpose() << "],"
    //              << "[" << aabb.max().transpose() << "]" << std::endl;
    //}

    //std::vector<detail::LBVHNode> nodes_host;
    //lbvh.m_nodes.copy_to(nodes_host);
    //for(auto&& [i, node] : enumerate(nodes_host))
    //{
    //    std::cout << "node=" << i << "[" << aabbs_host[i].min().transpose() << "],"
    //              << "[" << aabbs_host[i].max().transpose() << "]"
    //              << ", parent=" << node.parent_idx
    //              << ", left=" << node.left_idx << ", right=" << node.right_idx
    //              << ", obj=" << node.object_idx << std::endl;
    //}

    return pairs_host;
}

std::vector<Vector2i> brute_froce_cp(span<const LBVH::AABB> aabbs)
{
    std::vector<Vector2i> pairs;
    for(auto&& [i, aabb0] : enumerate(aabbs))
    {
        for(int j = i + 1; j < aabbs.size(); ++j)
        {
            auto aabb1 = aabbs[j];
            if(aabb1.intersects(aabb0))
            {
                pairs.push_back(Vector2i(i, j));
            }
        }
    }
    return pairs;
}


void brute_froce_gpu(span<const LBVH::AABB> aabbs)
{
    DeviceBuffer<LBVH::AABB> d_aabbs(aabbs.size());
    d_aabbs.view().copy_from(aabbs.data());

    ParallelFor()
        .kernel_name("BruteForce::Query")
        .apply(aabbs.size(),
               [aabbs = d_aabbs.viewer().name("aabbs")] __device__(int i) mutable
               {
                   auto N = aabbs.total_size();

                   auto aabb0 = aabbs(i);
                   for(int j = i + 1; j < N; ++j)
                   {
                       auto aabb1 = aabbs(j);
                       if(aabb1.intersects(aabb0))
                       {
                           printf("i=%d, j=%d\n", i, j);
                       }
                   }
               });
}

SimplicialComplex tet()
{
    std::vector           Vs = {Vector3{0.0, 0.0, 0.0},
                                Vector3{1.0, 0.0, 0.0},
                                Vector3{0.0, 1.0, 0.0},
                                Vector3{0.0, 0.0, 1.0}};
    std::vector<Vector4i> Ts = {Vector4i{0, 1, 2, 3}};

    return tetmesh(Vs, Ts);
}


void lbvh_test(const SimplicialComplex& mesh)
{
    std::cout << "num_aabb=" << mesh.triangles().size() << std::endl;


    auto pos_view = mesh.positions().view();
    auto tri_view = mesh.triangles().topo().view();

    //tri_view = tri_view.subspan(0, 5);

    std::vector<LBVH::AABB> aabbs(tri_view.size());
    for(auto&& [i, tri] : enumerate(tri_view))
    {
        auto p0 = pos_view[tri[0]];
        auto p1 = pos_view[tri[1]];
        auto p2 = pos_view[tri[2]];
        aabbs[i].extend(p0).extend(p1).extend(p2);
    }


    auto lbvh_pairs = lbvh_cp(aabbs);


    auto bf_pairs = brute_froce_cp(aabbs);

    // brute_froce_gpu(aabbs);


    auto compare = [](const Vector2i& lhs, const Vector2i& rhs)
    { return lhs[0] < rhs[0] || (lhs[0] == rhs[0] && lhs[1] < rhs[1]); };

    std::ranges::sort(lbvh_pairs, compare);
    std::ranges::sort(bf_pairs, compare);


    auto check_unique = [](auto begin, auto end)
    {
        for(auto it = begin; it != end; ++it)
        {
            if(it + 1 != end && *it == *(it + 1))
            {
                return false;
            }
        }
        return true;
    };

    CHECK(check_unique(lbvh_pairs.begin(), lbvh_pairs.end()));


    std::list<Vector2i> diff;

    std::set_difference(bf_pairs.begin(),
                        bf_pairs.end(),
                        lbvh_pairs.begin(),
                        lbvh_pairs.end(),
                        std::back_inserter(diff),
                        compare);

    CHECK(diff.empty());

    if (!diff.empty())
    {
        std::cout << "lbvh_pairs.size()=" << lbvh_pairs.size() << std::endl;
        std::cout << "bf_pairs.size()=" << bf_pairs.size() << std::endl;
        std::cout << "diff:" << std::endl;
        for(auto&& d : diff)
        {
            std::cout << d.transpose() << std::endl;
        }
    }
}


TEST_CASE("lbvh", "[muda]")
{
    SECTION("tet")
    {
        lbvh_test(tet());
    }

    SECTION("cube.obj")
    {
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}cube.obj", AssetDir::trimesh_path()));
        lbvh_test(mesh);
    }

    SECTION("cube.msh")
    {
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}cube.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }

    SECTION("cylinder_hole.msh")
    {
        SimplicialComplexIO io;
        auto                mesh =
            io.read(fmt::format("{}cylinder_hole.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }

    SECTION("simple_axle.msh")
    {
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}simple_axle.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }

    SECTION("wheel_axle.msh")
    {
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}wheel_axle.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }

    SECTION("bunny0.msh")
    {
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}bunny0.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }
}
