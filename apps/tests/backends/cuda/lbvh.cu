#include "hip/hip_runtime.h"
#include <muda/ext/eigen/eigen_core_cxx20.h>  // to use Eigen in CUDA
#include <app/test_common.h>
#include <app/asset_dir.h>
#include <linear_bvh.h>
#include <uipc/geometry.h>
#include <muda/cub/device/device_scan.h>
#include <uipc/common/enumerate.h>
#include <muda/viewer/viewer_base.h>

using namespace muda;
using namespace uipc;
using namespace uipc::geometry;
using namespace uipc::backend::cuda;

void tree_consistency_test(const DeviceBuffer<LinearBVHNode>& d_a,
                           const DeviceBuffer<LinearBVHNode>& d_b,
                           const DeviceBuffer<LinearBVHAABB>& d_a_AABB,
                           const DeviceBuffer<LinearBVHAABB>& d_b_AABB)
{
    std::vector<LinearBVHNode> a;
    d_a.copy_to(a);

    std::vector<LinearBVHNode> b;
    d_b.copy_to(b);

    std::vector<LinearBVHAABB> a_AABB;
    d_a_AABB.copy_to(a_AABB);

    std::vector<LinearBVHAABB> b_AABB;
    d_b_AABB.copy_to(b_AABB);

    {
        auto it = std::mismatch(a.begin(),
                                a.end(),
                                b.begin(),
                                b.end(),
                                [](const auto& lhs, const auto& rhs)
                                {
                                    return lhs.parent_idx == rhs.parent_idx
                                           && lhs.left_idx == rhs.left_idx
                                           && lhs.right_idx == rhs.right_idx
                                           && lhs.object_idx == rhs.object_idx;
                                });

        REQUIRE(it.first == a.end());
        REQUIRE(it.second == b.end());

        if(it.first != a.end() || it.second != b.end())
        {
            std::cout << "tree inconsistency detected:" << std::endl;

            for(int i = 0; i < a.size(); ++i)
            {
                auto f_node = a[i];
                auto s_node = b[i];

                std::cout
                    << "node=" << i << "parent=(" << f_node.parent_idx << ", "
                    << s_node.parent_idx << ")"
                    << "left=(" << f_node.left_idx << ", " << s_node.left_idx << ")"
                    << "right=(" << f_node.right_idx << ", " << s_node.right_idx << ")"
                    << "obj=(" << f_node.object_idx << ", " << s_node.object_idx
                    << ")" << std::endl;
            }
        }
    }

    {
        auto it = std::mismatch(a_AABB.begin(),
                                a_AABB.end(),
                                b_AABB.begin(),
                                b_AABB.end(),
                                [](const auto& lhs, const auto& rhs) {
                                    return lhs.min() == rhs.min()
                                           && lhs.max() == rhs.max();
                                });

        CHECK(it.first == a_AABB.end());
        CHECK(it.second == b_AABB.end());

        while(it.first != a_AABB.end() || it.second != b_AABB.end())
        {
            std::cout << "AABB inconsistency detected: id="
                      << std::distance(a_AABB.begin(), it.first) << std::endl;

            auto f_aabb = *it.first;
            auto s_aabb = *it.second;

            std::cout << "aabb=(" << f_aabb.min().transpose() << ", "
                      << f_aabb.max().transpose() << ")\n"
                      << "aabb=(" << s_aabb.min().transpose() << ", "
                      << s_aabb.max().transpose() << ")" << std::endl;

            it = std::mismatch(++it.first,
                               a_AABB.end(),
                               ++it.second,
                               b_AABB.end(),
                               [](const auto& lhs, const auto& rhs) {
                                   return lhs.min() == rhs.min()
                                          && lhs.max() == rhs.max();
                               });
        }
    }
}


std::vector<Vector2i> LinearBVH_cp(span<const LinearBVHAABB> aabbs)
{
    DeviceBuffer<LinearBVHAABB> d_aabbs(aabbs.size());
    d_aabbs.view().copy_from(aabbs.data());

    // enlarge the aabbs by a 0.1 * diagonal length
    ParallelFor()
        .kernel_name("LinearBVHTest::Enlarge")
        .apply(aabbs.size(),
               [aabbs = d_aabbs.viewer().name("aabbs")] __device__(int i) mutable
               {
                   auto aabb = aabbs(i);
                   auto diag = aabb.sizes().norm();
                   //aabb.min().array() -= 0.1 * diag;
                   //aabb.max().array() += 0.1 * diag;
               });


    LinearBVH lbvh;
    lbvh.build(d_aabbs, muda::Stream::Default());

    DeviceBuffer<IndexT> counts(aabbs.size() + 1ull);
    DeviceBuffer<IndexT> offsets(aabbs.size() + 1ull);

    //for(int i = 0; i < aabbs.size(); ++i)
    //{
    //    auto aabb = aabbs[i];
    //    std::cout << "[" << aabb.min().transpose() << "],"
    //              << "[" << aabb.max().transpose() << "]" << std::endl;
    //}

    ParallelFor()
        .kernel_name("LinearBVHTest::Query")
        .apply(aabbs.size(),
               [LinearBVH = lbvh.viewer().name("LinearBVH"),
                aabbs     = d_aabbs.viewer().name("aabbs"),
                counts = counts.viewer().name("counts")] __device__(int i) mutable
               {
                   auto N = aabbs.total_size();

                   auto aabb  = aabbs(i);
                   auto count = 0;
                   LinearBVH.query(aabb,
                                   [&](uint32_t id)
                                   {
                                       if(id > i)
                                           count++;
                                   });
                   counts(i) = count;

                   if(i == 0)
                   {
                       counts(N) = 0;
                   }
               });

    DeviceScan().ExclusiveSum(counts.data(), offsets.data(), counts.size());
    IndexT total;
    offsets.view(aabbs.size()).copy_to(&total);

    DeviceBuffer<Vector2i> pairs(total);


    ParallelFor()
        .kernel_name("LinearBVHTest::Query")
        .apply(aabbs.size(),
               [LinearBVH = lbvh.viewer().name("LinearBVH"),
                aabbs     = d_aabbs.viewer().name("aabbs"),
                counts    = counts.viewer().name("counts"),
                offsets   = offsets.viewer().name("offsets"),
                pairs = pairs.viewer().name("pairs")] __device__(int i) mutable
               {
                   auto N = aabbs.total_size();

                   auto aabb   = aabbs(i);
                   auto count  = counts(i);
                   auto offset = offsets(i);

                   auto pair = pairs.subview(offset, count);
                   int  j    = 0;
                   LinearBVH.query(aabb,
                                   [&](uint32_t id)
                                   {
                                       if(id > i)
                                           pair(j++) = Vector2i(i, id);
                                   });
                   MUDA_ASSERT(j == count, "j = %d, count=%d", j, count);
               });

    DeviceBuffer<LinearBVHNode> nodes_1 = LinearBVHVisitor(lbvh).nodes();
    DeviceBuffer<LinearBVHAABB> aabbs_1 = LinearBVHVisitor(lbvh).aabbs();

    lbvh.build(d_aabbs);  // build again, the internal nodes should be the same.
    DeviceBuffer<LinearBVHNode> nodes_2 = LinearBVHVisitor(lbvh).nodes();
    DeviceBuffer<LinearBVHAABB> aabbs_2 = LinearBVHVisitor(lbvh).aabbs();

    tree_consistency_test(nodes_1, nodes_2, aabbs_1, aabbs_2);

    LinearBVH lbvh2;
    lbvh2.build(d_aabbs);

    DeviceBuffer<LinearBVHNode> nodes_3 = LinearBVHVisitor(lbvh2).nodes();
    DeviceBuffer<LinearBVHAABB> aabbs_3 = LinearBVHVisitor(lbvh2).aabbs();

    tree_consistency_test(nodes_1, nodes_3, aabbs_1, aabbs_3);

    std::vector<Vector2i> pairs_host;
    pairs.copy_to(pairs_host);

    //std::vector<LinearBVHAABB> aabbs_host(aabbs.size());
    //LinearBVHVisitor(lbvh).aabbs().copy_to(aabbs_host.data());
    //for(auto&& [i, aabb] : enumerate(aabbs_host))
    //{
    //    std::cout << "[" << aabb.min().transpose() << "],"
    //              << "[" << aabb.max().transpose() << "]" << std::endl;
    //}

    //std::vector<LinearBVHNode> nodes_host(2 * aabbs.size() - 1);
    //LinearBVHVisitor(lbvh).nodes().copy_to(nodes_host.data());
    //for(auto&& [i, node] : enumerate(nodes_host))
    //{
    //    std::cout << "node=" << i << "[" << aabbs_host[i].min().transpose() << "],"
    //              << "[" << aabbs_host[i].max().transpose() << "]"
    //              << ", parent=" << node.parent_idx
    //              << ", left=" << node.left_idx << ", right=" << node.right_idx
    //              << ", obj=" << node.object_idx << std::endl;
    //}

    return pairs_host;
}

std::vector<Vector2i> brute_froce_cp(span<const LinearBVHAABB> aabbs)
{
    std::vector<Vector2i> pairs;
    for(auto&& [i, aabb0] : enumerate(aabbs))
    {
        for(int j = i + 1; j < aabbs.size(); ++j)
        {
            auto aabb1 = aabbs[j];
            if(aabb1.intersects(aabb0))
            {
                pairs.push_back(Vector2i(i, j));
            }
        }
    }
    return pairs;
}


void brute_froce_gpu(span<const LinearBVHAABB> aabbs)
{
    DeviceBuffer<LinearBVHAABB> d_aabbs(aabbs.size());
    d_aabbs.view().copy_from(aabbs.data());

    ParallelFor()
        .kernel_name("BruteForce::Query")
        .apply(aabbs.size(),
               [aabbs = d_aabbs.viewer().name("aabbs")] __device__(int i) mutable
               {
                   auto N = aabbs.total_size();

                   auto aabb0 = aabbs(i);
                   for(int j = i + 1; j < N; ++j)
                   {
                       auto aabb1 = aabbs(j);
                       if(aabb1.intersects(aabb0))
                       {
                           printf("i=%d, j=%d\n", i, j);
                       }
                   }
               });
}

SimplicialComplex tet()
{
    std::vector           Vs = {Vector3{0.0, 0.0, 0.0},
                                Vector3{1.0, 0.0, 0.0},
                                Vector3{0.0, 1.0, 0.0},
                                Vector3{0.0, 0.0, 1.0}};
    std::vector<Vector4i> Ts = {Vector4i{0, 1, 2, 3}};

    return tetmesh(Vs, Ts);
}


void lbvh_test(const SimplicialComplex& mesh)
{
    std::cout << "num_aabb=" << mesh.triangles().size() << std::endl;

    auto pos_view = mesh.positions().view();
    auto tri_view = mesh.triangles().topo().view();

    //tri_view = tri_view.subspan(0, 5);

    std::vector<LinearBVHAABB> aabbs(tri_view.size());
    for(auto&& [i, tri] : enumerate(tri_view))
    {
        auto p0 = pos_view[tri[0]];
        auto p1 = pos_view[tri[1]];
        auto p2 = pos_view[tri[2]];
        aabbs[i].extend(p0).extend(p1).extend(p2);
    }


    auto LinearBVH_pairs = LinearBVH_cp(aabbs);


    auto bf_pairs = brute_froce_cp(aabbs);

    // brute_froce_gpu(aabbs);


    auto compare = [](const Vector2i& lhs, const Vector2i& rhs)
    { return lhs[0] < rhs[0] || (lhs[0] == rhs[0] && lhs[1] < rhs[1]); };

    std::ranges::sort(LinearBVH_pairs, compare);
    std::ranges::sort(bf_pairs, compare);


    auto check_unique = [](auto begin, auto end)
    {
        for(auto it = begin; it != end; ++it)
        {
            if(it + 1 != end && *it == *(it + 1))
            {
                return false;
            }
        }
        return true;
    };

    CHECK(check_unique(LinearBVH_pairs.begin(), LinearBVH_pairs.end()));


    std::list<Vector2i> diff;

    std::set_difference(bf_pairs.begin(),
                        bf_pairs.end(),
                        LinearBVH_pairs.begin(),
                        LinearBVH_pairs.end(),
                        std::back_inserter(diff),
                        compare);

    CHECK(diff.empty());

    if(!diff.empty())
    {
        std::cout << "LinearBVH_pairs.size()=" << LinearBVH_pairs.size() << std::endl;
        std::cout << "bf_pairs.size()=" << bf_pairs.size() << std::endl;
        std::cout << "diff:" << std::endl;
        for(auto&& d : diff)
        {
            std::cout << d.transpose() << std::endl;
        }
    }
}

TEST_CASE("LinearBVH", "[muda]")
{
    SECTION("tet")
    {
        fmt::println("tet:");
        lbvh_test(tet());
    }

    SECTION("cube.obj")
    {
        fmt::println("cube.obj:");
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}cube.obj", AssetDir::trimesh_path()));
        lbvh_test(mesh);
    }

    SECTION("cube.msh")
    {
        fmt::println("cube.msh:");
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}cube.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }

    SECTION("cylinder_hole.msh")
    {
        fmt::println("cylinder_hole.msh:");
        SimplicialComplexIO io;
        auto                mesh =
            io.read(fmt::format("{}cylinder_hole.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }

    SECTION("simple_axle.msh")
    {
        fmt::println("simple_axle.msh:");
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}simple_axle.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }

    SECTION("wheel_axle.msh")
    {
        fmt::println("wheel_axle.msh:");
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}wheel_axle.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }

    SECTION("bunny0.msh")
    {
        fmt::println("bunny0.msh:");
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}bunny0.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }
}
