#include "hip/hip_runtime.h"
#include <type_define.h>  // to use Eigen in CUDA
#include <app/test_common.h>
#include <app/asset_dir.h>
#include <collision_detection/linear_bvh.h>
#include <uipc/geometry.h>
#include <muda/cub/device/device_scan.h>
#include <uipc/common/enumerate.h>
#include <muda/viewer/viewer_base.h>
#include <uipc/uipc.h>
#include <uipc/common/timer.h>

using namespace muda;
using namespace uipc;
using namespace uipc::geometry;
using namespace uipc::backend::cuda;

// check if two trees are the same
void tree_consistency_test(const DeviceBuffer<LinearBVHNode>& d_a,
                           const DeviceBuffer<LinearBVHNode>& d_b,
                           const DeviceBuffer<LinearBVHAABB>& d_a_AABB,
                           const DeviceBuffer<LinearBVHAABB>& d_b_AABB)
{
    std::vector<LinearBVHNode> a;
    d_a.copy_to(a);

    std::vector<LinearBVHNode> b;
    d_b.copy_to(b);

    std::vector<LinearBVHAABB> a_AABB;
    d_a_AABB.copy_to(a_AABB);

    std::vector<LinearBVHAABB> b_AABB;
    d_b_AABB.copy_to(b_AABB);

    {
        auto it = std::mismatch(a.begin(),
                                a.end(),
                                b.begin(),
                                b.end(),
                                [](const auto& lhs, const auto& rhs)
                                {
                                    return lhs.parent_idx == rhs.parent_idx
                                           && lhs.left_idx == rhs.left_idx
                                           && lhs.right_idx == rhs.right_idx
                                           && lhs.object_idx == rhs.object_idx;
                                });

        REQUIRE(it.first == a.end());
        REQUIRE(it.second == b.end());

        if(it.first != a.end() || it.second != b.end())
        {
            std::cout << "tree inconsistency detected:" << std::endl;

            for(int i = 0; i < a.size(); ++i)
            {
                auto f_node = a[i];
                auto s_node = b[i];

                std::cout
                    << "node=" << i << "parent=(" << f_node.parent_idx << ", "
                    << s_node.parent_idx << ")"
                    << "left=(" << f_node.left_idx << ", " << s_node.left_idx << ")"
                    << "right=(" << f_node.right_idx << ", " << s_node.right_idx << ")"
                    << "obj=(" << f_node.object_idx << ", " << s_node.object_idx
                    << ")" << std::endl;
            }
        }
    }

    {
        auto it = std::mismatch(a_AABB.begin(),
                                a_AABB.end(),
                                b_AABB.begin(),
                                b_AABB.end(),
                                [](const auto& lhs, const auto& rhs) {
                                    return lhs.min() == rhs.min()
                                           && lhs.max() == rhs.max();
                                });

        CHECK(it.first == a_AABB.end());
        CHECK(it.second == b_AABB.end());

        while(it.first != a_AABB.end() || it.second != b_AABB.end())
        {
            std::cout << "AABB inconsistency detected: id="
                      << std::distance(a_AABB.begin(), it.first) << std::endl;

            auto f_aabb = *it.first;
            auto s_aabb = *it.second;

            std::cout << "aabb=(" << f_aabb.min().transpose() << ", "
                      << f_aabb.max().transpose() << ")\n"
                      << "aabb=(" << s_aabb.min().transpose() << ", "
                      << s_aabb.max().transpose() << ")" << std::endl;

            it = std::mismatch(++it.first,
                               a_AABB.end(),
                               ++it.second,
                               b_AABB.end(),
                               [](const auto& lhs, const auto& rhs) {
                                   return lhs.min() == rhs.min()
                                          && lhs.max() == rhs.max();
                               });
        }
    }
}

// check if the test result is conservative (i.e., no false negative)
void check_cp_conservative(span<Vector2i> test, span<Vector2i> gd)
{
    auto compare = [](const Vector2i& lhs, const Vector2i& rhs)
    { return lhs[0] < rhs[0] || (lhs[0] == rhs[0] && lhs[1] < rhs[1]); };

    std::ranges::sort(test, compare);
    std::ranges::sort(gd, compare);

    std::list<Vector2i> diff;
    std::set_difference(
        test.begin(), test.end(), gd.begin(), gd.end(), std::back_inserter(diff), compare);

    CHECK(diff.empty());

    if(!diff.empty())
    {
        fmt::println("test.size()={}", test.size());
        fmt::println("ground_truth.size()={}", gd.size());
        fmt::println("diff:");
        for(auto&& d : diff)
        {
            fmt::println("{} {}", d[0], d[1]);
        }
    }
}


std::vector<Vector2i> two_step_lbvh_cp(span<const LinearBVHAABB> aabbs)
{
    DeviceBuffer<LinearBVHAABB> d_aabbs(aabbs.size());
    d_aabbs.view().copy_from(aabbs.data());

    LinearBVH              lbvh;
    DeviceBuffer<Vector2i> pairs;

    {
        Timer timer{"2_step_lbvh"};

        {
            Timer timer{"build_tree"};
            lbvh.build(d_aabbs);
        }
        {
            Timer timer{"rebuild_tree"};
            lbvh.build(d_aabbs);
        }
        {
            Timer timer{"update_aabbs"};
            lbvh.update(d_aabbs);
        }
        DeviceBuffer<IndexT> counts(aabbs.size() + 1ull);
        DeviceBuffer<IndexT> offsets(aabbs.size() + 1ull);

        //for(int i = 0; i < aabbs.size(); ++i)
        //{
        //    auto aabb = aabbs[i];
        //    std::cout << "[" << aabb.min().transpose() << "],"
        //              << "[" << aabb.max().transpose() << "]" << std::endl;
        //}


        {
            Timer timer{"unlucky_build_cp"};

            ParallelFor()
                .kernel_name("LinearBVHTest::Query")
                .apply(aabbs.size(),
                       [LinearBVH = lbvh.viewer().name("LinearBVH"),
                        aabbs     = d_aabbs.viewer().name("aabbs"),
                        counts = counts.viewer().name("counts")] __device__(int i) mutable
                       {
                           auto N = aabbs.total_size();

                           auto aabb  = aabbs(i);
                           auto count = 0;
                           LinearBVH.query(aabb,
                                           [&](uint32_t id)
                                           {
                                               if(id > i)
                                                   count++;
                                           });
                           counts(i) = count;

                           if(i == 0)
                           {
                               counts(N) = 0;
                           }
                       });

            DeviceScan().ExclusiveSum(counts.data(), offsets.data(), counts.size());
            IndexT total;
            offsets.view(aabbs.size()).copy_to(&total);

            pairs.resize(total);


            ParallelFor()
                .kernel_name("LinearBVHTest::Query")
                .apply(aabbs.size(),
                       [LinearBVH = lbvh.viewer().name("LinearBVH"),
                        aabbs     = d_aabbs.viewer().name("aabbs"),
                        counts    = counts.viewer().name("counts"),
                        offsets   = offsets.viewer().name("offsets"),
                        pairs = pairs.viewer().name("pairs")] __device__(int i) mutable
                       {
                           auto N = aabbs.total_size();

                           auto aabb   = aabbs(i);
                           auto count  = counts(i);
                           auto offset = offsets(i);

                           auto pair = pairs.subview(offset, count);
                           int  j    = 0;
                           LinearBVH.query(aabb,
                                           [&](uint32_t id)
                                           {
                                               if(id > i)
                                                   pair(j++) = Vector2i(i, id);
                                           });
                           MUDA_ASSERT(j == count, "j = %d, count=%d", j, count);
                       });
        }

        {
            Timer timer{"lucky_build_cp"};

            ParallelFor()
                .kernel_name("LinearBVHTest::Query")
                .apply(aabbs.size(),
                       [LinearBVH = lbvh.viewer().name("LinearBVH"),
                        aabbs     = d_aabbs.viewer().name("aabbs"),
                        counts = counts.viewer().name("counts")] __device__(int i) mutable
                       {
                           auto N = aabbs.total_size();

                           auto aabb  = aabbs(i);
                           auto count = 0;
                           LinearBVH.query(aabb,
                                           [&](uint32_t id)
                                           {
                                               if(id > i)
                                                   count++;
                                           });
                           counts(i) = count;

                           if(i == 0)
                           {
                               counts(N) = 0;
                           }
                       });

            DeviceScan().ExclusiveSum(counts.data(), offsets.data(), counts.size());
            IndexT total;
            offsets.view(aabbs.size()).copy_to(&total);

            pairs.resize(total);


            ParallelFor()
                .kernel_name("LinearBVHTest::Query")
                .apply(aabbs.size(),
                       [LinearBVH = lbvh.viewer().name("LinearBVH"),
                        aabbs     = d_aabbs.viewer().name("aabbs"),
                        counts    = counts.viewer().name("counts"),
                        offsets   = offsets.viewer().name("offsets"),
                        pairs = pairs.viewer().name("pairs")] __device__(int i) mutable
                       {
                           auto N = aabbs.total_size();

                           auto aabb   = aabbs(i);
                           auto count  = counts(i);
                           auto offset = offsets(i);

                           auto pair = pairs.subview(offset, count);
                           int  j    = 0;
                           LinearBVH.query(aabb,
                                           [&](uint32_t id)
                                           {
                                               if(id > i)
                                                   pair(j++) = Vector2i(i, id);
                                           });
                           MUDA_ASSERT(j == count, "j = %d, count=%d", j, count);
                       });
        }
    }

    DeviceBuffer<LinearBVHNode> nodes_1 = LinearBVHVisitor(lbvh).nodes();
    DeviceBuffer<LinearBVHAABB> aabbs_1 = LinearBVHVisitor(lbvh).aabbs();

    lbvh.build(d_aabbs);  // do_build again, the internal nodes should be the same.
    DeviceBuffer<LinearBVHNode> nodes_2 = LinearBVHVisitor(lbvh).nodes();
    DeviceBuffer<LinearBVHAABB> aabbs_2 = LinearBVHVisitor(lbvh).aabbs();

    tree_consistency_test(nodes_1, nodes_2, aabbs_1, aabbs_2);

    LinearBVH lbvh2;
    lbvh2.build(d_aabbs);

    DeviceBuffer<LinearBVHNode> nodes_3 = LinearBVHVisitor(lbvh2).nodes();
    DeviceBuffer<LinearBVHAABB> aabbs_3 = LinearBVHVisitor(lbvh2).aabbs();

    tree_consistency_test(nodes_1, nodes_3, aabbs_1, aabbs_3);

    std::vector<Vector2i> pairs_host;
    pairs.copy_to(pairs_host);

    //std::vector<LinearBVHAABB> aabbs_host(aabbs.size());
    //LinearBVHVisitor(lbvh).aabbs().copy_to(aabbs_host.data());
    //for(auto&& [i, aabb] : enumerate(aabbs_host))
    //{
    //    std::cout << "[" << aabb.min().transpose() << "],"
    //              << "[" << aabb.max().transpose() << "]" << std::endl;
    //}

    //std::vector<LinearBVHNode> nodes_host(2 * aabbs.size() - 1);
    //LinearBVHVisitor(lbvh).nodes().copy_to(nodes_host.data());
    //for(auto&& [i, node] : enumerate(nodes_host))
    //{
    //    std::cout << "node=" << i << "[" << aabbs_host[i].min().transpose() << "],"
    //              << "[" << aabbs_host[i].max().transpose() << "]"
    //              << ", parent=" << node.parent_idx
    //              << ", left=" << node.left_idx << ", right=" << node.right_idx
    //              << ", obj=" << node.object_idx << std::endl;
    //}

    return pairs_host;
}

std::vector<Vector2i> brute_froce_cp(span<const LinearBVHAABB> aabbs)
{
    std::vector<Vector2i> pairs;
    for(auto&& [i, aabb0] : enumerate(aabbs))
    {
        for(int j = i + 1; j < aabbs.size(); ++j)
        {
            auto aabb1 = aabbs[j];
            if(aabb1.intersects(aabb0))
            {
                pairs.push_back(Vector2i(i, j));
            }
        }
    }
    return pairs;
}

std::vector<Vector2i> lbvh_query_point(span<const LinearBVHAABB> aabbs)
{
    DeviceBuffer<LinearBVHAABB> d_aabbs(aabbs.size());
    d_aabbs.view().copy_from(aabbs.data());

    LinearBVH m_lbvh;
    m_lbvh.build(d_aabbs, muda::Stream::Default());

    DeviceBuffer<IndexT> counts(aabbs.size() + 1ull);
    DeviceBuffer<IndexT> offsets(aabbs.size() + 1ull);


    DeviceBuffer<Vector3> points(aabbs.size());
    ParallelFor()
        .kernel_name("LinearBVHTest::Points")
        .apply(aabbs.size(),
               [points = points.viewer().name("points"),
                aabbs = d_aabbs.viewer().name("aabbs")] __device__(int i) mutable
               { points(i) = aabbs(i).center(); });

    ParallelFor()
        .kernel_name("LinearBVHTest::Query")
        .apply(aabbs.size(),
               [LinearBVH = m_lbvh.viewer().name("LinearBVH"),
                points    = points.viewer().name("points"),
                counts = counts.viewer().name("counts")] __device__(int i) mutable
               {
                   auto N = points.total_size();

                   auto point = points(i);
                   counts(i)  = LinearBVH.query(point);

                   if(i == 0)
                   {
                       counts(N) = 0;
                   }
               });

    DeviceScan().ExclusiveSum(counts.data(), offsets.data(), counts.size());
    IndexT total;
    offsets.view(aabbs.size()).copy_to(&total);

    DeviceBuffer<Vector2i> pairs(total);


    ParallelFor()
        .kernel_name("LinearBVHTest::Query")
        .apply(aabbs.size(),
               [m_lbvh    = m_lbvh.viewer().name("LinearBVH"),
                points  = points.viewer().name("points"),
                counts  = counts.viewer().name("counts"),
                offsets = offsets.viewer().name("offsets"),
                m_pairs = pairs.viewer().name("pairs")] __device__(int i) mutable
               {
                   auto point  = points(i);
                   auto count  = counts(i);
                   auto offset = offsets(i);
                   auto pair   = m_pairs.subview(offset, count);
                   int  j      = 0;
                   m_lbvh.query(point,
                              [&](uint32_t id) { pair(j++) = Vector2i(i, id); });
                   MUDA_ASSERT(j == count, "j = %d, count=%d", j, count);
               });

    std::vector<Vector2i> pairs_host;
    pairs.copy_to(pairs_host);

    return pairs_host;
}

std::vector<Vector2i> brute_froce_query_point(span<const LinearBVHAABB> aabbs)
{
    std::vector<Vector2i> pairs;

    std::vector<Vector3> points(aabbs.size());

    std::ranges::transform(
        aabbs, points.begin(), [](const auto& aabb) { return aabb.center(); });

    for(auto&& [i, point0] : enumerate(points))
    {
        for(auto&& [j, aabb] : enumerate(aabbs))
        {
            if(aabb.contains(point0))
            {
                pairs.push_back(Vector2i(i, j));
            }
        }
    }

    return pairs;
}

std::vector<Vector2i> adaptive_lbvh_cp(span<const LinearBVHAABB> aabbs)
{
    DeviceBuffer<LinearBVHAABB> d_aabbs(aabbs.size());
    d_aabbs.view().copy_from(aabbs.data());

    LinearBVH m_lbvh;

    DeviceVar<int>         cp_num = 0;
    DeviceBuffer<Vector2i> pairs;
    // prepare size with aabbs.size()
    pairs.resize(aabbs.size());
    fmt::println("adaptive_lbvh, prepared_size={}", pairs.size());


    {
        Timer timer{"adaptive_lbvh"};
        {
            Timer timer{"build_tree"};
            m_lbvh.build(d_aabbs);
        }
        {
            Timer timer{"rebuild_tree"};
            m_lbvh.build(d_aabbs);
        }
        {
            Timer timer{"update_aabbs"};
            m_lbvh.update(d_aabbs);
        }


        auto do_query = [&]
        {
            cp_num = 0;
            ParallelFor()
                .kernel_name("LinearBVHTest::Query")
                .apply(aabbs.size(),
                       [m_lbvh   = m_lbvh.viewer().name("LinearBVH"),
                        aabbs  = d_aabbs.viewer().name("aabbs"),
                        cp_num = cp_num.viewer().name("cp_num"),
                        pairs = pairs.viewer().name("pairs")] __device__(int i) mutable
                       {
                           auto N = aabbs.total_size();

                           auto aabb  = aabbs(i);
                           auto count = 0;
                           m_lbvh.query(aabb,
                                      [&](uint32_t id)
                                      {
                                          if(id > i)
                                          {
                                              auto last =
                                                  muda::atomic_add(cp_num.data(), 1);
                                              if(last < pairs.total_size())
                                              {
                                                  pairs(last) = Vector2i(i, id);
                                              }
                                          }
                                      });
                       });
        };

        {
            Timer timer{"unlucky_build_cp"};

            {
                Timer timer{"try_build_cp"};
                // try to query with prepared size
                do_query();
            }

            {
                Timer timer{"rebuild_cp"};

                int h_cp_num = cp_num;
                if(h_cp_num > pairs.size())  // if failed, resize and retry
                {
                    fmt::println("try query with prepared_size={} (but too small), we resize it to the detected cp_num={}",
                                 pairs.size(),
                                 h_cp_num);

                    pairs.resize(h_cp_num);
                    do_query();
                    h_cp_num = cp_num;
                    CHECK(h_cp_num == pairs.size());
                }
            }
        }


        {
            Timer timer{"lucky_build_cp"};
            do_query();
        }
    }

    std::vector<Vector2i> pairs_host;
    pairs.copy_to(pairs_host);

    return pairs_host;
}


void lbvh_test(const SimplicialComplex& mesh)
{
    Timer::set_sync_func([]() { muda::wait_device(); });

    std::cout << "num_aabb=" << mesh.triangles().size() << std::endl;

    auto pos_view = mesh.positions().view();
    auto tri_view = mesh.triangles().topo().view();

    std::vector<LinearBVHAABB> aabbs(tri_view.size());
    for(auto&& [i, tri] : enumerate(tri_view))
    {
        auto p0 = pos_view[tri[0]];
        auto p1 = pos_view[tri[1]];
        auto p2 = pos_view[tri[2]];
        aabbs[i].extend(p0).extend(p1).extend(p2);
    }

    auto lbvh_pairs = two_step_lbvh_cp(aabbs);
    auto bf_pairs   = brute_froce_cp(aabbs);

    check_cp_conservative(lbvh_pairs, bf_pairs);

    auto lbvh_qp = lbvh_query_point(aabbs);
    auto bf_qp   = brute_froce_query_point(aabbs);
    check_cp_conservative(lbvh_qp, bf_qp);

    auto adaptive = adaptive_lbvh_cp(aabbs);
    check_cp_conservative(adaptive, bf_pairs);

    Timer::set_sync_func(nullptr);

    GlobalTimer::current()->print_timings();
    GlobalTimer::current()->clear();
}

SimplicialComplex tet()
{
    std::vector           Vs = {Vector3{0.0, 0.0, 0.0},
                                Vector3{1.0, 0.0, 0.0},
                                Vector3{0.0, 1.0, 0.0},
                                Vector3{0.0, 0.0, 1.0}};
    std::vector<Vector4i> Ts = {Vector4i{0, 1, 2, 3}};

    return tetmesh(Vs, Ts);
}


TEST_CASE("lbvh", "[collision detection]")
{
    SECTION("tet")
    {
        fmt::println("tet:");
        lbvh_test(tet());
    }

    SECTION("cube.obj")
    {
        fmt::println("cube.obj:");
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}cube.obj", AssetDir::trimesh_path()));
        lbvh_test(mesh);
    }

    SECTION("cube.msh")
    {
        fmt::println("cube.msh:");
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}cube.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }

    SECTION("cylinder_hole.msh")
    {
        fmt::println("cylinder_hole.msh:");
        SimplicialComplexIO io;
        auto                mesh =
            io.read(fmt::format("{}cylinder_hole.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }

    SECTION("simple_axle.msh")
    {
        fmt::println("simple_axle.msh:");
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}simple_axle.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }

    SECTION("wheel_axle.msh")
    {
        fmt::println("wheel_axle.msh:");
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}wheel_axle.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }

    SECTION("bunny0.msh")
    {
        fmt::println("bunny0.msh:");
        SimplicialComplexIO io;
        auto mesh = io.read(fmt::format("{}bunny0.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }
}
