#include "hip/hip_runtime.h"
#include <muda/ext/eigen/eigen_core_cxx20.h>  // to use Eigen in CUDA
#include <app/test_common.h>
#include <app/asset_dir.h>
#include <uipc/common/type_define.h>
#include <Eigen/Geometry>
#include <muda/buffer/device_buffer.h>
#include <muda/cub/device/device_reduce.h>
#include <muda/cub/device/device_radix_sort.h>
#include <uipc/geometry.h>
#include <uipc/common/enumerate.h>
#include <muda/atomic.h>
#include <muda/cub/device/device_scan.h>

namespace uipc::test::backend::cuda
{
namespace detail
{
    MUDA_DEVICE MUDA_INLINE int common_upper_bits(const unsigned int lhs,
                                                  const unsigned int rhs) noexcept
    {
        return ::__clz(lhs ^ rhs);
    }
    MUDA_DEVICE MUDA_INLINE int common_upper_bits(const unsigned long long int lhs,
                                                  const unsigned long long int rhs) noexcept
    {
        return ::__clzll(lhs ^ rhs);
    }

    MUDA_GENERIC MUDA_INLINE std::uint32_t expand_bits(std::uint32_t v) noexcept
    {
        v = (v * 0x00010001u) & 0xFF0000FFu;
        v = (v * 0x00000101u) & 0x0F00F00Fu;
        v = (v * 0x00000011u) & 0xC30C30C3u;
        v = (v * 0x00000005u) & 0x49249249u;
        return v;
    }

    MUDA_GENERIC MUDA_INLINE std::uint32_t morton_code(Vector3 xyz) noexcept
    {
        xyz = xyz.cwiseMin(1.0).cwiseMax(0.0);
        const std::uint32_t xx =
            expand_bits(static_cast<std::uint32_t>(xyz.x() * 1024.0));
        const std::uint32_t yy =
            expand_bits(static_cast<std::uint32_t>(xyz.y() * 1024.0));
        const std::uint32_t zz =
            expand_bits(static_cast<std::uint32_t>(xyz.z() * 1024.0));
        return xx * 4 + yy * 2 + zz;
    }

    struct LBVHNode
    {
        uint32_t parent_idx = 0xFFFFFFFF;  // parent node
        uint32_t left_idx   = 0xFFFFFFFF;  // index of left  child node
        uint32_t right_idx  = 0xFFFFFFFF;  // index of right child node
        uint32_t object_idx = 0xFFFFFFFF;  // == 0xFFFFFFFF if internal node.
    };

    struct LBVHMortonIndex
    {
        MUDA_GENERIC LBVHMortonIndex(uint32_t m, uint32_t idx) noexcept
        {
            morton = m;
            morton <<= 32;
            morton |= idx;
        }

        MUDA_GENERIC LBVHMortonIndex() noexcept = default;

        uint64_t morton = ~0ull;
        // uint32_t idx;
    };

    MUDA_GENERIC bool operator==(const LBVHMortonIndex& lhs, const LBVHMortonIndex& rhs) noexcept
    {
        return lhs.morton == rhs.morton;
        //&&lhs.idx == rhs.idx;
    }

    MUDA_DEVICE uint2 determine_range(muda::Dense1D<LBVHMortonIndex> node_code,
                                      const uint32_t                 num_leaves,
                                      uint32_t                       idx)
    {
        if(idx == 0)
        {
            return make_uint2(0, num_leaves - 1);
        }

        // determine direction of the range
        const auto self_code = node_code(idx);
        const int  L_delta =
            common_upper_bits(self_code.morton, node_code(idx - 1).morton);
        const int R_delta =
            common_upper_bits(self_code.morton, node_code(idx + 1).morton);
        const int d = (R_delta > L_delta) ? 1 : -1;

        // Compute upper bound for the length of the range

        const int delta_min = thrust::min(L_delta, R_delta);
        int       l_max     = 2;
        int       delta     = -1;
        int       i_tmp     = idx + d * l_max;
        if(0 <= i_tmp && i_tmp < num_leaves)
        {
            delta = common_upper_bits(self_code.morton, node_code(i_tmp).morton);
        }
        while(delta > delta_min)
        {
            l_max <<= 1;
            i_tmp = idx + d * l_max;
            delta = -1;
            if(0 <= i_tmp && i_tmp < num_leaves)
            {
                delta = common_upper_bits(self_code.morton, node_code(i_tmp).morton);
            }
        }

        // Find the other end by binary search
        int l = 0;
        int t = l_max >> 1;
        while(t > 0)
        {
            i_tmp = idx + (l + t) * d;
            delta = -1;
            if(0 <= i_tmp && i_tmp < num_leaves)
            {
                delta = common_upper_bits(self_code.morton, node_code(i_tmp).morton);
            }
            if(delta > delta_min)
            {
                l += t;
            }
            t >>= 1;
        }
        uint32_t jdx = idx + l * d;
        if(d < 0)
        {
            thrust::swap(idx, jdx);  // make it sure that idx < jdx
        }
        return make_uint2(idx, jdx);
    }


    MUDA_DEVICE uint32_t find_split(muda::Dense1D<LBVHMortonIndex> node_code,
                                    const uint32_t                 num_leaves,
                                    const uint32_t                 first,
                                    const uint32_t last) noexcept
    {
        const auto first_code = node_code(first);
        const auto last_code  = node_code(last);
        if(first_code == last_code)
        {
            return (first + last) >> 1;
        }
        const int delta_node = common_upper_bits(first_code.morton, last_code.morton);

        // binary search...
        int split  = first;
        int stride = last - first;
        do
        {
            stride           = (stride + 1) >> 1;
            const int middle = split + stride;
            if(middle < last)
            {
                const int delta =
                    common_upper_bits(first_code.morton, node_code(middle).morton);
                if(delta > delta_node)
                {
                    split = middle;
                }
            }
        } while(stride > 1);

        return split;
    }
}  // namespace detail

class LBVH;

template <bool IsConst>
class LBVHViewerT : muda::ViewerBase<IsConst>
{
    MUDA_VIEWER_COMMON_NAME(LBVHViewerT);

    using Base = muda::ViewerBase<IsConst>;
    template <typename U>
    using auto_const_t = typename Base::template auto_const_t<U>;

    friend class LBVH;
    using Node = detail::LBVHNode;

  public:
    using ConstViewer    = LBVHViewerT<true>;
    using NonConstViewer = LBVHViewerT<false>;
    using ThisViewer     = LBVHViewerT<IsConst>;
    using AABB           = Eigen::AlignedBox<Float, 3>;

    struct DefaultQueryCallback
    {
        MUDA_GENERIC void operator()(uint32_t obj_idx) const noexcept {}
    };

    MUDA_GENERIC LBVHViewerT(const uint32_t      num_nodes,
                             const uint32_t      num_objects,
                             auto_const_t<Node>* nodes,
                             auto_const_t<AABB>* aabbs)
        : m_num_nodes(num_nodes)
        , m_num_objects(num_objects)
        , m_nodes(nodes)
        , m_aabbs(aabbs)
    {
        MUDA_KERNEL_ASSERT(m_nodes && m_aabbs,
                           "BVHViewerBase[%s:%s]: nullptr is passed,"
                           "nodes=%p,"
                           "aabbs=%p,"
                           "objects=%p\n",
                           this->name(),
                           this->kernel_name(),
                           m_nodes,
                           m_aabbs);
    }

    MUDA_GENERIC auto as_const() const noexcept
    {
        return ConstViewer{m_num_nodes, m_num_objects, m_nodes, m_aabbs};
    }

    MUDA_GENERIC operator ConstViewer() const noexcept { return as_const(); }

    MUDA_GENERIC auto num_nodes() const noexcept { return m_num_nodes; }
    MUDA_GENERIC auto num_objects() const noexcept { return m_num_objects; }

    /**
     * @brief query AABBs that intersect with the given point q.
     * 
     * @param q query point
     * @param callback callback function that is called when an AABB is found (may be called multiple times)
     * 
     * @return the number of found AABBs
     */
    template <uint32_t StackNum = 64, std::invocable<uint32_t> CallbackF = DefaultQueryCallback>
    MUDA_GENERIC uint32_t query(const Vector3& q,
                                CallbackF callback = DefaultQueryCallback{}) const noexcept
    {
        uint32_t stack[StackNum];
        return this->query(q, stack, StackNum, callback);
    }

    template <std::invocable<uint32_t> CallbackF = DefaultQueryCallback>
    MUDA_GENERIC uint32_t query(const Vector3& q,
                                uint32_t*      stack,
                                uint32_t       stack_num,
                                CallbackF callback = DefaultQueryCallback{}) const noexcept
    {
        return this->query(
            q,
            [](const AABB& aabb, const Vector3& q) { return aabb.contains(q); },
            stack,
            stack_num,
            callback);
    }

    /**
     * @brief query AABBs that intersect with the given AABB q.
     * 
     * @param q query AABB
     * @param callback callback function that is called when an AABB is found (may be called multiple times)
     * 
     * @return the number of found AABBs
     */
    template <uint32_t StackNum = 64, std::invocable<uint32_t> CallbackF = DefaultQueryCallback>
    MUDA_GENERIC uint32_t query(const AABB& aabb,
                                CallbackF callback = DefaultQueryCallback{}) const noexcept
    {
        uint32_t stack[StackNum];
        return this->query(aabb, stack, StackNum, callback);
    }

    template <std::invocable<uint32_t> CallbackF = DefaultQueryCallback>
    MUDA_GENERIC uint32_t query(const AABB& aabb,
                                uint32_t*   stack,
                                uint32_t    stack_num,
                                CallbackF callback = DefaultQueryCallback{}) const noexcept
    {
        return this->query(
            aabb, [](const AABB& A, const AABB& B) { return A.intersects(B); }, stack, stack_num, callback);
    }


    /**
     * @brief check if the stack overflow occurs during the query.
     */
    bool stack_overflow() const noexcept { return m_stack_overflow; }

  private:
    uint32_t m_num_nodes;    // (# of internal node) + (# of leaves), 2N+1
    uint32_t m_num_objects;  // (# of leaves), the same as the number of objects

    auto_const_t<Node>* m_nodes;
    auto_const_t<AABB>* m_aabbs;

    MUDA_INLINE MUDA_GENERIC void check_index(const uint32_t idx) const noexcept
    {
        MUDA_KERNEL_ASSERT(idx < m_num_objects,
                           "BVHViewer[%s:%s]: index out of range, idx=%u, num_objects=%u",
                           this->name(),
                           this->kernel_name(),
                           idx,
                           m_num_objects);
    }

    MUDA_INLINE MUDA_GENERIC void stack_overflow_warning(uint32_t num_found,
                                                         uint32_t stack_num) const noexcept
    {
        if constexpr(muda::RUNTIME_CHECK_ON)
        {
            MUDA_KERNEL_WARN_WITH_LOCATION("BVHViewer[%s:%s]: stack overflow, num_found=%u, stack_num=%u, the return value may be invalid, enlarge the stack please.",
                                           this->name(),
                                           this->kernel_name(),
                                           num_found,
                                           stack_num);
        }
    }

    mutable bool m_stack_overflow = false;

    template <typename QueryType, typename IntersectF, typename CallbackF>
    MUDA_GENERIC uint32_t query(const QueryType& Q,
                                IntersectF       Intersect,
                                uint32_t*        stack,
                                uint32_t         stack_num,
                                CallbackF        Callback) const noexcept
    {
        uint32_t* stack_ptr = stack;
        uint32_t* stack_end = stack + stack_num;
        *stack_ptr++        = 0;  // root node is always 0

        if(m_num_objects == 1)
        {
            if(Intersect(m_aabbs[0], Q))
            {
                Callback(0);
                return 1;
            }
        }

        uint32_t num_found = 0;
        do
        {
            const uint32_t node  = *--stack_ptr;
            const uint32_t L_idx = m_nodes[node].left_idx;
            const uint32_t R_idx = m_nodes[node].right_idx;

            if(Intersect(m_aabbs[L_idx], Q))
            {
                const auto obj_idx = m_nodes[L_idx].object_idx;
                if(obj_idx != 0xFFFFFFFF)
                {
                    Callback(obj_idx);
                    ++num_found;
                }
                else  // the node is not a leaf.
                {
                    *stack_ptr++ = L_idx;
                }
            }
            if(Intersect(m_aabbs[R_idx], Q))
            {
                const auto obj_idx = m_nodes[R_idx].object_idx;
                if(obj_idx != 0xFFFFFFFF)
                {
                    Callback(obj_idx);
                    ++num_found;
                }
                else  // the node is not a leaf.
                {
                    *stack_ptr++ = R_idx;
                }
            }
            if(stack_ptr >= stack_end)
            {
                stack_overflow_warning(num_found, stack_num);
                break;
            }
        } while(stack < stack_ptr);
        return num_found;
    }
};

using LBVHViewer  = LBVHViewerT<false>;
using CLBVHViewer = LBVHViewerT<true>;

class LBVH
{
    using Node = detail::LBVHNode;

  public:
    using AABB        = Eigen::AlignedBox<Float, 3>;
    using MortonIndex = detail::LBVHMortonIndex;

    // now we only use default stream
    void build(muda::CBufferView<AABB> aabbs, muda::Stream& s = muda::Stream::Default())
    {
        using namespace muda;

        if(aabbs.size() == 0)
            return;

        const uint32_t num_objects        = aabbs.size();
        const uint32_t num_internal_nodes = num_objects - 1;
        const uint32_t leaf_start         = num_internal_nodes;
        const uint32_t num_nodes          = num_objects * 2 - 1;

        AABB default_aabb;
        resize(s, m_aabbs, num_nodes);
        BufferLaunch(s).fill(m_aabbs.view(), default_aabb);

        resize(s, m_sorted_mortons, num_objects);
        resize(s, m_sorted_mortons, num_objects);

        resize(s, m_indices, num_objects);
        resize(s, m_new_to_old, num_objects);

        resize(s, m_mortons, num_objects);
        resize(s, m_morton_idx, num_objects);

        Node default_node;
        m_nodes.resize(num_nodes, default_node);
        resize(s, m_nodes, num_nodes);

        resize(s, m_flags, num_objects);
        BufferLaunch(s).fill(m_flags.view(), 0);

        // 1) setup aabbs
        //BufferLaunch(s).copy(aabbs, aabbs);

        // 2) get max aabb
        DeviceReduce(s).Reduce(
            aabbs.data(),
            m_max_aabb.data(),
            aabbs.size(),
            [] HIPCUB_RUNTIME_FUNCTION(const AABB& a, const AABB& b)
            { return a.merged(b); },
            default_aabb);

        // 3) calculate morton code
        on(s)
            .next<ParallelFor>()
            .kernel_name("LBVH::MortonCode")
            .apply(num_objects,
                   [max_aabb = m_max_aabb.viewer().name("max_aabb"),
                    aabbs    = aabbs.viewer().name("filled_aabbs"),
                    mortons = m_mortons.viewer().name("mortons")] __device__(int i) mutable
                   {
                       Vector3 p = aabbs(i).center();
                       p -= max_aabb->min();
                       p.array() /= max_aabb->sizes().array();
                       mortons(i) = detail::morton_code(p);
                   });

        // 4) sort morton code
        on(s)
            .next<ParallelFor>()
            .kernel_name("LBVH::Iota")
            .apply(m_indices.size(),
                   [indices = m_indices.viewer()] __device__(int i) mutable
                   { indices(i) = i; });

        // 5) sort morton code
        DeviceRadixSort(s).SortPairs(m_mortons.data(),
                                     m_sorted_mortons.data(),
                                     m_indices.data(),
                                     m_new_to_old.data(),
                                     num_objects);

        // 6) expand morton code to 64bit, the last 32bit is the index
        on(s)
            .next<ParallelFor>()
            .kernel_name("LBVH::ExpandMorton")
            .apply(m_mortons.size(),
                   [morton64s = m_morton_idx.viewer().name("morton64s"),
                    mortons   = m_sorted_mortons.viewer().name("mortons"),
                    indices = m_new_to_old.viewer().name("indices")] __device__(int i) mutable
                   {
                       MortonIndex morton{mortons(i), indices(i)};
                       morton64s(i) = morton;
                   });

        // 7) setup leaf nodes
        auto leaf_aabbs = m_aabbs.view(leaf_start);  // offset = leaf_start
        auto leaf_nodes = m_nodes.view(leaf_start);  // offset = leaf_start
        on(s)
            .next<ParallelFor>()
            .kernel_name("LBVH::SetupLeafNodes")
            .apply(num_objects,
                   [leaf_nodes = leaf_nodes.viewer().name("leaf_nodes"),
                    indices    = m_new_to_old.viewer().name("indices"),
                    aabbs      = aabbs.viewer().name("aabbs"),
                    sorted_aabbs = leaf_aabbs.viewer().name("sorted_aabbs")] __device__(int i) mutable
                   {
                       Node node;
                       node.parent_idx = 0xFFFFFFFF;
                       node.left_idx   = 0xFFFFFFFF;
                       node.right_idx  = 0xFFFFFFFF;
                       node.object_idx = indices(i);
                       leaf_nodes(i)   = node;
                       sorted_aabbs(i) = aabbs(node.object_idx);
                   });

        // 8) construct internal nodes
        on(s)
            .next<ParallelFor>()
            .kernel_name("LBVH::ConstructInternalNodes")
            .apply(num_internal_nodes,
                   [nodes     = m_nodes.viewer().name("nodes"),
                    morton64s = m_morton_idx.viewer().name("morton64s"),
                    num_objects] __device__(int idx) mutable
                   {
                       nodes(idx).object_idx = 0xFFFFFFFF;  //  internal nodes

                       const uint2 ij =
                           detail::determine_range(morton64s, num_objects, idx);
                       const int gamma =
                           detail::find_split(morton64s, num_objects, ij.x, ij.y);

                       nodes(idx).left_idx  = gamma;
                       nodes(idx).right_idx = gamma + 1;
                       if(thrust::min(ij.x, ij.y) == gamma)
                       {
                           nodes(idx).left_idx += num_objects - 1;
                       }
                       if(thrust::max(ij.x, ij.y) == gamma + 1)
                       {
                           nodes(idx).right_idx += num_objects - 1;
                       }
                       nodes(nodes(idx).left_idx).parent_idx  = idx;
                       nodes(nodes(idx).right_idx).parent_idx = idx;
                   });

        // 9) calculate the AABB of internal nodes
        auto internal_aabbs = m_aabbs.view(0, num_internal_nodes);
        on(s)
            .next<ParallelFor>()
            .kernel_name("LBVH::CalculateInternalAABB")
            .apply(num_objects,
                   [nodes = m_nodes.cviewer().name("nodes"),
                    aabbs = m_aabbs.viewer().name("aabbs"),
                    flags = m_flags.viewer().name("flags"),
                    leaf_start] __device__(int I) mutable
                   {
                       auto leaf_idx = I + leaf_start;
                       auto parent   = nodes(leaf_idx).parent_idx;

                       while(parent != 0xFFFFFFFF)  // means idx == 0
                       {
                           const int old = muda::atomic_cas(&flags(parent), 0, 1);
                           if(old == 0)
                           {
                               // this is the first thread entered here.
                               // wait the other thread from the other child node.
                               return;
                           }
                           MUDA_KERNEL_ASSERT(old == 1, "old=%d", old);
                           // here, the flag has already been 1. it means that this
                           // thread is the 2nd thread. merge AABB of both childlen.

                           const auto lidx = nodes(parent).left_idx;
                           const auto ridx = nodes(parent).right_idx;
                           const auto lbox = aabbs(lidx);
                           const auto rbox = aabbs(ridx);
                           aabbs(parent)   = lbox.merged(rbox);

                           // look the next parent...
                           parent = nodes(parent).parent_idx;
                       }
                   });
    }

    auto viewer() noexcept
    {
        return LBVHViewer{(uint32_t)m_nodes.size(),
                          (uint32_t)m_mortons.size(),
                          m_nodes.data(),
                          m_aabbs.data()};
    }

    auto viewer() const noexcept
    {
        return CLBVHViewer{(uint32_t)m_nodes.size(),
                           (uint32_t)m_mortons.size(),
                           m_nodes.data(),
                           m_aabbs.data()};
    }
    template <typename T>
    void resize(muda::Stream& s, muda::DeviceBuffer<T>& V, size_t size)
    {
        if(size > V.capacity())
            muda::BufferLaunch(s).reserve(V, size * m_resize_factor);
        muda::BufferLaunch(s).resize(V, size);
    }

    muda::DeviceBuffer<AABB>        m_aabbs;
    muda::DeviceBuffer<uint32_t>    m_mortons;
    muda::DeviceBuffer<uint32_t>    m_sorted_mortons;
    muda::DeviceBuffer<uint32_t>    m_indices;
    muda::DeviceBuffer<uint32_t>    m_new_to_old;
    muda::DeviceBuffer<MortonIndex> m_morton_idx;
    muda::DeviceBuffer<int>         m_flags;
    muda::DeviceBuffer<Node>        m_nodes;
    muda::DeviceVar<AABB>           m_max_aabb;

    Float m_resize_factor = 1.5;
};
}  // namespace uipc::test::backend::cuda


using namespace uipc;
using namespace uipc::test::backend::cuda;
using namespace uipc::geometry;
using namespace muda;


std::vector<Vector2i> lbvh_cp(span<const LBVH::AABB> aabbs)
{
    DeviceBuffer<LBVH::AABB> d_aabbs(aabbs.size());
    d_aabbs.view().copy_from(aabbs.data());

    LBVH lbvh;
    lbvh.build(d_aabbs);

    DeviceBuffer<IndexT> counts(aabbs.size() + 1ull);
    DeviceBuffer<IndexT> offsets(aabbs.size() + 1ull);

    for(int i = 0; i < aabbs.size(); ++i)
    {
        auto aabb = aabbs[i];
        std::cout << "[" << aabb.min().transpose() << "],"
                  << "[" << aabb.max().transpose() << "]" << std::endl;
    }

    ParallelFor()
        .kernel_name("LBVHTest::Query")
        .apply(aabbs.size(),
               [lbvh  = lbvh.viewer().name("lbvh"),
                aabbs = d_aabbs.viewer().name("aabbs"),
                counts = counts.viewer().name("counts")] __device__(int i) mutable
               {
                   auto N = aabbs.total_size();

                   auto aabb  = aabbs(i);
                   auto count = 0;
                   lbvh.query(aabb,
                              [&](uint32_t id)
                              {
                                  if(id > i)
                                      count++;
                              });
                   counts(i) = count;

                   if(i == 0)
                   {
                       counts(N) = 0;
                   }
               });

    DeviceScan().ExclusiveSum(counts.data(), offsets.data(), counts.size());
    IndexT total;
    offsets.view(aabbs.size()).copy_to(&total);

    DeviceBuffer<Vector2i> pairs(total);


    ParallelFor()
        .kernel_name("LBVHTest::Query")
        .apply(aabbs.size(),
               [lbvh    = lbvh.viewer().name("lbvh"),
                aabbs   = d_aabbs.viewer().name("aabbs"),
                counts  = counts.viewer().name("counts"),
                offsets = offsets.viewer().name("offsets"),
                pairs = pairs.viewer().name("pairs")] __device__(int i) mutable
               {
                   auto N = aabbs.total_size();

                   auto aabb   = aabbs(i);
                   auto count  = counts(i);
                   auto offset = offsets(i);

                   auto pair = pairs.subview(offset, count);
                   int  j    = 0;
                   lbvh.query(aabb,
                              [&](uint32_t id)
                              {
                                  if(id > i)
                                      pair(j++) = Vector2i(i, id);
                              });
               });

    std::vector<Vector2i> pairs_host;
    pairs.copy_to(pairs_host);

    //std::vector<LBVH::AABB> aabbs_host;
    //lbvh.m_aabbs.copy_to(aabbs_host);
    //for(auto&& [i, aabb] : enumerate(aabbs_host))
    //{
    //    std::cout << "[" << aabb.min().transpose() << "],"
    //              << "[" << aabb.max().transpose() << "]" << std::endl;
    //}

    //std::vector<detail::LBVHNode> nodes_host;
    //lbvh.m_nodes.copy_to(nodes_host);
    //for(auto&& [i, node] : enumerate(nodes_host))
    //{
    //    std::cout << "node=" << i << "[" << aabbs_host[i].min().transpose() << "],"
    //              << "[" << aabbs_host[i].max().transpose() << "]"
    //              << ", parent=" << node.parent_idx
    //              << ", left=" << node.left_idx << ", right=" << node.right_idx
    //              << ", obj=" << node.object_idx << std::endl;
    //}

    return pairs_host;
}

std::vector<Vector2i> brute_froce_cp(span<const LBVH::AABB> aabbs)
{
    std::vector<Vector2i> pairs;
    for(auto&& [i, aabb0] : enumerate(aabbs))
    {
        for(int j = i + 1; j < aabbs.size(); ++j)
        {
            auto aabb1 = aabbs[j];
            if(aabb1.intersects(aabb0))
            {
                pairs.push_back(Vector2i(i, j));
            }
        }
    }
    return pairs;
}


void brute_froce_gpu(span<const LBVH::AABB> aabbs)
{
    DeviceBuffer<LBVH::AABB> d_aabbs(aabbs.size());
    d_aabbs.view().copy_from(aabbs.data());

    ParallelFor()
        .kernel_name("BruteForce::Query")
        .apply(aabbs.size(),
               [aabbs = d_aabbs.viewer().name("aabbs")] __device__(int i) mutable
               {
                   auto N = aabbs.total_size();

                   auto aabb0 = aabbs(i);
                   for(int j = i + 1; j < N; ++j)
                   {
                       auto aabb1 = aabbs(j);
                       if(aabb1.intersects(aabb0))
                       {
                           printf("i=%d, j=%d\n", i, j);
                       }
                   }
               });
}

SimplicialComplex tet()
{
    std::vector           Vs = {Vector3{0.0, 0.0, 0.0},
                                Vector3{1.0, 0.0, 0.0},
                                Vector3{0.0, 1.0, 0.0},
                                Vector3{0.0, 0.0, 1.0}};
    std::vector<Vector4i> Ts = {Vector4i{0, 1, 2, 3}};

    return tetmesh(Vs, Ts);
}


void lbvh_test(const SimplicialComplex& mesh)
{
    auto pos_view = mesh.positions().view();
    auto tri_view = mesh.triangles().topo().view();

    //tri_view = tri_view.subspan(0, 5);

    std::vector<LBVH::AABB> aabbs(tri_view.size());
    for(auto&& [i, tri] : enumerate(tri_view))
    {
        auto p0 = pos_view[tri[0]];
        auto p1 = pos_view[tri[1]];
        auto p2 = pos_view[tri[2]];
        aabbs[i].extend(p0).extend(p1).extend(p2);
        aabbs[i].max() += Vector3{0.1, 0.1, 0.1};
        aabbs[i].min() -= Vector3{0.1, 0.1, 0.1};
    }


    auto lbvh_pairs = lbvh_cp(aabbs);


    auto bf_pairs = brute_froce_cp(aabbs);

    // brute_froce_gpu(aabbs);


    auto compare = [](const Vector2i& lhs, const Vector2i& rhs)
    { return lhs[0] < rhs[0] || (lhs[0] == rhs[0] && lhs[1] < rhs[1]); };

    std::ranges::sort(lbvh_pairs, compare);
    std::ranges::sort(bf_pairs, compare);


    auto check_unique = [](auto begin, auto end)
    {
        for(auto it = begin; it != end; ++it)
        {
            if(it + 1 != end && *it == *(it + 1))
            {
                return false;
            }
        }
        return true;
    };

    CHECK(check_unique(lbvh_pairs.begin(), lbvh_pairs.end()));

    bool same = lbvh_pairs == bf_pairs;
    CHECK(same);

    if(!same)
    {
        std::list<Vector2i> diff;

        std::set_difference(lbvh_pairs.begin(),
                            lbvh_pairs.end(),
                            bf_pairs.begin(),
                            bf_pairs.end(),
                            std::back_inserter(diff),
                            compare);

        std::cout << "lbvh_pairs.size()=" << lbvh_pairs.size() << std::endl;
        std::cout << "bf_pairs.size()=" << bf_pairs.size() << std::endl;
        std::cout << "diff:" << std::endl;
        for(auto&& [i, p] : enumerate(diff))
        {
            std::cout << "i=" << i << ", p=[" << p[0] << ", " << p[1] << "]" << std::endl;
        }
    }
}


TEST_CASE("lbvh", "[muda]")
{

    SECTION("tet")
    {
        lbvh_test(tet());
    }

    SECTION("cube.obj")
    {
        SimplicialComplexIO io;
        auto mesh = io.read_obj(fmt::format("{}cube.obj", AssetDir::trimesh_path()));
        lbvh_test(mesh);
    }

    SECTION("cube.msh")
    {
        SimplicialComplexIO io;
        auto mesh = io.read_msh(fmt::format("{}cube.msh", AssetDir::tetmesh_path()));
        lbvh_test(mesh);
    }
}
