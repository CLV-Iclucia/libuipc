#include "hip/hip_runtime.h"
#include <global_vertex_manager.h>
#include <uipc/common/enumerate.h>
#include <muda/cub/device/device_reduce.h>

/*************************************************************************************************
* Core Implementation
*************************************************************************************************/
namespace uipc::backend::cuda
{
void GlobalVertexManager::Impl::build_vertex_info()
{
    auto          N = vertex_registers.size();
    vector<SizeT> counts(N);
    vector<SizeT> offsets(N);

    for(auto&& [i, R] : enumerate(vertex_registers))
    {
        VertexCountInfo info;
        R.m_report_vertex_count(info);
        // get count back
        counts[i] = info.count();
    }

    std::exclusive_scan(counts.begin(), counts.end(), offsets.begin(), 0);

    auto total_count = offsets.back() + counts.back();

    // resize the global coindex buffer
    coindex.resize(total_count);
    positions.resize(total_count);
    displacements.resize(total_count, Vector3::Zero());

    // create the subviews for each attribute_reporter
    for(auto&& [i, R] : enumerate(vertex_registers))
    {
        VertexAttributes attributes;
        auto             offset = offsets[i];
        auto             count  = counts[i];

        attributes.m_coindex   = coindex.view(offset, count);
        attributes.m_positions = positions.view(offset, count);
        R.m_report_vertex_attributes(attributes);
    }
}

Float GlobalVertexManager::Impl::compute_max_displacement()
{
    muda::DeviceReduce().Reduce((Float*)displacements.data(),
                                max_disp.data(),
                                displacements.size() * 3,
                                [] HIPCUB_RUNTIME_FUNCTION(const Float& L, const Float& R)
                                {
                                    auto absL = std::abs(L);
                                    auto absR = std::abs(R);
                                    return absL > absR ? absL : absR;
                                },
                                0.0);
    return max_disp;
}

AABB GlobalVertexManager::Impl::compute_vertex_bounding_box()
{
    Float max_float = std::numeric_limits<Float>::max();
    muda::DeviceReduce()
        .Reduce(
            positions.data(),
            min_pos.data(),
            positions.size(),
            [] HIPCUB_RUNTIME_FUNCTION(const Vector3& L, const Vector3& R) -> Vector3
            { return L.cwiseMin(R); },
            Vector3{max_float, max_float, max_float})
        .Reduce(
            positions.data(),
            max_pos.data(),
            positions.size(),
            [] HIPCUB_RUNTIME_FUNCTION(const Vector3& L, const Vector3& R) -> Vector3
            { return L.cwiseMax(R); },
            Vector3{-max_float, -max_float, -max_float});

    Vector3 min_pos_host, max_pos_host;
    min_pos_host = min_pos;
    max_pos_host = max_pos;
    return AABB{min_pos_host, max_pos_host};
}
GlobalVertexManager::VertexRegister::VertexRegister(
    std::string_view                         name,
    std::function<void(VertexCountInfo&)>&&  report_vertex_count,
    std::function<void(VertexAttributes&)>&& report_vertex_attributes,
    std::function<void(VertexDisplacement&)>&& report_vertex_displacement) noexcept
    : m_name(name)
    , m_report_vertex_count(std::move(report_vertex_count))
    , m_report_vertex_attributes(std::move(report_vertex_attributes))
    , m_report_vertex_displacement(std::move(report_vertex_displacement))
{
}
}  // namespace uipc::backend::cuda


/*************************************************************************************************
* API Implementation
*************************************************************************************************/
namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(GlobalVertexManager);

void GlobalVertexManager::VertexCountInfo::count(SizeT count) noexcept
{
    m_count = count;
}

SizeT GlobalVertexManager::VertexCountInfo::count() const noexcept
{
    return m_count;
}

muda::BufferView<IndexT> GlobalVertexManager::VertexAttributes::coindex() const noexcept
{
    return m_coindex;
}

muda::BufferView<Vector3> GlobalVertexManager::VertexAttributes::positions() const noexcept
{
    return m_positions;
}

muda::BufferView<Vector3> GlobalVertexManager::VertexDisplacement::displacements() const noexcept
{
    return m_displacements;
}

void GlobalVertexManager::build_vertex_info()
{
    m_impl.build_vertex_info();
}

void GlobalVertexManager::on_update(std::string_view name,
                                    std::function<void(VertexCountInfo&)>&& report_vertex_count,
                                    std::function<void(VertexAttributes&)>&& report_vertex_attributes,
                                    std::function<void(VertexDisplacement&)>&& report_vertex_displacement)
{
    check_state(SimEngineState::BuildSystems, "on_update()");
    m_impl.on_update(VertexRegister{name,
                                    std::move(report_vertex_count),
                                    std::move(report_vertex_attributes),
                                    std::move(report_vertex_displacement)});
}

void GlobalVertexManager::Impl::on_update(VertexRegister&& vertex_register)
{
    vertex_registers.push_back(std::move(vertex_register));
}

muda::CBufferView<IndexT> GlobalVertexManager::coindex() const noexcept
{
    return m_impl.coindex;
}

muda::CBufferView<Vector3> GlobalVertexManager::positions() const noexcept
{
    return m_impl.positions;
}

muda::CBufferView<Vector3> GlobalVertexManager::displacements() const noexcept
{
    return m_impl.displacements;
}

Float GlobalVertexManager::compute_max_displacement()
{
    return m_impl.compute_max_displacement();
}

AABB GlobalVertexManager::compute_vertex_bounding_box()
{
    return m_impl.compute_vertex_bounding_box();
}
}  // namespace uipc::backend::cuda