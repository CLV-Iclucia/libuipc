#include "hip/hip_runtime.h"
#include <global_vertex_manager.h>
#include <uipc/common/enumerate.h>
#include <muda/cub/device/device_reduce.h>

/*************************************************************************************************
* Core Implementation
*************************************************************************************************/
namespace uipc::backend::cuda
{
void GlobalVertexManager::Impl::build_vertex_info()
{
    auto          N = vertex_count_reporter.size();
    vector<SizeT> counts(N);
    vector<SizeT> offsets(N);

    for(auto&& [i, reporter] : enumerate(vertex_count_reporter))
    {
        VertexCountInfo info;
        reporter(info);
        // get count back
        counts[i] = info.count();
    }

    std::exclusive_scan(counts.begin(), counts.end(), offsets.begin(), 0);

    auto total_count = offsets.back() + counts.back();

    // resize the global coindex buffer
    coindex.resize(total_count);
    positions.resize(total_count);
    displacements.resize(total_count);

    // create the subviews for each attribute_reporter
    for(auto&& [i, attribute_reporter] : enumerate(vertex_attribute_reporter))
    {
        VertexAttributes attributes;
        auto             offset = offsets[i];
        auto             count  = counts[i];

        attributes.m_coindex   = coindex.view(offset, count);
        attributes.m_positions = positions.view(offset, count);
        attribute_reporter(attributes);
    }
}

Float GlobalVertexManager::Impl::compute_max_displacement()
{
    muda::DeviceReduce().Reduce((Float*)displacements.data(),
                                max_disp.data(),
                                displacements.size() * 3,
                                [] HIPCUB_RUNTIME_FUNCTION(const Float& L, const Float& R)
                                {
                                    auto absL = std::abs(L);
                                    auto absR = std::abs(R);
                                    return absL > absR ? absL : absR;
                                },
                                0.0);
    return max_disp;
}

AABB GlobalVertexManager::Impl::compute_vertex_bounding_box()
{
    Float max_float = std::numeric_limits<Float>::max();
    muda::DeviceReduce()
        .Reduce(
            positions.data(),
            min_pos.data(),
            positions.size(),
            [] HIPCUB_RUNTIME_FUNCTION(const Vector3& L, const Vector3& R) -> Vector3
            { return L.cwiseMin(R); },
            Vector3{max_float, max_float, max_float})
        .Reduce(
            positions.data(),
            max_pos.data(),
            positions.size(),
            [] HIPCUB_RUNTIME_FUNCTION(const Vector3& L, const Vector3& R) -> Vector3
            { return L.cwiseMax(R); },
            Vector3{-max_float, -max_float, -max_float});

    Vector3 min_pos_host, max_pos_host;
    min_pos_host = min_pos;
    max_pos_host = max_pos;
    return AABB{min_pos_host, max_pos_host};
}
}  // namespace uipc::backend::cuda


/*************************************************************************************************
* API Implementation
*************************************************************************************************/
namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(GlobalVertexManager);

void GlobalVertexManager::VertexCountInfo::count(SizeT count) noexcept
{
    m_count = count;
}

SizeT GlobalVertexManager::VertexCountInfo::count() const noexcept
{
    return m_count;
}

muda::BufferView<IndexT> GlobalVertexManager::VertexAttributes::coindex() const noexcept
{
    return m_coindex;
}

muda::BufferView<Vector3> GlobalVertexManager::VertexAttributes::positions() const noexcept
{
    return m_positions;
}

muda::BufferView<Vector3> GlobalVertexManager::VertexDisplacement::displacements() const noexcept
{
    return m_displacements;
}

void GlobalVertexManager::build_vertex_info()
{
    m_impl.build_vertex_info();
}

void GlobalVertexManager::on_update(std::function<void(VertexCountInfo&)>&& report_vertex_count,
                                    std::function<void(VertexAttributes&)>&& report_vertex_attributes,
                                    std::function<void(VertexDisplacement&)>&& report_vertex_displacement)
{
    check_state(SimEngineState::BuildSystems, "on_update()");
    m_impl.on_update(std::move(report_vertex_count),
                     std::move(report_vertex_attributes),
                     std::move(report_vertex_displacement));
}

void GlobalVertexManager::Impl::on_update(
    std::function<void(VertexCountInfo&)>&&    report_vertex_count,
    std::function<void(VertexAttributes&)>&&   report_vertex_attributes,
    std::function<void(VertexDisplacement&)>&& report_vertex_displacement)
{
    vertex_count_reporter.emplace_back(std::move(report_vertex_count));
    vertex_attribute_reporter.emplace_back(std::move(report_vertex_attributes));
    vertex_displacement_reporter.emplace_back(std::move(report_vertex_displacement));
}

muda::CBufferView<IndexT> GlobalVertexManager::coindex() const noexcept
{
    return m_impl.coindex;
}

muda::CBufferView<Vector3> GlobalVertexManager::positions() const noexcept
{
    return m_impl.positions;
}

muda::CBufferView<Vector3> GlobalVertexManager::displacements() const noexcept
{
    return m_impl.displacements;
}

Float GlobalVertexManager::compute_max_displacement()
{
    return m_impl.compute_max_displacement();
}

AABB GlobalVertexManager::compute_vertex_bounding_box()
{
    return m_impl.compute_vertex_bounding_box();
}
}  // namespace uipc::backend::cuda