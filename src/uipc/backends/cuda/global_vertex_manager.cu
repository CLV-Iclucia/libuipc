#include "hip/hip_runtime.h"
#include <global_vertex_manager.h>
#include <uipc/common/enumerate.h>
#include <uipc/common/range.h>
#include <muda/cub/device/device_reduce.h>

/*************************************************************************************************
* Core Implementation
*************************************************************************************************/
namespace uipc::backend::cuda
{
void GlobalVertexManager::Impl::build_vertex_info()
{
    vertex_registers.reserve(vertex_registers_buffer.size());
    std::ranges::move(vertex_registers_buffer, std::back_inserter(vertex_registers));

    auto N = vertex_registers.size();
    register_vertex_counts.resize(N);
    register_vertex_offsets.resize(N);

    for(auto&& [i, R] : enumerate(vertex_registers))
    {
        VertexCountInfo info;
        R.m_report_vertex_count(info);
        // get count back
        register_vertex_counts[i] = info.count();
    }

    std::exclusive_scan(register_vertex_counts.begin(),
                        register_vertex_counts.end(),
                        register_vertex_offsets.begin(),
                        0);

    auto total_count = register_vertex_offsets.back() + register_vertex_counts.back();

    // resize the global coindex buffer
    coindex.resize(total_count);
    positions.resize(total_count);
    displacements.resize(total_count, Vector3::Zero());

    // create the subviews for each attribute_reporter
    for(auto&& [i, R] : enumerate(vertex_registers))
    {
        VertexAttributeInfo attributes{this, i};
        R.m_report_vertex_attributes(attributes);
    }
}

Float GlobalVertexManager::Impl::compute_max_displacement()
{
    collect_vertex_displacements();

    muda::DeviceReduce().Reduce((Float*)displacements.data(),
                                max_disp.data(),
                                displacements.size() * 3,
                                [] HIPCUB_RUNTIME_FUNCTION(const Float& L, const Float& R)
                                {
                                    auto absL = std::abs(L);
                                    auto absR = std::abs(R);
                                    return absL > absR ? absL : absR;
                                },
                                0.0);
    return max_disp;
}

AABB GlobalVertexManager::Impl::compute_vertex_bounding_box()
{
    Float max_float = std::numeric_limits<Float>::max();
    muda::DeviceReduce()
        .Reduce(
            positions.data(),
            min_pos.data(),
            positions.size(),
            [] HIPCUB_RUNTIME_FUNCTION(const Vector3& L, const Vector3& R) -> Vector3
            { return L.cwiseMin(R); },
            Vector3{max_float, max_float, max_float})
        .Reduce(
            positions.data(),
            max_pos.data(),
            positions.size(),
            [] HIPCUB_RUNTIME_FUNCTION(const Vector3& L, const Vector3& R) -> Vector3
            { return L.cwiseMax(R); },
            Vector3{-max_float, -max_float, -max_float});

    Vector3 min_pos_host, max_pos_host;
    min_pos_host = min_pos;
    max_pos_host = max_pos;
    return AABB{min_pos_host, max_pos_host};
}
GlobalVertexManager::VertexRegister::VertexRegister(
    std::string_view                            name,
    std::function<void(VertexCountInfo&)>&&     report_vertex_count,
    std::function<void(VertexAttributeInfo&)>&& report_vertex_attributes,
    std::function<void(VertexDisplacementInfo&)>&& report_vertex_displacement) noexcept
    : m_name(name)
    , m_report_vertex_count(std::move(report_vertex_count))
    , m_report_vertex_attributes(std::move(report_vertex_attributes))
    , m_report_vertex_displacement(std::move(report_vertex_displacement))
{
}
}  // namespace uipc::backend::cuda


/*************************************************************************************************
* API Implementation
*************************************************************************************************/
namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(GlobalVertexManager);

void GlobalVertexManager::VertexCountInfo::count(SizeT count) noexcept
{
    m_count = count;
}

SizeT GlobalVertexManager::VertexCountInfo::count() const noexcept
{
    return m_count;
}

GlobalVertexManager::VertexAttributeInfo::VertexAttributeInfo(Impl* impl, SizeT index) noexcept
    : m_impl(impl)
    , m_index(index)
{
}

muda::BufferView<IndexT> GlobalVertexManager::VertexAttributeInfo::coindex() const noexcept
{
    return m_impl->subview(m_impl->coindex, m_index);
}

muda::BufferView<Vector3> GlobalVertexManager::VertexAttributeInfo::positions() const noexcept
{
    return m_impl->subview(m_impl->positions, m_index);
}

GlobalVertexManager::VertexDisplacementInfo::VertexDisplacementInfo(Impl* impl, SizeT index) noexcept
    : m_impl(impl)
    , m_index(index)
{
}

muda::BufferView<Vector3> GlobalVertexManager::VertexDisplacementInfo::displacements() const noexcept
{
    return m_impl->subview(m_impl->displacements, m_index);
}

muda::CBufferView<IndexT> GlobalVertexManager::VertexDisplacementInfo::coindex() const noexcept
{
    return m_impl->subview(m_impl->coindex, m_index);
}

void GlobalVertexManager::build_vertex_info()
{
    m_impl.build_vertex_info();
}

void GlobalVertexManager::on_update(std::string_view name,
                                    std::function<void(VertexCountInfo&)>&& report_vertex_count,
                                    std::function<void(VertexAttributeInfo&)>&& report_vertex_attributes,
                                    std::function<void(VertexDisplacementInfo&)>&& report_vertex_displacement)
{
    check_state(SimEngineState::BuildSystems, "on_update()");
    m_impl.on_update(VertexRegister{name,
                                    std::move(report_vertex_count),
                                    std::move(report_vertex_attributes),
                                    std::move(report_vertex_displacement)});
}

void GlobalVertexManager::Impl::on_update(VertexRegister&& vertex_register)
{
    vertex_registers_buffer.push_back(std::move(vertex_register));
}

muda::CBufferView<IndexT> GlobalVertexManager::coindex() const noexcept
{
    return m_impl.coindex;
}

muda::CBufferView<Vector3> GlobalVertexManager::positions() const noexcept
{
    return m_impl.positions;
}

muda::CBufferView<Vector3> GlobalVertexManager::displacements() const noexcept
{
    return m_impl.displacements;
}

void GlobalVertexManager::Impl::collect_vertex_displacements()
{
    for(auto&& [i, R] : enumerate(vertex_registers))
    {
        VertexDisplacementInfo vd{this, i};
        R.m_report_vertex_displacement(vd);
    }
}

Float GlobalVertexManager::compute_max_displacement()
{
    return m_impl.compute_max_displacement();
}

AABB GlobalVertexManager::compute_vertex_bounding_box()
{
    return m_impl.compute_vertex_bounding_box();
}
}  // namespace uipc::backend::cuda