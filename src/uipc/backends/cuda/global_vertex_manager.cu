#include "hip/hip_runtime.h"
#include <global_vertex_manager.h>
#include <uipc/common/enumerate.h>
#include <uipc/common/range.h>
#include <muda/cub/device/device_reduce.h>
#include <vertex_reporter.h>

/*************************************************************************************************
* Core Implementation
*************************************************************************************************/
namespace uipc::backend::cuda
{
void GlobalVertexManager::Impl::init_vertex_info()
{
    vertex_reporters.resize(vertex_reporter_buffer.size());
    std::ranges::move(vertex_reporter_buffer, vertex_reporters.begin());

    auto N = vertex_reporters.size();
    reporter_vertex_counts.resize(N);
    reporter_vertex_offsets.resize(N);

    for(auto&& [i, R] : enumerate(vertex_reporters))
    {
        VertexCountInfo info;
        R->report_count(info);
        // get count back
        reporter_vertex_counts[i] = info.m_count;
        if(info.m_changable)
        {
            // record
            vertex_reporter_buffer.push_back(R);
        }
    }

    changable_vertex_reporters.resize(vertex_reporter_buffer.size());
    std::ranges::move(vertex_reporter_buffer, changable_vertex_reporters.begin());

    std::exclusive_scan(reporter_vertex_counts.begin(),
                        reporter_vertex_counts.end(),
                        reporter_vertex_offsets.begin(),
                        0);

    auto total_count = reporter_vertex_offsets.back() + reporter_vertex_counts.back();

    // resize the global coindex buffer
    coindex.resize(total_count);
    positions.resize(total_count);
    displacements.resize(total_count, Vector3::Zero());

    // create the subviews for each attribute_reporter
    for(auto&& [i, R] : enumerate(vertex_reporters))
    {
        VertexAttributeInfo attributes{this, i};
        R->report_attributes(attributes);
    }
}

void GlobalVertexManager::Impl::rebuild_vertex_info()
{
    // TODO: Implement this function
}

Float GlobalVertexManager::Impl::compute_max_displacement()
{
    collect_vertex_displacements();

    muda::DeviceReduce().Reduce((Float*)displacements.data(),
                                max_disp.data(),
                                displacements.size() * 3,
                                [] HIPCUB_RUNTIME_FUNCTION(const Float& L, const Float& R)
                                {
                                    auto absL = std::abs(L);
                                    auto absR = std::abs(R);
                                    return absL > absR ? absL : absR;
                                },
                                0.0);
    return max_disp;
}

AABB GlobalVertexManager::Impl::compute_vertex_bounding_box()
{
    Float max_float = std::numeric_limits<Float>::max();
    muda::DeviceReduce()
        .Reduce(
            positions.data(),
            min_pos.data(),
            positions.size(),
            [] HIPCUB_RUNTIME_FUNCTION(const Vector3& L, const Vector3& R) -> Vector3
            { return L.cwiseMin(R); },
            Vector3{max_float, max_float, max_float})
        .Reduce(
            positions.data(),
            max_pos.data(),
            positions.size(),
            [] HIPCUB_RUNTIME_FUNCTION(const Vector3& L, const Vector3& R) -> Vector3
            { return L.cwiseMax(R); },
            Vector3{-max_float, -max_float, -max_float});

    Vector3 min_pos_host, max_pos_host;
    min_pos_host = min_pos;
    max_pos_host = max_pos;
    return AABB{min_pos_host, max_pos_host};
}
GlobalVertexManager::VertexRegister::VertexRegister(
    std::string_view                            name,
    std::function<void(VertexCountInfo&)>&&     report_vertex_count,
    std::function<void(VertexAttributeInfo&)>&& report_vertex_attributes,
    std::function<void(VertexDisplacementInfo&)>&& report_vertex_displacement) noexcept
    : m_name(name)
    , m_report_vertex_count(std::move(report_vertex_count))
    , m_report_vertex_attributes(std::move(report_vertex_attributes))
    , m_report_vertex_displacement(std::move(report_vertex_displacement))
{
}
}  // namespace uipc::backend::cuda


/*************************************************************************************************
* API Implementation
*************************************************************************************************/
namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(GlobalVertexManager);

void GlobalVertexManager::VertexCountInfo::count(SizeT count) noexcept
{
    m_count = count;
}

void GlobalVertexManager::VertexCountInfo::changable(bool is_changable) noexcept
{
    m_changable = is_changable;
}

GlobalVertexManager::VertexAttributeInfo::VertexAttributeInfo(Impl* impl, SizeT index) noexcept
    : m_impl(impl)
    , m_index(index)
{
}

muda::BufferView<IndexT> GlobalVertexManager::VertexAttributeInfo::coindex() const noexcept
{
    return m_impl->subview(m_impl->coindex, m_index);
}

muda::BufferView<Vector3> GlobalVertexManager::VertexAttributeInfo::positions() const noexcept
{
    return m_impl->subview(m_impl->positions, m_index);
}

GlobalVertexManager::VertexDisplacementInfo::VertexDisplacementInfo(Impl* impl, SizeT index) noexcept
    : m_impl(impl)
    , m_index(index)
{
}

muda::BufferView<Vector3> GlobalVertexManager::VertexDisplacementInfo::displacements() const noexcept
{
    return m_impl->subview(m_impl->displacements, m_index);
}

muda::CBufferView<IndexT> GlobalVertexManager::VertexDisplacementInfo::coindex() const noexcept
{
    return m_impl->subview(m_impl->coindex, m_index);
}

void GlobalVertexManager::init_vertex_info()
{
    m_impl.init_vertex_info();
}

void GlobalVertexManager::rebuild_vertex_info()
{
    m_impl.rebuild_vertex_info();
}

muda::CBufferView<IndexT> GlobalVertexManager::coindex() const noexcept
{
    return m_impl.coindex;
}

muda::CBufferView<Vector3> GlobalVertexManager::positions() const noexcept
{
    return m_impl.positions;
}

muda::CBufferView<Vector3> GlobalVertexManager::displacements() const noexcept
{
    return m_impl.displacements;
}

void GlobalVertexManager::Impl::collect_vertex_displacements()
{
    for(auto&& [i, R] : enumerate(vertex_reporters))
    {
        VertexDisplacementInfo vd{this, i};
        R->report_displacements(vd);
    }
}

Float GlobalVertexManager::compute_max_displacement()
{
    return m_impl.compute_max_displacement();
}

AABB GlobalVertexManager::compute_vertex_bounding_box()
{
    return m_impl.compute_vertex_bounding_box();
}

void GlobalVertexManager::add_reporter(VertexReporter* reporter)
{
    check_state(SimEngineState::BuildSystems, "add_reporter()");
    reporter->m_index = m_impl.vertex_reporter_buffer.size();
    m_impl.vertex_reporter_buffer.push_back(reporter);
}
}  // namespace uipc::backend::cuda