#include "hip/hip_runtime.h"
#include <line_search/line_searcher.h>
#include <uipc/common/enumerate.h>
#include <uipc/common/zip.h>
#include <line_search/line_search_reporter.h>

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(LineSearcher);

void LineSearcher::do_build()
{
    on_init_scene([this]() { init(); });
}

void LineSearcher::init()
{
    auto scene = world().scene();

    m_reporters.build();

    m_energy_values.resize(m_reporters.view().size() + m_energy_reporters.size(), 0);

    for(auto&& [i, R] : enumerate(m_reporters.view()))
        R->m_index = i;


    m_report_energy = scene.info()["line_search"]["report_energy"];
    m_max_iter      = scene.info()["line_search"]["max_iter"];
    m_dt            = scene.info()["dt"];
}

void LineSearcher::record_start_point()
{
    for(auto&& R : m_reporters.view())
    {
        RecordInfo info;
        R->record_start_point(info);
    }
}

void LineSearcher::step_forward(Float alpha)
{
    for(auto&& R : m_reporters.view())
    {
        StepInfo info;
        info.alpha = alpha;
        R->step_forward(info);
    }
}

Float LineSearcher::compute_energy()
{
    auto reporter_energyes = span{m_energy_values}.subspan(0, m_reporters.view().size());

    for(auto&& [E, R] : zip(reporter_energyes, m_reporters.view()))
    {
        EnergyInfo info{this};
        R->compute_energy(info);
        UIPC_ASSERT(info.m_energy.has_value(),
                    "Energy[{}] not set by reporter, did you forget to call energy()?",
                    R->name());
        E = info.m_energy.value();
        UIPC_ASSERT(!std::isnan(E) && std::isfinite(E), "Energy [{}] is {}", R->name(), E);
    }

    auto energy_reporter_energyes =
        span{m_energy_values}.subspan(m_reporters.view().size());

    for(auto&& [E, ER, name] :
        zip(energy_reporter_energyes, m_energy_reporters, m_energy_reporter_names))
    {
        EnergyInfo info{this};
        ER(info);
        UIPC_ASSERT(info.m_energy.has_value(),
                    "Energy[{}] not set by reporter, did you forget to call energy()?",
                    name);
        E = info.m_energy.value();
        UIPC_ASSERT(!std::isnan(E) && std::isfinite(E), "Energy [{}] is {}", name, E);
    }

    Float total_energy =
        std::accumulate(m_energy_values.begin(), m_energy_values.end(), 0.0);

    if(m_report_energy)
    {
        m_report_stream << R"(
-------------------------------------------------------------------------------
*                             Compute Energy                                  *
-------------------------------------------------------------------------------
)";
        m_report_stream << "Total:" << total_energy << "\n";
        for(auto&& [R, value] : zip(m_reporters, reporter_energyes))
        {
            m_report_stream << "  > " << R->name() << "=" << value << "\n";
        }

        for(auto&& [ER, value, name] :
            zip(m_energy_reporters, energy_reporter_energyes, m_energy_reporter_names))
        {
            m_report_stream << "  * " << name << "=" << value << "\n";
        }

        m_report_stream << "-------------------------------------------------------------------------------";
        spdlog::info(m_report_stream.str());
        m_report_stream.str("");
    }

    return total_energy;
}

void LineSearcher::add_reporter(LineSearchReporter* reporter)
{
    check_state(SimEngineState::BuildSystems, "add_reporter()");
    m_reporters.register_subsystem(reporter);
}

void LineSearcher::add_reporter(std::string_view energy_name,
                                std::function<void(EnergyInfo)>&& energy_reporter)
{
    check_state(SimEngineState::BuildSystems, "add_reporter()");
    m_energy_reporters.push_back(std::move(energy_reporter));
    m_energy_reporter_names.push_back(std::string{energy_name});
}

LineSearcher::EnergyInfo::EnergyInfo(LineSearcher* impl) noexcept
    : m_impl(impl)
{
}

Float LineSearcher::EnergyInfo::dt() noexcept
{
    return m_impl->m_dt;
}

void LineSearcher::EnergyInfo::energy(Float e) noexcept
{
    m_energy = e;
}

SizeT LineSearcher::max_iter() const noexcept
{
    return m_max_iter;
}

}  // namespace uipc::backend::cuda
