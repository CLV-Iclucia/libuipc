#include "hip/hip_runtime.h"
#include <line_search/line_searcher.h>
#include <uipc/common/enumerate.h>
#include <uipc/common/zip.h>
#include <line_search/line_search_reporter.h>

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(LineSearcher);

void LineSearcher::add_reporter(LineSearchReporter* reporter)
{
    check_state(SimEngineState::BuildSystems, "add_reporter()");
    reporter->m_index = m_reporter_buffer.size();
    m_reporter_buffer.push_back(reporter);
}

void LineSearcher::add_reporter(std::string_view energy_name,
                                std::function<void(EnergyInfo)>&& energy_reporter)
{
    check_state(SimEngineState::BuildSystems, "add_reporter()");
    m_energy_reporters.push_back(std::move(energy_reporter));
    m_energy_reporter_names.push_back(std::string{energy_name});
}

void LineSearcher::do_build()
{
    on_init_scene([this]() { init(); });
}

void LineSearcher::init()
{
    auto scene = world().scene();

    m_reporters.resize(m_reporter_buffer.size());
    std::ranges::move(m_reporter_buffer, m_reporters.begin());

    m_energy_values.resize(m_reporters.size() + m_energy_reporters.size(), 0);

    m_report_energy = scene.info()["debug"]["report_energy"];
    m_dt            = scene.info()["dt"];
}

void LineSearcher::record_start_point()
{
    for(auto&& R : m_reporters)
    {
        RecordInfo info;
        R->record_start_point(info);
    }
}

void LineSearcher::step_forward(Float alpha)
{
    for(auto&& R : m_reporters)
    {
        StepInfo info;
        info.alpha = alpha;
        R->step_forward(info);
    }
}

Float LineSearcher::compute_energy()
{
    auto reporter_energyes = span{m_energy_values}.subspan(0, m_reporters.size());

    for(auto&& [E, R] : zip(reporter_energyes, m_reporters))
    {
        EnergyInfo info{this};
        R->compute_energy(info);
        UIPC_ASSERT(info.m_energy.has_value(),
                    "Energy[{}] not set by reporter, did you forget to call energy()?", R->name());
        E = info.m_energy.value();
        UIPC_ASSERT(!std::isnan(E) && std::isfinite(E), "Energy [{}] is {}", R->name(), E);
    }

    auto energy_reporter_energyes = span{m_energy_values}.subspan(m_reporters.size());

    for(auto&& [E, ER, name] :
        zip(energy_reporter_energyes, m_energy_reporters, m_energy_reporter_names))
    {
        EnergyInfo info{this};
        ER(info);
        UIPC_ASSERT(info.m_energy.has_value(),
                    "Energy[{}] not set by reporter, did you forget to call energy()?", name);
        E = info.m_energy.value();
        UIPC_ASSERT(!std::isnan(E) && std::isfinite(E), "Energy [{}] is {}", name, E);
    }

    Float total_energy =
        std::accumulate(m_energy_values.begin(), m_energy_values.end(), 0.0);

    if(m_report_energy)
    {
        m_report_stream << R"(
-------------------------------------------------------------------------------
*                             Compute Energy                                  *
-------------------------------------------------------------------------------
)";
        m_report_stream << "Total:" << total_energy << "\n";
        for(auto&& [R, value] : zip(m_reporters, reporter_energyes))
        {
            m_report_stream << "  > " << R->name() << "=" << value << "\n";
        }

        for(auto&& [ER, value, name] :
            zip(m_energy_reporters, energy_reporter_energyes, m_energy_reporter_names))
        {
            m_report_stream << "  * " << name << "=" << value << "\n";
        }

        m_report_stream << "-------------------------------------------------------------------------------";
        spdlog::info(m_report_stream.str());
        m_report_stream.str("");
    }

    return total_energy;
}

LineSearcher::EnergyInfo::EnergyInfo(LineSearcher* impl) noexcept
    : m_impl(impl)
{
}

Float LineSearcher::EnergyInfo::dt() noexcept
{
    return m_impl->m_dt;
}
void LineSearcher::EnergyInfo::energy(Float e) noexcept
{
    m_energy = e;
}
}  // namespace uipc::backend::cuda
