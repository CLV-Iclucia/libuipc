#include "hip/hip_runtime.h"
#include <finite_element/finite_element_method.h>
#include <Eigen/Dense>
#include <Eigen/Geometry>
#include <finite_element/finite_element_constitution.h>
#include <uipc/builtin/attribute_name.h>
#include <uipc/geometry/simplicial_complex.h>
#include <uipc/common/map.h>
#include <uipc/common/zip.h>
#include <finite_element/fem_utils.h>
#include <uipc/common/algorithm/run_length_encode.h>
#include <uipc/common/json_eigen.h>
#include <muda/ext/eigen/inverse.h>
#include <ranges>

// constitutions
#include <finite_element/fem_3d_constitution.h>
#include <finite_element/codim_2d_constitution.h>
#include <finite_element/codim_1d_constitution.h>
#include <finite_element/codim_0d_constitution.h>

bool operator<(const uipc::backend::cuda::FiniteElementMethod::DimUID& a,
               const uipc::backend::cuda::FiniteElementMethod::DimUID& b)
{
    return a.dim < b.dim || (a.dim == b.dim && a.uid < b.uid);
}

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(FiniteElementMethod);

void FiniteElementMethod::add_constitution(FiniteElementConstitution* constitution)
{
    check_state(SimEngineState::BuildSystems, "add_constitution()");
    m_impl.constitutions.register_subsystem(*constitution);
}

void FiniteElementMethod::do_build()
{
    const auto& scene = world().scene();
    auto&       types = scene.constitution_tabular().types();
    if(types.find(constitution::ConstitutionType::FiniteElement) == types.end())
    {
        throw SimSystemException("No Finite Element Constitution found in the scene");
    }

    m_impl.gravity = scene.info()["gravity"].get<Vector3>();

    m_impl.global_vertex_manager = &require<GlobalVertexManager>();

    auto& dof_predictor = require<DoFPredictor>();

    dof_predictor.on_predict(*this,
                             [this](DoFPredictor::PredictInfo& info)
                             { m_impl.compute_x_tilde(info); });

    dof_predictor.on_compute_velocity(*this,
                                      [this](DoFPredictor::ComputeVelocityInfo& info)
                                      { m_impl.compute_velocity(info); });

    // Register the action to initialize the finite element geometry
    on_init_scene([this] { m_impl.init(world()); });

    // Register the action to write the scene
    on_write_scene([this] { m_impl.write_scene(world()); });
}

void FiniteElementMethod::Impl::init(WorldVisitor& world)
{
    _init_constitutions();
    _build_geo_infos(world);
    _build_constitution_infos();
    _build_on_host(world);
    _build_on_device();
    _distribute_constitution_filtered_info();
}

void FiniteElementMethod::Impl::_init_constitutions()
{
    auto constitution_view = constitutions.view();

    // 1) sort the constitutions by (dim, uid)
    std::sort(constitution_view.begin(),
              constitution_view.end(),
              [](const FiniteElementConstitution* a, const FiniteElementConstitution* b)
              {
                  auto   uida = a->constitution_uid();
                  auto   uidb = b->constitution_uid();
                  auto   dima = a->dimension();
                  auto   dimb = b->dimension();
                  DimUID uid_dim_a{dima, uida};
                  DimUID uid_dim_b{dimb, uidb};
                  return uid_dim_a < uid_dim_b;
              });

    for(auto&& [i, c] : enumerate(constitution_view))
        c->m_index = i;

    // 2) classify the constitutions
    codim_0d_constitutions.reserve(constitution_view.size());
    codim_1d_constitutions.reserve(constitution_view.size());
    codim_2d_constitutions.reserve(constitution_view.size());
    fem_3d_constitutions.reserve(constitution_view.size());

    for(auto&& constitution : constitution_view)
    {
        auto dim = constitution->dimension();
        switch(dim)
        {
            case 0: {
                auto derived = dynamic_cast<Codim0DConstitution*>(constitution);
                UIPC_ASSERT(derived, "The constitution is not a Codim0DConstitution, its dim = {}", dim);
                derived->m_index_in_dim = codim_0d_constitutions.size();
                codim_0d_constitutions.push_back(derived);
                codim_0d_uid_to_index.insert(
                    {derived->constitution_uid(), derived->m_index_in_dim});
            }
            break;
            case 1: {
                auto derived = dynamic_cast<Codim1DConstitution*>(constitution);
                UIPC_ASSERT(derived, "The constitution is not a Codim1DConstitution, its dim = {}", dim);
                derived->m_index_in_dim = codim_1d_constitutions.size();
                codim_1d_constitutions.push_back(derived);
                codim_1d_uid_to_index.insert(
                    {derived->constitution_uid(), derived->m_index_in_dim});
            }
            break;
            case 2: {
                auto derived = dynamic_cast<Codim2DConstitution*>(constitution);
                UIPC_ASSERT(derived, "The constitution is not a Codim2DConstitution, its dim = {}", dim);
                derived->m_index_in_dim = codim_2d_constitutions.size();
                codim_2d_constitutions.push_back(derived);
                codim_2d_uid_to_index.insert(
                    {derived->constitution_uid(), derived->m_index_in_dim});
            }
            break;
            case 3: {
                auto derived = dynamic_cast<FEM3DConstitution*>(constitution);
                UIPC_ASSERT(derived, "The constitution is not a FEM3DConstitution, its dim = {}", dim);
                derived->m_index_in_dim = fem_3d_constitutions.size();
                fem_3d_constitutions.push_back(derived);
                fem_3d_uid_to_index.insert({derived->constitution_uid(), derived->m_index_in_dim});
            }
            break;
            default:
                break;
        }
    }
}

void FiniteElementMethod::Impl::_build_geo_infos(WorldVisitor& world)
{
    set<U64> filter_uids;

    for(auto&& filter : constitutions.view())
        filter_uids.insert(filter->constitution_uid());

    // 1) find all the finite element constitutions
    auto geo_slots = world.scene().geometries();
    geo_infos.reserve(geo_slots.size());

    for(auto&& [i, geo_slot] : enumerate(geo_slots))
    {
        auto& geo  = geo_slot->geometry();
        auto  cuid = geo.meta().find<U64>(builtin::constitution_uid);
        if(cuid)
        {
            auto uid = cuid->view()[0];
            if(filter_uids.find(uid) != filter_uids.end())  // if exists
            {
                auto* sc = geo.as<geometry::SimplicialComplex>();
                UIPC_ASSERT(sc,
                            "The geometry is not a simplicial complex (it's {}). Why can it happen?",
                            geo.type());

                GeoInfo info;
                info.geo_slot_index = i;
                info.vertex_count   = sc->vertices().size();
                info.dim_uid.dim    = sc->dim();
                info.dim_uid.uid    = uid;

                switch(sc->dim())
                {
                    case 0:
                        info.primitive_count = sc->vertices().size();
                        break;
                    case 1:
                        info.primitive_count = sc->edges().size();
                        break;
                    case 2:
                        info.primitive_count = sc->triangles().size();
                        break;
                    case 3:
                        info.primitive_count = sc->tetrahedra().size();
                        break;
                    default:
                        break;
                }

                geo_infos.push_back(info);
            }
        }
    }

    // 2) sort geometry by (dim, uid)
    std::sort(geo_infos.begin(),
              geo_infos.end(),
              [](const GeoInfo& a, const GeoInfo& b)
              { return a.dim_uid < b.dim_uid; });

    // 3) setup vertex offsets
    auto count = geo_infos.size() + 1;  // add one to calculate the total size

    vector<SizeT> vertex_counts(count, 0);
    vector<SizeT> vertex_offsets(count, 0);

    std::transform(geo_infos.begin(),
                   geo_infos.end(),
                   vertex_counts.begin(),
                   [](const GeoInfo& info) { return info.vertex_count; });

    std::exclusive_scan(
        vertex_counts.begin(), vertex_counts.end(), vertex_offsets.begin(), 0);

    for(auto&& [i, info] : enumerate(geo_infos))
        info.vertex_offset = vertex_offsets[i];

    h_positions.resize(vertex_offsets.back());

    // 4) setup dim infos
    {
        std::array<SizeT, 4> dim_geo_counts;
        std::array<SizeT, 4> dim_geo_offsets;
        dim_geo_counts.fill(0);

        vector<SizeT> offsets;
        offsets.reserve(dim_geo_counts.size());
        vector<SizeT> counts;
        counts.reserve(dim_geo_counts.size());

        // encode the dimension
        encode_offset_count(geo_infos.begin(),
                            geo_infos.end(),
                            std::back_inserter(offsets),
                            std::back_inserter(counts),
                            [](const GeoInfo& current, const GeoInfo& value)
                            { return current.dim_uid.dim == value.dim_uid.dim; });

        for(auto&& [offset, count] : zip(offsets, counts))
        {
            auto& info                       = geo_infos[offset];
            dim_geo_counts[info.dim_uid.dim] = count;
        }

        std::exclusive_scan(
            dim_geo_counts.begin(), dim_geo_counts.end(), dim_geo_offsets.begin(), 0);

        for(auto&& [i, dim_info] : enumerate(dim_infos))
        {
            dim_info.geo_info_offset = dim_geo_offsets[i];
            dim_info.geo_info_count  = dim_geo_counts[i];
        }
    }


    // 4) setup primitive offsets
    for(auto&& [i, dim_info] : enumerate(dim_infos))
    {
        auto it = std::find_if(geo_infos.begin(),
                               geo_infos.end(),
                               [i](const GeoInfo& info)
                               { return info.dim_uid.dim == i; });

        if(it == geo_infos.end())
            continue;

        auto count = dim_info.geo_info_count + 1;  // + 1 to calculate the total size

        vector<SizeT> primitive_counts(count, 0);
        vector<SizeT> primitive_offsets(count, 0);

        auto geo_span =
            span{geo_infos}.subspan(dim_info.geo_info_offset, dim_info.geo_info_count);

        std::ranges::transform(geo_span,
                               primitive_counts.begin(),
                               [](const GeoInfo& info)
                               { return info.primitive_count; });

        std::exclusive_scan(primitive_counts.begin(),
                            primitive_counts.end(),
                            primitive_offsets.begin(),
                            0);

        for(auto&& [j, info] : enumerate(geo_span))
        {
            info.primitive_offset = primitive_offsets[j];
            info.primitive_count  = primitive_counts[j];
        }


        dim_info.primitive_count = primitive_offsets.back();
        dim_info.primitive_offset = geo_infos[dim_info.geo_info_offset].primitive_offset;
    }

    h_codim_0ds.resize(dim_infos[0].primitive_count);
    h_codim_1ds.resize(dim_infos[1].primitive_count);
    h_codim_2ds.resize(dim_infos[2].primitive_count);
    h_tets.resize(dim_infos[3].primitive_count);
}

void FiniteElementMethod::Impl::_build_constitution_infos()
{
    auto build_infos = [&]<std::derived_from<FiniteElementConstitution> ConstitutionT>(
                           vector<ConstitutionInfo>& infos,
                           span<ConstitutionT*>      constitutions,
                           IndexT                    dim,
                           unordered_map<U64, SizeT> uid_to_index)
    {
        infos.resize(constitutions.size());
        vector<SizeT> vertex_counts(infos.size(), 0);
        vector<SizeT> primitive_counts(infos.size(), 0);
        vector<SizeT> geometry_counts(infos.size(), 0);

        const auto& dim_info = dim_infos[dim];

        auto geo_info_subspan =
            span{geo_infos}.subspan(dim_info.geo_info_offset, dim_info.geo_info_count);


        for(auto&& geo_info : geo_info_subspan)
        {
            auto index = uid_to_index[geo_info.dim_uid.uid];
            geometry_counts[index]++;
            vertex_counts[index] += geo_info.vertex_count;
            primitive_counts[index] += geo_info.primitive_count;
        }

        vector<SizeT> vertex_offsets(infos.size(), 0);
        vector<SizeT> primitive_offsets(infos.size(), 0);
        vector<SizeT> geometry_offsets(infos.size(), 0);

        SizeT dim_geo_offset    = dim_info.geo_info_offset;
        SizeT dim_vertex_offset = 0;

        if(geo_infos.size() > 0)
        {
            const auto& begin_geo         = geo_infos[dim_geo_offset];
            SizeT       dim_vertex_offset = begin_geo.vertex_offset;
        }

        std::exclusive_scan(vertex_counts.begin(),
                            vertex_counts.end(),
                            vertex_offsets.begin(),
                            dim_vertex_offset);

        std::exclusive_scan(primitive_counts.begin(),
                            primitive_counts.end(),
                            primitive_offsets.begin(),
                            0);

        std::exclusive_scan(geometry_counts.begin(),
                            geometry_counts.end(),
                            geometry_offsets.begin(),
                            dim_geo_offset);

        for(auto&& [i, info] : enumerate(infos))
        {
            info.vertex_count     = vertex_counts[i];
            info.vertex_offset    = vertex_offsets[i];
            info.primitive_count  = primitive_counts[i];
            info.primitive_offset = primitive_offsets[i];
            info.geo_info_count   = geometry_counts[i];
            info.geo_info_offset  = geometry_offsets[i];
        }
    };


    build_infos(codim_0d_constitution_infos, span{codim_0d_constitutions}, 0, codim_0d_uid_to_index);
    build_infos(codim_1d_constitution_infos, span{codim_1d_constitutions}, 1, codim_1d_uid_to_index);
    build_infos(codim_2d_constitution_infos, span{codim_2d_constitutions}, 2, codim_2d_uid_to_index);
    build_infos(fem_3d_constitution_infos, span{fem_3d_constitutions}, 3, fem_3d_uid_to_index);
}

void FiniteElementMethod::Impl::_build_on_host(WorldVisitor& world)
{
    auto geo_slots      = world.scene().geometries();
    auto rest_geo_slots = world.scene().rest_geometries();

    // resize buffers
    h_rest_positions.resize(h_positions.size());
    h_thicknesses.resize(h_positions.size(), 0);  // fill 0 for default
    h_masses.resize(h_positions.size());
    h_vertex_contact_element_ids.resize(h_positions.size(), 0);  // fill 0 for default
    h_vertex_is_fixed.resize(h_positions.size(), 0);  // fill 0 for default

    for(auto&& [i, info] : enumerate(geo_infos))
    {
        auto& geo_slot      = geo_slots[info.geo_slot_index];
        auto& rest_geo_slot = rest_geo_slots[info.geo_slot_index];
        auto& geo           = geo_slot->geometry();
        auto& rest_geo      = rest_geo_slot->geometry();
        auto* sc            = geo.as<geometry::SimplicialComplex>();
        UIPC_ASSERT(sc,
                    "The geometry is not a simplicial complex (it's {}). Why can it happen?",
                    geo.type());
        auto* rest_sc = rest_geo.as<geometry::SimplicialComplex>();
        UIPC_ASSERT(rest_sc,
                    "The geometry is not a simplicial complex (it's {}). Why can it happen?",
                    rest_geo.type());

        // 1) setup primitives
        switch(sc->dim())
        {
            case 0: {
                auto dst_codim_0d_span =
                    span{h_codim_0ds}.subspan(info.primitive_offset, info.primitive_count);
                std::iota(dst_codim_0d_span.begin(), dst_codim_0d_span.end(), info.vertex_offset);
            }
            break;
            case 1: {
                auto dst_codim_1d_span =
                    span{h_codim_1ds}.subspan(info.primitive_offset, info.primitive_count);

                auto edge_view = sc->edges().topo().view();
                UIPC_ASSERT(edge_view.size() == dst_codim_1d_span.size(), "edge size mismatching");

                std::transform(edge_view.begin(),
                               edge_view.end(),
                               dst_codim_1d_span.begin(),
                               [&](const Vector2i& edge) -> Vector2i
                               { return edge.array() + info.vertex_offset; });
            }
            break;
            case 2: {
                auto dst_codim_2d_span =
                    span{h_codim_2ds}.subspan(info.primitive_offset, info.primitive_count);

                auto tri_view = sc->triangles().topo().view();
                UIPC_ASSERT(tri_view.size() == dst_codim_2d_span.size(),
                            "triangle size mismatching");

                std::transform(tri_view.begin(),
                               tri_view.end(),
                               dst_codim_2d_span.begin(),
                               [&](const Vector3i& tri) -> Vector3i
                               { return tri.array() + info.vertex_offset; });
            }
            break;
            case 3: {
                auto dst_tet_span =
                    span{h_tets}.subspan(info.primitive_offset, info.primitive_count);

                auto tet_view = sc->tetrahedra().topo().view();
                UIPC_ASSERT(tet_view.size() == dst_tet_span.size(), "tetrahedra size mismatching");

                std::transform(tet_view.begin(),
                               tet_view.end(),
                               dst_tet_span.begin(),
                               [&](const Vector4i& tet) -> Vector4i
                               { return tet.array() + info.vertex_offset; });
            }
            break;
            default:
                break;
        }

        {  // 2) fill backend_fem_vertex_offset in geometry
            auto vertex_offset = sc->meta().find<IndexT>(builtin::backend_fem_vertex_offset);
            if(!vertex_offset)
                vertex_offset =
                    sc->meta().create<IndexT>(builtin::backend_fem_vertex_offset, -1);
            auto vertex_offset_view = geometry::view(*vertex_offset);
            std::ranges::fill(vertex_offset_view, info.vertex_offset);
        }

        {  // 3) setup positions
            auto pos_view = sc->positions().view();
            auto dst_pos_span =
                span{h_positions}.subspan(info.vertex_offset, info.vertex_count);
            UIPC_ASSERT(pos_view.size() == dst_pos_span.size(), "position size mismatching");
            std::copy(pos_view.begin(), pos_view.end(), dst_pos_span.begin());

            auto rest_pos_view = rest_sc->positions().view();
            auto dst_rest_pos_span =
                span{h_rest_positions}.subspan(info.vertex_offset, info.vertex_count);
            UIPC_ASSERT(rest_pos_view.size() == dst_rest_pos_span.size(),
                        "rest position size mismatching");
            std::ranges::copy(rest_pos_view, dst_rest_pos_span.begin());
        }

        {  // 4) setup mass
            auto mass      = sc->vertices().find<Float>(builtin::mass);
            auto mass_view = mass->view();
            auto dst_mass_span =
                span{h_masses}.subspan(info.vertex_offset, info.vertex_count);
            UIPC_ASSERT(mass_view.size() == dst_mass_span.size(), "mass size mismatching");
            std::ranges::copy(mass_view, dst_mass_span.begin());
        }

        {  // 5) setup thickness
            auto thickness = sc->vertices().find<Float>(builtin::thickness);
            auto dst_thickness_span =
                span{h_thicknesses}.subspan(info.vertex_offset, info.vertex_count);

            if(thickness)
            {
                auto thickness_view = thickness->view();
                UIPC_ASSERT(thickness_view.size() == dst_thickness_span.size(),
                            "thickness size mismatching");
                std::ranges::copy(thickness_view, dst_thickness_span.begin());
            }
        }


        {  // 6) setup vertex contact element id
            auto ceid = sc->vertices().find<IndexT>(builtin::contact_element_id);
            auto dst_eid_span =
                span{h_vertex_contact_element_ids}.subspan(info.vertex_offset,
                                                           info.vertex_count);

            if(ceid)
            {
                auto eid_view = ceid->view();
                UIPC_ASSERT(eid_view.size() == dst_eid_span.size(),
                            "contact_element_id size mismatching");
                std::ranges::copy(eid_view, dst_eid_span.begin());
            }
        }

        {  // 7) setup vertex is_fixed

            auto is_fixed = sc->vertices().find<IndexT>(builtin::is_fixed);
            auto dst_is_fixed_span =
                span{h_vertex_is_fixed}.subspan(info.vertex_offset, info.vertex_count);

            if(is_fixed)
            {
                auto is_fixed_view = is_fixed->view();
                UIPC_ASSERT(is_fixed_view.size() == dst_is_fixed_span.size(),
                            "is_fixed size mismatching");
                std::ranges::copy(is_fixed_view, dst_is_fixed_span.begin());
            }
        }
    }
}

void FiniteElementMethod::Impl::_build_on_device()
{
    using namespace muda;

    // 1) Vertex States
    xs.resize(h_positions.size());
    xs.view().copy_from(h_positions.data());

    x_bars.resize(h_rest_positions.size());
    x_bars.view().copy_from(h_rest_positions.data());

    x_temps  = xs;
    x_tildes = xs;
    x_prevs  = xs;

    is_fixed.resize(h_vertex_is_fixed.size());
    is_fixed.view().copy_from(h_vertex_is_fixed.data());

    dxs.resize(xs.size(), Vector3::Zero());
    vs.resize(xs.size(), Vector3::Zero());

    masses.resize(h_masses.size());
    masses.view().copy_from(h_masses.data());

    thicknesses.resize(h_thicknesses.size());
    thicknesses.view().copy_from(h_thicknesses.data());

    diag_hessians.resize(xs.size());

    // 2) Elements
    codim_0ds.resize(h_codim_0ds.size());
    codim_0ds.view().copy_from(h_codim_0ds.data());

    codim_1ds.resize(h_codim_1ds.size());
    codim_1ds.view().copy_from(h_codim_1ds.data());
    rest_lengths.resize(codim_1ds.size());

    codim_2ds.resize(h_codim_2ds.size());
    codim_2ds.view().copy_from(h_codim_2ds.data());
    rest_areas.resize(codim_2ds.size());

    tets.resize(h_tets.size());
    tets.view().copy_from(h_tets.data());
    rest_volumes.resize(tets.size());

    // 3) Material Space Attribute
    // Rod
    ParallelFor()
        .kernel_name("Rod Basis")
        .apply(codim_1ds.size(),
               [codim_1ds = codim_1ds.viewer().name("codim_1ds"),
                x_bars    = x_bars.viewer().name("x_bars"),
                rest_lengths = rest_lengths.viewer().name("rest_lengths")] __device__(int i) mutable
               {
                   const Vector2i& edge = codim_1ds(i);
                   const Vector3&  x0   = x_bars(edge[0]);
                   const Vector3&  x1   = x_bars(edge[1]);

                   rest_lengths(i) = (x1 - x0).norm();
               });


    // Shell
    ParallelFor()
        .kernel_name("Shell Basis")
        .apply(codim_2ds.size(),
               [codim_2ds = codim_2ds.viewer().name("codim_2ds"),
                x_bars    = x_bars.viewer().name("x_bars"),
                rest_areas = rest_areas.viewer().name("rest_areas")] __device__(int i) mutable
               {
                   const Vector3i& tri = codim_2ds(i);
                   const Vector3&  x0  = x_bars(tri[0]);
                   const Vector3&  x1  = x_bars(tri[1]);
                   const Vector3&  x2  = x_bars(tri[2]);

                   Vector3 E01 = x1 - x0;
                   Vector3 E02 = x2 - x0;

                   rest_areas(i) = 0.5 * E01.cross(E02).norm();
               });

    // FEM3D Material Basis
    Dm3x3_invs.resize(tets.size());
    ParallelFor()
        .kernel_name("FEM3D Material Basis")
        .apply(tets.size(),
               [tets      = tets.viewer().name("tets"),
                x_bars    = x_bars.viewer().name("x_bars"),
                Dm9x9_inv = Dm3x3_invs.viewer().name("Dm3x3_inv"),
                rest_volumes = rest_volumes.viewer().name("rest_volumes")] __device__(int i) mutable
               {
                   const Vector4i& tet = tets(i);
                   const Vector3&  x0  = x_bars(tet[0]);
                   const Vector3&  x1  = x_bars(tet[1]);
                   const Vector3&  x2  = x_bars(tet[2]);
                   const Vector3&  x3  = x_bars(tet[3]);

                   Dm9x9_inv(i) = fem::Dm_inv(x0, x1, x2, x3);
                   Float V      = fem::Ds(x0, x1, x2, x3).determinant();
                   MUDA_ASSERT(V > 0.0,
                               "Negative volume tetrahedron (%d, %d, %d, %d)",
                               tet[0],
                               tet[1],
                               tet[2],
                               tet[3]);
                   rest_volumes(i) = V;
               });

    // 4) Allocate memory for energy, gradient and hessian
    vertex_kinetic_energies.resize(xs.size());
    G3s.resize(xs.size());
    H3x3s.resize(xs.size());

    auto constitution_count = constitutions.view().size();

    codim_1d_elastic_energy = 0;
    codim_1d_elastic_energies.resize(codim_1ds.size());
    G6s.resize(codim_1ds.size());
    H6x6s.resize(codim_1ds.size());

    codim_2d_elastic_energy = 0;
    codim_2d_elastic_energies.resize(codim_2ds.size());
    G9s.resize(codim_2ds.size());
    H9x9s.resize(codim_2ds.size());

    fem_3d_elastic_energy = 0;
    fem_3d_elastic_energies.resize(tets.size());
    G12s.resize(tets.size());
    H12x12s.resize(tets.size());
}

void FiniteElementMethod::Impl::_distribute_constitution_filtered_info()
{
    for(auto&& [i, c] : enumerate(codim_0d_constitutions))
    {
        Codim0DFilteredInfo filtered_info{this};
        filtered_info.m_index_in_dim = i;
        c->retrieve(filtered_info);
    }

    for(auto&& [i, c] : enumerate(codim_1d_constitutions))
    {
        Codim1DFilteredInfo filtered_info{this};
        filtered_info.m_index_in_dim = i;
        c->retrieve(filtered_info);
    }

    for(auto&& [i, c] : enumerate(codim_2d_constitutions))
    {
        Codim2DFilteredInfo filtered_info{this};
        filtered_info.m_index_in_dim = i;
        c->retrieve(filtered_info);
    }

    for(auto&& [i, c] : enumerate(fem_3d_constitutions))
    {
        FEM3DFilteredInfo filtered_info{this};
        filtered_info.m_index_in_dim = i;
        c->retrieve(filtered_info);
    }
}

void FiniteElementMethod::Impl::write_scene(WorldVisitor& world)
{
    _download_geometry_to_host();

    auto geo_slots = world.scene().geometries();

    auto position_span = span{h_positions};

    for(auto&& [i, info] : enumerate(geo_infos))
    {
        auto& geo_slot = geo_slots[info.geo_slot_index];
        auto& geo      = geo_slot->geometry();
        auto* sc       = geo.as<geometry::SimplicialComplex>();
        UIPC_ASSERT(sc,
                    "The geometry is not a simplicial complex (it's {}). Why can it happen?",
                    geo.type());

        // 1) write positions back
        auto pos_view = geometry::view(sc->positions());
        auto src_pos_span = position_span.subspan(info.vertex_offset, info.vertex_count);
        UIPC_ASSERT(pos_view.size() == src_pos_span.size(), "position size mismatching");
        std::copy(src_pos_span.begin(), src_pos_span.end(), pos_view.begin());

        // 2) write primitives back
        // TODO:
        // Now there is no topology modification, so no need to write back
        // In the future, we may need to write back the topology if the topology is modified
    }
}

void FiniteElementMethod::Impl::_download_geometry_to_host()
{
    xs.view().copy_to(h_positions.data());
}

void FiniteElementMethod::Impl::compute_x_tilde(DoFPredictor::PredictInfo& info)
{
    using namespace muda;
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(xs.size(),
               [is_fixed = is_fixed.cviewer().name("fixed"),
                x_prevs  = x_prevs.cviewer().name("x_prevs"),
                vs       = vs.cviewer().name("vs"),
                x_tildes = x_tildes.viewer().name("x_tildes"),
                g        = gravity,
                dt       = info.dt()] __device__(int i) mutable
               {
                   const Vector3& x_prev = x_prevs(i);
                   const Vector3& v      = vs(i);
                   // TODO: this time, we only consider gravity
                   if(is_fixed(i))
                   {
                       x_tildes(i) = x_prev;
                   }
                   else
                   {
                       x_tildes(i) = x_prev + v * dt + g * (dt * dt);
                   }
               });
}

void FiniteElementMethod::Impl::compute_velocity(DoFPredictor::ComputeVelocityInfo& info)
{
    using namespace muda;
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(xs.size(),
               [is_fixed = is_fixed.cviewer().name("fixed"),
                xs       = xs.cviewer().name("xs"),
                vs       = vs.viewer().name("vs"),
                x_prevs  = x_prevs.viewer().name("x_prevs"),
                dt       = info.dt()] __device__(int i) mutable
               {
                   Vector3& v      = vs(i);
                   Vector3& x_prev = x_prevs(i);

                   const Vector3& x = xs(i);

                   if(is_fixed(i))
                       v = Vector3::Zero();
                   else
                       v = (x - x_prev) * (1.0 / dt);

                   x_prev = x;
               });
}
}  // namespace uipc::backend::cuda


// Info:
namespace uipc::backend::cuda
{
Float FiniteElementMethod::ComputeEnergyInfo::dt() const noexcept
{
    return m_dt;
}

FiniteElementMethod::ComputeGradientHessianInfo::ComputeGradientHessianInfo(Float dt) noexcept
    : m_dt(dt)
{
}
}  // namespace uipc::backend::cuda