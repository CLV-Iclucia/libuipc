#include <finite_element/fem_utils.h>
#include <muda/ext/eigen/inverse.h>
#include <Eigen/Geometry>

namespace uipc::backend::cuda::fem
{
MUDA_GENERIC Float invariant2(const Matrix3x3& F)
{
    return ddot(F, F);
}

MUDA_GENERIC Float invariant2(const Vector3& Sigma)
{
    return Sigma[0] * Sigma[0] + Sigma[1] * Sigma[1] + Sigma[2] * Sigma[2];
}

MUDA_GENERIC Float invariant3(const Matrix3x3& F)
{
    return F.determinant();
}

MUDA_GENERIC Float invariant3(const Vector3& Sigma)
{
    return Sigma[0] * Sigma[1] * Sigma[2];
}

MUDA_GENERIC Float invariant4(const Matrix3x3& F, const Vector3& a)
{
    Matrix3x3 U, V;
    Vector3   Sigma;
    svd(F, U, Sigma, V);
    const Matrix3x3 S = V * Sigma.asDiagonal() * V.transpose();
    return (S * a).dot(a);
}

MUDA_GENERIC Float invariant5(const Matrix3x3& F, const Vector3& a)
{
    return (F * a).squaredNorm();
}

MUDA_GENERIC Matrix3x3 dJdF(const Matrix3x3& F)
{
    Matrix3x3 dJdF;
    //tex:
    //$$
    //\frac{\partial I_{3}}{\partial \mathbf{F}}=\frac{\partial J}{\partial \mathbf{F}}=\left[\begin{array}{l|l|l}
    //\mathbf{f}_{1} \times \mathbf{f}_{2} & \mathbf{f}_{2} \times \mathbf{f}_{0} & \mathbf{f}_{0} \times \mathbf{f}_{1}
    //\end{array}\right]
    //$$
    dJdF.col(0) = F.col(1).cross(F.col(2));
    dJdF.col(1) = F.col(2).cross(F.col(0));
    dJdF.col(2) = F.col(0).cross(F.col(1));
    return dJdF;
}

MUDA_GENERIC Matrix3x3 Dm_inv(const Vector3& X0, const Vector3& X1, const Vector3& X2, const Vector3& X3)
{
    Matrix3x3 Dm = Ds(X0, X1, X2, X3);
    return muda::eigen::inverse(Dm);
}

MUDA_GENERIC Matrix3x3 Ds(const Vector3& x0, const Vector3& x1, const Vector3& x2, const Vector3& x3)
{
    Matrix3x3 Ds;
    Ds.col(0) = x1 - x0;
    Ds.col(1) = x2 - x0;
    Ds.col(2) = x3 - x0;
    return Ds;
}

MUDA_GENERIC Matrix9x12 dFdx(const Matrix3x3& DmInv)
{
    // clang-format off
    
    //tex:
    //$$
    //\begin{array}{l}
    //\frac{\partial \mathbf{D}_{s}}{\partial x_{0}}=\left[\begin{array}{ccc}
    //-1 & -1 & -1 \\
    //0 & 0 & 0 \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{D}_{s}}{\partial x_{1}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //-1 & -1 & -1 \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{D}_{s}}{\partial x_{2}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //0 & 0 & 0 \\
    //-1 & -1 & -1
    //\end{array}\right] \\
    //\frac{\partial \mathbf{D}_{s}}{\partial x_{3}}=\left[\begin{array}{ccc}
    //1 & 0 & 0 \\
    //0 & 0 & 0 \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{D}_{s}}{\partial x_{4}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //1 & 0 & 0 \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{D}_{s}}{\partial x_{5}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //0 & 0 & 0 \\
    //1 & 0 & 0
    //\end{array}\right] \\
    //\frac{\partial \mathbf{D}_{s}}{\partial x_{6}}=\left[\begin{array}{ccc}
    //0 & 1 & 0 \\
    //0 & 0 & 0 \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{D}_{s}}{\partial x_{7}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //0 & 1 & 0 \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{D}_{s}}{\partial x_{8}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //0 & 0 & 0 \\
    //0 & 1 & 0
    //\end{array}\right] \\
    //\frac{\partial \mathbf{D}_{s}}{\partial x_{9}}=\left[\begin{array}{ccc}
    //0 & 0 & 1 \\
    //0 & 0 & 0 \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{D}_{s}}{\partial x_{10}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //0 & 0 & 1 \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{D}_{s}}{\partial x_{11}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //0 & 0 & 0 \\
    //0 & 0 & 1
    //\end{array}\right] \\
    //\end{array}
    //$$
    

    //tex:
    //$$ \mathbf{D}_m^{-1}=\left[\begin{array}{l}
    //\mathbf{r}_{0} \\
    //\hline \mathbf{r}_{1} \\
    //\hline \mathbf{r}_{2}
    //\end{array}\right]
    //=
    //\left[\begin{array}{l|l|l} 
    //\mathbf{c}_{0} & \mathbf{c}_{1} & \mathbf{c}_{2}
    //\end{array}\right]
	//$$


    //tex:
    //$$
    //\begin{array}{l}
    //\frac{\partial \mathbf{F}}{\partial x_{0}}=\left[\begin{array}{ccc}
    //-s_{0} & -s_{1} & -s_{2} \\
    //0 & 0 & 0 \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{F}}{\partial x_{1}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //-s_{0} & -s_{1} & -s_{2} \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{F}}{\partial x_{2}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //0 & 0 & 0 \\
    //-s_{0} & -s_{1} & -s_{2}
    //\end{array}\right] \\
    //\frac{\partial \mathbf{F}}{\partial x_{3}}=\left[\begin{array}{ccc} 
    //& \mathbf{r}_{0} \\
    //0 & 0 & 0 \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{F}}{\partial x_{4}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //& \mathbf{r}_{0} & \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{F}}{\partial x_{5}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //0 & 0 & 0 \\
    //& \mathbf{r}_{0} &
    //\end{array}\right] \\
    //\frac{\partial \mathbf{F}}{\partial x_{6}}=\left[\begin{array}{ccc} 
    //& \mathbf{r}_{1} \\
    //0 & 0 & 0 \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{F}}{\partial x_{7}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //& \mathbf{r}_{1} & \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{F}}{\partial x_{8}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //0 & 0 & 0 \\
    //& \mathbf{r}_{1} &
    //\end{array}\right] \\
    //\frac{\partial \mathbf{F}}{\partial x_{9}}=\left[\begin{array}{ccc} 
    //& \mathbf{r}_{2} & 0 \\
    //0 & 0 & \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{F}}{\partial x_{10}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //& \mathbf{r}_{2} & \\
    //0 & 0 & 0
    //\end{array}\right] \quad \frac{\partial \mathbf{F}}{\partial x_{11}}=\left[\begin{array}{ccc}
    //0 & 0 & 0 \\
    //0 & 0 & 0 \\
    //& \mathbf{r}_{2} &
    //\end{array}\right] \\
    //\end{array}
    //$$


    // clang-format on
    const Float m = DmInv(0, 0);
    const Float n = DmInv(0, 1);
    const Float o = DmInv(0, 2);
    const Float p = DmInv(1, 0);
    const Float q = DmInv(1, 1);
    const Float r = DmInv(1, 2);
    const Float s = DmInv(2, 0);
    const Float t = DmInv(2, 1);
    const Float u = DmInv(2, 2);

    const Float t1 = -m - p - s;
    const Float t2 = -n - q - t;
    const Float t3 = -o - r - u;

    Matrix9x12 PFPu = Matrix9x12::Zero();
    PFPu(0, 0)      = t1;
    PFPu(0, 3)      = m;
    PFPu(0, 6)      = p;
    PFPu(0, 9)      = s;
    PFPu(1, 1)      = t1;
    PFPu(1, 4)      = m;
    PFPu(1, 7)      = p;
    PFPu(1, 10)     = s;
    PFPu(2, 2)      = t1;
    PFPu(2, 5)      = m;
    PFPu(2, 8)      = p;
    PFPu(2, 11)     = s;
    PFPu(3, 0)      = t2;
    PFPu(3, 3)      = n;
    PFPu(3, 6)      = q;
    PFPu(3, 9)      = t;
    PFPu(4, 1)      = t2;
    PFPu(4, 4)      = n;
    PFPu(4, 7)      = q;
    PFPu(4, 10)     = t;
    PFPu(5, 2)      = t2;
    PFPu(5, 5)      = n;
    PFPu(5, 8)      = q;
    PFPu(5, 11)     = t;
    PFPu(6, 0)      = t3;
    PFPu(6, 3)      = o;
    PFPu(6, 6)      = r;
    PFPu(6, 9)      = u;
    PFPu(7, 1)      = t3;
    PFPu(7, 4)      = o;
    PFPu(7, 7)      = r;
    PFPu(7, 10)     = u;
    PFPu(8, 2)      = t3;
    PFPu(8, 5)      = o;
    PFPu(8, 8)      = r;
    PFPu(8, 11)     = u;

    return PFPu;
}

MUDA_GENERIC Matrix3x3 F(const Vector3&   x0,
                         const Vector3&   x1,
                         const Vector3&   x2,
                         const Vector3&   x3,
                         const Matrix3x3& DmInv)
{
    auto ds = Ds(x0, x1, x2, x3);
    return ds * DmInv;
}


MUDA_GENERIC void EG_to_lame(Float E, Float G, Float& lambda, Float& mu, Float& poisson)
{
    // ref: https://en.wikipedia.org/wiki/Lam%C3%A9_parameters

    //tex: $\mu = G$
    mu = G;
    //tex: $\lambda = \frac{G(E-2 G)}{3 G-E}$
    lambda = G * (E - 2 * G) / (3 * G - E);
    //tex: $\nu = {\frac {E}{2G}}-1$
    poisson = E / (2 * G) - 1;
}

MUDA_GENERIC void lame_to_EG(Float lambda, Float mu, Float& E, Float& G, Float& poisson)
{
    // ref: https://en.wikipedia.org/wiki/Lam%C3%A9_parameters

    //tex: $G = \mu$
    G = mu;
    //tex: $ E = \frac{G(3 \lambda+2 G)}{\lambda+G}$
    E = G * (3 * lambda + 2 * G) / (lambda + G);
    //tex: $\nu = \frac{\lambda}{2(\lambda+G)}$
    poisson = lambda / (2 * (lambda + G));
}
}  // namespace uipc::backend::cuda
