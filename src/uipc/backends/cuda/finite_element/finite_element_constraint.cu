#include <finite_element/finite_element_constraint.h>

namespace uipc::backend::cuda
{
void FiniteElementConstraint::do_build()
{
    auto& fem_animator = require<FiniteElementAnimator>();

    auto all_uids = world().scene().constitution_tabular().uids();
    if(!std::binary_search(all_uids.begin(), all_uids.end(), uid()))
    {
        throw SimSystemException(fmt::format("Requires Constraint UID={}", uid()));
    }

    BuildInfo info;
    do_build(info);

    fem_animator.add_constraint(this);
}

void FiniteElementConstraint::init(FiniteElementAnimator::FilteredInfo& info)
{
    do_init(info);
}

void FiniteElementConstraint::step(FiniteElementAnimator::FilteredInfo& info)
{
    do_step(info);
}

void FiniteElementConstraint::report_extent(FiniteElementAnimator::ReportExtentInfo& info)
{
    do_report_extent(info);
}

void FiniteElementConstraint::compute_energy(FiniteElementAnimator::ComputeEnergyInfo& info)
{
    do_compute_energy(info);
}

void FiniteElementConstraint::compute_gradient_hessian(FiniteElementAnimator::ComputeGradientHessianInfo& info)
{
    do_compute_gradient_hessian(info);
}

U64 FiniteElementConstraint::uid() const noexcept
{
    return get_uid();
}
}  // namespace uipc::backend::cuda
