#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <uipc/backends/module.h>
#include <uipc/common/log.h>
#include <sim_system_auto_register.h>
#include <log_pattern_guard.h>
#include <global_surface_manager.h>
#include <global_vertex_manager.h>
#include <dof_predictor.h>
#include <line_search/line_searcher.h>
#include <gradient_hessian_computer.h>
#include <linear_system/global_linear_system.h>

namespace uipc::backend::cuda
{
void SimEngine::build()
{
    // find those engine-aware topo systems
    m_global_vertex_manager     = find<GlobalVertexManager>();
    m_global_surface_manager    = find<GlobalSurfaceManager>();
    m_dof_predictor             = find<DoFPredictor>();
    m_line_searcher             = find<LineSearcher>();
    m_gradient_hessian_computer = find<GradientHessianComputer>();
    m_global_linear_system      = find<GlobalLinearSystem>();
}

void SimEngine::init_scene()
{
    auto& info        = m_world_visitor->scene().info();
    m_newton_tol      = info["newton"]["tolerance"];
    m_newton_max_iter = info["newton"]["max_iter"];
    Vector3 gravity   = info["gravity"];
    Float   dt        = info["dt"];
    m_abs_tol         = gravity.norm() * dt * dt / 2;
}

void SimEngine::do_init(backend::WorldVisitor v)
{
    LogGuard guard;

    m_world_visitor = make_unique<backend::WorldVisitor>(v);

    // 1) ConstitutionRegister all systems
    m_state = SimEngineState::RegisterSystems;
    register_all_systems();

    // 2) Build the relationships between systems
    m_state = SimEngineState::BuildSystems;
    for(auto&& [k, s] : m_system_collection.m_sim_systems)
        s->build();
    build();

    // 3) Trigger the init_scene event, systems register their actions will be called here
    m_state = SimEngineState::InitScene;
    {
        init_scene();
        event_init_scene();

        m_global_vertex_manager->init_vertex_info();
        m_global_surface_manager->init_surface_info();
    }

    // 4) Any creation and deletion of objects after this point will be pending
    auto scene_visitor = m_world_visitor->scene();
    scene_visitor.begin_pending();
}
}  // namespace uipc::backend::cuda