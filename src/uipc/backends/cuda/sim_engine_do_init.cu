#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <uipc/backends/module.h>
#include <uipc/common/log.h>
#include <sim_system_auto_register.h>
#include <log_pattern_guard.h>

namespace uipc::backend::cuda
{
void SimEngine::do_init(backend::WorldVisitor v)
{
    LogGuard guard;

    m_world_visitor = std::make_unique<backend::WorldVisitor>(v);

    // 1) Register all systems
    m_state = SimEngineState::RegisterSystems;
    register_all_systems();

    // 2) Build the relationships between systems
    m_state = SimEngineState::BuildSystems;
    for(auto&& [k, s] : m_system_collection.m_sim_systems)
        s->build();

    // 3) Trigger the init_scene event, systems register their actions will be called here
    m_state = SimEngineState::InitScene;
    event_init_scene();

    // 4) Any creation and deletion of objects after this point will be pending
    auto scene_visitor = m_world_visitor->scene();
    scene_visitor.begin_pending();
}
}  // namespace uipc::backend::cuda