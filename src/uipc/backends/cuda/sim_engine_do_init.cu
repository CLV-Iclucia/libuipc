#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <uipc/backends/module.h>
#include <uipc/common/log.h>
#include <sim_system_auto_register.h>
#include <log_pattern_guard.h>

namespace uipc::backend::cuda
{
void SimEngine::do_init(backend::WorldVisitor v)
{
    LogGuard guard;

    spdlog::info("do_init() called.");

    m_world_visitor = std::make_unique<backend::WorldVisitor>(v);

    // 1) Register all systems
    auto& funcs = SimSystemAutoRegister::internal_data().m_entries;
    for(auto& f : funcs)
    {
        auto uptr = f(*this);
        if(uptr)
            m_system_collection.create(std::move(uptr));
    }

    spdlog::info("Registered Systems:\n{}", m_system_collection);

    // 2) Build the relationships between systems
    m_state = SimEngineState::BuildSystems;
    for(auto&& [k, s] : m_system_collection.m_sim_systems)
        s->build();

    // 3) trigger the init_scene event, systems register their actions will be called here
    m_state = SimEngineState::InitScene;
    event_init_scene();
}
}  // namespace uipc::backend::cuda