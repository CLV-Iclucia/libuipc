#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <uipc/backends/module.h>
#include <uipc/common/log.h>
#include <sim_system_auto_register.h>
#include <log_pattern_guard.h>
#include <global_geometry/global_surface_manager.h>
#include <global_geometry/global_vertex_manager.h>
#include <contact_system/global_contact_manager.h>
#include <collision_detection/global_dcd_filter.h>
#include <collision_detection/global_ccd_filter.h>
#include <dof_predictor.h>
#include <line_search/line_searcher.h>
#include <gradient_hessian_computer.h>
#include <linear_system/global_linear_system.h>
#include <uipc/backends/module.h>
#include <fstream>
namespace uipc::backend::cuda
{
void SimEngine::build()
{
    namespace fs = std::filesystem;

    // 1) build all systems
    m_system_collection.build_systems();

    // 2) find those engine-aware topo systems
    m_global_vertex_manager     = find<GlobalVertexManager>();
    m_global_surface_manager    = find<GlobalSimpicialSurfaceManager>();
    m_global_contact_manager    = find<GlobalContactManager>();
    m_global_dcd_filter         = find<GlobalDCDFilter>();
    m_global_ccd_filter         = find<GlobalCCDFilter>();
    m_dof_predictor             = find<DoFPredictor>();
    m_line_searcher             = find<LineSearcher>();
    m_gradient_hessian_computer = find<GradientHessianComputer>();
    m_global_linear_system      = find<GlobalLinearSystem>();

    // 3) dump system info
    auto          workspace = ModuleInfo::instance().workspace();
    fs::path      p = fs::absolute(fs::path{workspace} / "systems.json");
    std::ofstream ofs(p);
    ofs << to_json().dump(4);
    spdlog::info("System info dumped to {}", p.string());

    // 4) clean up invalid systems
    spdlog::info("Cleaning up invalid systems...");
    m_system_collection.cleanup_invalid_systems();
    spdlog::info("Built Systems:\n{}", m_system_collection);
}

void SimEngine::init_scene()
{
    auto& info        = m_world_visitor->scene().info();
    m_newton_tol      = info["newton"]["tolerance"];
    m_newton_max_iter = info["newton"]["max_iter"];
    Vector3 gravity   = info["gravity"];
    Float   dt        = info["dt"];
    m_abs_tol         = gravity.norm() * dt * dt;
    if(m_abs_tol == 0.0)
        m_abs_tol = std::numeric_limits<Float>::max();
}

void SimEngine::do_init(backend::WorldVisitor v)
{
    LogGuard guard;

    m_world_visitor = make_unique<backend::WorldVisitor>(v);

    // 1) ConstitutionRegister all systems
    m_state = SimEngineState::RegisterSystems;
    register_all_systems();

    // 2) Build the relationships between systems
    m_state = SimEngineState::BuildSystems;
    build();

    // 3) Trigger the init_scene event, systems register their actions will be called here
    m_state = SimEngineState::InitScene;
    {
        init_scene();
        event_init_scene();

        m_global_vertex_manager->init_vertex_info();
        m_global_surface_manager->init_surface_info();
        if(m_global_contact_manager)
            m_global_contact_manager->compute_d_hat();
    }

    // 4) Any creation and deletion of objects after this point will be pending
    auto scene_visitor = m_world_visitor->scene();
    scene_visitor.begin_pending();
}
}  // namespace uipc::backend::cuda