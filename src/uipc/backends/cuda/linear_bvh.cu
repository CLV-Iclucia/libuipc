#include "hip/hip_runtime.h"
#include <linear_bvh.h>

#include <muda/cub/device/device_scan.h>
#include <muda/cub/device/device_reduce.h>
#include <muda/cub/device/device_radix_sort.h>

#include <cub/util_ptx.cuh>
#include <cuda/atomic>
#include <muda/atomic.h>
#include <muda/ext/eigen/atomic.h>
#include <thrust/detail/minmax.h>

/*****************************************************************************************
 * Core Implementation
 *****************************************************************************************/
namespace uipc::backend::cuda::detail
{
MUDA_DEVICE MUDA_INLINE int common_upper_bits(const unsigned long long int lhs,
                                              const unsigned long long int rhs) noexcept
{
    return ::__clzll(lhs ^ rhs);
}

MUDA_GENERIC MUDA_INLINE std::uint32_t expand_bits(std::uint32_t v) noexcept
{
    v = (v * 0x00010001u) & 0xFF0000FFu;
    v = (v * 0x00000101u) & 0x0F00F00Fu;
    v = (v * 0x00000011u) & 0xC30C30C3u;
    v = (v * 0x00000005u) & 0x49249249u;
    return v;
}

MUDA_GENERIC MUDA_INLINE std::uint32_t morton_code(Vector3 xyz) noexcept
{
    xyz = xyz.cwiseMin(1.0).cwiseMax(0.0);
    const std::uint32_t xx = expand_bits(static_cast<std::uint32_t>(xyz.x() * 1024.0));
    const std::uint32_t yy = expand_bits(static_cast<std::uint32_t>(xyz.y() * 1024.0));
    const std::uint32_t zz = expand_bits(static_cast<std::uint32_t>(xyz.z() * 1024.0));
    return xx * 4 + yy * 2 + zz;
}

MUDA_DEVICE uint2 determine_range(muda::Dense1D<LinearBVHMortonIndex> node_code,
                                  const uint32_t num_leaves,
                                  uint32_t       idx)
{
    if(idx == 0)
    {
        return make_uint2(0, num_leaves - 1);
    }

    // determine direction of the range
    const auto self_code = node_code(idx);
    const int  L_delta   = common_upper_bits(self_code, node_code(idx - 1));
    const int  R_delta   = common_upper_bits(self_code, node_code(idx + 1));
    const int  d         = (R_delta > L_delta) ? 1 : -1;

    // Compute upper bound for the length of the range

    const int delta_min = thrust::min(L_delta, R_delta);
    int       l_max     = 2;
    int       delta     = -1;
    int       i_tmp     = idx + d * l_max;
    if(0 <= i_tmp && i_tmp < num_leaves)
    {
        delta = common_upper_bits(self_code, node_code(i_tmp));
    }
    while(delta > delta_min)
    {
        l_max <<= 1;
        i_tmp = idx + d * l_max;
        delta = -1;
        if(0 <= i_tmp && i_tmp < num_leaves)
        {
            delta = common_upper_bits(self_code, node_code(i_tmp));
        }
    }

    // Find the other end by binary search
    int l = 0;
    int t = l_max >> 1;
    while(t > 0)
    {
        i_tmp = idx + (l + t) * d;
        delta = -1;
        if(0 <= i_tmp && i_tmp < num_leaves)
        {
            delta = common_upper_bits(self_code, node_code(i_tmp));
        }
        if(delta > delta_min)
        {
            l += t;
        }
        t >>= 1;
    }
    uint32_t jdx = idx + l * d;
    if(d < 0)
    {
        thrust::swap(idx, jdx);  // make it sure that idx < jdx
    }
    return make_uint2(idx, jdx);
}


MUDA_DEVICE uint32_t find_split(muda::Dense1D<LinearBVHMortonIndex> node_code,
                                const uint32_t                      num_leaves,
                                const uint32_t                      first,
                                const uint32_t last) noexcept
{
    const auto first_code = node_code(first);
    const auto last_code  = node_code(last);

    if(first_code == last_code)
    {
        return (first + last) >> 1;
    }
    const int delta_node = common_upper_bits(first_code, last_code);

    // binary search...
    int split  = first;
    int stride = last - first;
    do
    {
        stride           = (stride + 1) >> 1;
        const int middle = split + stride;
        if(middle < last)
        {
            const int delta = common_upper_bits(first_code, node_code(middle));
            if(delta > delta_node)
            {
                split = middle;
            }
        }
    } while(stride > 1);

    return split;
}
}  // namespace uipc::backend::cuda::detail

namespace uipc::backend::cuda
{
void LinearBVH::build(muda::CBufferView<LinearBVHAABB> aabbs, muda::Stream& s)
{
    using namespace muda;

    if(aabbs.size() == 0)
        return;

    const uint32_t num_objects        = aabbs.size();
    const uint32_t num_internal_nodes = num_objects - 1;
    const uint32_t leaf_start         = num_internal_nodes;
    const uint32_t num_nodes          = num_objects * 2 - 1;

    LinearBVHAABB default_aabb;
    resize(s, m_aabbs, num_nodes);
    BufferLaunch(s).fill(m_aabbs.view(), default_aabb);

    resize(s, m_sorted_mortons, num_objects);
    resize(s, m_sorted_mortons, num_objects);

    resize(s, m_indices, num_objects);
    resize(s, m_new_to_old, num_objects);

    resize(s, m_mortons, num_objects);
    resize(s, m_morton_idx, num_objects);

    LinearBVHNode default_node;
    m_nodes.resize(num_nodes, default_node);
    resize(s, m_nodes, num_nodes);

    resize(s, m_flags, num_internal_nodes);
    BufferLaunch(s).fill(m_flags.view(), 0);

    // 1) get max aabb
    DeviceReduce(s).Reduce(
        aabbs.data(),
        m_max_aabb.data(),
        aabbs.size(),
        [] HIPCUB_RUNTIME_FUNCTION(const LinearBVHAABB& a, const LinearBVHAABB& b) noexcept -> LinearBVHAABB
        { return a.merged(b); },
        default_aabb);

    // 2) calculate m_morton_index code
    on(s)
        .next<ParallelFor>()
        .kernel_name("LBVH::MortonCode")
        .apply(num_objects,
               [max_aabb = m_max_aabb.viewer().name("max_aabb"),
                aabbs    = aabbs.viewer().name("filled_aabbs"),
                mortons = m_mortons.viewer().name("mortons")] __device__(int i) mutable
               {
                   Vector3 p = aabbs(i).center();
                   p -= max_aabb->min();
                   p.array() /= max_aabb->sizes().array();
                   mortons(i) = detail::morton_code(p);
               });

    // 3) sort m_morton_index code
    on(s)
        .next<ParallelFor>()
        .kernel_name("LBVH::Iota")
        .apply(m_indices.size(),
               [indices = m_indices.viewer()] __device__(int i) mutable
               { indices(i) = i; });

    // 4) sort m_morton_index code
    DeviceRadixSort(s).SortPairs(m_mortons.data(),
                                 m_sorted_mortons.data(),
                                 m_indices.data(),
                                 m_new_to_old.data(),
                                 num_objects);

    // 5) expand m_morton_index code to 64bit, the last 32bit is the index
    on(s)
        .next<ParallelFor>()
        .kernel_name("LBVH::ExpandMorton")
        .apply(m_mortons.size(),
               [morton64s = m_morton_idx.viewer().name("morton64s"),
                mortons   = m_sorted_mortons.viewer().name("mortons"),
                indices = m_new_to_old.viewer().name("indices")] __device__(int i) mutable
               {
                   MortonIndex morton{mortons(i), indices(i)};
                   morton64s(i) = morton;
               });

    // 6) setup leaf nodes
    auto leaf_aabbs = m_aabbs.view(leaf_start);  // offset = leaf_start
    auto leaf_nodes = m_nodes.view(leaf_start);  // offset = leaf_start
    on(s)
        .next<ParallelFor>()
        .kernel_name("LBVH::SetupLeafNodes")
        .apply(num_objects,
               [leaf_nodes = leaf_nodes.viewer().name("leaf_nodes"),
                indices    = m_new_to_old.viewer().name("indices"),
                aabbs      = aabbs.viewer().name("aabbs"),
                sorted_aabbs = leaf_aabbs.viewer().name("sorted_aabbs")] __device__(int i) mutable
               {
                   LinearBVHNode node;
                   node.parent_idx = 0xFFFFFFFF;
                   node.left_idx   = 0xFFFFFFFF;
                   node.right_idx  = 0xFFFFFFFF;
                   node.object_idx = indices(i);
                   leaf_nodes(i)   = node;
                   sorted_aabbs(i) = aabbs(node.object_idx);
               });

    // 7) construct internal nodes
    on(s)
        .next<ParallelFor>()
        .kernel_name("LBVH::ConstructInternalNodes")
        .apply(num_internal_nodes,
               [nodes      = m_nodes.viewer().name("nodes"),
                morton_idx = m_morton_idx.viewer().name("morton_idx"),
                num_objects] __device__(int idx) mutable
               {
                   nodes(idx).object_idx = 0xFFFFFFFF;  //  internal nodes

                   const uint2 ij = detail::determine_range(morton_idx, num_objects, idx);
                   const int gamma =
                       detail::find_split(morton_idx, num_objects, ij.x, ij.y);

                   nodes(idx).left_idx  = gamma;
                   nodes(idx).right_idx = gamma + 1;
                   if(thrust::min(ij.x, ij.y) == gamma)
                   {
                       nodes(idx).left_idx += num_objects - 1;
                   }
                   if(thrust::max(ij.x, ij.y) == gamma + 1)
                   {
                       nodes(idx).right_idx += num_objects - 1;
                   }
                   nodes(nodes(idx).left_idx).parent_idx  = idx;
                   nodes(nodes(idx).right_idx).parent_idx = idx;
               });

    // 8) calculate the AABB of internal nodes
    auto internal_aabbs = m_aabbs.view(0, num_internal_nodes);
    on(s)
        .next<ParallelFor>()
        .kernel_name("LBVH::CalculateInternalAABB")
        .apply(num_objects,
               [nodes = m_nodes.cviewer().name("nodes"),
                aabbs = m_aabbs.viewer().name("aabbs"),
                flags = m_flags.viewer().name("flags"),
                leaf_start] __device__(int I) mutable
               {
                   auto leaf_idx = I + leaf_start;
                   auto parent   = nodes(leaf_idx).parent_idx;

                   while(parent != 0xFFFFFFFF)  // means idx == 0
                   {
                       const int old = muda::atomic_add(&flags(parent), 1);

                       // the memory fence is necessary to disable reordering of the memory access.
                       // we need to ensure that this thread can get the updated value of AABB.
                       ::cuda::atomic_thread_fence(::cuda::memory_order_acquire,
                                                   ::cuda::thread_scope_system);

                       if(old == 0)
                       {
                           // this is the first thread entered here.
                           // wait the other thread from the other child node.
                           return;
                       }
                       MUDA_KERNEL_ASSERT(old == 1, "old=%d", old);
                       //here, the flag has already been 1. it means that this
                       //thread is the 2nd thread. merge AABB of both childlen.

                       const auto lidx = nodes(parent).left_idx;
                       const auto ridx = nodes(parent).right_idx;
                       auto&      lbox = aabbs(lidx);
                       auto&      rbox = aabbs(ridx);

                       // to avoid cache coherency problem, we must use atomic operation.
                       auto atomic_fetch = [](LinearBVHAABB& aabb) -> LinearBVHAABB
                       {
                           Vector3       zero  = Vector3::Zero();
                           LinearBVHAABB aabb_ = aabb;

                           // without atomic_thread_fence, this loop may be infinite.
                           while(aabb_.isEmpty())
                           {
                               Vector3 min_ = eigen::atomic_add(aabb.min(), zero);
                               Vector3 max_ = eigen::atomic_add(aabb.max(), zero);
                               aabb_ = LinearBVHAABB{min_, max_};
                           };

                           return aabb_;
                       };

                       auto L = atomic_fetch(lbox);
                       auto R = atomic_fetch(rbox);

                       aabbs(parent) = L.merged(R);

                       // look the next parent...
                       parent = nodes(parent).parent_idx;
                   }
               })
        .wait();
}
}  // namespace uipc::backend::cuda


/*****************************************************************************************
 * API Implementation
 *****************************************************************************************/
namespace uipc::backend::cuda
{
LinearBVH::LinearBVH(const LinearBVHConfig& config) noexcept {}

LinearBVHViewer LinearBVH::viewer() noexcept
{
    return LinearBVHViewer{(uint32_t)m_nodes.size(),
                           (uint32_t)m_mortons.size(),
                           m_nodes.data(),
                           m_aabbs.data()};
}

CLinearBVHViewer LinearBVH::viewer() const noexcept
{
    return CLinearBVHViewer{(uint32_t)m_nodes.size(),
                            (uint32_t)m_mortons.size(),
                            m_nodes.data(),
                            m_aabbs.data()};
}

LinearBVHVisitor::LinearBVHVisitor(LinearBVH& bvh) noexcept
    : m_bvh(bvh)
{
}

muda::CBufferView<LinearBVHNode> LinearBVHVisitor::nodes() const noexcept
{
    return m_bvh.m_nodes;
}

muda::CBufferView<LinearBVHNode> LinearBVHVisitor::object_nodes() const noexcept
{
    auto object_count = m_bvh.m_indices.size();
    auto node_offset  = m_bvh.m_nodes.size() - object_count;
    return m_bvh.m_nodes.view(node_offset);
}

muda::CBufferView<LinearBVHAABB> LinearBVHVisitor::aabbs() const noexcept
{
    return m_bvh.m_aabbs;
}

muda::CVarView<LinearBVHAABB> LinearBVHVisitor::top_aabb() const noexcept
{
    return m_bvh.m_max_aabb;
}
}  // namespace uipc::backend::cuda