#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <log_pattern_guard.h>
#include <dof_predictor.h>
#include <global_geometry/global_vertex_manager.h>
#include <global_geometry/global_simplicial_surface_manager.h>
#include <contact_system/global_contact_manager.h>
#include <collision_detection/global_trajectory_filter.h>
#include <line_search/line_searcher.h>
#include <gradient_hessian_computer.h>
#include <linear_system/global_linear_system.h>
#include <uipc/common/range.h>

namespace uipc::backend::cuda
{
void SimEngine::do_advance()
{
    auto pipeline = [this]
    {
        LogGuard guard;

        ++m_current_frame;

        spdlog::info(R"(>>> Begin Frame: {})", m_current_frame);

        Float alpha     = 1.0;
        Float ccd_alpha = 1.0;
        Float cfl_alpha = 1.0;

        auto detect_dcd_candidates = [this]
        {
            if(m_global_trajectory_filter)
            {
                m_global_trajectory_filter->detect(0.0);
                m_global_trajectory_filter->filter_active();
            }
        };

        auto detect_trajectory_candidates = [this](Float alpha)
        {
            if(m_global_trajectory_filter)
            {
                m_global_trajectory_filter->detect(alpha);
            }
        };

        auto filter_dcd_candidates = [this]
        {
            if(m_global_trajectory_filter)
            {
                m_global_trajectory_filter->filter_active();
            }
        };

        auto record_friction_candidates = [this]
        {
            if(m_global_trajectory_filter)
            {
                m_global_trajectory_filter->record_friction_candidates();
            }
        };

        auto compute_adaptive_kappa = [this]
        {
            if(m_global_contact_manager)
                m_global_contact_manager->compute_adaptive_kappa();
        };

        auto compute_contact = [this]
        {
            if(m_global_contact_manager)
                m_global_contact_manager->compute_contact();
        };

        auto cfl_condition = [&cfl_alpha, this](Float alpha)
        {
            //if(m_global_contact_manager)
            //{
            //    auto max_disp = m_global_vertex_manager->compute_max_displacement_norm();
            //    auto d_hat = m_global_contact_manager->d_hat();

            //    cfl_alpha = d_hat / max_disp;
            //    spdlog::info("CFL Condition: {} / {}, max dx: {}", cfl_alpha, alpha, max_disp);

            //    if(cfl_alpha < alpha)
            //    {
            //        return cfl_alpha;
            //    }
            //}
            return alpha;
        };

        auto filter_toi = [&ccd_alpha, this](Float alpha)
        {
            if(m_global_trajectory_filter)
            {
                ccd_alpha = m_global_trajectory_filter->filter_toi(alpha);
                if(ccd_alpha < alpha)
                {
                    spdlog::info("CCD Filter: {} < {}", ccd_alpha, alpha);
                    return ccd_alpha;
                }
            }
            // return ccd_alpha;
            return alpha;
        };

        auto compute_energy = [this, filter_dcd_candidates](Float alpha) -> Float
        {
            // Step Forward => x = x_0 + alpha * dx
            spdlog::info("Step Forward: {}", alpha);
            m_global_vertex_manager->step_forward(alpha);
            m_line_searcher->step_forward(alpha);

            // Update the collision pairs
            filter_dcd_candidates();

            // Compute New Energy => E
            return m_line_searcher->compute_energy(false);
        };

        /***************************************************************************************
        *                                  Core Pipeline
        ***************************************************************************************/

        {
            // 1. Adaptive Parameter Calculation
            AABB vertex_bounding_box =
                m_global_vertex_manager->compute_vertex_bounding_box();
            detect_dcd_candidates();
            compute_adaptive_kappa();

            // 2. Record Friction Candidates at the beginning of the frame
            record_friction_candidates();

            // 3. Predict Motion => x_tilde = x + v * dt
            m_state = SimEngineState::PredictMotion;
            m_dof_predictor->predict();

            // 4. Nonlinear-Newton Iteration
            Float box_size = vertex_bounding_box.diagonal().norm();
            Float tol      = m_newton_tol * box_size;
            Float res0     = 0.0;

            for(auto&& newton_iter : range(m_newton_max_iter))
            {
                // 1) Build Collision Pairs
                if(newton_iter > 0)
                    detect_dcd_candidates();

                // 2) Compute Contact Gradient and Hessian => G:Vector3, H:Matrix3x3
                m_state = SimEngineState::ComputeContact;
                compute_contact();

                // 3) Compute System Gradient and Hessian
                m_state = SimEngineState::ComputeGradientHessian;
                m_gradient_hessian_computer->compute_gradient_hessian();

                // 4) Solve Global Linear System => dx = A^-1 * b
                m_state = SimEngineState::SolveGlobalLinearSystem;
                m_global_linear_system->solve();

                // 5) Get Max Movement => dx_max = max(|dx|), if dx_max < tol, break
                m_global_vertex_manager->collect_vertex_displacements();
                Float res = m_global_vertex_manager->compute_axis_max_displacement();

                if(newton_iter == 0)
                    res0 = res;

                Float rel_tol = res == 0.0 ? 0.0 : res / res0;

                spdlog::info(">> Newton Iteration: {}. Residual/Tol/AbsTol/RelTol: {}/{}/{}/{}",
                             newton_iter,
                             res,
                             tol,
                             m_abs_tol,
                             rel_tol);

                // 6) Check Termination Condition
                // TODO: Maybe we can implement a class for termination condition in the future
                if(res <= tol && (res < m_abs_tol || res <= 1e-2 * res0))
                    break;

                // 7) Begin Line Search
                m_state = SimEngineState::LineSearch;
                {
                    alpha = 1.0;

                    // Record Current State x to x_0
                    m_line_searcher->record_start_point();
                    m_global_vertex_manager->record_start_point();
                    detect_trajectory_candidates(alpha);

                    // Compute Current Energy => E_0
                    Float E0 = m_line_searcher->compute_energy(true);  // initial energy

                    // CCD filter
                    alpha = filter_toi(alpha);

                    // CFL condition
                    alpha = cfl_condition(alpha);

                    // Compute Test Energy => E
                    Float E = compute_energy(alpha);

                    SizeT line_search_iter = 0;
                    while(line_search_iter++ < m_line_searcher->max_iter())  // Energy Test
                    {
                        bool energy_decrease = E <= E0;  // Check Energy Decrease

                        // TODO: Intersection & Inversion Check
                        bool no_inversion = true;

                        spdlog::info("Line Search Iteration: {} Alpha: {}, E/E0: {}, E0: {}",
                                     line_search_iter,
                                     alpha,
                                     E / E0,
                                     E0);

                        bool success = energy_decrease && no_inversion;
                        if(success)
                            break;

                        // If not success, then shrink alpha
                        alpha /= 2;
                        E = compute_energy(alpha);
                    }

                    if(line_search_iter >= m_line_searcher->max_iter())
                    {
                        spdlog::warn(
                            "Line Search Exits with Max Iteration: {} (Frame={}, Newton={})\n"
                            "E/E0: {}, E0:{}",
                            m_line_searcher->max_iter(),
                            m_current_frame,
                            newton_iter,
                            E / E0,
                            E0);
                    }
                }
            }

            // 5. Update Velocity => v = (x - x_0) / dt
            m_state = SimEngineState::UpdateVelocity;
            m_dof_predictor->compute_velocity();
            m_global_vertex_manager->record_prev_positions();
        }

        // Trigger the rebuild_scene event, systems register their actions will be called here
        m_state = SimEngineState::RebuildScene;
        {
            event_rebuild_scene();

            // TODO: rebuild the vertex and surface info
            // m_global_vertex_manager->rebuild_vertex_info();
            // m_global_surface_manager->rebuild_surface_info();
        }

        // After the rebuild_scene event, the pending creation or deletion can be solved
        auto scene = m_world_visitor->scene();
        scene.solve_pending();

        spdlog::info("<<< End Frame: {}", m_current_frame);
    };

    try
    {
        pipeline();
    }
    catch(const SimEngineException& e)
    {
        spdlog::error("Simulation Engine Exception: {}", e.what());
    }
}
}  // namespace uipc::backend::cuda
