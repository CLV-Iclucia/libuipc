#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <log_pattern_guard.h>
#include <dof_predictor.h>
#include <global_geometry/global_vertex_manager.h>
#include <global_geometry/global_surface_manager.h>
#include <contact_system/global_contact_manager.h>
#include <collision_detection/global_dcd_filter.h>
#include <collision_detection/global_ccd_filter.h>
#include <line_search/line_searcher.h>
#include <gradient_hessian_computer.h>
#include <linear_system/global_linear_system.h>
#include <uipc/common/range.h>

namespace uipc::backend::cuda
{
void SimEngine::do_advance()
{
    auto pipeline = [this]
    {
        LogGuard guard;

        ++m_current_frame;

        spdlog::info(R"(>>> Begin Frame: {})", m_current_frame);

        Float alpha     = 1.0;
        Float ccd_alpha = 1.0;
        Float cfl_alpha = 1.0;

        auto detect_dcd_candidates = [this]
        {
            if(m_global_dcd_filter)
                m_global_dcd_filter->detect();
        };

        auto compute_contact = [this]
        {
            if(m_global_contact_manager)
                m_global_contact_manager->compute_contact();
        };

        auto cfl_condition = [&cfl_alpha, this](Float alpha)
        {
            if(m_global_contact_manager)
            {
                auto max_disp = m_global_vertex_manager->compute_max_displacement_norm();
                auto d_hat = m_global_contact_manager->d_hat();

                cfl_alpha = d_hat / max_disp;
                spdlog::info("CFL Condition: {} / {}, max dx: {}", cfl_alpha, alpha, max_disp);

                if(cfl_alpha < alpha)
                {
                    return cfl_alpha;
                }
            }
            return alpha;
        };

        auto filter_toi = [&ccd_alpha, this](Float alpha)
        {
            if(m_global_ccd_filter)
            {
                ccd_alpha = m_global_ccd_filter->filter_toi(alpha);
                if(ccd_alpha < alpha)
                {
                    spdlog::info("CCD Filter: {} < {}", ccd_alpha, alpha);
                    return ccd_alpha;
                }
            }
            return alpha;
        };

        auto compute_energy = [this, detect_dcd_candidates](Float alpha) -> Float
        {
            // Step Forward => x = x_0 + alpha * dx
            spdlog::info("Step Forward : {}", alpha);
            m_global_vertex_manager->step_forward(alpha);
            m_line_searcher->step_forward(alpha);

            // Update the collision pairs
            detect_dcd_candidates();

            // Compute New Energy => E
            return m_line_searcher->compute_energy();
        };

        /***************************************************************************************
        *                                  Core Pipeline
        ***************************************************************************************/

        {
            // 1. Adaptive Parameter Calculation
            AABB vertex_bounding_box =
                m_global_vertex_manager->compute_vertex_bounding_box();
            detect_dcd_candidates();
            if(m_global_contact_manager)
                m_global_contact_manager->compute_adaptive_kappa();

            // 2. Predict Motion => x_tilde = x + v * dt
            m_state = SimEngineState::PredictMotion;
            m_dof_predictor->predict();

            // 3. Nonlinear-Newton Iteration
            Float box_size = vertex_bounding_box.diagonal().norm();
            Float tol      = m_newton_tol * box_size;
            Float res0     = 0.0;

            for(auto&& iter : range(m_newton_max_iter))
            {
                // 1) Build Collision Pairs
                if(iter > 0)
                    detect_dcd_candidates();


                // 2) Compute Contact Gradient and Hessian => G:Vector3, H:Matrix3x3
                m_state = SimEngineState::ComputeContact;
                compute_contact();

                // 3) Compute System Gradient and Hessian
                m_state = SimEngineState::ComputeGradientHessian;
                m_gradient_hessian_computer->compute_gradient_hessian();


                // 4) Solve Global Linear System => dx = A^-1 * b
                m_state = SimEngineState::SolveGlobalLinearSystem;
                m_global_linear_system->solve();

                // 5) Get Max Movement => dx_max = max(|dx|), if dx_max < tol, break
                m_global_vertex_manager->collect_vertex_displacements();
                Float res = m_global_vertex_manager->compute_axis_max_displacement();

                if(iter == 0)
                    res0 = res;

                Float rel_tol = res == 0.0 ? 0.0 : res / res0;

                spdlog::info(">> Newton Iteration: {}. Residual/Tol/AbsTol/RelTol: {}/{}/{}/{}",
                             iter,
                             res,
                             tol,
                             m_abs_tol,
                             rel_tol);

                // 6) Check Termination Condition
                // TODO: Maybe we can implement a class for termination condition in the future
                if(res <= tol && (res < m_abs_tol || res <= 1e-2 * res0))
                    break;

                // 7) Begin Line Search
                m_state = SimEngineState::LineSearch;
                {
                    alpha = 1.0;

                    // Record Current State x to x_0
                    m_line_searcher->record_start_point();
                    m_global_vertex_manager->record_start_point();

                    // Compute Current Energy => E_0
                    Float E0 = m_line_searcher->compute_energy();

                    // CCD filter
                    alpha = filter_toi(alpha);

                    // CFL condition
                    alpha = cfl_condition(alpha);

                    // Compute Test Energy => E
                    Float E = compute_energy(alpha);

                    SizeT max_line_search_iter = 1000;  // now just hard code it
                    SizeT line_search_iter     = 0;
                    while(line_search_iter++ < max_line_search_iter)  // Energy Test
                    {
                        bool energy_decrease = E <= E0;  // Check Energy Decrease
                        bool no_inversion = true;        // Check Inversion

                        spdlog::info("Line Search Iteration: {} Alpha: {}, E/E0: {}",
                                     line_search_iter,
                                     alpha,
                                     E / E0);

                        bool success = energy_decrease && no_inversion;
                        if(success)
                            break;

                        // If not success, then shrink alpha
                        alpha /= 2;
                        E = compute_energy(alpha);
                    }
                }
            }

            // 4. Update Velocity => v = (x - x_0) / dt
            m_state = SimEngineState::UpdateVelocity;
            m_dof_predictor->compute_velocity();
            m_global_vertex_manager->record_prev_positions();
        }

        // Trigger the rebuild_scene event, systems register their actions will be called here
        m_state = SimEngineState::RebuildScene;
        {
            event_rebuild_scene();

            // TODO: rebuild the vertex and surface info
            // m_global_vertex_manager->rebuild_vertex_info();
            // m_global_surface_manager->rebuild_surface_info();
        }

        // After the rebuild_scene event, the pending creation or deletion can be solved
        auto scene = m_world_visitor->scene();
        scene.solve_pending();

        spdlog::info("<<< End Frame: {}", m_current_frame);
    };

    try
    {
        pipeline();
    }
    catch(const SimEngineException& e)
    {
        spdlog::error("Exception: {}", e.what());
    }
}
}  // namespace uipc::backend::cuda
