#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <uipc/backends/module.h>
#include <uipc/common/log.h>
#include <log_pattern_guard.h>
#include <global_geometry/global_surface_manager.h>
#include <global_geometry/global_vertex_manager.h>
#include <contact_system/global_contact_manager.h>
#include <collision_detection/global_dcd_filter.h>
#include <collision_detection/global_ccd_filter.h>
#include <dof_predictor.h>
#include <line_search/line_searcher.h>
#include <gradient_hessian_computer.h>
#include <linear_system/global_linear_system.h>
#include <uipc/backends/module.h>
#include <fstream>

namespace uipc::backend::cuda
{
void SimEngine::build()
{
    // 1) build all systems
    build_systems();

    m_on_init_scene.init();
    m_on_rebuild_scene.init();
    m_on_write_scene.init();

    // 2) find those engine-aware topo systems
    m_global_vertex_manager     = &require<GlobalVertexManager>();
    m_dof_predictor             = &require<DoFPredictor>();
    m_line_searcher             = &require<LineSearcher>();
    m_gradient_hessian_computer = &require<GradientHessianComputer>();
    m_global_linear_system      = &require<GlobalLinearSystem>();

    m_global_surface_manager = find<GlobalSimpicialSurfaceManager>();
    m_global_contact_manager = find<GlobalContactManager>();
    m_global_dcd_filter      = find<GlobalDCDFilter>();
    m_global_ccd_filter      = find<GlobalCCDFilter>();

    // 3) dump system info
    dump_system_info();
}

void SimEngine::init_scene()
{
    auto& info        = m_world_visitor->scene().info();
    m_newton_tol      = info["newton"]["tolerance"];
    m_newton_max_iter = info["newton"]["max_iter"];
    Vector3 gravity   = info["gravity"];
    Float   dt        = info["dt"];

    m_abs_tol         = gravity.norm() * dt * dt;
    if(m_abs_tol == 0.0)
        m_abs_tol = std::numeric_limits<Float>::max();
}

void SimEngine::do_init(backend::WorldVisitor v)
{
    LogGuard guard;

    m_world_visitor = make_unique<backend::WorldVisitor>(v);

    // 1. Build all the systems and their dependencies
    m_state = SimEngineState::BuildSystems;
    build();

    // 2. Trigger the init_scene event, systems register their actions will be called here
    m_state = SimEngineState::InitScene;
    {
        init_scene();
        event_init_scene();

        m_global_vertex_manager->init_vertex_info();
        m_global_surface_manager->init_surface_info();
        if(m_global_contact_manager)
            m_global_contact_manager->compute_d_hat();
    }

    // 3. Any creation and deletion of objects after this point will be pending
    auto scene_visitor = m_world_visitor->scene();
    scene_visitor.begin_pending();
}
}  // namespace uipc::backend::cuda