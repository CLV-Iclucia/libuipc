#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <uipc/common/log.h>
#include <log_pattern_guard.h>
#include <global_geometry/global_simplicial_surface_manager.h>
#include <global_geometry/global_vertex_manager.h>
#include <contact_system/global_contact_manager.h>
#include <collision_detection/global_trajectory_filter.h>
#include <dof_predictor.h>
#include <line_search/line_searcher.h>
#include <gradient_hessian_computer.h>
#include <linear_system/global_linear_system.h>
#include <fstream>

namespace uipc::backend::cuda
{
void SimEngine::build()
{
    // 1) build all systems
    build_systems();

    // 2) find those engine-aware topo systems
    m_global_vertex_manager     = &require<GlobalVertexManager>();
    m_dof_predictor             = &require<DoFPredictor>();
    m_line_searcher             = &require<LineSearcher>();
    m_gradient_hessian_computer = &require<GradientHessianComputer>();
    m_global_linear_system      = &require<GlobalLinearSystem>();

    m_global_simplicial_surface_manager   = find<GlobalSimpicialSurfaceManager>();
    m_global_contact_manager   = find<GlobalContactManager>();
    m_global_trajectory_filter = find<GlobalTrajectoryFilter>();

    // 3) dump system info
    dump_system_info();
}

void SimEngine::init_scene()
{
    auto& info            = m_world_visitor->scene().info();
    m_newton_velocity_tol = info["newton"]["velocity_tol"];
    m_newton_max_iter     = info["newton"]["max_iter"];
    m_friction_enabled    = info["contact"]["friction"]["enable"];
    Vector3 gravity       = info["gravity"];
    Float   dt            = info["dt"];

    m_abs_tol = m_newton_velocity_tol * dt;

    // early init:
    [[maybe_unuse]] m_on_init_scene.view();
    [[maybe_unuse]] m_on_rebuild_scene.view();
    [[maybe_unuse]] m_on_write_scene.view();

    event_init_scene();

    // some systems need to be initialized after the scene is built
    m_global_vertex_manager->init_vertex_info();
    m_global_simplicial_surface_manager->init_surface_info();
    if(m_global_contact_manager)
        m_global_contact_manager->compute_d_hat();
}

void SimEngine::do_init(backend::WorldVisitor v)
{
    LogGuard guard;
    try
    {
        m_world_visitor = make_unique<backend::WorldVisitor>(v);

        // 1. Build all the systems and their dependencies
        m_state = SimEngineState::BuildSystems;
        build();

        // 2. Trigger the init_scene event, systems register their actions will be called here
        m_state = SimEngineState::InitScene;
        init_scene();

        // 3. Any creation and deletion of objects after this point will be pending
        auto scene_visitor = m_world_visitor->scene();
        scene_visitor.begin_pending();
    }
    catch(const SimEngineException& e)
    {
        spdlog::error("SimEngine init error: {}", e.what());
    }
}
}  // namespace uipc::backend::cuda