#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <uipc/common/log.h>
#include <muda/muda.h>
#include <kernel_cout.h>
#include <sim_engine_device_common.h>
#include <log_pattern_guard.h>

namespace uipc::backend::cuda
{
void say_hello_from_muda()
{
    using namespace muda;

    Launch()
        .apply([] __device__()
               { cout << "CUDA Backend Kernel Console Init Success!\n"; })
        .wait();
}

SimEngine::SimEngine()
    : m_device_impl(make_unique<DeviceImpl>())
{
    LogGuard guard;
    try
    {
        using namespace muda;

        spdlog::info("Cuda Backend Init Success.");

        auto viewer_ptr       = device_logger_viewer_ptr();
        m_device_impl->logger = make_unique<muda::Logger>(viewer_ptr);

        Debug::set_sync_callback(
            [this]
            {
                m_string_stream.str("");
                m_device_impl->logger->retrieve(m_string_stream);
                if(m_string_stream.str().empty())
                    return;

                spdlog::info(R"( 
-------------------------------------------------------------------------------
*                               Kernel  Console                               *
-------------------------------------------------------------------------------
{}
-------------------------------------------------------------------------------)",
                             m_string_stream.str());
            });

        say_hello_from_muda();

#ifndef NDEBUG
        // if in debug mode, sync all the time to check for errors
        muda::Debug::debug_sync_all(true);
#endif
    }
    catch(SimEngineException& e)
    {
        spdlog::error("Cuda Backend Init Failed: {}", e.what());
    }
}

SimEngine::~SimEngine()
{
    LogGuard guard;

    muda::wait_device();

    // remove the sync callback
    muda::Debug::set_sync_callback(nullptr);

    spdlog::info("Cuda Backend Shutdown Success.");
}

auto SimEngine::device_impl() noexcept -> DeviceImpl&
{
    return *m_device_impl;
}

WorldVisitor& SimEngine::world() noexcept
{
    UIPC_ASSERT(m_world_visitor, "WorldVisitor is not initialized.");
    return *m_world_visitor;
}

SimEngineState SimEngine::state() const noexcept
{
    return m_state;
}

void SimEngine::event_init_scene()
{
    for(auto& action : m_on_init_scene.view())
        action();
}

void SimEngine::event_rebuild_scene()
{
    for(auto& action : m_on_rebuild_scene.view())
        action();
}

void SimEngine::event_write_scene()
{
    for(auto& action : m_on_write_scene.view())
        action();
}
}  // namespace uipc::backend::cuda
