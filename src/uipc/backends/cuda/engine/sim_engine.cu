#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <uipc/common/log.h>
#include <muda/muda.h>
#include <kernel_cout.h>
#include <sim_engine_device_common.h>
#include <log_pattern_guard.h>
#include <uipc/backends/common/module.h>
#include <global_geometry/global_vertex_manager.h>
#include <global_geometry/global_surface_manager.h>
#include <fstream>

namespace uipc::backend::cuda
{
void say_hello_from_muda()
{
    using namespace muda;

    Launch()
        .apply([] __device__()
               { cout << "CUDA Backend Kernel Console Init Success!\n"; })
        .wait();
}

SimEngine::SimEngine()
    : m_device_impl(make_unique<DeviceImpl>())
{
    LogGuard guard;
    try
    {
        using namespace muda;

        spdlog::info("Initializing Cuda Backend...");

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        spdlog::info("Device: {}", prop.name);
        spdlog::info("Compute Capability: {}.{}", prop.major, prop.minor);

        auto viewer_ptr       = device_logger_viewer_ptr();
        m_device_impl->logger = make_unique<muda::Logger>(viewer_ptr);

        Debug::set_sync_callback(
            [this]
            {
                m_string_stream.str("");
                m_device_impl->logger->retrieve(m_string_stream);
                if(m_string_stream.str().empty())
                    return;

                spdlog::info(R"( 
-------------------------------------------------------------------------------
*                               Kernel  Console                               *
-------------------------------------------------------------------------------
{}
-------------------------------------------------------------------------------)",
                             m_string_stream.str());
            });

        say_hello_from_muda();

#ifndef NDEBUG
        // if in debug mode, sync all the time to check for errors
        muda::Debug::debug_sync_all(true);
#endif
        spdlog::info("Cuda Backend Init Success.");
    }
    catch(const SimEngineException& e)
    {
        spdlog::error("Cuda Backend Init Failed: {}", e.what());
    }
}

SimEngine::~SimEngine()
{
    LogGuard guard;

    muda::wait_device();

    // remove the sync callback
    muda::Debug::set_sync_callback(nullptr);

    spdlog::info("Cuda Backend Shutdown Success.");
}

auto SimEngine::device_impl() noexcept -> DeviceImpl&
{
    return *m_device_impl;
}

WorldVisitor& SimEngine::world() noexcept
{
    UIPC_ASSERT(m_world_visitor, "WorldVisitor is not initialized.");
    return *m_world_visitor;
}

SimEngineState SimEngine::state() const noexcept
{
    return m_state;
}

void SimEngine::event_init_scene()
{
    for(auto& action : m_on_init_scene.view())
        action();
}

void SimEngine::event_rebuild_scene()
{
    for(auto& action : m_on_rebuild_scene.view())
        action();
}

void SimEngine::event_write_scene()
{
    for(auto& action : m_on_write_scene.view())
        action();
}

void uipc::backend::cuda::SimEngine::dump_global_surface(std::string_view name)
{
    auto path      = ModuleInfo::instance().workspace();
    auto file_path = fmt::format("{}{}.obj", path, name);

    std::vector<Vector3> positions;
    auto                 src_ps = m_global_vertex_manager->positions();
    positions.resize(src_ps.size());
    src_ps.copy_to(positions.data());

    std::vector<Vector3i> faces;
    auto                  src_fs = m_global_surface_manager->surf_triangles();
    faces.resize(src_fs.size());
    src_fs.copy_to(faces.data());

    std::ofstream file(file_path);

    for(auto& pos : positions)
        file << fmt::format("v {} {} {}\n", pos.x(), pos.y(), pos.z());

    for(auto& face : faces)
        file << fmt::format("f {} {} {}\n", face.x() + 1, face.y() + 1, face.z() + 1);

    spdlog::info("Dumped global surface to {}", file_path);
}

bool SimEngine::do_dump()
{
    auto path = dump_path();

    Json j     = Json::object();
    j["frame"] = m_current_frame;

    {
        std::ofstream file(path + "state.json");
        file << j.dump(4);
    }

    return backend::SimEngine::do_dump();
}

bool SimEngine::do_recover()
{
    auto path = dump_path();

    bool success = false;

    do
    {
        Json j;
        {
            std::ifstream file(path + "state.json");
            if(file.is_open())
            {
                file >> j;
            }
            else
            {
                spdlog::warn("Failed to open state.json when recovering, so skip.");
                break;
            }
        }

        if(!backend::SimEngine::do_recover())
            break;

        bool has_error = false;
        try
        {
            j["frame"].get<SizeT>();
        }
        catch(std::exception e)
        {
            has_error = true;
            spdlog::warn("Failed to retrieve data from state.json when recovering, so skip. Reason: {}",
                         e.what());
        }
        if(has_error)
            break;

        m_current_frame = j["frame"];
        spdlog::info("Recover at frame: {}", m_current_frame);

        success = true;
    } while(0);

    return success;
}
}  // namespace uipc::backend::cuda
