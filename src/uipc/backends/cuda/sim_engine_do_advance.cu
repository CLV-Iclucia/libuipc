#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <log_pattern_guard.h>
#include <dof_predictor.h>
#include <global_geometry/global_vertex_manager.h>
#include <global_geometry/global_surface_manager.h>
#include <contact_system/global_contact_manager.h>
#include <collision_detection/global_collision_detector.h>
#include <collision_detection/global_ccd_filter.h>
#include <line_search/line_searcher.h>
#include <gradient_hessian_computer.h>
#include <linear_system/global_linear_system.h>
#include <uipc/common/range.h>

namespace uipc::backend::cuda
{
void SimEngine::do_advance()
{
    LogGuard guard;

    auto detect_candiates = [this](Float alpha)
    {
        if(m_global_collision_detector)
            m_global_collision_detector->detect_candidates(alpha);
    };

    auto compute_energy = [this](Float alpha) -> Float
    {
        // Step Forward => x = x_0 + alpha * dx
        m_line_searcher->step_forward(alpha);
        m_global_vertex_manager->step_forward(alpha);
        // Compute New Energy => E
        return m_line_searcher->compute_energy();
    };

    auto filter_toi = [this](Float alpha) -> Float
    {
        if(m_global_ccd_filter)
            return m_global_ccd_filter->filter_toi(alpha);
        return alpha;
    };

    // The Pipeline
    {
        // 1. Adaptive Parameter Calculation
        AABB vertex_bounding_box = m_global_vertex_manager->compute_vertex_bounding_box();
        detect_candiates(0.0);
        if(m_global_contact_manager)
            m_global_contact_manager->compute_adaptive_kappa();

        // 2. Predict Motion => x_tilde = x + v * dt
        m_state = SimEngineState::PredictMotion;
        m_dof_predictor->predict();

        // 3. Nonlinear-Newton Iteration
        Float box_size = vertex_bounding_box.diagonal().norm();
        Float tol      = std::min(m_newton_tol * box_size, m_abs_tol);

        for(auto&& iter : range(m_newton_max_iter))
        {
            // 1) Build Collision Pairs
            if(iter > 0)
                detect_candiates(0.0);

            m_state = SimEngineState::ComputeGradientHassian;
            // 2) Compute Contact Gradient and Hessian => G:Vector3, H:Matrix3x3
            if(m_global_contact_manager)
                m_global_contact_manager->compute_contact();

            // 3) Compute System Gradient and Hessian => G:Vector3, H:Matrix3x3
            // E.g. FEM/ABD ...
            m_gradient_hessian_computer->compute_gradient_hessian();


            m_state = SimEngineState::SolveGlobalLinearSystem;
            // 4) Solve Global Linear System => dx = A^-1 * b
            m_global_linear_system->solve();

            // 5) Get Max Movement => dx_max = max(|dx|), if dx_max < tol, break
            Float res = m_global_vertex_manager->compute_max_displacement();
            spdlog::info("Newton Iteration: {} Residual: {}/{}", iter, res, tol);
            if(res < tol)
                break;

            m_state = SimEngineState::LineSearch;
            // 6) Begin Line Search
            {
                Float alpha = 1.0;

                // update the candidates by the step length
                // so now we take in all the candidates that
                // can happen in the whole step
                detect_candiates(alpha);

                // Record Current State x to x_0
                m_line_searcher->record_start_point();
                m_global_vertex_manager->record_start_point();

                // Compute Current Energy => E_0
                Float E0 = m_line_searcher->compute_energy();

                // ccd filter
                alpha = filter_toi(alpha);

                SizeT max_line_search_iter = 1000;  // now just hard code it
                SizeT line_search_iter     = 0;
                while(line_search_iter++ < max_line_search_iter)  // Energy Test
                {
                    Float E = compute_energy(alpha);

                    bool energy_decrease = E < E0;  // Check Energy Decrease
                    bool no_inversion    = true;    // Check Inversion

                    bool success = energy_decrease && no_inversion;
                    if(success)
                        break;

                    // If not success, then shrink alpha
                    alpha /= 2;
                    E = compute_energy(alpha);
                }
            }
        }

        // 4. Update Velocity => v = (x - x_0) / dt
        m_state = SimEngineState::UpdateVelocity;
        m_dof_predictor->compute_velocity();
    }

    // Trigger the rebuild_scene event, systems register their actions will be called here
    m_state = SimEngineState::RebuildScene;
    {
        event_rebuild_scene();

        // TODO: rebuild the vertex and surface info
        // m_global_vertex_manager->rebuild_vertex_info();
        // m_global_surface_manager->rebuild_surface_info();
    }

    // After the rebuild_scene event, the pending creation or deletion can be solved
    auto scene = m_world_visitor->scene();
    scene.solve_pending();
}
}  // namespace uipc::backend::cuda
