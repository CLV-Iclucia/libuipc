#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <log_pattern_guard.h>

namespace uipc::backend::cuda
{
void SimEngine::do_advance()
{
    LogGuard guard;

    // 1. Trigger the rebuild_scene event, systems register their actions will be called here
    m_state = SimEngineState::RebuildScene;
    event_rebuild_scene();

    // 2. After the rebuild_scene event, the pending creation or deletion can be solved
    auto scene = m_world_visitor->scene();
    scene.solve_pending();

    // 3. Begin the pipeline
    {
        // a. Predict Motion => x_tilde = x + v * dt

        // b. Build Collision Pairs

        // c. Select Adaptive Kappa

        // d. Begin Newton Iteration
        Float tol  = 1e-6;     // now just hard code it
        Float res  = 2 * tol;  // now just hard code it
        SizeT iter = 1000;     // now just hard code it
        while(iter--)
        {
            // 1) Build Collision Pairs

            // 2) Compute Contact Gradient and Hessian => G:Vector3, H:Matrix3x3

            // 3) Compute System Gradient and Hessian => G:Vector3, H:Matrix3x3
            // E.g. FEM/ABD ...

            // 4) Assemble Global Linear System => A:SparseMatrix of H, b:DenseVector of G

            // 5) Solve Global Linear System => dx = A^-1 * b

            // 6) Get Max Movement => dx_max = max(|dx|), if dx_max < tol, break
            res = 0;  // = dx_max
            if(res < tol)
                break;

            // 8) Begin Line Search
            {
                // Record Current State x to x_0

                auto compute_energy = [&](Float alpha) -> Float
                {
                    // Continuous Collision Detection => alpha
                    // Step Forward => x = x_0 + alpha * dx
                    // Compute New Energy => E
                    return 0.0;
                };

                // Compute Current Energy => E_0

                Float E0 = compute_energy(0.0);

                // Build Continuous Collision Pairs with alpha = 1.0
                Float alpha = 1.0;

                SizeT max_line_search_iter = 1000;  // now just hard code it
                while(max_line_search_iter--)       // Energy Test
                {
                    // Check some conditions
                    Float E               = compute_energy(alpha);
                    bool  energy_decrease = E < E0;  // Check Energy Decrease
                    bool  no_inversion    = true;    // Check Inversion

                    bool success = energy_decrease && no_inversion;
                    if(success)
                        break;

                    // If not success, then shrink alpha

                    alpha /= 2;
                    E = compute_energy(alpha);
                }
            }
        }

        // e. Update Velocity => v = (x - x_0) / dt
    }
}
}  // namespace uipc::backend::cuda
