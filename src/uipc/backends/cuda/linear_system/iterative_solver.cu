#include <linear_system/iterative_solver.h>
#include <linear_system/global_linear_system.h>
namespace uipc::backend::cuda
{
void IterativeSolver::spmv(Float                         a,
                           muda::CDenseVectorView<Float> x,
                           Float                         b,
                           muda::DenseVectorView<Float>  y)
{
    m_system->m_impl.spmv(a, x, b, y);
}
void IterativeSolver::spmv(muda::CDenseVectorView<Float> x, muda::DenseVectorView<Float> y)
{
    spmv(1.0, x, 0.0, y);
}
void IterativeSolver::apply_preconditioner(muda::DenseVectorView<Float>  z,
                                           muda::CDenseVectorView<Float> r)
{
    m_system->m_impl.apply_preconditioner(z, r);
}
bool IterativeSolver::accuracy_statisfied(muda::DenseVectorView<Float> r)
{
    return m_system->m_impl.accuracy_statisfied(r);
}
muda::LinearSystemContext& IterativeSolver::ctx() const
{
    return m_system->m_impl.ctx;
}
void IterativeSolver::solve(GlobalLinearSystem::SolvingInfo& info)
{
    do_solve(info);
}
}  // namespace uipc::backend::cuda
