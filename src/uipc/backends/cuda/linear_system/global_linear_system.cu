#include "hip/hip_runtime.h"
#include <linear_system/global_linear_system.h>
#include <linear_system/diag_linear_subsystem.h>
#include <linear_system/off_diag_linear_subsystem.h>
#include <uipc/common/range.h>
#include <linear_system/iterative_solver.h>
#include <linear_system/global_preconditioner.h>
#include <linear_system/local_preconditioner.h>

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(GlobalLinearSystem);

void GlobalLinearSystem::do_build()
{
    on_init_scene([this] { m_impl.init(); });
}

void GlobalLinearSystem::solve()
{
    m_impl.build_linear_system();
    // if the system is empty, skip the following steps
    if(m_impl.empty_system) [[unlikely]]
        return;
    m_impl.solve_linear_system();
    m_impl.distribute_solution();
}

void GlobalLinearSystem::Impl::init()
{
    UIPC_ASSERT(!diag_subsystem_buffer.empty() || !off_diag_subsystem_buffer.empty(),
                "No linear subsystems added to the global linear system.");

    // build the linear subsystem infos
    diag_subsystems.resize(diag_subsystem_buffer.size());
    off_diag_subsystems.resize(off_diag_subsystem_buffer.size());
    local_preconditioners.resize(local_preconditioner_buffer.size());

    std::ranges::move(diag_subsystem_buffer, diag_subsystems.begin());
    std::ranges::move(off_diag_subsystem_buffer, off_diag_subsystems.begin());
    std::ranges::move(local_preconditioner_buffer, local_preconditioners.begin());

    auto total_count = diag_subsystems.size() + off_diag_subsystems.size();

    subsystem_infos.resize(total_count);

    // put the diag subsystems in the front
    auto diag_span = span{subsystem_infos}.subspan(0, diag_subsystems.size());
    // then the off diag subsystems
    auto off_diag_span = span{subsystem_infos}.subspan(diag_subsystems.size(),
                                                       off_diag_subsystems.size());

    auto offset = 0;
    for(auto i : range(diag_span.size()))
    {
        auto& dst_diag              = diag_span[i];
        dst_diag.is_diag            = true;
        dst_diag.local_index        = i;
        auto index                  = offset + i;
        dst_diag.index              = index;
        diag_subsystems[i]->m_index = index;
    }

    offset += diag_subsystems.size();
    for(auto i : range(off_diag_span.size()))
    {
        auto& dst_off_diag       = off_diag_span[i];
        dst_off_diag.is_diag     = false;
        dst_off_diag.local_index = i;
        dst_off_diag.index       = offset + i;
    }

    // prepare the storage for dof and matrix triplet
    subsystem_triplet_offsets.resize(total_count, ~0ull);
    subsystem_triplet_offsets.resize(total_count, ~0ull);

    diag_dof_offsets.resize(diag_subsystems.size());
    diag_dof_counts.resize(diag_subsystems.size());
    accuracy_statisfied_flags.resize(diag_subsystems.size());

    off_diag_lr_triplet_counts.resize(off_diag_subsystems.size());
}

void GlobalLinearSystem::Impl::build_linear_system()
{
    empty_system = !_update_subsystem_extent();
    // if empty, skip the following steps
    if(empty_system) [[unlikely]]
        return;

    _assemble_linear_system();

    converter.convert(triplet_A, bcoo_A);
    converter.ge2sym(bcoo_A);

    _assemble_preconditioner();
}

bool GlobalLinearSystem::Impl::_update_subsystem_extent()
{
    bool dof_count_changed     = false;
    bool triplet_count_changed = false;

    for(const auto& subsystem_info : subsystem_infos)
    {
        if(subsystem_info.is_diag)
        {
            auto           dof_i          = subsystem_info.local_index;
            auto           triplet_i      = subsystem_info.index;
            auto&          diag_subsystem = diag_subsystems[dof_i];
            DiagExtentInfo info;
            info.m_storage_type = HessianStorageType::Full;
            diag_subsystem->report_extent(info);

            dof_count_changed |= diag_dof_counts[dof_i] != info.m_dof_count;
            diag_dof_counts[dof_i] = info.m_dof_count;


            triplet_count_changed |= subsystem_triplet_counts[triplet_i] != info.m_block_count;
            subsystem_triplet_counts[triplet_i] = info.m_block_count;
        }
        else
        {
            auto triplet_i = subsystem_info.index;
            auto& off_diag_subsystem = off_diag_subsystems[subsystem_info.local_index];
            OffDiagExtentInfo info;
            info.m_storage_type = HessianStorageType::Full;
            off_diag_subsystem->report_extent(info);

            auto total_block_count = info.m_lr_block_count + info.m_rl_block_count;

            triplet_count_changed |= subsystem_triplet_counts[triplet_i] != total_block_count;
            subsystem_triplet_counts[triplet_i] = total_block_count;
            off_diag_lr_triplet_counts[subsystem_info.local_index] =
                ulonglong2{info.m_lr_block_count, info.m_rl_block_count};
        }
    }

    SizeT total_dof     = 0;
    SizeT total_triplet = 0;

    if(dof_count_changed)
    {
        std::exclusive_scan(
            diag_dof_counts.begin(), diag_dof_counts.end(), diag_dof_offsets.begin(), 0);
        total_dof = diag_dof_offsets.back() + diag_dof_counts.back();
        if(x.size() < total_dof)
        {
            auto reserve_count = total_dof * reserve_ratio;
            x.reserve(reserve_count);
            x.resize(total_dof);

            b.reserve(reserve_count);
            b.resize(total_dof);

            auto blocked_dof = total_dof / DoFBlockSize;

            triplet_A.reshape(blocked_dof, blocked_dof);
            bsr_A.reserve_offsets(reserve_count / DoFBlockSize + 1);
        }
    }
    else
    {
        total_dof = diag_dof_offsets.back() + diag_dof_counts.back();
    }

    if(triplet_count_changed) [[likely]]
    {
        std::exclusive_scan(subsystem_triplet_counts.begin(),
                            subsystem_triplet_counts.end(),
                            subsystem_triplet_offsets.begin(),
                            0);
        total_triplet =
            subsystem_triplet_offsets.back() + subsystem_triplet_counts.back();
        if(triplet_A.triplet_count() < total_triplet)
        {
            auto reserve_count = total_triplet * reserve_ratio;
            triplet_A.reserve_triplets(reserve_count);
            triplet_A.resize_triplets(total_triplet);

            bcoo_A.reserve_triplets(reserve_count);
            bsr_A.reserve(reserve_count);
        }
    }
    else
    {
        total_triplet =
            subsystem_triplet_offsets.back() + subsystem_triplet_counts.back();
    }

    if(total_dof == 0 || total_triplet == 0) [[unlikely]]
    {
        spdlog::warn("The global linear system is empty, skip *assembling, *solving and *solution distributing phase.");
        return false;
    }

    return true;
}

void GlobalLinearSystem::Impl::_assemble_linear_system()
{
    auto HA = triplet_A.view();
    auto X  = x.view();
    for(const auto& subsystem_info : subsystem_infos)
    {
        if(subsystem_info.is_diag)
        {
            auto  dof_i          = subsystem_info.local_index;
            auto  triplet_i      = subsystem_info.index;
            auto& diag_subsystem = diag_subsystems[dof_i];

            int  dof_offset = diag_dof_offsets[dof_i];
            int  dof_count  = diag_dof_counts[dof_i];
            int2 ij_offset  = {dof_offset, dof_offset};
            int2 ij_count   = {dof_count, dof_count};

            DiagInfo info{this};

            info.m_index        = triplet_i;
            info.m_storage_type = HessianStorageType::Full;
            info.m_gradient     = X.subview(dof_offset, dof_count);
            info.m_hessian = HA.subview(subsystem_triplet_offsets[triplet_i],
                                        subsystem_triplet_counts[triplet_i])
                                 .submatrix(ij_offset, ij_count);

            diag_subsystem->assemble(info);
        }
        else
        {
            auto triplet_i   = subsystem_info.index;
            auto local_index = subsystem_info.local_index;
            auto& off_diag_subsystem = off_diag_subsystems[subsystem_info.local_index];
            auto& l_diag_index = off_diag_subsystem->m_l->m_index;
            auto& r_diag_index = off_diag_subsystem->m_r->m_index;


            int l_blocked_dof_offset = diag_dof_offsets[l_diag_index] / DoFBlockSize;
            int l_blocked_dof_count = diag_dof_counts[l_diag_index] / DoFBlockSize;

            int r_blocked_dof_offset = diag_dof_offsets[r_diag_index] / DoFBlockSize;
            int r_blocked_dof_count = diag_dof_counts[r_diag_index] / DoFBlockSize;

            auto lr_offset = subsystem_triplet_offsets[triplet_i];
            auto lr_count  = off_diag_lr_triplet_counts[local_index].x;
            auto rl_offset = lr_offset + lr_count;
            auto rl_count  = off_diag_lr_triplet_counts[local_index].y;

            OffDiagInfo info{this};
            info.m_index        = triplet_i;
            info.m_storage_type = HessianStorageType::Full;

            info.m_lr_hessian =
                HA.subview(lr_offset, lr_count)
                    .submatrix(int2{l_blocked_dof_offset, r_blocked_dof_offset},
                               int2{l_blocked_dof_count, r_blocked_dof_count});

            info.m_rl_hessian =
                HA.subview(rl_offset, rl_count)
                    .submatrix(int2{r_blocked_dof_offset, l_blocked_dof_offset},
                               int2{r_blocked_dof_count, l_blocked_dof_count});

            off_diag_subsystem->assemble(info);
        }
    }
}

void GlobalLinearSystem::Impl::_assemble_preconditioner()
{
    if(global_preconditioner)
    {
        GlobalPreconditionerAssemblyInfo info{this};
        info.symmetric = true;
        global_preconditioner->assemble(info);
    }

    for(auto&& [i, preconditioner] : enumerate(local_preconditioners))
    {
        LocalPreconditionerAssemblyInfo info{this};
        info.m_index = i;
        preconditioner->assemble(info);
    }
}

void GlobalLinearSystem::Impl::solve_linear_system()
{
    if(iterative_solver)
    {
        SolvingInfo info{this};
        info.m_b = b.cview();
        info.m_x = x.view();
        iterative_solver->solve(info);
    }
}

void GlobalLinearSystem::Impl::distribute_solution()
{
    // distribute the solution to all diag subsystems
    for(auto&& [i, diag_subsystem] : enumerate(diag_subsystems))
    {
        SolutionInfo info{this};
        info.m_solution = x.view().subview(diag_dof_offsets[i], diag_dof_counts[i]);
        diag_subsystem->retrieve_solution(info);
    }
}

void GlobalLinearSystem::Impl::apply_preconditioner(muda::DenseVectorView<Float> z,
                                                    muda::CDenseVectorView<Float> r)
{
    if(global_preconditioner)
    {
        ApplyPreconditionerInfo info{this};
        info.m_z = z;
        info.m_r = r;
        global_preconditioner->apply(info);
    }

    for(auto& preconditioner : local_preconditioners)
    {
        ApplyPreconditionerInfo info{this};
        info.m_z = z;
        info.m_r = r;
        preconditioner->apply(info);
    }
}

void GlobalLinearSystem::Impl::spmv(Float                         a,
                                    muda::CDenseVectorView<Float> x,
                                    Float                         b,
                                    muda::DenseVectorView<Float>  y)
{
    spmver.rbk_sym_spmv(a, bcoo_A.cview(), x, b, y);
}

bool GlobalLinearSystem::Impl::accuracy_statisfied(muda::DenseVectorView<Float> r)
{
    for(auto&& [i, diag_subsystems] : enumerate(diag_subsystems))
    {
        AccuracyInfo info{this};
        info.m_r = r.subview(diag_dof_offsets[i], diag_dof_counts[i]);
        diag_subsystems->accuracy_check(info);

        accuracy_statisfied_flags[i] = info.m_statisfied ? 1 : 0;
    }

    return std::ranges::all_of(accuracy_statisfied_flags,
                               [](bool flag) { return flag; });
}
}  // namespace uipc::backend::cuda

namespace uipc::backend::cuda
{
void GlobalLinearSystem::add_subsystem(DiagLinearSubsystem* subsystem)
{
    check_state(SimEngineState::BuildSystems, "add_subsystem()");
    m_impl.diag_subsystem_buffer.push_back(subsystem);
}

void GlobalLinearSystem::add_subsystem(OffDiagLinearSubsystem* subsystem)
{
    check_state(SimEngineState::BuildSystems, "add_subsystem()");
    subsystem->check_dep_systems();
    m_impl.off_diag_subsystem_buffer.push_back(subsystem);
}

void GlobalLinearSystem::add_solver(IterativeSolver* solver)
{
    check_state(SimEngineState::BuildSystems, "add_solver()");
    UIPC_ASSERT(m_impl.iterative_solver == nullptr,
                "Only support one linear system solver, {} already added before.",
                m_impl.iterative_solver->name());
    m_impl.iterative_solver = solver;
}

void GlobalLinearSystem::add_preconditioner(LocalPreconditioner* preconditioner)
{
    check_state(SimEngineState::BuildSystems, "add_preconditioner()");
    m_impl.local_preconditioner_buffer.push_back(preconditioner);
}

void GlobalLinearSystem::add_preconditioner(GlobalPreconditioner* preconditioner)
{
    check_state(SimEngineState::BuildSystems, "add_preconditioner()");
    UIPC_ASSERT(m_impl.global_preconditioner == nullptr,
                "Only support one global preconditioner, {} already added before.",
                m_impl.global_preconditioner->name());
    m_impl.global_preconditioner = preconditioner;
}
}  // namespace uipc::backend::cuda
