#include "hip/hip_runtime.h"
#include <linear_system/global_linear_system.h>
#include <linear_system/diag_linear_subsystem.h>
#include <linear_system/off_diag_linear_subsystem.h>
#include <uipc/common/range.h>
#include <linear_system/iterative_solver.h>
#include <linear_system/global_preconditioner.h>
#include <linear_system/local_preconditioner.h>

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(GlobalLinearSystem);

void GlobalLinearSystem::do_build()
{
    on_init_scene([this] { m_impl.init(); });
}

void GlobalLinearSystem::solve()
{
    m_impl.build_linear_system();
    // if the system is empty, skip the following steps
    if(m_impl.empty_system) [[unlikely]]
        return;
    m_impl.solve_linear_system();
    m_impl.distribute_solution();
}

void GlobalLinearSystem::Impl::init()
{
    UIPC_ASSERT(!diag_subsystem_buffer.empty(),
                "No linear subsystems added to the global linear system.");

    // build the linear subsystem infos
    diag_subsystems.resize(diag_subsystem_buffer.size());
    off_diag_subsystems.resize(off_diag_subsystem_buffer.size());
    local_preconditioners.resize(local_preconditioner_buffer.size());

    std::ranges::move(diag_subsystem_buffer, diag_subsystems.begin());
    std::ranges::move(off_diag_subsystem_buffer, off_diag_subsystems.begin());
    std::ranges::move(local_preconditioner_buffer, local_preconditioners.begin());

    auto total_count = diag_subsystems.size() + off_diag_subsystems.size();

    subsystem_infos.resize(total_count);

    // put the diag subsystems in the front
    auto diag_span = span{subsystem_infos}.subspan(0, diag_subsystems.size());
    // then the off diag subsystems
    auto off_diag_span = span{subsystem_infos}.subspan(diag_subsystems.size(),
                                                       off_diag_subsystems.size());

    auto offset = 0;
    for(auto i : range(diag_span.size()))
    {
        auto& dst_diag              = diag_span[i];
        dst_diag.is_diag            = true;
        dst_diag.local_index        = i;
        auto index                  = offset + i;
        dst_diag.index              = index;
        diag_subsystems[i]->m_index = index;
    }

    offset += diag_subsystems.size();
    for(auto i : range(off_diag_span.size()))
    {
        auto& dst_off_diag       = off_diag_span[i];
        dst_off_diag.is_diag     = false;
        dst_off_diag.local_index = i;
        dst_off_diag.index       = offset + i;
    }

    // prepare the storage for dof and matrix triplet
    subsystem_triplet_offsets.resize(total_count, ~0ull);
    subsystem_triplet_counts.resize(total_count, ~0ull);


    diag_dof_offsets.resize(diag_subsystems.size());
    diag_dof_counts.resize(diag_subsystems.size());
    accuracy_statisfied_flags.resize(diag_subsystems.size());

    off_diag_lr_triplet_counts.resize(off_diag_subsystems.size());
}

void GlobalLinearSystem::Impl::build_linear_system()
{
    empty_system = !_update_subsystem_extent();
    // if empty, skip the following steps
    if(empty_system) [[unlikely]]
        return;

    _assemble_linear_system();

    converter.convert(triplet_A, bcoo_A);
    converter.ge2sym(bcoo_A);

    _assemble_preconditioner();
}

bool GlobalLinearSystem::Impl::_update_subsystem_extent()
{
    bool dof_count_changed     = false;
    bool triplet_count_changed = false;

    for(const auto& subsystem_info : subsystem_infos)
    {
        if(subsystem_info.is_diag)
        {
            auto           dof_i          = subsystem_info.local_index;
            auto           triplet_i      = subsystem_info.index;
            auto&          diag_subsystem = diag_subsystems[dof_i];
            DiagExtentInfo info;
            info.m_storage_type = HessianStorageType::Full;
            diag_subsystem->report_extent(info);

            dof_count_changed |= diag_dof_counts[dof_i] != info.m_dof_count;
            diag_dof_counts[dof_i] = info.m_dof_count;


            triplet_count_changed |= subsystem_triplet_counts[triplet_i] != info.m_block_count;
            subsystem_triplet_counts[triplet_i] = info.m_block_count;
        }
        else
        {
            auto triplet_i = subsystem_info.index;
            auto& off_diag_subsystem = off_diag_subsystems[subsystem_info.local_index];
            OffDiagExtentInfo info;
            info.m_storage_type = HessianStorageType::Full;
            off_diag_subsystem->report_extent(info);

            auto total_block_count = info.m_lr_block_count + info.m_rl_block_count;

            triplet_count_changed |= subsystem_triplet_counts[triplet_i] != total_block_count;
            subsystem_triplet_counts[triplet_i] = total_block_count;
            off_diag_lr_triplet_counts[subsystem_info.local_index] =
                ulonglong2{info.m_lr_block_count, info.m_rl_block_count};
        }
    }

    SizeT total_dof     = 0;
    SizeT total_triplet = 0;

    if(dof_count_changed)
    {
        std::exclusive_scan(
            diag_dof_counts.begin(), diag_dof_counts.end(), diag_dof_offsets.begin(), 0);
        total_dof = diag_dof_offsets.back() + diag_dof_counts.back();
        if(x.capacity() < total_dof)
        {
            auto reserve_count = total_dof * reserve_ratio;
            x.reserve(reserve_count);
            b.reserve(reserve_count);
        }
        auto blocked_dof = total_dof / DoFBlockSize;
        triplet_A.reshape(blocked_dof, blocked_dof);
        x.resize(total_dof);
        b.resize(total_dof);
    }
    else
    {
        total_dof = diag_dof_offsets.back() + diag_dof_counts.back();
    }

    if(triplet_count_changed) [[likely]]
    {
        std::exclusive_scan(subsystem_triplet_counts.begin(),
                            subsystem_triplet_counts.end(),
                            subsystem_triplet_offsets.begin(),
                            0);
        total_triplet =
            subsystem_triplet_offsets.back() + subsystem_triplet_counts.back();
        if(triplet_A.triplet_capacity() < total_triplet)
        {
            auto reserve_count = total_triplet * reserve_ratio;
            triplet_A.reserve_triplets(reserve_count);
            bcoo_A.reserve_triplets(reserve_count);
        }
        triplet_A.resize_triplets(total_triplet);
    }
    else
    {
        total_triplet =
            subsystem_triplet_offsets.back() + subsystem_triplet_counts.back();
    }

    if(total_dof == 0 || total_triplet == 0) [[unlikely]]
    {
        spdlog::warn("The global linear system is empty, skip *assembling, *solving and *solution distributing phase.");
        return false;
    }

    return true;
}

void GlobalLinearSystem::Impl::_assemble_linear_system()
{
    auto HA = triplet_A.view();
    auto B  = b.view();
    for(const auto& subsystem_info : subsystem_infos)
    {
        if(subsystem_info.is_diag)
        {
            auto  dof_i          = subsystem_info.local_index;
            auto  triplet_i      = subsystem_info.index;
            auto& diag_subsystem = diag_subsystems[dof_i];

            int  dof_offset         = diag_dof_offsets[dof_i];
            int  dof_count          = diag_dof_counts[dof_i];
            int  blocked_dof_offset = dof_offset / DoFBlockSize;
            int  blocked_dof_count  = dof_count / DoFBlockSize;
            int2 ij_offset          = {blocked_dof_offset, blocked_dof_offset};
            int2 ij_count           = {blocked_dof_count, blocked_dof_count};

            DiagInfo info{this};

            info.m_index        = triplet_i;
            info.m_storage_type = HessianStorageType::Full;
            info.m_gradient     = B.subview(dof_offset, dof_count);
            info.m_hessian = HA.subview(subsystem_triplet_offsets[triplet_i],
                                        subsystem_triplet_counts[triplet_i])
                                 .submatrix(ij_offset, ij_count);

            diag_subsystem->assemble(info);
        }
        else
        {
            auto triplet_i   = subsystem_info.index;
            auto local_index = subsystem_info.local_index;
            auto& off_diag_subsystem = off_diag_subsystems[subsystem_info.local_index];
            auto& l_diag_index = off_diag_subsystem->m_l->m_index;
            auto& r_diag_index = off_diag_subsystem->m_r->m_index;


            int l_blocked_dof_offset = diag_dof_offsets[l_diag_index] / DoFBlockSize;
            int l_blocked_dof_count = diag_dof_counts[l_diag_index] / DoFBlockSize;

            int r_blocked_dof_offset = diag_dof_offsets[r_diag_index] / DoFBlockSize;
            int r_blocked_dof_count = diag_dof_counts[r_diag_index] / DoFBlockSize;

            auto lr_triplet_offset = subsystem_triplet_offsets[triplet_i];
            auto lr_triplet_count  = off_diag_lr_triplet_counts[local_index].x;
            auto rl_triplet_offset = lr_triplet_offset + lr_triplet_count;
            auto rl_triplet_count  = off_diag_lr_triplet_counts[local_index].y;

            OffDiagInfo info{this};
            info.m_index        = triplet_i;
            info.m_storage_type = HessianStorageType::Full;

            info.m_lr_hessian =
                HA.subview(lr_triplet_offset, lr_triplet_count)
                    .submatrix(int2{l_blocked_dof_offset, r_blocked_dof_offset},
                               int2{l_blocked_dof_count, r_blocked_dof_count});

            info.m_rl_hessian =
                HA.subview(rl_triplet_offset, rl_triplet_count)
                    .submatrix(int2{r_blocked_dof_offset, l_blocked_dof_offset},
                               int2{r_blocked_dof_count, l_blocked_dof_count});

            off_diag_subsystem->assemble(info);
        }
    }
}

void GlobalLinearSystem::Impl::_assemble_preconditioner()
{
    if(global_preconditioner)
    {
        GlobalPreconditionerAssemblyInfo info{this};
        info.symmetric = true;
        global_preconditioner->assemble(info);
    }

    for(auto&& [i, preconditioner] : enumerate(local_preconditioners))
    {
        LocalPreconditionerAssemblyInfo info{this};
        info.m_index = i;
        preconditioner->assemble(info);
    }
}

void GlobalLinearSystem::Impl::solve_linear_system()
{
    if(iterative_solver)
    {
        SolvingInfo info{this};
        info.m_b = b.cview();
        info.m_x = x.view();
        iterative_solver->solve(info);
    }
}

void GlobalLinearSystem::Impl::distribute_solution()
{
    // _distribute the solution to all diag subsystems
    for(auto&& [i, diag_subsystem] : enumerate(diag_subsystems))
    {
        SolutionInfo info{this};
        info.m_solution = x.view().subview(diag_dof_offsets[i], diag_dof_counts[i]);
        diag_subsystem->retrieve_solution(info);
    }
}

void GlobalLinearSystem::Impl::apply_preconditioner(muda::DenseVectorView<Float> z,
                                                    muda::CDenseVectorView<Float> r)
{
    if(global_preconditioner)
    {
        ApplyPreconditionerInfo info{this};
        info.m_z = z;
        info.m_r = r;
        global_preconditioner->apply(info);
    }

    for(auto& preconditioner : local_preconditioners)
    {
        ApplyPreconditionerInfo info{this};
        auto                    index  = preconditioner->m_subsystem->m_index;
        auto                    offset = diag_dof_offsets[index];
        auto                    count  = diag_dof_counts[index];
        info.m_z                       = z.subview(offset, count);
        info.m_r                       = r.subview(offset, count);
        preconditioner->apply(info);
    }

    if(!global_preconditioner && local_preconditioners.empty())
    {
        muda::BufferLaunch().copy<Float>(z.buffer_view(), r.buffer_view());
    }
}

void GlobalLinearSystem::Impl::spmv(Float                         a,
                                    muda::CDenseVectorView<Float> x,
                                    Float                         b,
                                    muda::DenseVectorView<Float>  y)
{
    spmver.rbk_sym_spmv(a, bcoo_A.cview(), x, b, y);
}

bool GlobalLinearSystem::Impl::accuracy_statisfied(muda::DenseVectorView<Float> r)
{
    for(auto&& [i, diag_subsystems] : enumerate(diag_subsystems))
    {
        AccuracyInfo info{this};
        info.m_r = r.subview(diag_dof_offsets[i], diag_dof_counts[i]);
        diag_subsystems->accuracy_check(info);

        accuracy_statisfied_flags[i] = info.m_statisfied ? 1 : 0;
    }

    return std::ranges::all_of(accuracy_statisfied_flags,
                               [](bool flag) { return flag; });
}
}  // namespace uipc::backend::cuda

namespace uipc::backend::cuda
{
void GlobalLinearSystem::add_subsystem(DiagLinearSubsystem* subsystem)
{
    check_state(SimEngineState::BuildSystems, "add_subsystem()");
    m_impl.diag_subsystem_buffer.push_back(subsystem);
}

void GlobalLinearSystem::add_subsystem(OffDiagLinearSubsystem* subsystem,
                                       DiagLinearSubsystem*    depend_l,
                                       DiagLinearSubsystem*    depend_r)
{
    check_state(SimEngineState::BuildSystems, "add_subsystem()");
    UIPC_ASSERT(depend_l != nullptr && depend_r != nullptr,
                "The depend_l and depend_r should not be nullptr.");
    subsystem->depend_on(depend_l, depend_r);
    m_impl.off_diag_subsystem_buffer.push_back(subsystem);
}

void GlobalLinearSystem::add_solver(IterativeSolver* solver)
{
    check_state(SimEngineState::BuildSystems, "add_solver()");
    UIPC_ASSERT(m_impl.iterative_solver == nullptr,
                "Only support one linear system solver, {} already added before.",
                m_impl.iterative_solver->name());
    m_impl.iterative_solver = solver;
    solver->m_system        = this;
}

void GlobalLinearSystem::add_preconditioner(LocalPreconditioner* preconditioner,
                                            DiagLinearSubsystem* depend_subsystem)
{
    check_state(SimEngineState::BuildSystems, "add_preconditioner()");
    UIPC_ASSERT(depend_subsystem != nullptr, "The depend_subsystem should not be nullptr.");
    preconditioner->m_subsystem = depend_subsystem;
    m_impl.local_preconditioner_buffer.push_back(preconditioner);
}

void GlobalLinearSystem::add_preconditioner(GlobalPreconditioner* preconditioner)
{
    check_state(SimEngineState::BuildSystems, "add_preconditioner()");
    UIPC_ASSERT(m_impl.global_preconditioner == nullptr,
                "Only support one global preconditioner, {} already added before.",
                m_impl.global_preconditioner->name());
    m_impl.global_preconditioner = preconditioner;
}
}  // namespace uipc::backend::cuda