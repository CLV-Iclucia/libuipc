#include "hip/hip_runtime.h"
#include <linear_system/linear_pcg.h>
#include <sim_engine.h>

namespace uipc::backend::cuda
{
template <>
class SimSystemCreator<LinearPCG>
{
  public:
    static U<LinearPCG> create(SimEngine& engine)
    {
        auto& info = engine.world().scene().info();
        if(info["linear_system"]["solver"] == "linear_pcg")
            return make_unique<LinearPCG>(engine);
        else
            return nullptr;
    }
};

REGISTER_SIM_SYSTEM(LinearPCG);

void LinearPCG::do_build()
{
    on_init_scene(
        [this]
        {
            // TODO: get info from the scene, now we just use the default value
            max_iter_ratio  = 2.0;
            global_tol_rate = 1e-4;
            spdlog::info("LinearPCG: max_iter_ratio = {}, global_tol_rate = {}",
                         max_iter_ratio,
                         global_tol_rate);
        });
}

void LinearPCG::do_solve(GlobalLinearSystem::SolvingInfo& info)
{
    auto x = info.x();
    auto b = info.b();

    x.buffer_view().fill(0);

    auto N = x.size();
    if(z.size() < N)
    {
        auto M = reserve_ratio * N;
        z.reserve(M);
        p.reserve(M);
        r.reserve(M);
        Ap.reserve(M);
    }

    z.resize(N);
    p.resize(N);
    r.resize(N);
    Ap.resize(N);

    auto iter = pcg(x, b, max_iter_ratio * b.size());

    info.iter_count(iter);
}

SizeT LinearPCG::pcg(muda::DenseVectorView<Float> x, muda::CDenseVectorView<Float> b, SizeT max_iter)
{
    SizeT k = 0;
    // r = b - A * x
    {
        // r = b;
        r.buffer_view().copy_from(b.buffer_view());

        // x == 0, so we don't need to do the following
        // r = - A * x + r
        //spmv(-1.0, x.as_const(), 1.0, r.view());
    }

    Float alpha, beta, rz, rz0;

    // z = P * r (apply preconditioner)
    apply_preconditioner(z, r);

    // p = z
    p = z;

    // init rz
    // rz = r^T * z
    rz = ctx().dot(r.cview(), z.cview());

    rz0 = std::abs(rz);

    // check convergence
    if(accuracy_statisfied(r) && std::abs(rz) <= global_tol_rate * rz0)
        return k;

    for(k = 1; k < max_iter; ++k)
    {
        spmv(p.cview(), Ap.view());

        // alpha = rz / dot(p.cview(), Ap.cview());
        alpha = rz / ctx().dot(p.cview(), Ap.cview());

        // x = x + alpha * p
        ctx().axpby(alpha, p.cview(), 1.0, x);

        // r = r - alpha * Ap
        ctx().axpby(-alpha, Ap.cview(), 1.0, r.view());


        // check convergence
        if(accuracy_statisfied(r) && std::abs(rz) <= global_tol_rate * rz0)
            break;

        // z = P * r (apply preconditioner)
        apply_preconditioner(z, r);

        // rz_new = r^T * z
        // rz_new = dot(r.cview(), z.cview());
        Float rz_new = ctx().dot(r.cview(), z.cview());

        // beta = rz_new / rz
        beta = rz_new / rz;

        // p = z + beta * p
        ctx().axpby(1.0, z.cview(), beta, p.view());

        // update rz
        rz = rz_new;
    }

    return k;
}
}  // namespace uipc::backend::cuda
