#include "hip/hip_runtime.h"
#include <linear_system/off_diag_linear_subsystem.h>
#include <uipc/common/log.h>
namespace uipc::backend::cuda
{
void OffDiagLinearSubsystem::depend_on(DiagLinearSubsystem* L, DiagLinearSubsystem* R)
{
    m_l = L;
    m_r = R;
}
void OffDiagLinearSubsystem::do_build()
{
    auto&     global_linear_system = require<GlobalLinearSystem>();
    BuildInfo info;
    do_build(info);

    UIPC_ASSERT(info.m_diag_l != nullptr && info.m_diag_r != nullptr,
                "Did you forget to call BuildInfo::connect() in {}'s do_build()?",
                this->name());

    global_linear_system.add_subsystem(this, info.m_diag_l, info.m_diag_r);
}
}  // namespace uipc::backend::cuda
