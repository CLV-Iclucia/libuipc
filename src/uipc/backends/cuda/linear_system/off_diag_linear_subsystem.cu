#include "hip/hip_runtime.h"
#include <linear_system/off_diag_linear_subsystem.h>
#include <uipc/common/log.h>
namespace uipc::backend::cuda
{
void OffDiagLinearSubsystem::depend_on(DiagLinearSubsystem* L, DiagLinearSubsystem* R)
{
    m_l = L;
    m_r = R;
}
void OffDiagLinearSubsystem::check_dep_systems() const
{
    UIPC_ASSERT(m_l && m_r, "Dependent DiagLinearSubsystems are nullptr, call `depend_on()` in `do_build()` to setup them");
}
}  // namespace uipc::backend::cuda
