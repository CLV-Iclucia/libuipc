#include "hip/hip_runtime.h"
#include <linear_system/off_diag_linear_subsystem.h>
#include <uipc/common/log.h>
namespace uipc::backend::cuda
{
void OffDiagLinearSubsystem::depend_on(DiagLinearSubsystem* L, DiagLinearSubsystem* R)
{
    m_l = L;
    m_r = R;
}
}  // namespace uipc::backend::cuda
