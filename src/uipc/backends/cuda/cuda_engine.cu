#include "hip/hip_runtime.h"
#include <uipc/backends/cuda/cuda_engine.h>
#include <uipc/backends/module.h>
#include <uipc/common/log.h>
#include <muda/muda.h>
#include <uipc/backends/cuda/kernel_cout.h>
#include <uipc/backends/cuda/cuda_engine_device_common.h>

namespace uipc::backend
{
void say_hello_from_muda()
{
    using namespace muda;

    Launch()
        .apply([] __device__()
               { cout << "Hello from CudaEngine CUDA Kernel! \n "; })
        .wait();
}

CudaEngine::CudaEngine()
    : m_device_common(std::make_unique<DeviceCommon>())
{
    spdlog::info("[CudaEngine] Cuda Backend Init Success.");

    using namespace muda;

    if(!cout)
    {
        auto viewer_ptr         = device_logger_viewer_ptr();
        m_device_common->logger = std::make_unique<muda::Logger>(viewer_ptr);

        Debug::set_sync_callback(
            [this]
            {
                m_string_stream.str("");
                m_device_common->logger->retrieve(m_string_stream);
                if(m_string_stream.str().empty())
                    return;

                std::string str = m_string_stream.str();
                if(str.back() == '\n')
                    str.pop_back();
                spdlog::info(R"([CudaEngine Kernel Console]:
-------------------------------------------------------------------------------
{}
-------------------------------------------------------------------------------)",
                             str);
            });
    }

    say_hello_from_muda();
}

auto CudaEngine::device_common() noexcept -> DeviceCommon&
{
    return *m_device_common;
}

CudaEngine::~CudaEngine()
{
    muda::wait_device();

    // remove the sync callback
    muda::Debug::set_sync_callback(nullptr);
    cout = {};

    spdlog::info("[CudaEngine] Cuda Backend Shutdown Success.");
}
}  // namespace uipc::backend
