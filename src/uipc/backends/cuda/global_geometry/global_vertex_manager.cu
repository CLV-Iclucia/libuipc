#include "hip/hip_runtime.h"
#include <global_geometry/global_vertex_manager.h>
#include <uipc/common/enumerate.h>
#include <uipc/common/range.h>
#include <muda/cub/device/device_reduce.h>
#include <global_geometry/vertex_reporter.h>

/*************************************************************************************************
* Core Implementation
*************************************************************************************************/
namespace uipc::backend::cuda
{
void GlobalVertexManager::Impl::init_vertex_info()
{
    vertex_reporters.init();
    auto vertex_reporter_view = vertex_reporters.view();
    for(auto&& [i, R] : enumerate(vertex_reporter_view))
        R->m_index = i;

    auto N = vertex_reporter_view.size();
    reporter_vertex_counts.resize(N);
    reporter_vertex_offsets.resize(N);

    for(auto&& [i, R] : enumerate(vertex_reporter_view))
    {
        VertexCountInfo info;
        R->report_count(info);
        // get count back
        reporter_vertex_counts[i] = info.m_count;
    }

    std::exclusive_scan(reporter_vertex_counts.begin(),
                        reporter_vertex_counts.end(),
                        reporter_vertex_offsets.begin(),
                        0);

    auto total_count = reporter_vertex_offsets.back() + reporter_vertex_counts.back();

    // resize the global coindices buffer
    coindices.resize(total_count);
    positions.resize(total_count);
    rest_positions.resize(total_count);
    safe_positions.resize(total_count);
    contact_element_ids.resize(total_count, 0);
    displacements.resize(total_count, Vector3::Zero());
    displacement_norms.resize(total_count, 0.0);

    // create the subviews for each attribute_reporter
    for(auto&& [i, R] : enumerate(vertex_reporter_view))
    {
        VertexAttributeInfo attributes{this, i};
        R->report_attributes(attributes);
    }

    // TODO: now just copy at the first time
    // latter, we need to check if user fill rest_positions
    // if not, then copy, otherwise, just use it
    rest_positions = positions;
    prev_positions = positions;
}

void GlobalVertexManager::Impl::rebuild_vertex_info()
{
    UIPC_ASSERT(false, "Not implemented yet");
}

void GlobalVertexManager::add_reporter(VertexReporter* reporter)
{
    check_state(SimEngineState::BuildSystems, "add_reporter()");
    m_impl.vertex_reporters.register_subsystem(*reporter);
}

void GlobalVertexManager::Impl::step_forward(Float alpha)
{
    using namespace muda;

    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(positions.size(),
               [pos      = positions.viewer().name("pos"),
                safe_pos = safe_positions.viewer().name("safe_pos"),
                disp     = displacements.viewer().name("disp"),
                alpha    = alpha] __device__(int i) mutable
               { pos(i) = safe_pos(i) + alpha * disp(i); });
}

void GlobalVertexManager::Impl::collect_vertex_displacements()
{
    for(auto&& [i, R] : enumerate(vertex_reporters.view()))
    {
        VertexDisplacementInfo vd{this, i};
        R->report_displacements(vd);
    }
}

void GlobalVertexManager::Impl::record_prev_positions()
{
    using namespace muda;
    BufferLaunch().copy<Vector3>(prev_positions.view(), std::as_const(positions).view());
}

void GlobalVertexManager::Impl::record_start_point()
{
    using namespace muda;
    BufferLaunch().copy<Vector3>(safe_positions.view(), std::as_const(positions).view());
}

Float GlobalVertexManager::Impl::compute_axis_max_displacement()
{
    muda::DeviceReduce().Reduce((Float*)displacements.data(),
                                axis_max_disp.data(),
                                displacements.size() * 3,
                                [] HIPCUB_RUNTIME_FUNCTION(const Float& L, const Float& R)
                                {
                                    auto absL = std::abs(L);
                                    auto absR = std::abs(R);
                                    return absL > absR ? absL : absR;
                                },
                                0.0);
    return axis_max_disp;
}

Float GlobalVertexManager::Impl::compute_max_displacement_norm()
{
    using namespace muda;
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(displacements.size(),
               [disps = displacements.cviewer().name("disp"),
                disp_norms = displacement_norms.viewer().name("disp_norm")] __device__(int i) mutable
               {
                   auto d        = disps(i).norm();
                   disp_norms(i) = d;
               });

    Float max_float = std::numeric_limits<Float>::max();

    DeviceReduce().Max(displacement_norms.data(),
                       max_disp_norm.data(),
                       displacement_norms.size());

    return max_disp_norm;
}

AABB GlobalVertexManager::Impl::compute_vertex_bounding_box()
{
    Float max_float = std::numeric_limits<Float>::max();
    muda::DeviceReduce()
        .Reduce(
            positions.data(),
            min_pos.data(),
            positions.size(),
            [] HIPCUB_RUNTIME_FUNCTION(const Vector3& L, const Vector3& R) -> Vector3
            { return L.cwiseMin(R); },
            Vector3{max_float, max_float, max_float})
        .Reduce(
            positions.data(),
            max_pos.data(),
            positions.size(),
            [] HIPCUB_RUNTIME_FUNCTION(const Vector3& L, const Vector3& R) -> Vector3
            { return L.cwiseMax(R); },
            Vector3{-max_float, -max_float, -max_float});

    Vector3 min_pos_host, max_pos_host;
    min_pos_host = min_pos;
    max_pos_host = max_pos;

    vertex_bounding_box = AABB{min_pos_host, max_pos_host};
    return vertex_bounding_box;
}
}  // namespace uipc::backend::cuda


/*************************************************************************************************
* API Implementation
*************************************************************************************************/
namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(GlobalVertexManager);

void GlobalVertexManager::VertexCountInfo::count(SizeT count) noexcept
{
    m_count = count;
}

void GlobalVertexManager::VertexCountInfo::changable(bool is_changable) noexcept
{
    m_changable = is_changable;
}

GlobalVertexManager::VertexAttributeInfo::VertexAttributeInfo(Impl* impl, SizeT index) noexcept
    : m_impl(impl)
    , m_index(index)
{
}

muda::BufferView<Vector3> GlobalVertexManager::VertexAttributeInfo::rest_positions() const noexcept
{
    return m_impl->subview(m_impl->rest_positions, m_index);
}

muda::BufferView<IndexT> GlobalVertexManager::VertexAttributeInfo::coindices() const noexcept
{
    return m_impl->subview(m_impl->coindices, m_index);
}

muda::BufferView<Vector3> GlobalVertexManager::VertexAttributeInfo::positions() const noexcept
{
    return m_impl->subview(m_impl->positions, m_index);
}

muda::BufferView<IndexT> GlobalVertexManager::VertexAttributeInfo::contact_element_ids() const noexcept
{
    return m_impl->subview(m_impl->contact_element_ids, m_index);
}

GlobalVertexManager::VertexDisplacementInfo::VertexDisplacementInfo(Impl* impl, SizeT index) noexcept
    : m_impl(impl)
    , m_index(index)
{
}

muda::BufferView<Vector3> GlobalVertexManager::VertexDisplacementInfo::displacements() const noexcept
{
    return m_impl->subview(m_impl->displacements, m_index);
}

muda::CBufferView<IndexT> GlobalVertexManager::VertexDisplacementInfo::coindices() const noexcept
{
    return m_impl->subview(m_impl->coindices, m_index);
}

void GlobalVertexManager::do_build() {}

void GlobalVertexManager::init_vertex_info()
{
    m_impl.init_vertex_info();
}

void GlobalVertexManager::rebuild_vertex_info()
{
    m_impl.rebuild_vertex_info();
}

void GlobalVertexManager::record_prev_positions()
{
    m_impl.record_prev_positions();
}

void GlobalVertexManager::collect_vertex_displacements()
{
    m_impl.collect_vertex_displacements();
}

muda::CBufferView<IndexT> GlobalVertexManager::coindices() const noexcept
{
    return m_impl.coindices;
}

muda::CBufferView<Vector3> GlobalVertexManager::positions() const noexcept
{
    return m_impl.positions;
}

muda::CBufferView<Vector3> GlobalVertexManager::prev_positions() const noexcept
{
    return m_impl.prev_positions;
}

muda::CBufferView<Vector3> GlobalVertexManager::rest_positions() const noexcept
{
    return m_impl.rest_positions;
}

muda::CBufferView<Vector3> GlobalVertexManager::safe_positions() const noexcept
{
    return m_impl.safe_positions;
}

muda::CBufferView<IndexT> GlobalVertexManager::contact_element_ids() const noexcept
{
    return m_impl.contact_element_ids;
}

muda::CBufferView<Vector3> GlobalVertexManager::displacements() const noexcept
{
    return m_impl.displacements;
}

Float GlobalVertexManager::compute_axis_max_displacement()
{
    return m_impl.compute_axis_max_displacement();
}

Float GlobalVertexManager::compute_max_displacement_norm()
{
    return m_impl.compute_max_displacement_norm();
}

AABB GlobalVertexManager::compute_vertex_bounding_box()
{
    return m_impl.compute_vertex_bounding_box();
}

void GlobalVertexManager::step_forward(Float alpha)
{
    m_impl.step_forward(alpha);
}

void GlobalVertexManager::record_start_point()
{
    m_impl.record_start_point();
}

AABB GlobalVertexManager::vertex_bounding_box() const noexcept
{
    return m_impl.vertex_bounding_box;
}
}  // namespace uipc::backend::cuda