#include "hip/hip_runtime.h"
#include <contact_system/contact_models/ipc_simplex_frictional_contact.h>
#include <contact_system/contact_models/ipc_simplex_frictional_contact_function.h>

#include <kernel_cout.h>

namespace uipc::backend::cuda
{
constexpr bool DebugTest = false;

REGISTER_SIM_SYSTEM(IPCSimplexFrictionalContact);

void IPCSimplexFrictionalContact::do_build(BuildInfo& info) {}

void IPCSimplexFrictionalContact::do_compute_energy(EnergyInfo& info)
{
    using namespace muda;

    // Compute Point-Triangle energy
    auto PT_count = info.friction_PTs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(PT_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PTs = info.friction_PTs().viewer().name("PTs"),
                Es  = info.friction_PT_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PT = PTs(i);

                   auto cid_L = contact_ids(PT[0]);
                   auto cid_R = contact_ids(PT[1]);

                   const auto& P  = Ps(PT[0]);
                   const auto& T0 = Ps(PT[1]);
                   const auto& T1 = Ps(PT[2]);
                   const auto& T2 = Ps(PT[3]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;

                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;


                   Float D_hat = d_hat * d_hat;

                   Float          prev_D;
                   const Vector3& prev_P  = prev_Ps(PT[0]);
                   const Vector3& prev_T0 = prev_Ps(PT[1]);
                   const Vector3& prev_T1 = prev_Ps(PT[2]);
                   const Vector3& prev_T2 = prev_Ps(PT[3]);
                   distance::point_triangle_distance(prev_P, prev_T0, prev_T1, prev_T2, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);

                   Float D;
                   distance::point_triangle_distance(P, T0, T1, T2, D);
                   // NOTE: D can be larger than D_hat

                   // cout << "PT: " << PT.transpose().eval() << " D/prev_D: " << D
                   //      << "/" << prev_D << "\n";

                   Es(i) = sym::ipc_simplex_contact::PT_friction_energy(
                       kappa, D_hat, friction_rate, dt, P, T0, T1, T2, prev_P, prev_T0, prev_T1, prev_T2, eps_v);
               });

    // Compute Edge-Edge energy
    auto EE_count = info.friction_EEs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(EE_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                EEs = info.friction_EEs().viewer().name("EEs"),
                Es  = info.friction_EE_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v   = info.eps_velocity(),
                rest_Ps = info.rest_positions().viewer().name("rest_Ps"),
                d_hat   = info.d_hat(),
                dt      = info.dt()] __device__(int i) mutable
               {
                   const auto& EE = EEs(i);

                   auto cid_L = contact_ids(EE[0]);
                   auto cid_R = contact_ids(EE[2]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   const Vector3& prev_E0 = prev_Ps(EE[0]);
                   const Vector3& prev_E1 = prev_Ps(EE[1]);
                   const Vector3& prev_E2 = prev_Ps(EE[2]);
                   const Vector3& prev_E3 = prev_Ps(EE[3]);

                   Float D_hat = d_hat * d_hat;

                   Float prev_D;
                   distance::edge_edge_distance(prev_E0, prev_E1, prev_E2, prev_E3, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);


                   const Vector3& E0 = Ps(EE[0]);
                   const Vector3& E1 = Ps(EE[1]);
                   const Vector3& E2 = Ps(EE[2]);
                   const Vector3& E3 = Ps(EE[3]);

                   // cout << "EE: " << EE.transpose().eval() << " D/prev_D: " << D
                   //      << "/" << prev_D << "\n";

                   Es(i) = sym::ipc_simplex_contact::EE_friction_energy(
                       kappa, D_hat, friction_rate, dt, E0, E1, E2, E3, prev_E0, prev_E1, prev_E2, prev_E3, eps_v);
               });

    // Compute Point-Edge energy
    auto PE_count = info.friction_PEs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(PE_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PEs = info.friction_PEs().viewer().name("PEs"),
                Es  = info.friction_PE_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PE = PEs(i);

                   auto cid_L = contact_ids(PE[0]);
                   auto cid_R = contact_ids(PE[1]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Float D_hat = d_hat * d_hat;


                   const Vector3& prev_P  = prev_Ps(PE[0]);
                   const Vector3& prev_E0 = prev_Ps(PE[1]);
                   const Vector3& prev_E1 = prev_Ps(PE[2]);
                   Float          prev_D;
                   distance::point_edge_distance(prev_P, prev_E0, prev_E1, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);

                   const Vector3& P  = Ps(PE[0]);
                   const Vector3& E0 = Ps(PE[1]);
                   const Vector3& E1 = Ps(PE[2]);

                   // cout << "PE: " << PE.transpose().eval() << " D/prev_D: " << D
                   //      << "/" << prev_D << "\n";

                   Es(i) = sym::ipc_simplex_contact::PE_friction_energy(
                       kappa, D_hat, friction_rate, dt, P, E0, E1, prev_P, prev_E0, prev_E1, eps_v);
               });

    // Compute Point-Point energy
    auto PP_count = info.friction_PPs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(PP_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PPs = info.friction_PPs().viewer().name("PPs"),
                Es  = info.friction_PP_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PP = PPs(i);

                   auto cid_L = contact_ids(PP[0]);
                   auto cid_R = contact_ids(PP[1]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Float D_hat = d_hat * d_hat;

                   const Vector3& prev_P0 = prev_Ps(PP[0]);
                   const Vector3& prev_P1 = prev_Ps(PP[1]);
                   Float          prev_D;
                   distance::point_point_distance(prev_P0, prev_P1, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);


                   const Vector3& P0 = Ps(PP[0]);
                   const Vector3& P1 = Ps(PP[1]);
                   Float          D  = D_hat;

                   // cout << "PP: " << PP.transpose().eval() << " D/prev_D: " << D
                   //      << "/" << prev_D << "\n";

                   Es(i) = sym::ipc_simplex_contact::PP_friction_energy(
                       kappa, D_hat, friction_rate, dt, P0, P1, prev_P0, prev_P1, eps_v);
               });
}

namespace sym::ipc_simplex_contact
{
    __device__ void PT_friction_test(Float           kappa,
                                     Float           d_hat,
                                     Float           friction_rate,
                                     Float           dt,
                                     Float           eps_v,
                                     const Vector4i& PT,
                                     const Vector3&  P,
                                     const Vector3&  T0,
                                     const Vector3&  T1,
                                     const Vector3&  T2,
                                     const muda::CDense1D<Vector3>& prev_Ps,
                                     const Vector12&               G_friction,
                                     const Matrix12x12&            H_friction);

    __device__ void EE_friction_test(Float           kappa,
                                     Float           d_hat,
                                     Float           friction_rate,
                                     Float           dt,
                                     Float           eps_v,
                                     const Vector4i& EE,
                                     const Vector3&  E0,
                                     const Vector3&  E1,
                                     const Vector3&  E2,
                                     const Vector3&  E3,
                                     const muda::CDense1D<Vector3>& prev_Ps,
                                     const Vector12&                G_friction,
                                     const Matrix12x12&             H_friction);

    __device__ void PE_friction_test(Float           kappa,
                                     Float           d_hat,
                                     Float           friction_rate,
                                     Float           dt,
                                     Float           eps_v,
                                     const Vector3i& PE,
                                     const Vector3&  P,
                                     const Vector3&  E0,
                                     const Vector3&  E1,
                                     const muda::CDense1D<Vector3>& prev_Ps,
                                     const Vector9&                 G_friction,
                                     const Matrix9x9&               H_friction);

    __device__ void PP_friction_test(Float            kappa,
                                     Float            d_hat,
                                     Float            friction_rate,
                                     Float            dt,
                                     Float            eps_v,
                                     const Vector2i&  PP,
                                     const Vector3&   P0,
                                     const Vector3&   P1,
                                     const Vector3&   prev_P0,
                                     const Vector3&   prev_P1,
                                     const Vector6&   G_friction,
                                     const Matrix6x6& H_friction);
}  // namespace sym::ipc_simplex_contact

void IPCSimplexFrictionalContact::do_assemble(ContactInfo& info)
{
    using namespace muda;

    // Compute Point-Triangle Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.friction_PTs().size(),
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PTs = info.friction_PTs().viewer().name("PTs"),
                Gs  = info.friction_PT_gradients().viewer().name("Gs"),
                Hs  = info.friction_PT_hessians().viewer().name("Hs"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PT = PTs(i);

                   auto cid_L = contact_ids(PT[0]);
                   auto cid_R = contact_ids(PT[1]);

                   const auto& P  = Ps(PT[0]);
                   const auto& T0 = Ps(PT[1]);
                   const auto& T1 = Ps(PT[2]);
                   const auto& T2 = Ps(PT[3]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;

                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Vector12    G_friction;
                   Matrix12x12 H_friction;
                   sym::ipc_simplex_contact::PT_friction_gradient_hessian(
                       G_friction,
                       H_friction,
                       kappa,
                       d_hat * d_hat,
                       friction_rate,
                       dt,
                       P,
                       T0,
                       T1,
                       T2,
                       prev_Ps(PT[0]),
                       prev_Ps(PT[1]),
                       prev_Ps(PT[2]),
                       prev_Ps(PT[3]),
                       eps_v);


                   if constexpr(DebugTest)
                   {
                       // Gradient check
                       sym::ipc_simplex_contact::PT_friction_test(
                           kappa, d_hat, friction_rate, dt, eps_v, PT, P, T0, T1, T2, prev_Ps, G_friction, H_friction);
                   }

                   Gs(i) = G_friction;
                   Hs(i) = H_friction;
               });

    // Compute Edge-Edge Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(
            info.friction_EEs().size(),
            [table = info.contact_tabular().viewer().name("contact_tabular"),
             contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
             EEs = info.friction_EEs().viewer().name("EEs"),
             Gs  = info.friction_EE_gradients().viewer().name("Gs"),
             Hs  = info.friction_EE_hessians().viewer().name("Hs"),
             Ps  = info.positions().viewer().name("Ps"),
             prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
             eps_v   = info.eps_velocity(),
             rest_Ps = info.rest_positions().viewer().name("rest_Ps"),
             d_hat   = info.d_hat(),
             dt      = info.dt()] __device__(int i) mutable
            {
                const auto& EE = EEs(i);

                auto cid_L = contact_ids(EE[0]);
                auto cid_R = contact_ids(EE[2]);

                auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                // Use this to compute friction
                auto friction_rate = table(cid_L, cid_R).mu;

                Float D_hat = d_hat * d_hat;

                const Vector3& prev_E0 = prev_Ps(EE[0]);
                const Vector3& prev_E1 = prev_Ps(EE[1]);
                const Vector3& prev_E2 = prev_Ps(EE[2]);
                const Vector3& prev_E3 = prev_Ps(EE[3]);

                Float prev_D;
                distance::edge_edge_distance(prev_E0, prev_E1, prev_E2, prev_E3, prev_D);
                MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);


                const Vector3& E0 = Ps(EE[0]);
                const Vector3& E1 = Ps(EE[1]);
                const Vector3& E2 = Ps(EE[2]);
                const Vector3& E3 = Ps(EE[3]);

                Vector12    G_friction;
                Matrix12x12 H_friction;
                sym::ipc_simplex_contact::EE_friction_gradient_hessian(G_friction,
                                                                       H_friction,
                                                                       kappa,
                                                                       d_hat * d_hat,
                                                                       friction_rate,
                                                                       dt,
                                                                       E0,
                                                                       E1,
                                                                       E2,
                                                                       E3,
                                                                       prev_E0,
                                                                       prev_E1,
                                                                       prev_E2,
                                                                       prev_E3,
                                                                       eps_v);

                if constexpr(DebugTest)
                {
                    // Gradient check
                    sym::ipc_simplex_contact::EE_friction_test(
                        kappa, d_hat, friction_rate, dt, eps_v, EE, E0, E1, E2, E3, prev_Ps, G_friction, H_friction);
                }

                Gs(i) = G_friction;
                Hs(i) = H_friction;
            });

    // Compute Point-Edge Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(
            info.friction_PEs().size(),
            [table = info.contact_tabular().viewer().name("contact_tabular"),
             contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
             PEs = info.friction_PEs().viewer().name("PEs"),
             Gs  = info.friction_PE_gradients().viewer().name("Gs"),
             Hs  = info.friction_PE_hessians().viewer().name("Hs"),
             Ps  = info.positions().viewer().name("Ps"),
             prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
             eps_v = info.eps_velocity(),
             d_hat = info.d_hat(),
             dt    = info.dt()] __device__(int i) mutable
            {
                const auto& PE = PEs(i);

                auto cid_L = contact_ids(PE[0]);
                auto cid_R = contact_ids(PE[1]);

                auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                // Use this to compute friction
                auto friction_rate = table(cid_L, cid_R).mu;

                Float D_hat = d_hat * d_hat;

                const Vector3& prev_P  = prev_Ps(PE[0]);
                const Vector3& prev_E0 = prev_Ps(PE[1]);
                const Vector3& prev_E1 = prev_Ps(PE[2]);
                Float          prev_D;
                distance::point_edge_distance(prev_P, prev_E0, prev_E1, prev_D);
                MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);

                const Vector3& P  = Ps(PE[0]);
                const Vector3& E0 = Ps(PE[1]);
                const Vector3& E1 = Ps(PE[2]);

                Vector9   G_friction = Eigen::Matrix<Float, 9, 1>::Zero();
                Matrix9x9 H_friction = Eigen::Matrix<Float, 9, 9>::Zero();
                sym::ipc_simplex_contact::PE_friction_gradient_hessian(
                    G_friction, H_friction, kappa, d_hat * d_hat, friction_rate, dt, P, E0, E1, prev_P, prev_E0, prev_E1, eps_v);

                if constexpr(DebugTest)
                {
                    // Gradient check
                    sym::ipc_simplex_contact::PE_friction_test(
                        kappa, d_hat, friction_rate, dt, eps_v, PE, P, E0, E1, prev_Ps, G_friction, H_friction);
                }

                Gs(i) = G_friction;
                Hs(i) = H_friction;
            });

    // Compute Point-Point Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(
            info.friction_PPs().size(),
            [table = info.contact_tabular().viewer().name("contact_tabular"),
             contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
             PPs = info.friction_PPs().viewer().name("PPs"),
             Gs  = info.friction_PP_gradients().viewer().name("Gs"),
             Hs  = info.friction_PP_hessians().viewer().name("Hs"),
             Ps  = info.positions().viewer().name("Ps"),
             prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
             eps_v = info.eps_velocity(),
             d_hat = info.d_hat(),
             dt    = info.dt()] __device__(int i) mutable
            {
                const auto& PP = PPs(i);

                auto cid_L = contact_ids(PP[0]);
                auto cid_R = contact_ids(PP[1]);

                auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                // Use this to compute friction
                auto friction_rate = table(cid_L, cid_R).mu;

                Float D_hat = d_hat * d_hat;

                const Vector3& prev_P0 = prev_Ps(PP[0]);
                const Vector3& prev_P1 = prev_Ps(PP[1]);
                Float          prev_D;
                distance::point_point_distance(prev_P0, prev_P1, prev_D);
                MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);

                const Vector3& P0 = Ps(PP[0]);
                const Vector3& P1 = Ps(PP[1]);
                Float          D  = D_hat;
                distance::point_point_distance(P0, P1, D);
                // NOTE: D can be larger than D_hat, if so, ignore this friction
                Vector6   G_friction;
                Matrix6x6 H_friction;
                sym::ipc_simplex_contact::PP_friction_gradient_hessian(
                    G_friction, H_friction, kappa, d_hat * d_hat, friction_rate, dt, P0, P1, prev_P0, prev_P1, eps_v);


                if constexpr(DebugTest)
                {
                    // Gradient check
                    sym::ipc_simplex_contact::PP_friction_test(
                        kappa, d_hat, friction_rate, dt, eps_v, PP, P0, P1, prev_P0, prev_P1, G_friction, H_friction);
                }


                Gs(i) = G_friction;
                Hs(i) = H_friction;
            });
}

namespace sym::ipc_simplex_contact
{
    __device__ void PT_friction_test(Float           kappa,
                                     Float           d_hat,
                                     Float           friction_rate,
                                     Float           dt,
                                     Float           eps_v,
                                     const Vector4i& PT,
                                     const Vector3&  P,
                                     const Vector3&  T0,
                                     const Vector3&  T1,
                                     const Vector3&  T2,
                                     const muda::CDense1D<Vector3>& prev_Ps,
                                     const Vector12&                G_friction,
                                     const Matrix12x12&             H_friction)
    {
        Vector3 test;
        test(0)  = 1e-8;
        test(1)  = 1e-8;
        test(2)  = 1e-8;
        Float E1 = sym::ipc_simplex_contact::PT_friction_energy(kappa,
                                                                d_hat * d_hat,
                                                                friction_rate,
                                                                dt,
                                                                P + test,
                                                                T0,
                                                                T1,
                                                                T2,
                                                                prev_Ps(PT[0]),
                                                                prev_Ps(PT[1]),
                                                                prev_Ps(PT[2]),
                                                                prev_Ps(PT[3]),
                                                                eps_v);
        Float E2 = sym::ipc_simplex_contact::PT_friction_energy(kappa,
                                                                d_hat * d_hat,
                                                                friction_rate,
                                                                dt,
                                                                P - test,
                                                                T0,
                                                                T1,
                                                                T2,
                                                                prev_Ps(PT[0]),
                                                                prev_Ps(PT[1]),
                                                                prev_Ps(PT[2]),
                                                                prev_Ps(PT[3]),
                                                                eps_v);
        Float numerical_diff = (E1 - E2) / 2;
        cout << "numerical_diff: " << numerical_diff << "\n";
        Float analytical_diff = G_friction(0) * test(0) + G_friction(1) * test(1)
                                + G_friction(2) * test(2);
        cout << "analytical_diff: " << analytical_diff << "\n";
        cout << "numerical_diff - analytical_diff: " << numerical_diff - analytical_diff
             << "\n";
        cout << "numerical_diff / analytical_diff: " << numerical_diff / analytical_diff
             << "\n";

        Vector12    G_friction1 = Eigen::Matrix<Float, 12, 1>::Zero();
        Vector12    G_friction2 = Eigen::Matrix<Float, 12, 1>::Zero();
        Matrix12x12 H_friction0 = Eigen::Matrix<Float, 12, 12>::Zero();
        Vector12    test12      = Eigen::Matrix<Float, 12, 1>::Zero();
        test12(0)               = 1e-8;
        test12(1)               = 1e-8;
        test12(2)               = 1e-8;
        sym::ipc_simplex_contact::PT_friction_gradient_hessian(G_friction1,
                                                               H_friction0,
                                                               kappa,
                                                               d_hat * d_hat,
                                                               friction_rate,
                                                               dt,
                                                               P + test12.segment<3>(0),
                                                               T0,
                                                               T1,
                                                               T2,
                                                               prev_Ps(PT[0]),
                                                               prev_Ps(PT[1]),
                                                               prev_Ps(PT[2]),
                                                               prev_Ps(PT[3]),
                                                               eps_v);
        sym::ipc_simplex_contact::PT_friction_gradient_hessian(G_friction2,
                                                               H_friction0,
                                                               kappa,
                                                               d_hat * d_hat,
                                                               friction_rate,
                                                               dt,
                                                               P - test12.segment<3>(0),
                                                               T0,
                                                               T1,
                                                               T2,
                                                               prev_Ps(PT[0]),
                                                               prev_Ps(PT[1]),
                                                               prev_Ps(PT[2]),
                                                               prev_Ps(PT[3]),
                                                               eps_v);

        Vector12 G_friction_numerical_diff  = (G_friction1 - G_friction2) / 2;
        Vector12 G_friction_analytical_diff = H_friction * test12;
        cout << "G_friction_numerical_diff: " << G_friction_numerical_diff.norm() << "\n";
        cout << "G_friction_analytical_diff: " << G_friction_analytical_diff.norm() << "\n";
        cout << "PT_grad_diff: "
             << (G_friction_numerical_diff - G_friction_analytical_diff).norm() << "\n";
    }

    __device__ void EE_friction_test(Float           kappa,
                                     Float           d_hat,
                                     Float           friction_rate,
                                     Float           dt,
                                     Float           eps_v,
                                     const Vector4i& EE,
                                     const Vector3&  E0,
                                     const Vector3&  E1,
                                     const Vector3&  E2,
                                     const Vector3&  E3,
                                     const muda::CDense1D<Vector3>& prev_Ps,
                                     const Vector12&                G_friction,
                                     const Matrix12x12&             H_friction)
    {
        Vector3 test;
        test(0)   = 1e-6;
        test(1)   = 1e-6;
        test(2)   = 1e-6;
        Float E1_ = sym::ipc_simplex_contact::EE_friction_energy(kappa,
                                                                 d_hat * d_hat,
                                                                 friction_rate,
                                                                 dt,
                                                                 E0 + test,
                                                                 E1,
                                                                 E2,
                                                                 E3,
                                                                 prev_Ps(EE[0]),
                                                                 prev_Ps(EE[1]),
                                                                 prev_Ps(EE[2]),
                                                                 prev_Ps(EE[3]),
                                                                 eps_v);
        Float E2_ = sym::ipc_simplex_contact::EE_friction_energy(kappa,
                                                                 d_hat * d_hat,
                                                                 friction_rate,
                                                                 dt,
                                                                 E0 - test,
                                                                 E1,
                                                                 E2,
                                                                 E3,
                                                                 prev_Ps(EE[0]),
                                                                 prev_Ps(EE[1]),
                                                                 prev_Ps(EE[2]),
                                                                 prev_Ps(EE[3]),
                                                                 eps_v);
        Float numerical_diff = (E1_ - E2_) / 2;
        cout << "numerical_diff: " << numerical_diff << "\n";
        Float analytical_diff = G_friction(0) * test(0) + G_friction(1) * test(1)
                                + G_friction(2) * test(2);
        cout << "analytical_diff: " << analytical_diff << "\n";
        cout << "numerical_diff - analytical_diff: " << numerical_diff - analytical_diff
             << "\n";
        cout << "numerical_diff / analytical_diff: " << numerical_diff / analytical_diff
             << "\n";

        Vector12    G_friction1 = Eigen::Matrix<Float, 12, 1>::Zero();
        Vector12    G_friction2 = Eigen::Matrix<Float, 12, 1>::Zero();
        Matrix12x12 H_friction0 = Eigen::Matrix<Float, 12, 12>::Zero();
        Vector12    test12      = Eigen::Matrix<Float, 12, 1>::Zero();
        test12(0)               = 1e-8;
        test12(1)               = 1e-8;
        test12(2)               = 1e-8;
        sym::ipc_simplex_contact::EE_friction_gradient_hessian(G_friction1,
                                                               H_friction0,
                                                               kappa,
                                                               d_hat * d_hat,
                                                               friction_rate,
                                                               dt,
                                                               E0 + test12.segment<3>(0),
                                                               E1,
                                                               E2,
                                                               E3,
                                                               prev_Ps(EE[0]),
                                                               prev_Ps(EE[1]),
                                                               prev_Ps(EE[2]),
                                                               prev_Ps(EE[3]),
                                                               eps_v);
        sym::ipc_simplex_contact::EE_friction_gradient_hessian(G_friction2,
                                                               H_friction0,
                                                               kappa,
                                                               d_hat * d_hat,
                                                               friction_rate,
                                                               dt,
                                                               E0 - test12.segment<3>(0),
                                                               E1,
                                                               E2,
                                                               E3,
                                                               prev_Ps(EE[0]),
                                                               prev_Ps(EE[1]),
                                                               prev_Ps(EE[2]),
                                                               prev_Ps(EE[3]),
                                                               eps_v);

        Vector12 G_friction_numerical_diff  = (G_friction1 - G_friction2) / 2;
        Vector12 G_friction_analytical_diff = H_friction * test12;
        cout << "G_friction_numerical_diff: " << G_friction_numerical_diff.norm() << "\n";
        cout << "G_friction_analytical_diff: " << G_friction_analytical_diff.norm() << "\n";
        cout << "EE_grad_diff: "
             << (G_friction_numerical_diff - G_friction_analytical_diff).norm() << "\n";
    }

    __device__ void PE_friction_test(Float           kappa,
                                     Float           d_hat,
                                     Float           friction_rate,
                                     Float           dt,
                                     Float           eps_v,
                                     const Vector3i& PE,
                                     const Vector3&  P,
                                     const Vector3&  E0,
                                     const Vector3&  E1,
                                     const muda::CDense1D<Vector3>& prev_Ps,
                                     const Vector9&                 G_friction,
                                     const Matrix9x9&               H_friction)
    {
        Vector3 test;
        test(0)   = 1e-6;
        test(1)   = 1e-6;
        test(2)   = 1e-6;
        Float E1_ = sym::ipc_simplex_contact::PE_friction_energy(kappa,
                                                                 d_hat * d_hat,
                                                                 friction_rate,
                                                                 dt,
                                                                 P + test,
                                                                 E0,
                                                                 E1,
                                                                 prev_Ps(PE[0]),
                                                                 prev_Ps(PE[1]),
                                                                 prev_Ps(PE[2]),
                                                                 eps_v);
        Float E2_ = sym::ipc_simplex_contact::PE_friction_energy(kappa,
                                                                 d_hat * d_hat,
                                                                 friction_rate,
                                                                 dt,
                                                                 P - test,
                                                                 E0,
                                                                 E1,
                                                                 prev_Ps(PE[0]),
                                                                 prev_Ps(PE[1]),
                                                                 prev_Ps(PE[2]),
                                                                 eps_v);
        Float numerical_diff = (E1_ - E2_) / 2;
        cout << "numerical_diff: " << numerical_diff << "\n";
        Float analytical_diff = G_friction(0) * test(0) + G_friction(1) * test(1)
                                + G_friction(2) * test(2);
        cout << "analytical_diff: " << analytical_diff << "\n";
        cout << "numerical_diff - analytical_diff: " << numerical_diff - analytical_diff
             << "\n";
        cout << "numerical_diff / analytical_diff: " << numerical_diff / analytical_diff
             << "\n";

        Vector9   G_friction1 = Eigen::Matrix<Float, 9, 1>::Zero();
        Vector9   G_friction2 = Eigen::Matrix<Float, 9, 1>::Zero();
        Matrix9x9 H_friction0 = Eigen::Matrix<Float, 9, 9>::Zero();
        Vector9   test9       = Eigen::Matrix<Float, 9, 1>::Zero();
        test9(0)              = 1e-8;
        test9(1)              = 1e-8;
        test9(2)              = 1e-8;
        sym::ipc_simplex_contact::PE_friction_gradient_hessian(G_friction1,
                                                               H_friction0,
                                                               kappa,
                                                               d_hat * d_hat,
                                                               friction_rate,
                                                               dt,
                                                               P + test9.segment<3>(0),
                                                               E0,
                                                               E1,
                                                               prev_Ps(PE[0]),
                                                               prev_Ps(PE[1]),
                                                               prev_Ps(PE[2]),
                                                               eps_v);
        sym::ipc_simplex_contact::PE_friction_gradient_hessian(G_friction2,
                                                               H_friction0,
                                                               kappa,
                                                               d_hat * d_hat,
                                                               friction_rate,
                                                               dt,
                                                               P - test9.segment<3>(0),
                                                               E0,
                                                               E1,
                                                               prev_Ps(PE[0]),
                                                               prev_Ps(PE[1]),
                                                               prev_Ps(PE[2]),
                                                               eps_v);

        Vector9 G_friction_numerical_diff  = (G_friction1 - G_friction2) / 2;
        Vector9 G_friction_analytical_diff = H_friction * test9;
        cout << "G_friction_numerical_diff: " << G_friction_numerical_diff.norm() << "\n";
        cout << "G_friction_analytical_diff: " << G_friction_analytical_diff.norm() << "\n";
        cout << "E1_: " << E1_ << "\n";
        cout << "G_friction: " << G_friction.norm() << "\n";
        cout << "H_friction: " << H_friction.norm() << "\n";
        cout << "PE_grad_diff: "
             << (G_friction_numerical_diff - G_friction_analytical_diff).norm() << "\n";
    }

    __device__ void PP_friction_test(Float            kappa,
                                     Float            d_hat,
                                     Float            friction_rate,
                                     Float            dt,
                                     Float            eps_v,
                                     const Vector2i&  PP,
                                     const Vector3&   P0,
                                     const Vector3&   P1,
                                     const Vector3&   prev_P0,
                                     const Vector3&   prev_P1,
                                     const Vector6&   G_friction,
                                     const Matrix6x6& H_friction)
    {
        Vector3 test;
        test(0)   = 1e-6;
        test(1)   = 1e-6;
        test(2)   = 1e-6;
        Float E1_ = sym::ipc_simplex_contact::PP_friction_energy(
            kappa, d_hat * d_hat, friction_rate, dt, P0 + test, P1, prev_P0, prev_P1, eps_v);
        Float E2_ = sym::ipc_simplex_contact::PP_friction_energy(
            kappa, d_hat * d_hat, friction_rate, dt, P0 - test, P1, prev_P0, prev_P1, eps_v);
        Float numerical_diff = (E1_ - E2_) / 2;
        cout << "numerical_diff: " << numerical_diff << "\n";
        Float analytical_diff = G_friction(0) * test(0) + G_friction(1) * test(1)
                                + G_friction(2) * test(2);
        cout << "analytical_diff: " << analytical_diff << "\n";
        cout << "numerical_diff - analytical_diff: " << numerical_diff - analytical_diff
             << "\n";
        cout << "numerical_diff / analytical_diff: " << numerical_diff / analytical_diff
             << "\n";

        Vector6   G_friction1 = Eigen::Matrix<Float, 6, 1>::Zero();
        Vector6   G_friction2 = Eigen::Matrix<Float, 6, 1>::Zero();
        Matrix6x6 H_friction0 = Eigen::Matrix<Float, 6, 6>::Zero();
        Vector6   test6       = Eigen::Matrix<Float, 6, 1>::Zero();
        test6(0)              = 1e-8;
        test6(1)              = 1e-8;
        test6(2)              = 1e-8;
        sym::ipc_simplex_contact::PP_friction_gradient_hessian(G_friction1,
                                                               H_friction0,
                                                               kappa,
                                                               d_hat * d_hat,
                                                               friction_rate,
                                                               dt,
                                                               P0 + test6.segment<3>(0),
                                                               P1,
                                                               prev_P0,
                                                               prev_P1,
                                                               eps_v);
        sym::ipc_simplex_contact::PP_friction_gradient_hessian(G_friction2,
                                                               H_friction0,
                                                               kappa,
                                                               d_hat * d_hat,
                                                               friction_rate,
                                                               dt,
                                                               P0 - test6.segment<3>(0),
                                                               P1,
                                                               prev_P0,
                                                               prev_P1,
                                                               eps_v);

        Vector6 G_friction_numerical_diff  = (G_friction1 - G_friction2) / 2;
        Vector6 G_friction_analytical_diff = H_friction * test6;
        cout << "G_friction_numerical_diff: " << G_friction_numerical_diff.norm() << "\n";
        cout << "G_friction_analytical_diff: " << G_friction_analytical_diff.norm() << "\n";
        cout << "EE_grad_diff: "
             << (G_friction_numerical_diff - G_friction_analytical_diff).norm() << "\n";
    }
}  // namespace sym::ipc_simplex_contact
}  // namespace uipc::backend::cuda