#include "hip/hip_runtime.h"
#include <contact_system/contact_models/ipc_simplex_frictional_contact.h>
#include <contact_system/contact_models/ipc_simplex_contact_function.h>

#include <kernel_cout.h>

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(IPCSimplexFrictionalContact);

void IPCSimplexFrictionalContact::do_build(BuildInfo& info) {}

namespace sym::ipc_simplex_contact
{
#include "sym/ipc_friction.inl"
#include "sym/ipc_contact.inl"

    __device__ Float PT_friction_energy(Float          kappa,
                                        Float          squared_d_hat,
                                        Float          mu,
                                        Float          dt,
                                        const Vector3& P,
                                        const Vector3& T0,
                                        const Vector3& T1,
                                        const Vector3& T2,
                                        const Vector3& prev_P,
                                        const Vector3& prev_T0,
                                        const Vector3& prev_T1,
                                        const Vector3& prev_T2,
                                        Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        point_triangle_distance(prev_P, prev_T0, prev_T1, prev_T2, D);
        MUDA_ASSERT(D <= squared_d_hat);
        Vector12 GradD;
        point_triangle_distance_gradient(prev_P, prev_T0, prev_T1, prev_T2, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);
        Float lam = -dBdD * GradD.head(3).norm();
        Vector3 n =  (prev_T0 - prev_T1).cross(prev_T0 - prev_T2);
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;
        Vector3 v1 = (P - prev_P) / dt;

        // suppose P0 = t0 * T0 + t1 * T1 + t2 * T2
        Eigen::Matrix<Float, 3, 2> base;
        base.block<3, 1>(0, 0) = prev_T1 - prev_T0;
        base.block<3, 1>(0, 1) = prev_T2 - prev_T0;
        Eigen::Matrix<Float, 2, 2> Lhs = base.transpose() * base;
        Vector2 rhs = base.transpose() * (prev_P - prev_T0);
        Eigen::Matrix<Float, 2, 2> Lhs_inv;
        Float det = Lhs(0, 0) * Lhs(1, 1) - Lhs(0, 1) * Lhs(1, 0);
        Lhs_inv(0, 0) = Lhs(1, 1) / det;
        Lhs_inv(0, 1) = -Lhs(0, 1) / det;
        Lhs_inv(1, 0) = -Lhs(1, 0) / det;
        Lhs_inv(1, 1) = Lhs(0, 0) / det;
        Vector2 t = Lhs_inv * rhs;
        Float t1 = t(0);
        Float t2 = t(1);
        Float t0 = 1 - t1 - t2;

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = v1;
        V.segment<3>(3) = t0 * (T0 - prev_T0) / dt + t1 * (T1 - prev_T1) / dt + t2 * (T2 - prev_T2) / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        Float F;
        FrictionEnergy(F, lam * mu, eps_v, dt, y);
        return F;
    }
 
    __device__ void PT_friction_gradient_hessian(Vector12&      G,
                                                 Matrix12x12&   H,
                                                 Float          kappa,
                                                 Float          squared_d_hat,
                                                 Float          mu,
                                                 Float          dt,
                                                 const Vector3& P,
                                                 const Vector3& T0,
                                                 const Vector3& T1,
                                                 const Vector3& T2,
                                                 const Vector3& prev_P,
                                                 const Vector3& prev_T0,
                                                 const Vector3& prev_T1,
                                                 const Vector3& prev_T2,
                                                 Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        point_triangle_distance(prev_P, prev_T0, prev_T1, prev_T2, D);
        MUDA_ASSERT(D <= squared_d_hat);
        Vector12 GradD;
        point_triangle_distance_gradient(prev_P, prev_T0, prev_T1, prev_T2, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);

        Float lam = -dBdD * GradD.head(3).norm();
        Vector3 n =  (prev_T0 - prev_T1).cross(prev_T0 - prev_T2);
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;
        Vector3 v1 = (P - prev_P) / dt;

        // suppose prev_P0 = t0 * prev_T0 + t1 * prev_T1 + t2 * prev_T2
        Eigen::Matrix<Float, 3, 2> base;
        base.block<3, 1>(0, 0) = prev_T1 - prev_T0;
        base.block<3, 1>(0, 1) = prev_T2 - prev_T0;
        Eigen::Matrix<Float, 2, 2> Lhs = base.transpose() * base;
        Vector2 rhs = base.transpose() * (prev_P - prev_T0);
        Eigen::Matrix<Float, 2, 2> Lhs_inv;
        Float det = Lhs(0, 0) * Lhs(1, 1) - Lhs(0, 1) * Lhs(1, 0);
        Lhs_inv(0, 0) = Lhs(1, 1) / det;
        Lhs_inv(0, 1) = -Lhs(0, 1) / det;
        Lhs_inv(1, 0) = -Lhs(1, 0) / det;
        Lhs_inv(1, 1) = Lhs(0, 0) / det;
        Vector2 t = Lhs_inv * rhs;
        Float t1 = t(0);
        Float t2 = t(1);
        Float t0 = 1 - t1 - t2;

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = v1;
        V.segment<3>(3) = t0 * (T0 - prev_T0) / dt + t1 * (T1 - prev_T1) / dt + t2 * (T2 - prev_T2) / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        Eigen::Vector<Float, 6> dFdV;
        dFrictionEnergydV(dFdV, lam * mu, Tk,  eps_v, dt, vk);
        Eigen::Matrix<Float, 6, 12> GradV = Eigen::Matrix<Float, 6, 12>::Zero();
        GradV.block<3, 3>(0, 0) = I / dt;
        GradV.block<3, 3>(3, 3) = I * t0 / dt;
        GradV.block<3, 3>(3, 6) = I * t1 / dt;
        GradV.block<3, 3>(3, 9) = I * t2 / dt;
        Eigen::Matrix<Float, 12, 6> GradV_transpose = GradV.transpose();
        G = GradV_transpose * dFdV;

        Eigen::Matrix<Float, 6, 6> ddFddV;
        ddFrictionEnergyddV(ddFddV, lam * mu, Tk, eps_v, dt, vk);
        H = GradV_transpose * ddFddV * GradV;
    }

    __device__ Float EE_friction_energy(Float          kappa,
                                        Float          squared_d_hat,
                                        Float          mu,
                                        Float          dt,
                                        const Vector3& P0,
                                        const Vector3& P1,
                                        const Vector3& Q0,
                                        const Vector3& Q1,
                                        const Vector3& prev_P0,
                                        const Vector3& prev_P1,
                                        const Vector3& prev_Q0,
                                        const Vector3& prev_Q1,
                                        Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        edge_edge_distance(prev_P0, prev_P1, prev_Q0, prev_Q1, D);
        MUDA_ASSERT(D <= squared_d_hat);
        Vector12 GradD;
        edge_edge_distance_gradient(prev_P0, prev_P1, prev_Q0, prev_Q1, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);
        Vector3 n =  (prev_P0 - prev_P1).cross(prev_Q0 - prev_Q1);
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;

        // suppose prev_P_project = prev_P0 + t0 * (prev_P1 - prev_P0)
        //         prev_Q_project = prev_Q0 + t1 * (prev_Q1 - prev_Q0)
        Eigen::Matrix<Float, 3, 2> base;
        base.block<3, 1>(0, 0) = prev_P1 - prev_P0;
        base.block<3, 1>(0, 1) = prev_Q1 - prev_Q0;
        Eigen::Matrix<Float, 2, 2> X = Eigen::Matrix<Float, 2, 2>::Zero();
        X(0, 0) = -1;
        X(1, 1) = 1;
        Eigen::Matrix<Float, 2, 2> Lhs = base.transpose() * base * X;
        Vector2 rhs = base.transpose() * (prev_P0 - prev_Q0);
        Eigen::Matrix<Float, 2, 2> Lhs_inv;
        Float det = Lhs(0, 0) * Lhs(1, 1) - Lhs(0, 1) * Lhs(1, 0);
        Lhs_inv(0, 0) = Lhs(1, 1) / det;
        Lhs_inv(0, 1) = -Lhs(0, 1) / det;
        Lhs_inv(1, 0) = -Lhs(1, 0) / det;
        Lhs_inv(1, 1) = Lhs(0, 0) / det;
        Vector2 t = Lhs_inv * rhs;
        Float t0 = t(0);
        Float t1 = t(1);

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = (P0 - prev_P0) * (1 - t0) / dt + (P1 - prev_P1) * t0 / dt;
        V.segment<3>(3) = (Q0 - prev_Q0) * (1 - t1) / dt + (Q1 - prev_Q1) * t1 / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        Float lam = -dBdD * (GradD.segment<3>(0) * (1 - t0) + GradD.segment<3>(3) * t0).norm();
        Float F;
        FrictionEnergy(F, lam * mu, eps_v, dt, y);
        return F;
    }

    __device__ void EE_friction_gradient_hessian(Vector12&      G,
                                        Matrix12x12&   H,
                                        Float          kappa,
                                        Float          squared_d_hat,
                                        Float          mu,
                                        Float          dt,
                                        const Vector3& P0,
                                        const Vector3& P1,
                                        const Vector3& Q0,
                                        const Vector3& Q1,
                                        const Vector3& prev_P0,
                                        const Vector3& prev_P1,
                                        const Vector3& prev_Q0,
                                        const Vector3& prev_Q1,
                                        Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        edge_edge_distance(prev_P0, prev_P1, prev_Q0, prev_Q1, D);
        MUDA_ASSERT(D <= squared_d_hat);
        Vector12 GradD;
        edge_edge_distance_gradient(prev_P0, prev_P1, prev_Q0, prev_Q1, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);
        Vector3 n =  (prev_P0 - prev_P1).cross(prev_Q0 - prev_Q1);
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;

        // suppose prev_P_project = prev_P0 + t0 * (prev_P1 - prev_P0)
        //         prev_Q_project = prev_Q0 + t1 * (prev_Q1 - prev_Q0)
        Eigen::Matrix<Float, 3, 2> base;
        base.block<3, 1>(0, 0) = prev_P1 - prev_P0;
        base.block<3, 1>(0, 1) = prev_Q1 - prev_Q0;
        Eigen::Matrix<Float, 2, 2> X = Eigen::Matrix<Float, 2, 2>::Zero();
        X(0, 0) = -1;
        X(1, 1) = 1;
        Eigen::Matrix<Float, 2, 2> Lhs = base.transpose() * base * X;
        Vector2 rhs = base.transpose() * (prev_P0 - prev_Q0);
        Eigen::Matrix<Float, 2, 2> Lhs_inv;
        Float det = Lhs(0, 0) * Lhs(1, 1) - Lhs(0, 1) * Lhs(1, 0);
        Lhs_inv(0, 0) = Lhs(1, 1) / det;
        Lhs_inv(0, 1) = -Lhs(0, 1) / det;
        Lhs_inv(1, 0) = -Lhs(1, 0) / det;
        Lhs_inv(1, 1) = Lhs(0, 0) / det;
        Vector2 t = Lhs_inv * rhs;
        Float t0 = t(0);
        Float t1 = t(1);

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = (P0 - prev_P0) * (1 - t0) / dt + (P1 - prev_P1) * t0 / dt;
        V.segment<3>(3) = (Q0 - prev_Q0) * (1 - t1) / dt + (Q1 - prev_Q1) * t1 / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        Float lam = -dBdD * (GradD.segment<3>(0) * (1 - t0) + GradD.segment<3>(3) * t0).norm();

        Eigen::Vector<Float, 6> dFdV;
        dFrictionEnergydV(dFdV, lam * mu, Tk,  eps_v, dt, vk);
        Eigen::Matrix<Float, 6, 12> GradV = Eigen::Matrix<Float, 6, 12>::Zero();
        GradV.block<3, 3>(0, 0) = I * (1 - t0)/ dt;
        GradV.block<3, 3>(0, 3) = I * t0 / dt;
        GradV.block<3, 3>(3, 6) = I * (1 - t1) / dt;
        GradV.block<3, 3>(3, 9) = I * t1 / dt;
        Eigen::Matrix<Float, 12, 6> GradV_transpose = GradV.transpose();
        G = GradV_transpose * dFdV;

        Eigen::Matrix<Float, 6, 6> ddFddV;
        ddFrictionEnergyddV(ddFddV, lam * mu, Tk, eps_v, dt, vk);
        H = GradV_transpose * ddFddV * GradV;
    }

    __device__ Float PE_friction_energy(Float          kappa,
                                        Float          squared_d_hat,
                                        Float          mu,
                                        Float          dt,
                                        const Vector3& P,
                                        const Vector3& E0,
                                        const Vector3& E1,
                                        const Vector3& prev_P,
                                        const Vector3& prev_E0,
                                        const Vector3& prev_E1,
                                        Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        point_edge_distance(prev_P, prev_E0, prev_E1, D);
        MUDA_ASSERT(D <= squared_d_hat);
        Vector9 GradD;
        point_edge_distance_gradient(prev_P, prev_E0, prev_E1, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);
        Float lam = -dBdD * GradD.head(3).norm();
        // suppose prev_P0 = t0 * prev_E0 + t1 * prev_E1
        Float t0 = (prev_P - prev_E1).dot(prev_E0 - prev_E1) / (prev_E0 - prev_E1).squaredNorm();
        Float t1 = 1 - t0;

        Vector3 prev_P0 = t0 * prev_E0 + t1 * prev_E1;
        Vector3 n = prev_P0 - prev_P;
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;
        Vector3 v1 = (P - prev_P) / dt;

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = v1;
        V.segment<3>(3) = t0 * (E0 - prev_E0) / dt + t1 * (E1 - prev_E1) / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        Float F;
        FrictionEnergy(F, lam * mu, eps_v, dt, y);
        return F;
    }

    __device__ void PE_friction_gradient_hessian(Vector9&       G,
                                                 Matrix9x9&     H,
                                                 Float          kappa,
                                                 Float          squared_d_hat,
                                                 Float          mu,
                                                 Float          dt,
                                                 const Vector3& P,
                                                 const Vector3& E0,
                                                 const Vector3& E1,
                                                 const Vector3& prev_P,
                                                 const Vector3& prev_E0,
                                                 const Vector3& prev_E1,
                                                 Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        point_edge_distance(prev_P, prev_E0, prev_E1, D);
        MUDA_ASSERT(D <= squared_d_hat);
        Vector9 GradD = Vector9::Zero();
        point_edge_distance_gradient(prev_P, prev_E0, prev_E1, GradD);

        Float dBdD = 0;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);

        Float lam = -dBdD * GradD.head(3).norm();

        // suppose prev_P0 = t0 * prev_E0 + t1 * prev_E1
        Float t0 = (prev_P - prev_E1).dot(prev_E0 - prev_E1) / (prev_E0 - prev_E1).squaredNorm();
        Float t1 = 1 - t0;

        Vector3 prev_P0 = t0 * prev_E0 + t1 * prev_E1;
        Vector3 n = prev_P0 - prev_P;
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;
        Vector3 v1 = (P - prev_P) / dt;

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = v1;
        V.segment<3>(3) = t0 * (E0 - prev_E0) / dt + t1 * (E1 - prev_E1) / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        Eigen::Vector<Float, 6> dFdV;
        dFrictionEnergydV(dFdV, lam * mu, Tk,  eps_v, dt, vk);
        Eigen::Matrix<Float, 6, 9> GradV = Eigen::Matrix<Float, 6, 9>::Zero();
        GradV.block<3, 3>(0, 0) = I / dt;
        GradV.block<3, 3>(3, 3) = I * t0 / dt;
        GradV.block<3, 3>(3, 6) = I * t1 / dt;
        Eigen::Matrix<Float, 9, 6> GradV_transpose = GradV.transpose();
        G = GradV_transpose * dFdV;

        Eigen::Matrix<Float, 6, 6> ddFddV;
        ddFrictionEnergyddV(ddFddV, lam * mu, Tk, eps_v, dt, vk);
        H = GradV_transpose * ddFddV * GradV;
    }

    __device__ Float PP_friction_energy(Float          kappa,
                                        Float          squared_d_hat,
                                        Float          mu,
                                        Float          dt,
                                        const Vector3& P,
                                        const Vector3& Q,
                                        const Vector3& prev_P,
                                        const Vector3& prev_Q,
                                        Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        point_point_distance(prev_P, prev_Q, D);
        MUDA_ASSERT(D <= squared_d_hat);
        Vector6 GradD;
        point_point_distance_gradient(prev_P, prev_Q, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);
        Float lam = -dBdD * GradD.head(3).norm();

        Vector3 n = prev_Q - prev_P;
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = (P - prev_P) / dt;
        V.segment<3>(3) = (Q - prev_Q) / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        Float F;
        FrictionEnergy(F, lam * mu, eps_v, dt, y);
        return F;
    }

    __device__ void PP_friction_gradient_hessian(Vector6&      G,
                                                 Matrix6x6&   H,
                                                 Float          kappa,
                                                 Float          squared_d_hat,
                                                 Float          mu,
                                                 Float          dt,
                                                 const Vector3& P,
                                                 const Vector3& Q,
                                                 const Vector3& prev_P,
                                                 const Vector3& prev_Q,
                                                 Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        point_point_distance(prev_P, prev_Q, D);
        MUDA_ASSERT(D <= squared_d_hat);
        Vector6 GradD;
        point_point_distance_gradient(prev_P, prev_Q, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);
        Float lam = -dBdD * GradD.head(3).norm();

        Vector3 n = prev_Q - prev_P;
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = (P - prev_P) / dt;
        V.segment<3>(3) = (Q - prev_Q) / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        Eigen::Vector<Float, 6> dFdV;
        dFrictionEnergydV(dFdV, lam * mu, Tk,  eps_v, dt, vk);
        // GradV = Eigen::Matrix<Float, 6, 6>::Identity();
        G = dFdV;

        Eigen::Matrix<Float, 6, 6> ddFddV;
        ddFrictionEnergyddV(ddFddV, lam * mu, Tk, eps_v, dt, vk);
        H = ddFddV;
    }

}

void IPCSimplexFrictionalContact::do_compute_energy(EnergyInfo& info)
{
    using namespace muda;

    // Compute Point-Triangle energy
    auto PT_count = info.friction_PTs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(PT_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PTs = info.friction_PTs().viewer().name("PTs"),
                Es  = info.friction_PT_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PT = PTs(i);

                   auto cid_L = contact_ids(PT[0]);
                   auto cid_R = contact_ids(PT[1]);

                   const auto& P  = Ps(PT[0]);
                   const auto& T0 = Ps(PT[1]);
                   const auto& T1 = Ps(PT[2]);
                   const auto& T2 = Ps(PT[3]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;

                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;


                   Float D_hat = d_hat * d_hat;

                   Float          prev_D;
                   const Vector3& prev_P  = prev_Ps(PT[0]);
                   const Vector3& prev_T0 = prev_Ps(PT[1]);
                   const Vector3& prev_T1 = prev_Ps(PT[2]);
                   const Vector3& prev_T2 = prev_Ps(PT[3]);
                   distance::point_triangle_distance(prev_P, prev_T0, prev_T1, prev_T2, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);

                   Float          D;
                   distance::point_triangle_distance(P, T0, T1, T2, D);
                   // NOTE: D can be larger than D_hat

                   // cout << "PT: " << PT.transpose().eval() << " D/prev_D: " << D
                   //      << "/" << prev_D << "\n";

                   Es(i) = sym::ipc_simplex_contact::PT_friction_energy(kappa, D_hat, friction_rate, dt, P, T0, T1, T2, prev_P, prev_T0, prev_T1, prev_T2, eps_v);
               });

    // Compute Edge-Edge energy
    auto EE_count = info.friction_EEs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(EE_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                EEs = info.friction_EEs().viewer().name("EEs"),
                Es  = info.friction_EE_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v   = info.eps_velocity(),
                rest_Ps = info.rest_positions().viewer().name("rest_Ps"),
                d_hat   = info.d_hat(),
                dt      = info.dt()] __device__(int i) mutable
               {
                   const auto& EE = EEs(i);

                   auto cid_L = contact_ids(EE[0]);
                   auto cid_R = contact_ids(EE[2]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   const Vector3& prev_E0 = prev_Ps(EE[0]);
                   const Vector3& prev_E1 = prev_Ps(EE[1]);
                   const Vector3& prev_E2 = prev_Ps(EE[2]);
                   const Vector3& prev_E3 = prev_Ps(EE[3]);

                   Float D_hat = d_hat * d_hat;

                   Float prev_D;
                   distance::edge_edge_distance(prev_E0, prev_E1, prev_E2, prev_E3, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);


                   const Vector3& E0 = Ps(EE[0]);
                   const Vector3& E1 = Ps(EE[1]);
                   const Vector3& E2 = Ps(EE[2]);
                   const Vector3& E3 = Ps(EE[3]);

                   // cout << "EE: " << EE.transpose().eval() << " D/prev_D: " << D
                   //      << "/" << prev_D << "\n";

                   Es(i) = sym::ipc_simplex_contact::EE_friction_energy(kappa, D_hat, friction_rate, dt, E0, E1, E2, E3, prev_E0, prev_E1, prev_E2, prev_E3, eps_v);
               });

    // Compute Point-Edge energy
    auto PE_count = info.friction_PEs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(PE_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PEs = info.friction_PEs().viewer().name("PEs"),
                Es  = info.friction_PE_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PE = PEs(i);

                   auto cid_L = contact_ids(PE[0]);
                   auto cid_R = contact_ids(PE[1]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Float D_hat = d_hat * d_hat;


                   const Vector3& prev_P  = prev_Ps(PE[0]);
                   const Vector3& prev_E0 = prev_Ps(PE[1]);
                   const Vector3& prev_E1 = prev_Ps(PE[2]);
                   Float          prev_D;
                   distance::point_edge_distance(prev_P, prev_E0, prev_E1, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);

                   const Vector3& P  = Ps(PE[0]);
                   const Vector3& E0 = Ps(PE[1]);
                   const Vector3& E1 = Ps(PE[2]);

                   // cout << "PE: " << PE.transpose().eval() << " D/prev_D: " << D
                   //      << "/" << prev_D << "\n";

                   Es(i) = sym::ipc_simplex_contact::PE_friction_energy(kappa, D_hat, friction_rate, dt, P, E0, E1, prev_P, prev_E0, prev_E1, eps_v);
               });

    // Compute Point-Point energy
    auto PP_count = info.friction_PPs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(PP_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PPs = info.friction_PPs().viewer().name("PPs"),
                Es  = info.friction_PP_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PP = PPs(i);

                   auto cid_L = contact_ids(PP[0]);
                   auto cid_R = contact_ids(PP[1]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Float D_hat = d_hat * d_hat;

                   const Vector3& prev_P0 = prev_Ps(PP[0]);
                   const Vector3& prev_P1 = prev_Ps(PP[1]);
                   Float          prev_D;
                   distance::point_point_distance(prev_P0, prev_P1, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);


                   const Vector3& P0 = Ps(PP[0]);
                   const Vector3& P1 = Ps(PP[1]);
                   Float          D  = D_hat;

                   // cout << "PP: " << PP.transpose().eval() << " D/prev_D: " << D
                   //      << "/" << prev_D << "\n";

                   Es(i) = sym::ipc_simplex_contact::PP_friction_energy(kappa, D_hat, friction_rate, dt, P0, P1, prev_P0, prev_P1, eps_v);
               });
}

void IPCSimplexFrictionalContact::do_assemble(ContactInfo& info)
{
    using namespace muda;

    // Compute Point-Triangle Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.friction_PTs().size(),
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PTs = info.friction_PTs().viewer().name("PTs"),
                Gs  = info.friction_PT_gradients().viewer().name("Gs"),
                Hs  = info.friction_PT_hessians().viewer().name("Hs"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PT = PTs(i);

                   auto cid_L = contact_ids(PT[0]);
                   auto cid_R = contact_ids(PT[1]);

                   const auto& P  = Ps(PT[0]);
                   const auto& T0 = Ps(PT[1]);
                   const auto& T1 = Ps(PT[2]);
                   const auto& T2 = Ps(PT[3]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;

                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Vector12 G_friction;
                   Matrix12x12 H_friction;
                   sym::ipc_simplex_contact::PT_friction_gradient_hessian(
                          G_friction, H_friction, kappa, d_hat * d_hat, friction_rate, dt, P, T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v);
                   // Gradient check
                   /*
                   Vector3 test;
                   test(0) = 1e-8;
                   test(1) = 1e-8;
                   test(2) = 1e-8;
                   Float E1 = sym::ipc_simplex_contact::PT_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, P + test, T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v);
                   Float E2 = sym::ipc_simplex_contact::PT_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, P - test, T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v);
                   Float numerical_diff = (E1 - E2) / 2;
                     cout << "numerical_diff: " << numerical_diff << "\n";
                   Float analytical_diff = G_friction(0) * test(0) + G_friction(1) * test(1) + G_friction(2) * test(2);
                     cout << "analytical_diff: " << analytical_diff << "\n";
                   cout << "numerical_diff - analytical_diff: " << numerical_diff - analytical_diff << "\n";
                   cout << "numerical_diff / analytical_diff: " << numerical_diff / analytical_diff << "\n";

                   Vector12 G_friction1 = Eigen::Matrix<Float, 12, 1>::Zero();
                   Vector12 G_friction2 = Eigen::Matrix<Float, 12, 1>::Zero();
                   Matrix12x12 H_friction0 = Eigen::Matrix<Float, 12, 12>::Zero();
                   Vector12 test12 = Eigen::Matrix<Float, 12, 1>::Zero();
                   test12(0) = 1e-8;
                   test12(1) = 1e-8;
                   test12(2) = 1e-8;
                   sym::ipc_simplex_contact::PT_friction_gradient_hessian(
                              G_friction1, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, P + test12.segment<3>(0), T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v);
                   sym::ipc_simplex_contact::PT_friction_gradient_hessian(
                              G_friction2, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, P - test12.segment<3>(0), T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v);

                   Vector12 G_friction_numerical_diff = (G_friction1 - G_friction2) / 2;
                   Vector12 G_friction_analytical_diff = H_friction * test12;
                   cout << "G_friction_numerical_diff: " << G_friction_numerical_diff.norm() << "\n";
                   cout << "G_friction_analytical_diff: " << G_friction_analytical_diff.norm() << "\n";
                   cout << "PT_grad_diff: " << (G_friction_numerical_diff - G_friction_analytical_diff).norm() << "\n";
                   */
                   Gs(i) = G_friction;
                   Hs(i) = H_friction;
               });

    // Compute Edge-Edge Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.friction_EEs().size(),
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                EEs = info.friction_EEs().viewer().name("EEs"),
                Gs  = info.friction_EE_gradients().viewer().name("Gs"),
                Hs  = info.friction_EE_hessians().viewer().name("Hs"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v   = info.eps_velocity(),
                rest_Ps = info.rest_positions().viewer().name("rest_Ps"),
                d_hat   = info.d_hat(),
                dt      = info.dt()] __device__(int i) mutable
               {
                   const auto& EE = EEs(i);

                   auto cid_L = contact_ids(EE[0]);
                   auto cid_R = contact_ids(EE[2]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Float D_hat = d_hat * d_hat;

                   const Vector3& prev_E0 = prev_Ps(EE[0]);
                   const Vector3& prev_E1 = prev_Ps(EE[1]);
                   const Vector3& prev_E2 = prev_Ps(EE[2]);
                   const Vector3& prev_E3 = prev_Ps(EE[3]);

                   Float prev_D;
                   distance::edge_edge_distance(prev_E0, prev_E1, prev_E2, prev_E3, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);


                   const Vector3& E0 = Ps(EE[0]);
                   const Vector3& E1 = Ps(EE[1]);
                   const Vector3& E2 = Ps(EE[2]);
                   const Vector3& E3 = Ps(EE[3]);

                   Vector12 G_friction;
                   Matrix12x12 H_friction;
                   sym::ipc_simplex_contact::EE_friction_gradient_hessian(
                          G_friction, H_friction, kappa, d_hat * d_hat, friction_rate, dt, E0, E1, E2, E3, prev_E0, prev_E1, prev_E2, prev_E3, eps_v);
                   // Gradient check
                   /*
                   Vector3 test;
                   test(0) = 1e-6;
                   test(1) = 1e-6;
                   test(2) = 1e-6;
                   Float E1_ = sym::ipc_simplex_contact::EE_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, E0 + test, E1, E2, E3, prev_Ps(EE[0]), prev_Ps(EE[1]), prev_Ps(EE[2]), prev_Ps(EE[3]), eps_v);
                   Float E2_ = sym::ipc_simplex_contact::EE_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, E0 - test, E1, E2, E3, prev_Ps(EE[0]), prev_Ps(EE[1]), prev_Ps(EE[2]), prev_Ps(EE[3]), eps_v);
                   Float numerical_diff = (E1_ - E2_) / 2;
                     cout << "numerical_diff: " << numerical_diff << "\n";
                   Float analytical_diff = G_friction(0) * test(0) + G_friction(1) * test(1) + G_friction(2) * test(2);
                     cout << "analytical_diff: " << analytical_diff << "\n";
                   cout << "numerical_diff - analytical_diff: " << numerical_diff - analytical_diff << "\n";
                   cout << "numerical_diff / analytical_diff: " << numerical_diff / analytical_diff << "\n";

                   Vector12 G_friction1 = Eigen::Matrix<Float, 12, 1>::Zero();
                   Vector12 G_friction2 = Eigen::Matrix<Float, 12, 1>::Zero();
                   Matrix12x12 H_friction0 = Eigen::Matrix<Float, 12, 12>::Zero();
                   Vector12 test12 = Eigen::Matrix<Float, 12, 1>::Zero();
                   test12(0) = 1e-8;
                   test12(1) = 1e-8;
                   test12(2) = 1e-8;
                   sym::ipc_simplex_contact::EE_friction_gradient_hessian(
                              G_friction1, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, E0 + test12.segment<3>(0), E1, E2, E3, prev_Ps(EE[0]), prev_Ps(EE[1]), prev_Ps(EE[2]), prev_Ps(EE[3]), eps_v);
                   sym::ipc_simplex_contact::EE_friction_gradient_hessian(
                              G_friction2, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, E0 - test12.segment<3>(0), E1, E2, E3, prev_Ps(EE[0]), prev_Ps(EE[1]), prev_Ps(EE[2]), prev_Ps(EE[3]), eps_v);

                   Vector12 G_friction_numerical_diff = (G_friction1 - G_friction2) / 2;
                   Vector12 G_friction_analytical_diff = H_friction * test12;
                   cout << "G_friction_numerical_diff: " << G_friction_numerical_diff.norm() << "\n";
                   cout << "G_friction_analytical_diff: " << G_friction_analytical_diff.norm() << "\n";
                   cout << "EE_grad_diff: " << (G_friction_numerical_diff - G_friction_analytical_diff).norm() << "\n";
                   */
                   Gs(i) = G_friction;
                   Hs(i) = H_friction;
               });

    // Compute Point-Edge Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.friction_PEs().size(),
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PEs = info.friction_PEs().viewer().name("PEs"),
                Gs  = info.friction_PE_gradients().viewer().name("Gs"),
                Hs  = info.friction_PE_hessians().viewer().name("Hs"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PE = PEs(i);

                   auto cid_L = contact_ids(PE[0]);
                   auto cid_R = contact_ids(PE[1]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Float D_hat = d_hat * d_hat;

                   const Vector3& prev_P  = prev_Ps(PE[0]);
                   const Vector3& prev_E0 = prev_Ps(PE[1]);
                   const Vector3& prev_E1 = prev_Ps(PE[2]);
                   Float          prev_D;
                   distance::point_edge_distance(prev_P, prev_E0, prev_E1, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);

                   const Vector3& P  = Ps(PE[0]);
                   const Vector3& E0 = Ps(PE[1]);
                   const Vector3& E1 = Ps(PE[2]);

                   Vector9 G_friction = Eigen::Matrix<Float, 9, 1>::Zero();
                   Matrix9x9 H_friction = Eigen::Matrix<Float, 9, 9>::Zero();
                   sym::ipc_simplex_contact::PE_friction_gradient_hessian(
                          G_friction, H_friction, kappa, d_hat * d_hat, friction_rate, dt, P, E0, E1, prev_P, prev_E0, prev_E1, eps_v);
                   // Gradient check
                   /*
                   Vector3 test;
                   test(0) = 1e-6;
                   test(1) = 1e-6;
                   test(2) = 1e-6;
                   Float E1_ = sym::ipc_simplex_contact::PE_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, P + test, E0, E1, prev_Ps(PE[0]), prev_Ps(PE[1]), prev_Ps(PE[2]), eps_v);
                   Float E2_ = sym::ipc_simplex_contact::PE_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, P - test, E0, E1, prev_Ps(PE[0]), prev_Ps(PE[1]), prev_Ps(PE[2]), eps_v);
                   Float numerical_diff = (E1_ - E2_) / 2;
                     cout << "numerical_diff: " << numerical_diff << "\n";
                   Float analytical_diff = G_friction(0) * test(0) + G_friction(1) * test(1) + G_friction(2) * test(2);
                     cout << "analytical_diff: " << analytical_diff << "\n";
                   cout << "numerical_diff - analytical_diff: " << numerical_diff - analytical_diff << "\n";
                   cout << "numerical_diff / analytical_diff: " << numerical_diff / analytical_diff << "\n";

                   Vector9 G_friction1 = Eigen::Matrix<Float, 9, 1>::Zero();
                   Vector9 G_friction2 = Eigen::Matrix<Float, 9, 1>::Zero();
                   Matrix9x9 H_friction0 = Eigen::Matrix<Float, 9, 9>::Zero();
                   Vector9 test9 = Eigen::Matrix<Float, 9, 1>::Zero();
                   test9(0) = 1e-8;
                   test9(1) = 1e-8;
                   test9(2) = 1e-8;
                   sym::ipc_simplex_contact::PE_friction_gradient_hessian(
                              G_friction1, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, P + test9.segment<3>(0), E0, E1, prev_Ps(PE[0]), prev_Ps(PE[1]), prev_Ps(PE[2]), eps_v);
                   sym::ipc_simplex_contact::PE_friction_gradient_hessian(
                              G_friction2, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, P - test9.segment<3>(0), E0, E1, prev_Ps(PE[0]), prev_Ps(PE[1]), prev_Ps(PE[2]), eps_v);

                   Vector9 G_friction_numerical_diff = (G_friction1 - G_friction2) / 2;
                   Vector9 G_friction_analytical_diff = H_friction * test9;
                   cout << "G_friction_numerical_diff: " << G_friction_numerical_diff.norm() << "\n";
                   cout << "G_friction_analytical_diff: " << G_friction_analytical_diff.norm() << "\n";
                   cout << "E1_: " << E1_ << "\n";
                   cout << "G_friction: " << G_friction.norm() << "\n";
                   cout << "H_friction: " << H_friction.norm() << "\n";
                   cout << "PE_grad_diff: " << (G_friction_numerical_diff - G_friction_analytical_diff).norm() << "\n";
                   */
                   Gs(i) = G_friction;
                   Hs(i) = H_friction;
               });

    // Compute Point-Point Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.friction_PPs().size(),
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PPs = info.friction_PPs().viewer().name("PPs"),
                Gs  = info.friction_PP_gradients().viewer().name("Gs"),
                Hs  = info.friction_PP_hessians().viewer().name("Hs"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PP = PPs(i);

                   auto cid_L = contact_ids(PP[0]);
                   auto cid_R = contact_ids(PP[1]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Float D_hat = d_hat * d_hat;

                   const Vector3& prev_P0 = prev_Ps(PP[0]);
                   const Vector3& prev_P1 = prev_Ps(PP[1]);
                   Float          prev_D;
                   distance::point_point_distance(prev_P0, prev_P1, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);

                   const Vector3& P0 = Ps(PP[0]);
                   const Vector3& P1 = Ps(PP[1]);
                   Float          D  = D_hat;
                   distance::point_point_distance(P0, P1, D);
                   // NOTE: D can be larger than D_hat, if so, ignore this friction
                   Vector6 G_friction;
                   Matrix6x6 H_friction;
                   sym::ipc_simplex_contact::PP_friction_gradient_hessian(
                          G_friction, H_friction, kappa, d_hat * d_hat, friction_rate, dt, P0, P1, prev_P0, prev_P1, eps_v);
                   // Gradient check

                //    Vector3 test;
                //    test(0) = 1e-6;
                //    test(1) = 1e-6;
                //    test(2) = 1e-6;
                //    Float E1_ = sym::ipc_simplex_contact::PP_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, P0 + test, P1, prev_P0, prev_P1, eps_v);
                //    Float E2_ = sym::ipc_simplex_contact::PP_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, P0 - test, P1, prev_P0, prev_P1, eps_v);
                //    Float numerical_diff = (E1_ - E2_) / 2;
                //      cout << "numerical_diff: " << numerical_diff << "\n";
                //    Float analytical_diff = G_friction(0) * test(0) + G_friction(1) * test(1) + G_friction(2) * test(2);
                //      cout << "analytical_diff: " << analytical_diff << "\n";
                //    cout << "numerical_diff - analytical_diff: " << numerical_diff - analytical_diff << "\n";
                //    cout << "numerical_diff / analytical_diff: " << numerical_diff / analytical_diff << "\n";

                //    Vector6 G_friction1 = Eigen::Matrix<Float, 6, 1>::Zero();
                //    Vector6 G_friction2 = Eigen::Matrix<Float, 6, 1>::Zero();
                //    Matrix6x6 H_friction0 = Eigen::Matrix<Float, 6, 6>::Zero();
                //    Vector6 test6 = Eigen::Matrix<Float, 6, 1>::Zero();
                //    test6(0) = 1e-8;
                //    test6(1) = 1e-8;
                //    test6(2) = 1e-8;
                //    sym::ipc_simplex_contact::PP_friction_gradient_hessian(
                //               G_friction1, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, P0 + test6.segment<3>(0), P1, prev_P0, prev_P1, eps_v);
                //    sym::ipc_simplex_contact::PP_friction_gradient_hessian(
                //               G_friction2, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, P0 - test6.segment<3>(0), P1, prev_P0, prev_P1, eps_v);

                //    Vector6 G_friction_numerical_diff = (G_friction1 - G_friction2) / 2;
                //    Vector6 G_friction_analytical_diff = H_friction * test6;
                //    cout << "G_friction_numerical_diff: " << G_friction_numerical_diff.norm() << "\n";
                //    cout << "G_friction_analytical_diff: " << G_friction_analytical_diff.norm() << "\n";
                //    cout << "EE_grad_diff: " << (G_friction_numerical_diff - G_friction_analytical_diff).norm() << "\n";

                   Gs(i) = G_friction;
                   Hs(i) = H_friction;
               });
}
}  // namespace uipc::backend::cuda