#include "hip/hip_runtime.h"
#include <contact_system/contact_models/ipc_simplex_frictional_contact.h>
#include <contact_system/contact_models/ipc_simplex_contact_function.h>

#include <kernel_cout.h>

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(IPCSimplexFrictionalContact);

void IPCSimplexFrictionalContact::do_build(BuildInfo& info) {}

namespace sym::ipc_simplex_contact
{
#include "sym/ipc_friction.inl"
#include "sym/ipc_contact.inl"

    __device__ Float PT_friction_energy(Float          kappa,
                                        Float          squared_d_hat,
                                        Float          mu,
                                        Float          dt,
                                        const Vector3& P,
                                        const Vector3& T0,
                                        const Vector3& T1,
                                        const Vector3& T2,
                                        const Vector3& prev_P,
                                        const Vector3& prev_T0,
                                        const Vector3& prev_T1,
                                        const Vector3& prev_T2,
                                        Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        point_triangle_distance(prev_P, prev_T0, prev_T1, prev_T2, D);
        Float D_now;
        point_triangle_distance(P, T0, T1, T2, D_now);
        // cout << "D - squared_d_hat: " << D - squared_d_hat << "\n";
        // cout << "D_now - squared_d_hat: " << D_now - squared_d_hat << "\n";
        if (D >= squared_d_hat) {
            return 0;
        }
        Vector12 GradD;
        point_triangle_distance_gradient(prev_P, prev_T0, prev_T1, prev_T2, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);
        // cout << "dBdD: " << dBdD << "\n";
        // cout << "GradD.head(3).norm(): " << GradD.head(3).norm() << "\n";
        // cout << "D: " << D << "\n";
        // cout << "D_now: " << D_now << "\n";
        // cout << "squared_d_hat: " << squared_d_hat << "\n";
        Float lam = -dBdD * GradD.head(3).norm();
        Vector3 n =  (prev_T0 - prev_T1).cross(prev_T0 - prev_T2);
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;
        Vector3 v1 = (P - prev_P) / dt;
        // cout << "v1.norm(): " << v1.norm() << "\n";

        // suppose P0 = t(0) * T0 + t(1) * T1 + t(2) * T2
        Eigen::Matrix<Float, 3, 2> base;
        base.block<3, 1>(0, 0) = prev_T1 - prev_T0;
        base.block<3, 1>(0, 1) = prev_T2 - prev_T0;
        Eigen::Matrix<Float, 2, 2> Lhs = base.transpose() * base;
        Vector2 rhs = base.transpose() * (prev_P - prev_T0);
        Eigen::Matrix<Float, 2, 2> Lhs_inv;
        Float det = Lhs(0, 0) * Lhs(1, 1) - Lhs(0, 1) * Lhs(1, 0);
        Lhs_inv(0, 0) = Lhs(1, 1) / det;
        Lhs_inv(0, 1) = -Lhs(0, 1) / det;
        Lhs_inv(1, 0) = -Lhs(1, 0) / det;
        Lhs_inv(1, 1) = Lhs(0, 0) / det;
        Vector2 t = Lhs_inv * rhs;
        Float t1 = t(0);
        Float t2 = t(1);
        Float t0 = 1 - t1 - t2;

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = v1;
        V.segment<3>(3) = t0 * (T0 - prev_T0) / dt + t1 * (T1 - prev_T1) / dt + t2 * (T2 - prev_T2) / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        cout << "PT_lam: " << lam << "\n";
        cout << "PT_y: " << y << "\n";
        Float F;
        FrictionEnergy(F, lam * mu, eps_v, dt, y);
        return F;
    }
 
    __device__ void PT_friction_gradient_hessian(Vector12&      G,
                                                 Matrix12x12&   H,
                                                 Float          kappa,
                                                 Float          squared_d_hat,
                                                 Float          mu,
                                                 Float          dt,
                                                 const Vector3& P,
                                                 const Vector3& T0,
                                                 const Vector3& T1,
                                                 const Vector3& T2,
                                                 const Vector3& prev_P,
                                                 const Vector3& prev_T0,
                                                 const Vector3& prev_T1,
                                                 const Vector3& prev_T2,
                                                 Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        point_triangle_distance(prev_P, prev_T0, prev_T1, prev_T2, D);
        if (D >= squared_d_hat) {
            G.setZero();
            H = Matrix12x12::Zero();
            return;
        }
        Vector12 GradD;
        point_triangle_distance_gradient(prev_P, prev_T0, prev_T1, prev_T2, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);

        Float lam = -dBdD * GradD.head(3).norm();
        Vector3 n =  (prev_T0 - prev_T1).cross(prev_T0 - prev_T2);
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;
        Vector3 v1 = (P - prev_P) / dt;

        // suppose prev_P0 = t(0) * prev_T0 + t(1) * prev_T1 + t(2) * prev_T2
        Eigen::Matrix<Float, 3, 2> base;
        base.block<3, 1>(0, 0) = prev_T1 - prev_T0;
        base.block<3, 1>(0, 1) = prev_T2 - prev_T0;
        Eigen::Matrix<Float, 2, 2> Lhs = base.transpose() * base;
        Vector2 rhs = base.transpose() * (prev_P - prev_T0);
        Eigen::Matrix<Float, 2, 2> Lhs_inv;
        Float det = Lhs(0, 0) * Lhs(1, 1) - Lhs(0, 1) * Lhs(1, 0);
        Lhs_inv(0, 0) = Lhs(1, 1) / det;
        Lhs_inv(0, 1) = -Lhs(0, 1) / det;
        Lhs_inv(1, 0) = -Lhs(1, 0) / det;
        Lhs_inv(1, 1) = Lhs(0, 0) / det;
        Vector2 t = Lhs_inv * rhs;
        Float t1 = t(0);
        Float t2 = t(1);
        Float t0 = 1 - t1 - t2;
        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = v1;
        V.segment<3>(3) = t0 * (T0 - prev_T0) / dt + t1 * (T1 - prev_T1) / dt + t2 * (T2 - prev_T2) / dt;
        for (int i = 0; i < 6; i++) {
            // cout << "V(" << i << "): " << V(i) << "\n";
        }
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        // cout << "y: " << y << "\n";
        Eigen::Vector<Float, 6> dFdV;
        dFrictionEnergydV(dFdV, lam * mu, Tk,  eps_v, dt, vk);
        Vector3 test;
        test(0) = 1e-8;
        test(1) = 1e-8;
        test(2) = 1e-8;
        Float E1 = PT_friction_energy(kappa, squared_d_hat, mu, dt, P + test, T0, T1, T2, prev_P, prev_T0, prev_T1, prev_T2, eps_v);
        Float E2 = PT_friction_energy(kappa, squared_d_hat, mu, dt, P - test, T0, T1, T2, prev_P, prev_T0, prev_T1, prev_T2, eps_v);
        Float num_diff = (E1 - E2) / 2;
        for (int i = 0; i < 6; i++) {
            // cout << "dFdV(" << i << "): " << dFdV(i) << "\n";
        }
        Vector6 test6 = Vector6::Zero();
        test6(0) = 1e-8;
        test6(1) = 1e-8;
        test6(2) = 1e-8;
        Float E3;
        Float y1 = (Tk.transpose() * (V + test6)).norm() * dt;
        FrictionEnergy(E3, lam * mu, eps_v, dt, y1);
        Float E4;
        Float y2 = (Tk.transpose() * (V - test6)).norm() * dt;
        FrictionEnergy(E4, lam * mu, eps_v, dt, y2);
        Eigen::Matrix<Float, 6, 12> GradV = Eigen::Matrix<Float, 6, 12>::Zero();
        GradV.block<3, 3>(0, 0) = I / dt;
        GradV.block<3, 3>(3, 3) = I * t0 / dt;
        GradV.block<3, 3>(3, 6) = I * t1 / dt;
        GradV.block<3, 3>(3, 9) = I * t2 / dt;
        Eigen::Matrix<Float, 12, 6> GradV_transpose = GradV.transpose();
        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 6; j++) {
            // cout << "GradV_transpose(" << i << ", " << j << "): " << GradV_transpose(i, j) << "\n";
            }
        }
        G = GradV_transpose * dFdV;
        /*
        for (int i = 0; i < 12; i++) {
            G(i) = 0;
            for (int j = 0; j < 6; j++) {
                G(i) += GradV_transpose(i, j) * dFdV(j);
                // cout << "G(" << i << "): " << G(i) << "\n";
            }
        }
        Gradient check
        Float ana_diff = G(0) * test(0) + G(1) * test(1) + G(2) * test(2);
        Float ana_diff1 = dFdV.dot(test6);
        Float num_diff1 = (E3 - E4) / 2;
        cout << "ana_diff1: " << ana_diff1 << "\n";
        cout << "num_diff1: " << num_diff1 << "\n";
        cout << "num_diff: " << num_diff << "\n";
        cout << "ana_diff: " << ana_diff << "\n";
        cout << "num_diff - ana_diff: " << num_diff - ana_diff << "\n";
        */
        // G = GradV_transpose * dFdV;
        for (int i = 0; i < 6; i++) {
            // cout << "G_(" << i << "): " << G(i) << "\n";
        }
        for (int i = 0; i < 6; i++) {
            // cout << "dFdV(i): " << dFdV(i) << "\n";
        }

        Eigen::Matrix<Float, 6, 6> ddFddV;
        ddFrictionEnergyddV(ddFddV, lam * mu, Tk, eps_v, dt, vk);
        /*
        Vector6 dFdV1; 
        Vector3 vk_1 = Tk.transpose() * (V + test6);
        dFrictionEnergydV(dFdV1, lam * mu, Tk, eps_v, dt, vk_1);
        Vector6 dFdV2;
        Vector3 vk_2 = Tk.transpose() * (V - test6); 
        dFrictionEnergydV(dFdV2, lam * mu, Tk, eps_v, dt, vk_2);
        Vector6 num_diff6 = (dFdV1 - dFdV2) / 2;
        Vector6 ana_diff6 = ddFddV * test6;
        cout << "num_diff6: " << num_diff6.norm() << "\n";
        cout << "ana_diff6: " << ana_diff6.norm() << "\n";
        cout << "num_diff6 - ana_diff6: " << (num_diff6 - ana_diff6).norm() << "\n";

        The method is semi-implicit, where the t is explicit t_n compute with previous points, so the gradient dt is all 0.
        Vector3 offset = T0 - P + base * t;
        Eigen::Matrix<Float, 2, 3> dtdP = Lhs_inv * base.transpose();
        Eigen::Matrix<Float, 2, 3> dtdT1 = -Lhs_inv.col(0) * offset.transpose() - t1 * dtdP;
        Eigen::Matrix<Float, 2, 3> dtdT2 = -Lhs_inv.col(1) * offset.transpose() - t2 * dtdP;
        Eigen::Matrix<Float, 2, 3> dtdT0 = -dtdP - dtdT1 - dtdT2;
        Vector3 dt0dP = (-dtdP.row(0) - dtdP.row(1)).transpose();
        Vector3 dt0dT1 = (-dtdT1.row(0) - dtdT1.row(1)).transpose();
        Vector3 dt0dT2 = (-dtdT2.row(0) - dtdT2.row(1)).transpose();
        Vector3 dt0dT0 = (-dtdT0.row(0) - dtdT0.row(1)).transpose();
        // Grad(GradV.transpose()) * dFdV
        // a = (a1, a2), b = (b1, b2), c = (c1, c2) 
        // b0 = c0, b1 = b0 * c1
        // da1dc1 = da1
        H.block<3, 12>(0, 0) = Eigen::Matrix<Float, 3, 12>::Zero();
        H.block<3, 3>(3, 0) = dFdV.tail(3) * dt0dP.transpose() / dt;
        H.block<3, 3>(3, 3) = dFdV.tail(3) * dt0dT0.transpose() / dt;
        H.block<3, 3>(3, 6) = dFdV.tail(3) * dt0dT1.transpose() / dt;
        H.block<3, 3>(3, 9) = dFdV.tail(3) * dt0dT2.transpose() / dt;
        H.block<3, 3>(6, 0) = dFdV.tail(3) * dtdP.row(0) / dt;
        H.block<3, 3>(6, 3) = dFdV.tail(3) * dtdT0.row(0) / dt;
        H.block<3, 3>(6, 6) = dFdV.tail(3) * dtdT1.row(0) / dt;
        H.block<3, 3>(6, 9) = dFdV.tail(3) * dtdT2.row(0) / dt;
        H.block<3, 3>(9, 0) = dFdV.tail(3) * dtdP.row(1) / dt;
        H.block<3, 3>(9, 3) = dFdV.tail(3) * dtdT0.row(1) / dt;
        H.block<3, 3>(9, 6) = dFdV.tail(3) * dtdT1.row(1) / dt;
        H.block<3, 3>(9, 9) = dFdV.tail(3) * dtdT2.row(1) / dt;
        cout << "H(0,0): " << H(0, 0) << "\n";
        cout << "H - H.transpose(): " << (H - H.transpose()).norm() << "\n";
        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 12; j++) {
                cout << "H(" << i << ", " << j << "): " << H(i, j) << "\n";
            }
        }
        for (int i=0; i < 6; i++) {
            for (int j=0; j < 6; j++) {
                //cout << "ddFddV(" << i << ", " << j << "): " << ddFddV(i, j) << "\n";
            }
        }
        for (int i = 0; i < 6; i++) {
            for (int j = 0; j < 12; j++) {
                // cout << "GradV(" << i << ", " << j << "): " << GradV(i, j) << "\n";
            }
        }
        */
        H.block<3, 3>(0, 0) = ddFddV.block<3, 3>(0, 0) / dt / dt;
        H.block<3, 3>(0, 3) = ddFddV.block<3, 3>(0, 3) * t0 / dt / dt;
        H.block<3, 3>(0, 6) = ddFddV.block<3, 3>(0, 3) * t1 / dt / dt;
        H.block<3, 3>(0, 9) = ddFddV.block<3, 3>(0, 3) * t2 / dt / dt;
        H.block<3, 3>(3, 0) = ddFddV.block<3, 3>(3, 0) * t0 / dt / dt;
        H.block<3, 3>(3, 3) = ddFddV.block<3, 3>(3, 3) * t0 * t0 / dt / dt;
        H.block<3, 3>(3, 6) = ddFddV.block<3, 3>(3, 3) * t0 * t1 / dt / dt;
        H.block<3, 3>(3, 9) = ddFddV.block<3, 3>(3, 3) * t0 * t2 / dt / dt;
        H.block<3, 3>(6, 0) = ddFddV.block<3, 3>(3, 0) * t1 / dt / dt;
        H.block<3, 3>(6, 3) = ddFddV.block<3, 3>(3, 3) * t1 * t0 / dt / dt;
        H.block<3, 3>(6, 6) = ddFddV.block<3, 3>(3, 3) * t1 * t1 / dt / dt;
        H.block<3, 3>(6, 9) = ddFddV.block<3, 3>(3, 3) * t1 * t2 / dt / dt;
        H.block<3, 3>(9, 0) = ddFddV.block<3, 3>(3, 0) * t2 / dt / dt;
        H.block<3, 3>(9, 3) = ddFddV.block<3, 3>(3, 3) * t2 * t0 / dt / dt;
        H.block<3, 3>(9, 6) = ddFddV.block<3, 3>(3, 3) * t2 * t1 / dt / dt;
        H.block<3, 3>(9, 9) = ddFddV.block<3, 3>(3, 3) * t2 * t2 / dt / dt;
        /*
        Eigen::Matrix<Float, 6, 12> mat = Eigen::Matrix<Float, 6, 12>::Zero();
        for (int i = 0; i < 6; i++) {
            for (int j = 0; j < 12; j++) {
                for (int k = 0; k < 6; k++) {
                    mat(i, j) += ddFddV(i, k) * GradV(k, j);
                    cout << "mat(" << i << ", " << j << "): " << mat(i, j) << "\n";
                }
            }
        }
        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 12; j++) {
                H(i, j) = 0;
                for (int k = 0; k < 6; k++) {
                    H(i, j) += GradV.transpose()(i, k) * mat(k, j);
                    // cout << "GradV.transpose()(" << i << ", " << k << "): " << GradV.transpose()(i, k) << "\n";
                    // cout << "mat(" << k << ", " << j << "): " << mat(k, j) << "\n";
                    cout << "H(" << i << ", " << j << "): " << H(i, j) << "\n";
               }
            }
        }
        */
        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 12; j++) {
                // cout << "H(" << i << ", " << j << "): " << H(i, j) << "\n";
            }
        }
    }

    __device__ Float EE_friction_energy(Float          kappa,
                                        Float          squared_d_hat,
                                        Float          mu,
                                        Float          dt,
                                        const Vector3& P0,
                                        const Vector3& P1,
                                        const Vector3& Q0,
                                        const Vector3& Q1,
                                        const Vector3& prev_P0,
                                        const Vector3& prev_P1,
                                        const Vector3& prev_Q0,
                                        const Vector3& prev_Q1,
                                        Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        edge_edge_distance(prev_P0, prev_P1, prev_Q0, prev_Q1, D);
        Float D_now;
        edge_edge_distance(P0, P1, Q0, Q1, D_now);
        cout << "D - squared_d_hat: " << D - squared_d_hat << "\n";
        // cout << "D_now - squared_d_hat: " << D_now - squared_d_hat << "\n";
        if (D >= squared_d_hat) {
            return 0;
        }
        Vector12 GradD;
        edge_edge_distance_gradient(prev_P0, prev_P1, prev_Q0, prev_Q1, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);
        // cout << "dBdD: " << dBdD << "\n";
        // cout << "GradD.head(3).norm(): " << GradD.head(3).norm() << "\n";
        // cout << "D: " << D << "\n";
        // cout << "D_now: " << D_now << "\n";
        // cout << "squared_d_hat: " << squared_d_hat << "\n";
        Vector3 n =  (prev_P0 - prev_P1).cross(prev_Q0 - prev_Q1);
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;

        // suppose prev_P_project = prev_P0 + t0 * (prev_P1 - prev_P0)
        //         prev_Q_project = prev_Q0 + t1 * (prev_Q1 - prev_Q0)
        Eigen::Matrix<Float, 3, 2> base;
        base.block<3, 1>(0, 0) = prev_P1 - prev_P0;
        base.block<3, 1>(0, 1) = prev_Q1 - prev_Q0;
        Eigen::Matrix<Float, 2, 2> X = Eigen::Matrix<Float, 2, 2>::Zero();
        X(0, 0) = -1;
        X(1, 1) = 1;
        Eigen::Matrix<Float, 2, 2> Lhs = base.transpose() * base * X;
        Vector2 rhs = base.transpose() * (prev_P0 - prev_Q0);
        Eigen::Matrix<Float, 2, 2> Lhs_inv;
        Float det = Lhs(0, 0) * Lhs(1, 1) - Lhs(0, 1) * Lhs(1, 0);
        Lhs_inv(0, 0) = Lhs(1, 1) / det;
        Lhs_inv(0, 1) = -Lhs(0, 1) / det;
        Lhs_inv(1, 0) = -Lhs(1, 0) / det;
        Lhs_inv(1, 1) = Lhs(0, 0) / det;
        Vector2 t = Lhs_inv * rhs;
        Float t0 = t(0);
        Float t1 = t(1);

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = (P0 - prev_P0) * (1 - t0) / dt + (P1 - prev_P1) * t0 / dt;
        V.segment<3>(3) = (Q0 - prev_Q0) * (1 - t1) / dt + (Q1 - prev_Q1) * t1 / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        Float lam = -dBdD * (GradD.segment<3>(0) * (1 - t0) + GradD.segment<3>(3) * t0).norm();
        cout << "EE_lam: " << lam << "\n";
        cout << "EE_y: " << y << "\n";
        Float F;
        FrictionEnergy(F, lam * mu, eps_v, dt, y);
        return F;
    }

    __device__ void EE_friction_gradient_hessian(Vector12&      G,
                                        Matrix12x12&   H,
                                        Float          kappa,
                                        Float          squared_d_hat,
                                        Float          mu,
                                        Float          dt,
                                        const Vector3& P0,
                                        const Vector3& P1,
                                        const Vector3& Q0,
                                        const Vector3& Q1,
                                        const Vector3& prev_P0,
                                        const Vector3& prev_P1,
                                        const Vector3& prev_Q0,
                                        const Vector3& prev_Q1,
                                        Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        edge_edge_distance(prev_P0, prev_P1, prev_Q0, prev_Q1, D);
        Float D_now;
        edge_edge_distance(P0, P1, Q0, Q1, D_now);
        // cout << "D - squared_d_hat: " << D - squared_d_hat << "\n";
        // cout << "D_now - squared_d_hat: " << D_now - squared_d_hat << "\n";
        if (D >= squared_d_hat) {
            G.setZero();
            H = Matrix12x12::Zero();
            return;
        }
        Vector12 GradD;
        edge_edge_distance_gradient(prev_P0, prev_P1, prev_Q0, prev_Q1, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);
        // cout << "dBdD: " << dBdD << "\n";
        // cout << "GradD.head(3).norm(): " << GradD.head(3).norm() << "\n";
        // cout << "D: " << D << "\n";
        // cout << "D_now: " << D_now << "\n";
        // cout << "squared_d_hat: " << squared_d_hat << "\n";
        Vector3 n =  (prev_P0 - prev_P1).cross(prev_Q0 - prev_Q1);
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;

        // suppose prev_P_project = prev_P0 + t0 * (prev_P1 - prev_P0)
        //         prev_Q_project = prev_Q0 + t1 * (prev_Q1 - prev_Q0)
        Eigen::Matrix<Float, 3, 2> base;
        base.block<3, 1>(0, 0) = prev_P1 - prev_P0;
        base.block<3, 1>(0, 1) = prev_Q1 - prev_Q0;
        Eigen::Matrix<Float, 2, 2> X = Eigen::Matrix<Float, 2, 2>::Zero();
        X(0, 0) = -1;
        X(1, 1) = 1;
        Eigen::Matrix<Float, 2, 2> Lhs = base.transpose() * base * X;
        Vector2 rhs = base.transpose() * (prev_P0 - prev_Q0);
        Eigen::Matrix<Float, 2, 2> Lhs_inv;
        Float det = Lhs(0, 0) * Lhs(1, 1) - Lhs(0, 1) * Lhs(1, 0);
        Lhs_inv(0, 0) = Lhs(1, 1) / det;
        Lhs_inv(0, 1) = -Lhs(0, 1) / det;
        Lhs_inv(1, 0) = -Lhs(1, 0) / det;
        Lhs_inv(1, 1) = Lhs(0, 0) / det;
        Vector2 t = Lhs_inv * rhs;
        Float t0 = t(0);
        Float t1 = t(1);

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = (P0 - prev_P0) * (1 - t0) / dt + (P1 - prev_P1) * t0 / dt;
        V.segment<3>(3) = (Q0 - prev_Q0) * (1 - t1) / dt + (Q1 - prev_Q1) * t1 / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        Float lam = -dBdD * (GradD.segment<3>(0) * (1 - t0) + GradD.segment<3>(3) * t0).norm();
        // cout << "lam: " << lam << "\n";
        // cout << "y: " << y << "\n";
        Eigen::Vector<Float, 6> dFdV;
        dFrictionEnergydV(dFdV, lam * mu, Tk,  eps_v, dt, vk);
        Eigen::Matrix<Float, 6, 12> GradV = Eigen::Matrix<Float, 6, 12>::Zero();
        GradV.block<3, 3>(0, 0) = I * (1 - t0)/ dt;
        GradV.block<3, 3>(0, 3) = I * t0 / dt;
        GradV.block<3, 3>(3, 6) = I * (1 - t1) / dt;
        GradV.block<3, 3>(3, 9) = I * t1 / dt;
        Eigen::Matrix<Float, 12, 6> GradV_transpose = GradV.transpose();
        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 6; j++) {
            // cout << "GradV_transpose(" << i << ", " << j << "): " << GradV_transpose(i, j) << "\n";
            }
        }
        
        for (int i = 0; i < 12; i++) {
            G(i) = 0;
            for (int j = 0; j < 6; j++) {
                G(i) += GradV_transpose(i, j) * dFdV(j);
                // cout << "G(" << i << "): " << G(i) << "\n";
            }
        }
        Eigen::Matrix<Float, 6, 6> ddFddV;
        ddFrictionEnergyddV(ddFddV, lam * mu, Tk, eps_v, dt, vk);
        H.block<3, 3>(0, 0) = ddFddV.block<3, 3>(0, 0) * (1 - t0) * (1 - t0) / dt / dt;
        H.block<3, 3>(0, 3) = ddFddV.block<3, 3>(0, 3) * (1 - t0) * t0 / dt / dt;
        H.block<3, 3>(0, 6) = ddFddV.block<3, 3>(0, 3) * (1 - t0) * (1 - t1) / dt / dt;
        H.block<3, 3>(0, 9) = ddFddV.block<3, 3>(0, 3) * (1 - t0) * t1 / dt / dt;
        H.block<3, 3>(3, 0) = ddFddV.block<3, 3>(3, 0) * t0 * (1 - t0) / dt / dt;
        H.block<3, 3>(3, 3) = ddFddV.block<3, 3>(3, 3) * t0 * t0 / dt / dt;
        H.block<3, 3>(3, 6) = ddFddV.block<3, 3>(3, 3) * t0 * (1 - t1) / dt / dt;
        H.block<3, 3>(3, 9) = ddFddV.block<3, 3>(3, 3) * t0 * t1 / dt / dt;
        H.block<3, 3>(6, 0) = ddFddV.block<3, 3>(3, 0) * (1 - t1) * (1 - t0) / dt / dt;
        H.block<3, 3>(6, 3) = ddFddV.block<3, 3>(3, 3) * (1 - t1) * t0 / dt / dt;
        H.block<3, 3>(6, 6) = ddFddV.block<3, 3>(3, 3) * (1 - t1) * (1 - t1) / dt / dt;
        H.block<3, 3>(6, 9) = ddFddV.block<3, 3>(3, 3) * (1 - t1) * t1 / dt / dt;
        H.block<3, 3>(9, 0) = ddFddV.block<3, 3>(3, 0) * t1 * (1 - t0) / dt / dt;
        H.block<3, 3>(9, 3) = ddFddV.block<3, 3>(3, 3) * t1 * t0 / dt / dt;
        H.block<3, 3>(9, 6) = ddFddV.block<3, 3>(3, 3) * t1 * (1 - t1) / dt / dt;
        H.block<3, 3>(9, 9) = ddFddV.block<3, 3>(3, 3) * t1 * t1 / dt / dt;
        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 12; j++) {
                // cout << "H(" << i << ", " << j << "): " << H(i, j) << "\n";
            }
        }
    }

    __device__ Float PE_friction_energy(Float          kappa,
                                        Float          squared_d_hat,
                                        Float          mu,
                                        Float          dt,
                                        const Vector3& P,
                                        const Vector3& T0,
                                        const Vector3& T1,
                                        const Vector3& prev_P,
                                        const Vector3& prev_T0,
                                        const Vector3& prev_T1,
                                        Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        point_edge_distance(prev_P, prev_T0, prev_T1, D);
        Float D_now;
        point_edge_distance(P, T0, T1, D_now);
        // cout << "D - squared_d_hat: " << D - squared_d_hat << "\n";
        // cout << "D_now - squared_d_hat: " << D_now - squared_d_hat << "\n";
        if (D >= squared_d_hat) {
            return 0;
        }
        Vector9 GradD;
        point_edge_distance_gradient(prev_P, prev_T0, prev_T1, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);
        // cout << "dBdD: " << dBdD << "\n";
        // cout << "GradD.head(3).norm(): " << GradD.head(3).norm() << "\n";
        // cout << "D: " << D << "\n";
        // cout << "D_now: " << D_now << "\n";
        // cout << "squared_d_hat: " << squared_d_hat << "\n";
        Float lam = -dBdD * GradD.head(3).norm();
        cout << "PE_lam: " << lam << "\n";
        // suppose P0 = t(0) * T0 + t(1) * T1
        Float t0 = (prev_P - prev_T1).dot(prev_T0 - prev_T1) / (prev_T0 - prev_T1).squaredNorm();
        Float t1 = 1 - t0;

        Vector3 prev_P0 = t0 * prev_T0 + t1 * prev_T1;
        Vector3 n = prev_P0 - prev_P;
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;
        Vector3 v1 = (P - prev_P) / dt;

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = v1;
        V.segment<3>(3) = t0 * (T0 - prev_T0) / dt + t1 * (T1 - prev_T1) / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        // cout << "lam: " << lam << "\n";
        cout << "PE_y: " << y << "\n";
        Float F;
        FrictionEnergy(F, lam * mu, eps_v, dt, y);
        return F;
    }

    __device__ void PE_friction_gradient_hessian(Vector9&       G,
                                                 Matrix9x9&     H,
                                                 Float          kappa,
                                                 Float          squared_d_hat,
                                                 Float          mu,
                                                 Float          dt,
                                                 const Vector3& P,
                                                 const Vector3& T0,
                                                 const Vector3& T1,
                                                 const Vector3& prev_P,
                                                 const Vector3& prev_T0,
                                                 const Vector3& prev_T1,
                                                 Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        point_edge_distance(prev_P, prev_T0, prev_T1, D);
        if (D >= squared_d_hat) {
            MUDA_ASSERT(D >= squared_d_hat);
            G.setZero();
            H = Matrix9x9::Zero();
            return;
        }
        Vector9 GradD = Vector9::Zero();
        point_edge_distance_gradient(prev_P, prev_T0, prev_T1, GradD);

        Float dBdD = 0;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);

        Float lam = -dBdD * GradD.head(3).norm();

        // suppose P0 = t0 * T0 + t1 * T1
        Float t0 = (prev_P - prev_T1).dot(prev_T0 - prev_T1) / (prev_T0 - prev_T1).squaredNorm();
        Float t1 = 1 - t0;

        Vector3 prev_P0 = t0 * prev_T0 + t1 * prev_T1;
        Vector3 n = prev_P0 - prev_P;
        cout << "t0: " << t0 << "\n";
        cout << "n.dot(prev_T0 - prev_T1): " << n.dot(prev_T0 - prev_T1) << "\n";
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;
        Vector3 v1 = (P - prev_P) / dt;

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = v1;
        V.segment<3>(3) = t0 * (T0 - prev_T0) / dt + t1 * (T1 - prev_T1) / dt;
        for (int i = 0; i < 6; i++) {
            // cout << "V(" << i << "): " << V(i) << "\n";
        }
        cout << "V.norm(): " << V.norm() << "\n";
        Vector3 vk = Tk.transpose() * V;
        cout << "vk: " << vk.norm() << "\n";
        Float y = vk.norm() * dt;
        // cout << "y: " << y << "\n";
        Eigen::Vector<Float, 6> dFdV;
        dFrictionEnergydV(dFdV, lam * mu, Tk,  eps_v, dt, vk);
        cout << "dFdV: " << dFdV.norm() << "\n";
        Vector3 test;
        test(0) = 1e-8;
        test(1) = 1e-8;
        test(2) = 1e-8;
        Float E1 = PE_friction_energy(kappa, squared_d_hat, mu, dt, P + test, T0, T1, prev_P, prev_T0, prev_T1, eps_v);
        Float E2 = PE_friction_energy(kappa, squared_d_hat, mu, dt, P - test, T0, T1, prev_P, prev_T0, prev_T1, eps_v);
        Float num_diff = (E1 - E2) / 2;
        for (int i = 0; i < 6; i++) {
            // cout << "dFdV(" << i << "): " << dFdV(i) << "\n";
        }
        Vector6 test6 = Vector6::Zero();
        test6(0) = 1e-8;
        test6(1) = 1e-8;
        test6(2) = 1e-8;
        Float E3;
        Float y1 = (Tk.transpose() * (V + test6)).norm() * dt;
        FrictionEnergy(E3, lam * mu, eps_v, dt, y1);
        Float E4;
        Float y2 = (Tk.transpose() * (V - test6)).norm() * dt;
        FrictionEnergy(E4, lam * mu, eps_v, dt, y2);
        Eigen::Matrix<Float, 6, 9> GradV = Eigen::Matrix<Float, 6, 9>::Zero();
        GradV.block<3, 3>(0, 0) = I / dt;
        GradV.block<3, 3>(3, 3) = I * t0 / dt;
        GradV.block<3, 3>(3, 6) = I * t1 / dt;
        Eigen::Matrix<Float, 9, 6> GradV_transpose = GradV.transpose();
        for (int i = 0; i < 9; i++) {
            for (int j = 0; j < 6; j++) {
            // cout << "GradV_transpose(" << i << ", " << j << "): " << GradV_transpose(i, j) << "\n";
            }
        }
        G = GradV_transpose * dFdV;
        cout << "prev_P - P: " << (prev_P - P).norm() << "\n";
        cout << "prev_T0 - T0: " << (prev_T0 - T0).norm() << "\n";
        cout << "prev_T1 - T1: " << (prev_T1 - T1).norm() << "\n";
        cout << "G.norm(): " << G.norm() << "\n";
        /*
        for (int i = 0; i < 12; i++) {
            G(i) = 0;
            for (int j = 0; j < 6; j++) {
                G(i) += GradV_transpose(i, j) * dFdV(j);
                // cout << "G(" << i << "): " << G(i) << "\n";
            }
        }
        Gradient check
        Float ana_diff = G(0) * test(0) + G(1) * test(1) + G(2) * test(2);
        Float ana_diff1 = dFdV.dot(test6);
        Float num_diff1 = (E3 - E4) / 2;
        cout << "ana_diff1: " << ana_diff1 << "\n";
        cout << "num_diff1: " << num_diff1 << "\n";
        cout << "num_diff: " << num_diff << "\n";
        cout << "ana_diff: " << ana_diff << "\n";
        cout << "num_diff - ana_diff: " << num_diff - ana_diff << "\n";
        */
        // G = GradV_transpose * dFdV;
        for (int i = 0; i < 6; i++) {
            // cout << "G_(" << i << "): " << G(i) << "\n";
        }
        for (int i = 0; i < 6; i++) {
            // cout << "dFdV(i): " << dFdV(i) << "\n";
        }

        Eigen::Matrix<Float, 6, 6> ddFddV;
        ddFrictionEnergyddV(ddFddV, lam * mu, Tk, eps_v, dt, vk);
        /*
        Vector6 dFdV1; 
        Vector3 vk_1 = Tk.transpose() * (V + test6);
        dFrictionEnergydV(dFdV1, lam * mu, Tk, eps_v, dt, vk_1);
        Vector6 dFdV2;
        Vector3 vk_2 = Tk.transpose() * (V - test6); 
        dFrictionEnergydV(dFdV2, lam * mu, Tk, eps_v, dt, vk_2);
        Vector6 num_diff6 = (dFdV1 - dFdV2) / 2;
        Vector6 ana_diff6 = ddFddV * test6;
        cout << "num_diff6: " << num_diff6.norm() << "\n";
        cout << "ana_diff6: " << ana_diff6.norm() << "\n";
        cout << "num_diff6 - ana_diff6: " << (num_diff6 - ana_diff6).norm() << "\n";

        The method is semi-implicit, where the t is explicit t_n compute with previous points, so the gradient dt is all 0.
        Vector3 offset = T0 - P + base * t;
        Eigen::Matrix<Float, 2, 3> dtdP = Lhs_inv * base.transpose();
        Eigen::Matrix<Float, 2, 3> dtdT1 = -Lhs_inv.col(0) * offset.transpose() - t1 * dtdP;
        Eigen::Matrix<Float, 2, 3> dtdT2 = -Lhs_inv.col(1) * offset.transpose() - t2 * dtdP;
        Eigen::Matrix<Float, 2, 3> dtdT0 = -dtdP - dtdT1 - dtdT2;
        Vector3 dt0dP = (-dtdP.row(0) - dtdP.row(1)).transpose();
        Vector3 dt0dT1 = (-dtdT1.row(0) - dtdT1.row(1)).transpose();
        Vector3 dt0dT2 = (-dtdT2.row(0) - dtdT2.row(1)).transpose();
        Vector3 dt0dT0 = (-dtdT0.row(0) - dtdT0.row(1)).transpose();
        // Grad(GradV.transpose()) * dFdV
        // a = (a1, a2), b = (b1, b2), c = (c1, c2) 
        // b0 = c0, b1 = b0 * c1
        // da1dc1 = da1
        H.block<3, 12>(0, 0) = Eigen::Matrix<Float, 3, 12>::Zero();
        H.block<3, 3>(3, 0) = dFdV.tail(3) * dt0dP.transpose() / dt;
        H.block<3, 3>(3, 3) = dFdV.tail(3) * dt0dT0.transpose() / dt;
        H.block<3, 3>(3, 6) = dFdV.tail(3) * dt0dT1.transpose() / dt;
        H.block<3, 3>(3, 9) = dFdV.tail(3) * dt0dT2.transpose() / dt;
        H.block<3, 3>(6, 0) = dFdV.tail(3) * dtdP.row(0) / dt;
        H.block<3, 3>(6, 3) = dFdV.tail(3) * dtdT0.row(0) / dt;
        H.block<3, 3>(6, 6) = dFdV.tail(3) * dtdT1.row(0) / dt;
        H.block<3, 3>(6, 9) = dFdV.tail(3) * dtdT2.row(0) / dt;
        H.block<3, 3>(9, 0) = dFdV.tail(3) * dtdP.row(1) / dt;
        H.block<3, 3>(9, 3) = dFdV.tail(3) * dtdT0.row(1) / dt;
        H.block<3, 3>(9, 6) = dFdV.tail(3) * dtdT1.row(1) / dt;
        H.block<3, 3>(9, 9) = dFdV.tail(3) * dtdT2.row(1) / dt;
        cout << "H(0,0): " << H(0, 0) << "\n";
        cout << "H - H.transpose(): " << (H - H.transpose()).norm() << "\n";
        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 12; j++) {
                cout << "H(" << i << ", " << j << "): " << H(i, j) << "\n";
            }
        }
        for (int i=0; i < 6; i++) {
            for (int j=0; j < 6; j++) {
                //cout << "ddFddV(" << i << ", " << j << "): " << ddFddV(i, j) << "\n";
            }
        }
        for (int i = 0; i < 6; i++) {
            for (int j = 0; j < 12; j++) {
                // cout << "GradV(" << i << ", " << j << "): " << GradV(i, j) << "\n";
            }
        }
        */
        H.block<3, 3>(0, 0) = ddFddV.block<3, 3>(0, 0) / dt / dt;
        H.block<3, 3>(0, 3) = ddFddV.block<3, 3>(0, 3) * t0 / dt / dt;
        H.block<3, 3>(0, 6) = ddFddV.block<3, 3>(0, 3) * t1 / dt / dt;
        H.block<3, 3>(3, 0) = ddFddV.block<3, 3>(3, 0) * t0 / dt / dt;
        H.block<3, 3>(3, 3) = ddFddV.block<3, 3>(3, 3) * t0 * t0 / dt / dt;
        H.block<3, 3>(3, 6) = ddFddV.block<3, 3>(3, 3) * t0 * t1 / dt / dt;
        H.block<3, 3>(6, 0) = ddFddV.block<3, 3>(3, 0) * t1 / dt / dt;
        H.block<3, 3>(6, 3) = ddFddV.block<3, 3>(3, 3) * t1 * t0 / dt / dt;
        H.block<3, 3>(6, 6) = ddFddV.block<3, 3>(3, 3) * t1 * t1 / dt / dt;
        /*
        Eigen::Matrix<Float, 6, 12> mat = Eigen::Matrix<Float, 6, 12>::Zero();
        for (int i = 0; i < 6; i++) {
            for (int j = 0; j < 12; j++) {
                for (int k = 0; k < 6; k++) {
                    mat(i, j) += ddFddV(i, k) * GradV(k, j);
                    cout << "mat(" << i << ", " << j << "): " << mat(i, j) << "\n";
                }
            }
        }
        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 12; j++) {
                H(i, j) = 0;
                for (int k = 0; k < 6; k++) {
                    H(i, j) += GradV.transpose()(i, k) * mat(k, j);
                    // cout << "GradV.transpose()(" << i << ", " << k << "): " << GradV.transpose()(i, k) << "\n";
                    // cout << "mat(" << k << ", " << j << "): " << mat(k, j) << "\n";
                    cout << "H(" << i << ", " << j << "): " << H(i, j) << "\n";
               }
            }
        }
        */
        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 12; j++) {
                // cout << "H(" << i << ", " << j << "): " << H(i, j) << "\n";
            }
        }
    }

    __device__ Float PP_friction_energy(Float          kappa,
                                        Float          squared_d_hat,
                                        Float          mu,
                                        Float          dt,
                                        const Vector3& P,
                                        const Vector3& Q,
                                        const Vector3& prev_P,
                                        const Vector3& prev_Q,
                                        Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        point_point_distance(prev_P, prev_Q, D);
        Float D_now;
        point_point_distance(P, Q, D_now);
        // cout << "D - squared_d_hat: " << D - squared_d_hat << "\n";
        // cout << "D_now - squared_d_hat: " << D_now - squared_d_hat << "\n";
        if (D >= squared_d_hat) {
            return 0;
        }
        Vector6 GradD;
        point_point_distance_gradient(prev_P, prev_Q, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);
        // cout << "dBdD: " << dBdD << "\n";
        // cout << "GradD.head(3).norm(): " << GradD.head(3).norm() << "\n";
        // cout << "D: " << D << "\n";
        // cout << "D_now: " << D_now << "\n";
        // cout << "squared_d_hat: " << squared_d_hat << "\n";
        Float lam = -dBdD * GradD.head(3).norm();

        Vector3 n = prev_Q - prev_P;
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = (P - prev_P) / dt;
        V.segment<3>(3) = (Q - prev_Q) / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        // cout << "lam: " << lam << "\n";
        // cout << "y: " << y << "\n";
        Float F;
        FrictionEnergy(F, lam * mu, eps_v, dt, y);
        return F;
    }

    __device__ void PP_friction_gradient_hessian(Vector6&      G,
                                                 Matrix6x6&   H,
                                                 Float          kappa,
                                                 Float          squared_d_hat,
                                                 Float          mu,
                                                 Float          dt,
                                                 const Vector3& P,
                                                 const Vector3& Q,
                                                 const Vector3& prev_P,
                                                 const Vector3& prev_Q,
                                                 Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        point_point_distance(prev_P, prev_Q, D);
        Float D_now;
        point_point_distance(P, Q, D_now);
        if (D >= squared_d_hat) {
            return;
        }
        Vector6 GradD;
        point_point_distance_gradient(prev_P, prev_Q, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);
        // cout << "dBdD: " << dBdD << "\n";
        // cout << "GradD.head(3).norm(): " << GradD.head(3).norm() << "\n";
        // cout << "D: " << D << "\n";
        // cout << "D_now: " << D_now << "\n";
        // cout << "squared_d_hat: " << squared_d_hat << "\n";
        Float lam = -dBdD * GradD.head(3).norm();

        Vector3 n = prev_Q - prev_P;
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block<3, 3>(0, 0) = I - normal * normal.transpose();
        Tk.block<3, 3>(3, 0) = normal * normal.transpose() - I;

        Eigen::Vector<Float, 6> V;
        V.segment<3>(0) = (P - prev_P) / dt;
        V.segment<3>(3) = (Q - prev_Q) / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        // cout << "y: " << y << "\n";
        Eigen::Vector<Float, 6> dFdV;
        dFrictionEnergydV(dFdV, lam * mu, Tk,  eps_v, dt, vk);
        Vector3 test;
        test(0) = 1e-8;
        test(1) = 1e-8;
        test(2) = 1e-8;
        Float E1 = PP_friction_energy(kappa, squared_d_hat, mu, dt, P + test, Q, prev_P, prev_Q, eps_v);
        Float E2 = PP_friction_energy(kappa, squared_d_hat, mu, dt, P - test, Q, prev_P, prev_Q, eps_v);
        Float num_diff = (E1 - E2) / 2;
        for (int i = 0; i < 6; i++) {
            // cout << "dFdV(" << i << "): " << dFdV(i) << "\n";
        }
        Vector6 test6 = Vector6::Zero();
        test6(0) = 1e-8;
        test6(1) = 1e-8;
        test6(2) = 1e-8;
        Float E3;
        Float y1 = (Tk.transpose() * (V + test6)).norm() * dt;
        FrictionEnergy(E3, lam * mu, eps_v, dt, y1);
        Float E4;
        Float y2 = (Tk.transpose() * (V - test6)).norm() * dt;
        FrictionEnergy(E4, lam * mu, eps_v, dt, y2);
        // GradV = Eigen::Matrix<Float, 6, 6>::Identity();
        G = dFdV;
        /*
        Gradient check
        Float ana_diff = G(0) * test(0) + G(1) * test(1) + G(2) * test(2);
        Float ana_diff1 = dFdV.dot(test6);
        Float num_diff1 = (E3 - E4) / 2;
        cout << "ana_diff1: " << ana_diff1 << "\n";
        cout << "num_diff1: " << num_diff1 << "\n";
        cout << "num_diff: " << num_diff << "\n";
        cout << "ana_diff: " << ana_diff << "\n";
        cout << "num_diff - ana_diff: " << num_diff - ana_diff << "\n";
        */
        // G = GradV_transpose * dFdV;
        for (int i = 0; i < 6; i++) {
            // cout << "G_(" << i << "): " << G(i) << "\n";
        }
        for (int i = 0; i < 6; i++) {
            // cout << "dFdV(i): " << dFdV(i) << "\n";
        }

        Eigen::Matrix<Float, 6, 6> ddFddV;
        ddFrictionEnergyddV(ddFddV, lam * mu, Tk, eps_v, dt, vk);

        H = ddFddV;

        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 12; j++) {
                // cout << "H(" << i << ", " << j << "): " << H(i, j) << "\n";
            }
        }
    }

}



void IPCSimplexFrictionalContact::do_compute_energy(EnergyInfo& info)
{
    using namespace muda;

    // Compute Point-Triangle energy
    auto PT_count = info.friction_PTs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(PT_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PTs = info.friction_PTs().viewer().name("PTs"),
                Es  = info.friction_PT_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PT = PTs(i);

                   auto cid_L = contact_ids(PT[0]);
                   auto cid_R = contact_ids(PT[1]);

                   const auto& P  = Ps(PT[0]);
                   const auto& T0 = Ps(PT[1]);
                   const auto& T1 = Ps(PT[2]);
                   const auto& T2 = Ps(PT[3]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;

                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;


                   Float D_hat = d_hat * d_hat;

                   Float          prev_D;
                   const Vector3& prev_P  = prev_Ps(PT[0]);
                   const Vector3& prev_T0 = prev_Ps(PT[1]);
                   const Vector3& prev_T1 = prev_Ps(PT[2]);
                   const Vector3& prev_T2 = prev_Ps(PT[3]);
                   distance::point_triangle_distance(prev_P, prev_T0, prev_T1, prev_T2, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);

                   Float          D;
                   distance::point_triangle_distance(P, T0, T1, T2, D);
                   // NOTE: D can be larger than D_hat

                   cout << "PT: " << PT.transpose().eval() << " D/prev_D: " << D
                        << "/" << prev_D << "\n";

                   Es(i) = sym::ipc_simplex_contact::PT_friction_energy(kappa, D_hat, friction_rate, dt, P, T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v);
                   // cout << "PT_barrier_energy: " << sym::ipc_simplex_contact::PT_barrier_energy(kappa, D_hat, P, T0, T1, T2) << "\n";
                   // cout << "PT_friction_energy: " << sym::ipc_simplex_contact::PT_friction_energy(kappa, D_hat, friction_rate, dt, P, T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v) << "\n";
                   //cout << "PT energy: " << Es(i) << "\n";
               });

    // Compute Edge-Edge energy
    auto EE_count = info.friction_EEs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(EE_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                EEs = info.friction_EEs().viewer().name("EEs"),
                Es  = info.friction_EE_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v   = info.eps_velocity(),
                rest_Ps = info.rest_positions().viewer().name("rest_Ps"),
                d_hat   = info.d_hat(),
                dt      = info.dt()] __device__(int i) mutable
               {
                   const auto& EE = EEs(i);

                   auto cid_L = contact_ids(EE[0]);
                   auto cid_R = contact_ids(EE[2]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   const Vector3& t0_Ea0 = rest_Ps(EE[0]);
                   const Vector3& t0_Ea1 = rest_Ps(EE[1]);
                   const Vector3& t0_Eb0 = rest_Ps(EE[2]);
                   const Vector3& t0_Eb1 = rest_Ps(EE[3]);


                   const Vector3& prev_E0 = prev_Ps(EE[0]);
                   const Vector3& prev_E1 = prev_Ps(EE[1]);
                   const Vector3& prev_E2 = prev_Ps(EE[2]);
                   const Vector3& prev_E3 = prev_Ps(EE[3]);

                   Float D_hat = d_hat * d_hat;

                   Float prev_D;
                   distance::edge_edge_distance(prev_E0, prev_E1, prev_E2, prev_E3, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);


                   const Vector3& E0 = Ps(EE[0]);
                   const Vector3& E1 = Ps(EE[1]);
                   const Vector3& E2 = Ps(EE[2]);
                   const Vector3& E3 = Ps(EE[3]);

                   Float D = D_hat;
                   distance::edge_edge_distance(E0, E1, E2, E3, D);
                   // NOTE: D can be larger than D_hat


                   cout << "EE: " << EE.transpose().eval() << " D/prev_D: " << D
                        << "/" << prev_D << "\n";

                   Es(i) = sym::ipc_simplex_contact::EE_friction_energy(kappa, D_hat, friction_rate, dt, E0, E1, E2, E3, prev_E0, prev_E1, prev_E2, prev_E3, eps_v);
                   // cout << "sym::ipc_simplex_contact::EE_friction_energy(kappa, D_hat, friction_rate, dt, E0, E1, E2, E3, prev_E0, prev_E1, prev_E2, prev_E3, eps_v): "
                   //      << sym::ipc_simplex_contact::EE_friction_energy(kappa, D_hat, friction_rate, dt, E0, E1, E2, E3, prev_E0, prev_E1, prev_E2, prev_E3, eps_v) << "\n";
               });

    // Compute Point-Edge energy
    auto PE_count = info.friction_PEs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(PE_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PEs = info.friction_PEs().viewer().name("PEs"),
                Es  = info.friction_PE_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PE = PEs(i);

                   auto cid_L = contact_ids(PE[0]);
                   auto cid_R = contact_ids(PE[1]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Float D_hat = d_hat * d_hat;


                   const Vector3& prev_P  = prev_Ps(PE[0]);
                   const Vector3& prev_E0 = prev_Ps(PE[1]);
                   const Vector3& prev_E1 = prev_Ps(PE[2]);
                   Float          prev_D;
                   distance::point_edge_distance(prev_P, prev_E0, prev_E1, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);

                   const Vector3& P  = Ps(PE[0]);
                   const Vector3& E0 = Ps(PE[1]);
                   const Vector3& E1 = Ps(PE[2]);
                   Float          D;
                   distance::point_edge_distance(P, E0, E1, D);
                   // NOTE: D can be larger than D_hat

                   cout << "PE: " << PE.transpose().eval() << " D/prev_D: " << D
                        << "/" << prev_D << "\n";

                   Es(i) = sym::ipc_simplex_contact::PE_friction_energy(kappa, D_hat, friction_rate, dt, P, E0, E1, prev_P, prev_E0, prev_E1, eps_v);
               });

    // Compute Point-Point energy
    auto PP_count = info.friction_PPs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(PP_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PPs = info.friction_PPs().viewer().name("PPs"),
                Es  = info.friction_PP_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PP = PPs(i);

                   auto cid_L = contact_ids(PP[0]);
                   auto cid_R = contact_ids(PP[1]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Float D_hat = d_hat * d_hat;

                   const Vector3& prev_P0 = prev_Ps(PP[0]);
                   const Vector3& prev_P1 = prev_Ps(PP[1]);
                   Float          prev_D;
                   distance::point_point_distance(prev_P0, prev_P1, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);


                   const Vector3& P0 = Ps(PP[0]);
                   const Vector3& P1 = Ps(PP[1]);
                   Float          D  = D_hat;
                   distance::point_point_distance(P0, P1, D);
                   // NOTE: D can be larger than D_hat

                   cout << "PP: " << PP.transpose().eval() << " D/prev_D: " << D
                        << "/" << prev_D << "\n";

                   Es(i) = sym::ipc_simplex_contact::PP_friction_energy(kappa, D_hat, friction_rate, dt, P0, P1, prev_P0, prev_P1, eps_v);
               });
}

void IPCSimplexFrictionalContact::do_assemble(ContactInfo& info)
{
    using namespace muda;

    // Compute Point-Triangle Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.friction_PTs().size(),
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PTs = info.friction_PTs().viewer().name("PTs"),
                Gs  = info.friction_PT_gradients().viewer().name("Gs"),
                Hs  = info.friction_PT_hessians().viewer().name("Hs"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PT = PTs(i);

                   auto cid_L = contact_ids(PT[0]);
                   auto cid_R = contact_ids(PT[1]);

                   const auto& P  = Ps(PT[0]);
                   const auto& T0 = Ps(PT[1]);
                   const auto& T1 = Ps(PT[2]);
                   const auto& T2 = Ps(PT[3]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;


                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Vector12 G_friction;
                   Matrix12x12 H_friction;
                   Vector3 test;
                   test(0) = 1e-8;
                   test(1) = 1e-8;
                   test(2) = 1e-8;
                   Float E1 = sym::ipc_simplex_contact::PT_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, P + test, T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v);
                   Float E2 = sym::ipc_simplex_contact::PT_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, P - test, T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v);

                   sym::ipc_simplex_contact::PT_friction_gradient_hessian(
                          G_friction, H_friction, kappa, d_hat * d_hat, friction_rate, dt, P, T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v);

                   // Gradient check
                   /*
                   Float numerical_diff = (E1 - E2) / 2;
                     cout << "numerical_diff: " << numerical_diff << "\n";
                   Float analytical_diff = G_friction(0) * test(0) + G_friction(1) * test(1) + G_friction(2) * test(2);
                     cout << "analytical_diff: " << analytical_diff << "\n";
                   cout << "numerical_diff - analytical_diff: " << numerical_diff - analytical_diff << "\n";
                   cout << "numerical_diff / analytical_diff: " << numerical_diff / analytical_diff << "\n";

                   Vector12 G_friction1 = Eigen::Matrix<Float, 12, 1>::Zero();
                   Vector12 G_friction2 = Eigen::Matrix<Float, 12, 1>::Zero();
                   Matrix12x12 H_friction0 = Eigen::Matrix<Float, 12, 12>::Zero();
                   Vector12 test12 = Eigen::Matrix<Float, 12, 1>::Zero();
                   test12(0) = 1e-8;
                   test12(1) = 1e-8;
                   test12(2) = 1e-8;
                   sym::ipc_simplex_contact::PT_friction_gradient_hessian(
                              G_friction1, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, P + test12.segment<3>(0), T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v);
                   sym::ipc_simplex_contact::PT_friction_gradient_hessian(
                              G_friction2, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, P - test12.segment<3>(0), T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v);

                   Vector12 G_friction_numerical_diff = (G_friction1 - G_friction2) / 2;
                   Vector12 G_friction_analytical_diff = H_friction * test12;
                   cout << "G_friction_numerical_diff: " << G_friction_numerical_diff.norm() << "\n";
                   cout << "G_friction_analytical_diff: " << G_friction_analytical_diff.norm() << "\n";
                   cout << "PT_grad_diff: " << (G_friction_numerical_diff - G_friction_analytical_diff).norm() << "\n";
                   */
                   Gs(i) = G_friction;
                   Hs(i) = H_friction;
                    cout << "G_friction: " << G_friction.norm() << "\n";
                    cout << "H_friction: " << H_friction.norm() << "\n";
                   // Hs(i) = Eigen::Matrix<Float, 12, 12>::Identity();
               });

    // Compute Edge-Edge Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.friction_EEs().size(),
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                EEs = info.friction_EEs().viewer().name("EEs"),
                Gs  = info.friction_EE_gradients().viewer().name("Gs"),
                Hs  = info.friction_EE_hessians().viewer().name("Hs"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v   = info.eps_velocity(),
                rest_Ps = info.rest_positions().viewer().name("rest_Ps"),
                d_hat   = info.d_hat(),
                dt      = info.dt()] __device__(int i) mutable
               {
                   const auto& EE = EEs(i);

                   auto cid_L = contact_ids(EE[0]);
                   auto cid_R = contact_ids(EE[2]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   const Vector3& t0_Ea0 = rest_Ps(EE[0]);
                   const Vector3& t0_Ea1 = rest_Ps(EE[1]);
                   const Vector3& t0_Eb0 = rest_Ps(EE[2]);
                   const Vector3& t0_Eb1 = rest_Ps(EE[3]);

                   Float D_hat = d_hat * d_hat;

                   const Vector3& prev_E0 = prev_Ps(EE[0]);
                   const Vector3& prev_E1 = prev_Ps(EE[1]);
                   const Vector3& prev_E2 = prev_Ps(EE[2]);
                   const Vector3& prev_E3 = prev_Ps(EE[3]);

                   Float prev_D;
                   distance::edge_edge_distance(prev_E0, prev_E1, prev_E2, prev_E3, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);


                   const Vector3& E0 = Ps(EE[0]);
                   const Vector3& E1 = Ps(EE[1]);
                   const Vector3& E2 = Ps(EE[2]);
                   const Vector3& E3 = Ps(EE[3]);

                   Float D;
                   distance::edge_edge_distance(E0, E1, E2, E3, D);
                   // NOTE: D can be larger than D_hat, if so, ignore this friction

                   Vector12 G_friction;
                   Matrix12x12 H_friction;
                   Vector3 test;
                   test(0) = 1e-6;
                   test(1) = 1e-6;
                   test(2) = 1e-6;
                   Float E1_ = sym::ipc_simplex_contact::EE_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, E0 + test, E1, E2, E3, prev_Ps(EE[0]), prev_Ps(EE[1]), prev_Ps(EE[2]), prev_Ps(EE[3]), eps_v);
                   Float E2_ = sym::ipc_simplex_contact::EE_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, E0 - test, E1, E2, E3, prev_Ps(EE[0]), prev_Ps(EE[1]), prev_Ps(EE[2]), prev_Ps(EE[3]), eps_v);

                   sym::ipc_simplex_contact::EE_friction_gradient_hessian(
                          G_friction, H_friction, kappa, d_hat * d_hat, friction_rate, dt, E0, E1, E2, E3, prev_Ps(EE[0]), prev_Ps(EE[1]), prev_Ps(EE[2]), prev_Ps(EE[3]), eps_v);

                   // Gradient check
                   /*
                   Float numerical_diff = (E1_ - E2_) / 2;
                     cout << "numerical_diff: " << numerical_diff << "\n";
                   Float analytical_diff = G_friction(0) * test(0) + G_friction(1) * test(1) + G_friction(2) * test(2);
                     cout << "analytical_diff: " << analytical_diff << "\n";
                   cout << "numerical_diff - analytical_diff: " << numerical_diff - analytical_diff << "\n";
                   cout << "numerical_diff / analytical_diff: " << numerical_diff / analytical_diff << "\n";

                   Vector12 G_friction1 = Eigen::Matrix<Float, 12, 1>::Zero();
                   Vector12 G_friction2 = Eigen::Matrix<Float, 12, 1>::Zero();
                   Matrix12x12 H_friction0 = Eigen::Matrix<Float, 12, 12>::Zero();
                   Vector12 test12 = Eigen::Matrix<Float, 12, 1>::Zero();
                   test12(0) = 1e-8;
                   test12(1) = 1e-8;
                   test12(2) = 1e-8;
                   sym::ipc_simplex_contact::EE_friction_gradient_hessian(
                              G_friction1, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, E0 + test12.segment<3>(0), E1, E2, E3, prev_Ps(EE[0]), prev_Ps(EE[1]), prev_Ps(EE[2]), prev_Ps(EE[3]), eps_v);
                   sym::ipc_simplex_contact::EE_friction_gradient_hessian(
                              G_friction2, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, E0 - test12.segment<3>(0), E1, E2, E3, prev_Ps(EE[0]), prev_Ps(EE[1]), prev_Ps(EE[2]), prev_Ps(EE[3]), eps_v);

                   Vector12 G_friction_numerical_diff = (G_friction1 - G_friction2) / 2;
                   Vector12 G_friction_analytical_diff = H_friction * test12;
                   cout << "G_friction_numerical_diff: " << G_friction_numerical_diff.norm() << "\n";
                   cout << "G_friction_analytical_diff: " << G_friction_analytical_diff.norm() << "\n";
                   cout << "EE_grad_diff: " << (G_friction_numerical_diff - G_friction_analytical_diff).norm() << "\n";
                   */
                   cout << "EE_G_friction: " << G_friction.norm() << "\n";
                     cout << "EE_H_friction: " << H_friction.norm() << "\n";
                   Gs(i) = G_friction;
                   Hs(i) = H_friction;
               });

    // Compute Point-Edge Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.friction_PEs().size(),
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PEs = info.friction_PEs().viewer().name("PEs"),
                Gs  = info.friction_PE_gradients().viewer().name("Gs"),
                Hs  = info.friction_PE_hessians().viewer().name("Hs"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PE = PEs(i);

                   auto cid_L = contact_ids(PE[0]);
                   auto cid_R = contact_ids(PE[1]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Float D_hat = d_hat * d_hat;

                   const Vector3& prev_P  = prev_Ps(PE[0]);
                   const Vector3& prev_E0 = prev_Ps(PE[1]);
                   const Vector3& prev_E1 = prev_Ps(PE[2]);
                   Float          prev_D;
                   distance::point_edge_distance(prev_P, prev_E0, prev_E1, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);

                   const Vector3& P  = Ps(PE[0]);
                   const Vector3& E0 = Ps(PE[1]);
                   const Vector3& E1 = Ps(PE[2]);
                   Float          D;
                   distance::point_edge_distance(P, E0, E1, D);
                   // NOTE: D can be larger than D_hat, if so, ignore this friction
                   Vector9 G_friction = Eigen::Matrix<Float, 9, 1>::Zero();
                   Matrix9x9 H_friction = Eigen::Matrix<Float, 9, 9>::Zero();
                   Vector3 test;
                   test(0) = 1e-6;
                   test(1) = 1e-6;
                   test(2) = 1e-6;
                   Float E1_ = sym::ipc_simplex_contact::PE_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, P + test, E0, E1, prev_Ps(PE[0]), prev_Ps(PE[1]), prev_Ps(PE[2]), eps_v);
                   Float E2_ = sym::ipc_simplex_contact::PE_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, P - test, E0, E1, prev_Ps(PE[0]), prev_Ps(PE[1]), prev_Ps(PE[2]), eps_v);

                   sym::ipc_simplex_contact::PE_friction_gradient_hessian(
                          G_friction, H_friction, kappa, d_hat * d_hat, friction_rate, dt, P, E0, E1, prev_Ps(PE[0]), prev_Ps(PE[1]), prev_Ps(PE[2]), eps_v);

                   // Gradient check
                   
                   Float numerical_diff = (E1_ - E2_) / 2;
                     cout << "numerical_diff: " << numerical_diff << "\n";
                   Float analytical_diff = G_friction(0) * test(0) + G_friction(1) * test(1) + G_friction(2) * test(2);
                     cout << "analytical_diff: " << analytical_diff << "\n";
                   cout << "numerical_diff - analytical_diff: " << numerical_diff - analytical_diff << "\n";
                   cout << "numerical_diff / analytical_diff: " << numerical_diff / analytical_diff << "\n";

                   Vector9 G_friction1 = Eigen::Matrix<Float, 9, 1>::Zero();
                   Vector9 G_friction2 = Eigen::Matrix<Float, 9, 1>::Zero();
                   Matrix9x9 H_friction0 = Eigen::Matrix<Float, 9, 9>::Zero();
                   Vector9 test9 = Eigen::Matrix<Float, 9, 1>::Zero();
                   test9(0) = 1e-8;
                   test9(1) = 1e-8;
                   test9(2) = 1e-8;
                   sym::ipc_simplex_contact::PE_friction_gradient_hessian(
                              G_friction1, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, P + test9.segment<3>(0), E0, E1, prev_Ps(PE[0]), prev_Ps(PE[1]), prev_Ps(PE[2]), eps_v);
                   sym::ipc_simplex_contact::PE_friction_gradient_hessian(
                              G_friction2, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, P - test9.segment<3>(0), E0, E1, prev_Ps(PE[0]), prev_Ps(PE[1]), prev_Ps(PE[2]), eps_v);

                   Vector9 G_friction_numerical_diff = (G_friction1 - G_friction2) / 2;
                   Vector9 G_friction_analytical_diff = H_friction * test9;
                   cout << "G_friction_numerical_diff: " << G_friction_numerical_diff.norm() << "\n";
                   cout << "G_friction_analytical_diff: " << G_friction_analytical_diff.norm() << "\n";
                   cout << "E1_: " << E1_ << "\n";
                   cout << "G_friction: " << G_friction.norm() << "\n";
                   cout << "H_friction: " << H_friction.norm() << "\n";
                   cout << "PE_grad_diff: " << (G_friction_numerical_diff - G_friction_analytical_diff).norm() << "\n";
                   
                   Gs(i) = G_friction;
                   Hs(i) = H_friction;
               });

    // Compute Point-Point Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.friction_PPs().size(),
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PPs = info.friction_PPs().viewer().name("PPs"),
                Gs  = info.friction_PP_gradients().viewer().name("Gs"),
                Hs  = info.friction_PP_hessians().viewer().name("Hs"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PP = PPs(i);

                   auto cid_L = contact_ids(PP[0]);
                   auto cid_R = contact_ids(PP[1]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;
                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Float D_hat = d_hat * d_hat;

                   const Vector3& prev_P0 = prev_Ps(PP[0]);
                   const Vector3& prev_P1 = prev_Ps(PP[1]);
                   Float          prev_D;
                   distance::point_point_distance(prev_P0, prev_P1, prev_D);
                   MUDA_ASSERT(prev_D < D_hat, "prev_D(%f) out of range, (0,%f)", prev_D, D_hat);

                   const Vector3& P0 = Ps(PP[0]);
                   const Vector3& P1 = Ps(PP[1]);
                   Float          D  = D_hat;
                   distance::point_point_distance(P0, P1, D);
                   // NOTE: D can be larger than D_hat, if so, ignore this friction
                   Vector6 G_friction;
                   Matrix6x6 H_friction;
                   Vector3 test;
                   test(0) = 1e-6;
                   test(1) = 1e-6;
                   test(2) = 1e-6;
                   Float E1_ = sym::ipc_simplex_contact::PP_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, P0 + test, P1, prev_P0, prev_P1, eps_v);
                   Float E2_ = sym::ipc_simplex_contact::PP_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, P0 - test, P1, prev_P0, prev_P1, eps_v);

                   sym::ipc_simplex_contact::PP_friction_gradient_hessian(
                          G_friction, H_friction, kappa, d_hat * d_hat, friction_rate, dt, P0, P1, prev_P0, prev_P1, eps_v);

                   // Gradient check
                   
                   Float numerical_diff = (E1_ - E2_) / 2;
                     cout << "numerical_diff: " << numerical_diff << "\n";
                   Float analytical_diff = G_friction(0) * test(0) + G_friction(1) * test(1) + G_friction(2) * test(2);
                     cout << "analytical_diff: " << analytical_diff << "\n";
                   cout << "numerical_diff - analytical_diff: " << numerical_diff - analytical_diff << "\n";
                   cout << "numerical_diff / analytical_diff: " << numerical_diff / analytical_diff << "\n";

                   Vector6 G_friction1 = Eigen::Matrix<Float, 6, 1>::Zero();
                   Vector6 G_friction2 = Eigen::Matrix<Float, 6, 1>::Zero();
                   Matrix6x6 H_friction0 = Eigen::Matrix<Float, 6, 6>::Zero();
                   Vector6 test6 = Eigen::Matrix<Float, 6, 1>::Zero();
                   test6(0) = 1e-8;
                   test6(1) = 1e-8;
                   test6(2) = 1e-8;
                   sym::ipc_simplex_contact::PP_friction_gradient_hessian(
                              G_friction1, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, P0 + test6.segment<3>(0), P1, prev_P0, prev_P1, eps_v);
                   sym::ipc_simplex_contact::PP_friction_gradient_hessian(
                              G_friction2, H_friction0, kappa, d_hat * d_hat, friction_rate, dt, P0 - test6.segment<3>(0), P1, prev_P0, prev_P1, eps_v);

                   Vector6 G_friction_numerical_diff = (G_friction1 - G_friction2) / 2;
                   Vector6 G_friction_analytical_diff = H_friction * test6;
                   cout << "G_friction_numerical_diff: " << G_friction_numerical_diff.norm() << "\n";
                   cout << "G_friction_analytical_diff: " << G_friction_analytical_diff.norm() << "\n";
                   cout << "EE_grad_diff: " << (G_friction_numerical_diff - G_friction_analytical_diff).norm() << "\n";

                   Gs(i) = G_friction;
                   Hs(i) = H_friction;
               });
}
}  // namespace uipc::backend::cuda