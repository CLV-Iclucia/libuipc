#include "hip/hip_runtime.h"
#include <contact_system/constitutions/ipc_simplex_contact.h>
#include <muda/ext/geo/distance.h>
#include <kernel_cout.h>

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(IPCSimplexContact);

void IPCSimplexContact::do_build(BuildInfo& info)
{
}

namespace sym::ipc_simplex_contact
{
#include "sym/ipc_contact.inl"

    __device__ Float PT_barrier_energy(Float          kappa,
                                       Float          squared_d_hat,
                                       const Vector3& P,
                                       const Vector3& T0,
                                       const Vector3& T1,
                                       const Vector3& T2)
    {
        using namespace muda::distance;
        Float D_hat = squared_d_hat;
        Float D;
        point_triangle_distance(P, T0, T1, T2, D);
        Float B;
        KappaBarrier(B, kappa, D, D_hat);
        return B;
    }

    __device__ Float PT_friction_energy(Float          kappa,
                                        Float          squared_d_hat,
                                        Float          mu,
                                        Float          dt,
                                        const Vector3& P,
                                        const Vector3& T0,
                                        const Vector3& T1,
                                        const Vector3& T2,
                                        const Vector3& prev_P,
                                        const Vector3& prev_T0,
                                        const Vector3& prev_T1,
                                        const Vector3& prev_T2,
                                        Float          eps_v)
    {
        using namespace muda::distance;

        Float D;
        point_triangle_distance(prev_P, prev_T0, prev_T1, prev_T2, D);

        Vector12 GradD;
        point_triangle_distance_gradient(prev_P, prev_T0, prev_T1, prev_T2, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);

        Float lam = -dBdD * GradD.head(3).norm();
        Vector3 n =  (T0 - T1).cross(T0 - T2);
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block(0, 0, 3, 3) = I - normal * normal.transpose();
        Tk.block(3, 0, 3, 3) = normal * normal.transpose() - I;
        Vector3 v1 = (P - prev_P) / dt;

        // suppose P0 = t(0) * T0 + t(1) * T1 + t(2) * T2
        Eigen::Matrix<Float, 3, 2> base;
        base << T1 - T0, T2 - T0;
        Eigen::Matrix<Float, 2, 2> Lhs = base.transpose() * base;
        Vector2 rhs = base.transpose() * (P - T0);
        Eigen::Matrix<Float, 2, 2> Lhs_inv;
        Float det = Lhs(0, 0) * Lhs(1, 1) - Lhs(0, 1) * Lhs(1, 0);
        Lhs_inv << Lhs(1, 1) / det, -Lhs(0, 1) / det, -Lhs(1, 0) / det, Lhs(0, 0) / det;
        Vector2 t = Lhs_inv * rhs;
        Float t1 = t(0);
        Float t2 = t(1);
        Float t0 = 1 - t1 - t2;

        Eigen::Vector<Float, 6> V;
        V  << v1, t0 * (T0 - prev_T0) / dt + t1 * (T1 - prev_T1) / dt + t2 * (T2 - prev_T2) / dt;
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        // cout << "lam: " << lam << "\n";
        // cout << "y: " << y << "\n";
        Float F;
        FrictionEnergy(F, lam * mu, eps_v, dt, y);
        return F;
    }

    __device__ void PT_barrier_gradient_hessian(Vector12&      G,
                                                Matrix12x12&   H,
                                                Float          kappa,
                                                Float          squared_d_hat,
                                                const Vector3& P,
                                                const Vector3& T0,
                                                const Vector3& T1,
                                                const Vector3& T2)
    {
        using namespace muda::distance;

        Float D_hat = squared_d_hat;
        Float D;
        point_triangle_distance(P, T0, T1, T2, D);

        Vector12 GradD;
        point_triangle_distance_gradient(P, T0, T1, T2, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);

        //tex:
        //$$
        // G = \frac{\partial B}{\partial D} \frac{\partial D}{\partial x}
        //$$
        G = dBdD * GradD;

        Float ddBddD;
        ddKappaBarrierddD(ddBddD, kappa, D, squared_d_hat);

        Matrix12x12 HessD;
        point_triangle_distance_hessian(P, T0, T1, T2, HessD);

        //tex:
        //$$
        // H = \frac{\partial^2 B}{\partial D^2} \frac{\partial D}{\partial x} \frac{\partial D}{\partial x}^T + \frac{\partial B}{\partial D} \frac{\partial^2 D}{\partial x^2}
        //$$
        H = ddBddD * GradD * GradD.transpose() + dBdD * HessD;
    }

    __device__ void PT_friction_gradient_hessian(Vector12&      G,
                                                 Matrix12x12&   H,
                                                 Float          kappa,
                                                 Float          squared_d_hat,
                                                 Float          mu,
                                                 Float          dt,
                                                 const Vector3& P,
                                                 const Vector3& T0,
                                                 const Vector3& T1,
                                                 const Vector3& T2,
                                                 const Vector3& prev_P,
                                                 const Vector3& prev_T0,
                                                 const Vector3& prev_T1,
                                                 const Vector3& prev_T2,
                                                 Float          eps_v)
    {
        using namespace muda::distance;
        Float D;
        point_triangle_distance(prev_P, prev_T0, prev_T1, prev_T2, D);

        Vector12 GradD;
        point_triangle_distance_gradient(prev_P, prev_T0, prev_T1, prev_T2, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);

        Float lam = -dBdD * GradD.head(3).norm();
        Vector3 n =  (T0 - T1).cross(T0 - T2);
        Vector3 normal = n / n.norm();
        Eigen::Matrix<Float, 6, 3> Tk;
        Eigen::Matrix<Float, 3, 3> I = Eigen::Matrix<Float, 3, 3>::Identity();
        Tk.block(0, 0, 3, 3) = I - normal * normal.transpose();
        Tk.block(3, 0, 3, 3) = normal * normal.transpose() - I;
        Vector3 v1 = (P - prev_P) / dt;

        // suppose P0 = t(0) * T0 + t(1) * T1 + t(2) * T2
        Eigen::Matrix<Float, 3, 2> base;
        base << T1 - T0, T2 - T0;
        Eigen::Matrix<Float, 2, 2> Lhs = base.transpose() * base;
        Vector2 rhs = base.transpose() * (P - T0);
        Eigen::Matrix<Float, 2, 2> Lhs_inv;
        Float det = Lhs(0, 0) * Lhs(1, 1) - Lhs(0, 1) * Lhs(1, 0);
        Lhs_inv << Lhs(1, 1) / det, -Lhs(0, 1) / det, -Lhs(1, 0) / det, Lhs(0, 0) / det;
        Vector2 t = Lhs_inv * rhs;
        Float t1 = t(0);
        Float t2 = t(1);
        Float t0 = 1 - t1 - t2;
        Eigen::Vector<Float, 6> V;
        V << v1, t0 * (T0 - prev_T0) / dt + t1 * (T1 - prev_T1) / dt + t2 * (T2 - prev_T2) / dt;
        for (int i = 0; i < 6; i++) {
            // cout << "V(" << i << "): " << V(i) << "\n";
        }
        Vector3 vk = Tk.transpose() * V;
        Float y = vk.norm() * dt;
        cout << "y: " << y << "\n";
        Eigen::Vector<Float, 6> dFdV;
        dFrictionEnergydV(dFdV, lam * mu, Tk,  eps_v, dt, vk);
        Vector3 test;
        test << 1e-10, 1e-10, 1e-10;
        Float E1 = PT_friction_energy(kappa, squared_d_hat, mu, dt, P + test, T0, T1, T2, prev_P, prev_T0, prev_T1, prev_T2, eps_v);
        Float E2 = PT_friction_energy(kappa, squared_d_hat, mu, dt, P - test, T0, T1, T2, prev_P, prev_T0, prev_T1, prev_T2, eps_v);
        Float num_diff = (E1 - E2) / 2;
        for (int i = 0; i < 6; i++) {
            cout << "dFdV(" << i << "): " << dFdV(i) << "\n";
        }
        Vector6 test6;
        test6 << 1e-8, 1e-8, 1e-8, 0, 0, 0;
        Float E3;
        Float y1 = (Tk.transpose() * (V + test6)).norm() * dt;
        cout << "lam_: " << lam << "\n";
        cout << "y1: " << y1 << "\n";
        FrictionEnergy(E3, lam * mu, eps_v, dt, y1);
        Float E4;
        Float y2 = (Tk.transpose() * (V - test6)).norm() * dt;
        cout << "y2: " << y2 << "\n";
        FrictionEnergy(E4, lam * mu, eps_v, dt, y2);
        cout << "E1: " << E1 << "\n";
        cout << "E2: " << E2 << "\n";
        cout << "E3: " << E3 << "\n";
        cout << "E4: " << E4 << "\n";
        Eigen::Matrix<Float, 6, 12> GradV;
        GradV.block(0, 0, 3, 3) = I / dt;
        GradV.block(3, 3, 3, 3) = I * t0 / dt;
        GradV.block(3, 6, 3, 3) = I * t1 / dt;
        GradV.block(3, 9, 3, 3) = I * t2 / dt;
        Eigen::Matrix<Float, 12, 6> GradV_transpose = GradV.transpose();
        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 6; j++) {
            // cout << "GradV_transpose(" << i << ", " << j << "): " << GradV_transpose(i, j) << "\n";
            }
        }
        
        for (int i = 0; i < 12; i++) {
            G(i) = 0;
            for (int j = 0; j < 6; j++) {
                G(i) += GradV_transpose(i, j) * dFdV(j);
                cout << "G(" << i << "): " << G(i) << "\n";
            }
        }
        Float ana_diff = G(0) * test(0) + G(1) * test(1) + G(2) * test(2);
        Float ana_diff1 = dFdV.dot(test6);
        Float num_diff1 = (E3 - E4) / 2;
        cout << "ana_diff1: " << ana_diff1 << "\n";
        cout << "num_diff1: " << num_diff1 << "\n";
        cout << "num_diff: " << num_diff << "\n";
        cout << "ana_diff: " << ana_diff << "\n";
        cout << "num_diff - ana_diff: " << num_diff - ana_diff << "\n";
        // G = GradV_transpose * dFdV;
        for (int i = 0; i < 6; i++) {
            // cout << "G_(" << i << "): " << G(i) << "\n";
        }
        for (int i = 0; i < 6; i++) {
            // cout << "dFdV(i): " << dFdV(i) << "\n";
        }

        Eigen::Matrix<Float, 6, 6> ddFddV;
        ddFrictionEnergyddV(ddFddV, lam * mu, Tk, eps_v, dt, vk);
        Vector3 offset = T0 - P + base * t;
        Eigen::Matrix<Float, 2, 3> dtdP = Lhs_inv * base.transpose();
        Eigen::Matrix<Float, 2, 3> dtdT1 = -Lhs_inv.col(0) * offset.transpose() - t1 * dtdP;
        Eigen::Matrix<Float, 2, 3> dtdT2 = -Lhs_inv.col(1) * offset.transpose() - t2 * dtdP;
        Eigen::Matrix<Float, 2, 3> dtdT0 = -dtdP - dtdT1 - dtdT2;
        Vector3 dt0dP = (-dtdP.row(0) - dtdP.row(1)).transpose();
        Vector3 dt0dT1 = (-dtdT1.row(0) - dtdT1.row(1)).transpose();
        Vector3 dt0dT2 = (-dtdT2.row(0) - dtdT2.row(1)).transpose();
        Vector3 dt0dT0 = (-dtdT0.row(0) - dtdT0.row(1)).transpose();
        // Grad(GradV.transpose()) * dFdV
        H.block(0, 0, 3, 12) = Eigen::Matrix<Float, 3, 12>::Zero();
        H.block(3, 0, 3, 3) = dFdV.tail(3) * dt0dP.transpose() / dt;
        H.block(3, 3, 3, 3) = dFdV.tail(3) * dt0dT0.transpose() / dt;
        H.block(3, 6, 3, 3) = dFdV.tail(3) * dt0dT1.transpose() / dt;
        H.block(3, 9, 3, 3) = dFdV.tail(3) * dt0dT2.transpose() / dt;
        H.block(6, 0, 3, 3) = dFdV.tail(3) * dtdP.row(0) / dt;
        H.block(6, 3, 3, 3) = dFdV.tail(3) * dtdT0.row(0) / dt;
        H.block(6, 6, 3, 3) = dFdV.tail(3) * dtdT1.row(0) / dt;
        H.block(6, 9, 3, 3) = dFdV.tail(3) * dtdT2.row(0) / dt;
        H.block(9, 0, 3, 3) = dFdV.tail(3) * dtdP.row(1) / dt;
        H.block(9, 3, 3, 3) = dFdV.tail(3) * dtdT0.row(1) / dt;
        H.block(9, 6, 3, 3) = dFdV.tail(3) * dtdT1.row(1) / dt;
        H.block(9, 9, 3, 3) = dFdV.tail(3) * dtdT2.row(1) / dt;
        cout << "H(0,0): " << H(0, 0) << "\n";
        for (int i=0; i < 6; i++) {
            for (int j=0; j < 6; j++) {
                //cout << "ddFddV(" << i << ", " << j << "): " << ddFddV(i, j) << "\n";
            }
        }
        for (int i = 0; i < 6; i++) {
            for (int j = 0; j < 12; j++) {
                // cout << "GradV(" << i << ", " << j << "): " << GradV(i, j) << "\n";
            }
        }
        Eigen::Matrix<Float, 6, 12> mat = Eigen::Matrix<Float, 6, 12>::Zero();
        for (int i = 0; i < 6; i++) {
            for (int j = 0; j < 12; j++) {
                for (int k = 0; k < 6; k++) {
                    mat(i, j) += ddFddV(i, k) * GradV(k, j);
                }
            }
        }
        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 12; j++) {
               for (int k = 6; k < 6; k++) {
                   H(i, j) += GradV.transpose()(k, i) * mat(k, j);
               }
            }
        }
        cout << "H+(0, 0): " << H(0, 0) << "\n";
    }

    __device__ Float EE_barrier_energy(Float          kappa,
                                       Float          squared_d_hat,
                                       const Vector3& t0_Ea0,
                                       const Vector3& t0_Ea1,
                                       const Vector3& t0_Eb0,
                                       const Vector3& t0_Eb1,
                                       const Vector3& Ea0,
                                       const Vector3& Ea1,
                                       const Vector3& Eb0,
                                       const Vector3& Eb1)
    {
        // using mollifier to improve the smoothness of the edge-edge barrier

        using namespace muda::distance;
        Float D_hat = squared_d_hat;
        Float D;
        edge_edge_distance(Ea0, Ea1, Eb0, Eb1, D);
        Float B;
        KappaBarrier(B, kappa, D, D_hat);

        Float eps_x;
        edge_edge_mollifier_threshold(t0_Ea0, t0_Ea1, t0_Eb0, t0_Eb1, eps_x);

        Float ek;
        edge_edge_mollifier(Ea0, Ea1, Eb0, Eb1, eps_x, ek);

        return ek * B;
    }


    __device__ void EE_barrier_gradient_hessian(Vector12&      G,
                                                Matrix12x12&   H,
                                                Float          kappa,
                                                Float          squared_d_hat,
                                                const Vector3& t0_Ea0,
                                                const Vector3& t0_Ea1,
                                                const Vector3& t0_Eb0,
                                                const Vector3& t0_Eb1,
                                                const Vector3& Ea0,
                                                const Vector3& Ea1,
                                                const Vector3& Eb0,
                                                const Vector3& Eb1)
    {
        using namespace muda::distance;

        Float D_hat = squared_d_hat;


        Float D;
        edge_edge_distance(Ea0, Ea1, Eb0, Eb1, D);

        Float B;
        KappaBarrier(B, kappa, D, D_hat);

        //tex: $$ \epsilon_x $$
        Float eps_x;
        edge_edge_mollifier_threshold(t0_Ea0, t0_Ea1, t0_Eb0, t0_Eb1, eps_x);

        //tex: $$ e_k $$
        Float ek;
        edge_edge_mollifier(Ea0, Ea1, Eb0, Eb1, eps_x, ek);

        //tex: $$\nabla e_k$$
        Vector12 Gradek;
        edge_edge_mollifier_gradient(Ea0, Ea1, Eb0, Eb1, eps_x, Gradek);

        //tex: $$ \nabla D$$
        Vector12 GradD;
        edge_edge_distance_gradient(Ea0, Ea1, Eb0, Eb1, GradD);

        //tex: $$ \frac{\partial B}{\partial D} $$
        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, D_hat);

        //tex: $$ \nabla B = \frac{\partial B}{\partial D} \nabla D$$
        Vector12 GradB = dBdD * GradD;

        //tex:
        //$$
        // G = \nabla e_k B + e_k \nabla B
        //$$
        G = Gradek * B + ek * GradB;

        //tex: $$ \frac{\partial^2 B}{\partial D^2} $$
        Float ddBddD;
        ddKappaBarrierddD(ddBddD, kappa, D, squared_d_hat);

        //tex: $$ \nabla^2 D$$
        Matrix12x12 HessD;
        edge_edge_distance_hessian(Ea0, Ea1, Eb0, Eb1, HessD);

        //tex:
        //$$
        // \nabla^2 B = \frac{\partial^2 B}{\partial D^2} \nabla D \nabla D^T + \frac{\partial B}{\partial D} \nabla^2 D
        //$$
        Matrix12x12 HessB = ddBddD * GradD * GradD.transpose() + dBdD * HessD;

        //tex: $$ \nabla^2 e_k$$
        Matrix12x12 Hessek;
        edge_edge_mollifier_hessian(Ea0, Ea1, Eb0, Eb1, eps_x, Hessek);

        //tex: $$ \nabla^2 e_k B + \nabla e_k \nabla B^T + \nabla B \nabla e_k^T + e_k \nabla^2 B$$
        H = Hessek * B + Gradek * GradB.transpose() + GradB * Gradek.transpose() + ek * HessB;
    }

    __device__ Float PE_barrier_energy(Float          kappa,
                                       Float          squared_d_hat,
                                       const Vector3& P,
                                       const Vector3& E0,
                                       const Vector3& E1)
    {
        using namespace muda::distance;
        Float D_hat = squared_d_hat;
        Float D     = 0.0;
        point_edge_distance(P, E0, E1, D);
        Float E = 0.0;
        KappaBarrier(E, kappa, D, D_hat);
        return E;
    }

    __device__ void PE_barrier_gradient_hessian(Vector9&       G,
                                                Matrix9x9&     H,
                                                Float          kappa,
                                                Float          squared_d_hat,
                                                const Vector3& P,
                                                const Vector3& E0,
                                                const Vector3& E1)
    {
        using namespace muda::distance;

        Float D_hat = squared_d_hat;
        Float D     = 0.0;
        point_edge_distance(P, E0, E1, D);

        Vector9 GradD;
        point_edge_distance_gradient(P, E0, E1, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);

        //tex:
        //$$
        // G = \frac{\partial B}{\partial D} \frac{\partial D}{\partial x}
        //$$
        G = dBdD * GradD;

        Float ddBddD;
        ddKappaBarrierddD(ddBddD, kappa, D, squared_d_hat);

        Matrix9x9 HessD;
        point_edge_distance_hessian(P, E0, E1, HessD);

        //tex:
        //$$
        // H = \frac{\partial^2 B}{\partial D^2} \frac{\partial D}{\partial x} \frac{\partial D}{\partial x}^T + \frac{\partial B}{\partial D} \frac{\partial^2 D}{\partial x^2}
        //$$
        H = ddBddD * GradD * GradD.transpose() + dBdD * HessD;
    }

    __device__ Float PP_barrier_energy(Float          kappa,
                                       Float          squared_d_hat,
                                       const Vector3& P0,
                                       const Vector3& P1)
    {
        using namespace muda::distance;
        Float D_hat = squared_d_hat;
        Float D     = 0.0;
        point_point_distance(P0, P1, D);
        Float E = 0.0;
        KappaBarrier(E, kappa, D, D_hat);
        return E;
    }

    __device__ void PP_barrier_gradient_hessian(Vector6&       G,
                                                Matrix6x6&     H,
                                                Float          kappa,
                                                Float          squared_d_hat,
                                                const Vector3& P0,
                                                const Vector3& P1)
    {
        using namespace muda::distance;

        Float D_hat = squared_d_hat;
        Float D     = 0.0;
        point_point_distance(P0, P1, D);

        Vector6 GradD;
        point_point_distance_gradient(P0, P1, GradD);

        Float dBdD;
        dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);

        //tex:
        //$$
        // G = \frac{\partial B}{\partial D} \frac{\partial D}{\partial x}
        //$$
        G = dBdD * GradD;

        Float ddBddD;
        ddKappaBarrierddD(ddBddD, kappa, D, squared_d_hat);

        Matrix6x6 HessD;
        point_point_distance_hessian(P0, P1, HessD);

        //tex:
        //$$
        // H = \frac{\partial^2 B}{\partial D^2} \frac{\partial D}{\partial x} \frac{\partial D}{\partial x}^T + \frac{\partial B}{\partial D} \frac{\partial^2 D}{\partial x^2}
        //$$
        H = ddBddD * GradD * GradD.transpose() + dBdD * HessD;
    }
}  // namespace sym::ipc_simplex_contact


void IPCSimplexContact::do_compute_energy(EnergyInfo& info)
{
    using namespace muda;

    // Compute Point-Triangle energy
    auto PT_count = info.PTs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(PT_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PTs = info.PTs().viewer().name("PTs"),
                Es  = info.PT_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PT = PTs(i);

                   auto cid_L = contact_ids(PT[0]);
                   auto cid_R = contact_ids(PT[1]);

                   const auto& P  = Ps(PT[0]);
                   const auto& T0 = Ps(PT[1]);
                   const auto& T1 = Ps(PT[2]);
                   const auto& T2 = Ps(PT[3]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;

                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;


                   Float D_hat = d_hat * d_hat;
                   Float D     = D_hat;
                   distance::point_triangle_distance(P, T0, T1, T2, D);

                   MUDA_ASSERT(D < D_hat && D > 0,
                               "PT[%d,%d,%d,%d] d^2(%f) out of range, (0,%f)",
                               PT(0),
                               PT(1),
                               PT(2),
                               PT(3),
                               D,
                               D_hat);

                   Es(i) = sym::ipc_simplex_contact::PT_barrier_energy(
                       kappa, D_hat, P, T0, T1, T2);

                   //cout << "PT energy: " << Es(i) << "\n";
               });

    // Compute Edge-Edge energy
    auto EE_count = info.EEs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(EE_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                EEs = info.EEs().viewer().name("EEs"),
                Es  = info.EE_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v   = info.eps_velocity(),
                rest_Ps = info.rest_positions().viewer().name("rest_Ps"),
                d_hat   = info.d_hat(),
                dt      = info.dt()] __device__(int i) mutable
               {
                   const auto& EE = EEs(i);

                   auto cid_L = contact_ids(EE[0]);
                   auto cid_R = contact_ids(EE[2]);

                   const auto& E0 = Ps(EE[0]);
                   const auto& E1 = Ps(EE[1]);
                   const auto& E2 = Ps(EE[2]);
                   const auto& E3 = Ps(EE[3]);

                   const auto& t0_Ea0 = rest_Ps(EE[0]);
                   const auto& t0_Ea1 = rest_Ps(EE[1]);
                   const auto& t0_Eb0 = rest_Ps(EE[2]);
                   const auto& t0_Eb1 = rest_Ps(EE[3]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;

                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Float D_hat = d_hat * d_hat;
                   Float D     = D_hat;
                   distance::edge_edge_distance(E0, E1, E2, E3, D);

                   MUDA_ASSERT(D < D_hat && D > 0,
                               "EE[%d,%d,%d,%d] d^2(%f) out of range, (0,%f)",
                               EE(0),
                               EE(1),
                               EE(2),
                               EE(3),
                               D,
                               D_hat);

                   Es(i) = sym::ipc_simplex_contact::EE_barrier_energy(
                       kappa, D_hat, t0_Ea0, t0_Ea1, t0_Eb0, t0_Eb1, E0, E1, E2, E3);
               });

    // Compute Point-Edge energy
    auto PE_count = info.PEs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(PE_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PEs = info.PEs().viewer().name("PEs"),
                Es  = info.PE_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PE = PEs(i);

                   auto cid_L = contact_ids(PE[0]);
                   auto cid_R = contact_ids(PE[1]);

                   const auto& P  = Ps(PE[0]);
                   const auto& E0 = Ps(PE[1]);
                   const auto& E1 = Ps(PE[2]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;

                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Float D_hat = d_hat * d_hat;
                   Float D     = D_hat;
                   distance::point_edge_distance(P, E0, E1, D);

                   MUDA_ASSERT(D < D_hat && D > 0,
                               "PE[%d,%d,%d] d^2(%f) out of range, (0,%f)",
                               PE(0),
                               PE(1),
                               PE(2),
                               D,
                               D_hat);

                   Es(i) = sym::ipc_simplex_contact::PE_barrier_energy(kappa, D_hat, P, E0, E1);
               });

    // Compute Point-Point energy
    auto PP_count = info.PPs().size();
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(PP_count,
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PPs = info.PPs().viewer().name("PPs"),
                Es  = info.PP_energies().viewer().name("Es"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PP = PPs(i);

                   auto cid_L = contact_ids(PP[0]);
                   auto cid_R = contact_ids(PP[1]);

                   const auto& P0 = Ps(PP[0]);
                   const auto& P1 = Ps(PP[1]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;

                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Float D_hat = d_hat * d_hat;
                   Float D     = D_hat;
                   distance::point_point_distance(P0, P1, D);

                   MUDA_ASSERT(D < D_hat && D > 0,
                               "PP[%d,%d] d^2(%f) out of range, (0,%f)",
                               PP(0),
                               PP(1),
                               D,
                               D_hat);

                   Es(i) = sym::ipc_simplex_contact::PP_barrier_energy(kappa, D_hat, P0, P1);
               });
}

void IPCSimplexContact::do_assemble(ContactInfo& info)
{
    using namespace muda;

    // Compute Point-Triangle Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.PTs().size(),
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PTs = info.PTs().viewer().name("PTs"),
                Gs  = info.PT_gradients().viewer().name("Gs"),
                Hs  = info.PT_hessians().viewer().name("Hs"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PT = PTs(i);

                   auto cid_L = contact_ids(PT[0]);
                   auto cid_R = contact_ids(PT[1]);

                   const auto& P  = Ps(PT[0]);
                   const auto& T0 = Ps(PT[1]);
                   const auto& T1 = Ps(PT[2]);
                   const auto& T2 = Ps(PT[3]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;


                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   Vector12 G_contact;
                   Vector12 G_friction;
                   Matrix12x12 H_contact;
                   Matrix12x12 H_friction;
                   sym::ipc_contact::PT_barrier_gradient_hessian(
                       G_contact, H_contact, kappa, d_hat * d_hat, P, T0, T1, T2);
                   Vector3 test;
                   test << 1e-6, 1e-6, 1e-6;
                   Float E1 = sym::ipc_contact::PT_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, P + test, T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v);
                   Float E2 = sym::ipc_contact::PT_friction_energy(kappa, d_hat * d_hat, friction_rate, dt, P - test, T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v);

                   sym::ipc_contact::PT_friction_gradient_hessian(
                          G_friction, H_friction, kappa, d_hat * d_hat, friction_rate, dt, P, T0, T1, T2, prev_Ps(PT[0]), prev_Ps(PT[1]), prev_Ps(PT[2]), prev_Ps(PT[3]), eps_v);
                   for (int j = 0; j < 12; ++j)
                   {
                       // cout << "G_contact(" << j << "): " << G_friction(j) << "\n";
                       for (int k = 0; k < 12; ++k)
                       {
                           // cout << "H_contact(" << j << ", " << k << "): " << H_friction(j, k) << "\n";
                       }
                   }
                   Float numerical_diff = (E1 - E2) / 2;
                     cout << "numerical_diff: " << numerical_diff << "\n";
                   Float analytical_diff = G_friction(0) * test(0) + G_friction(1) * test(1) + G_friction(2) * test(2);
                     cout << "analytical_diff: " << analytical_diff << "\n";
                   cout << "numerical_diff - analytical_diff: " << numerical_diff - analytical_diff << "\n";
                   cout << "numerical_diff / analytical_diff: " << numerical_diff / analytical_diff << "\n";

                   Gs(i) = G_contact + G_friction;
                   // Hs(i) = H_contact + H_friction;
                   Hs(i) = Eigen::Matrix<Float, 12, 12>::Identity();
               });

    // Compute Edge-Edge Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.EEs().size(),
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                EEs = info.EEs().viewer().name("EEs"),
                Gs  = info.EE_gradients().viewer().name("Gs"),
                Hs  = info.EE_hessians().viewer().name("Hs"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v   = info.eps_velocity(),
                rest_Ps = info.rest_positions().viewer().name("rest_Ps"),
                d_hat   = info.d_hat(),
                dt      = info.dt()] __device__(int i) mutable
               {
                   const auto& EE = EEs(i);

                   auto cid_L = contact_ids(EE[0]);
                   auto cid_R = contact_ids(EE[2]);

                   const auto& E0 = Ps(EE[0]);
                   const auto& E1 = Ps(EE[1]);
                   const auto& E2 = Ps(EE[2]);
                   const auto& E3 = Ps(EE[3]);

                   const auto& t0_Ea0 = rest_Ps(EE[0]);
                   const auto& t0_Ea1 = rest_Ps(EE[1]);
                   const auto& t0_Eb0 = rest_Ps(EE[2]);
                   const auto& t0_Eb1 = rest_Ps(EE[3]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;

                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   sym::ipc_simplex_contact::EE_barrier_gradient_hessian(
                       Gs(i), Hs(i), kappa, d_hat * d_hat, t0_Ea0, t0_Ea1, t0_Eb0, t0_Eb1, E0, E1, E2, E3);
               });

    // Compute Point-Edge Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.PEs().size(),
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PEs = info.PEs().viewer().name("PEs"),
                Gs  = info.PE_gradients().viewer().name("Gs"),
                Hs  = info.PE_hessians().viewer().name("Hs"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PE = PEs(i);

                   auto cid_L = contact_ids(PE[0]);
                   auto cid_R = contact_ids(PE[1]);

                   const auto& P  = Ps(PE[0]);
                   const auto& E0 = Ps(PE[1]);
                   const auto& E1 = Ps(PE[2]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;

                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   sym::ipc_simplex_contact::PE_barrier_gradient_hessian(
                       Gs(i), Hs(i), kappa, d_hat * d_hat, P, E0, E1);
               });

    // Compute Point-Point Gradient and Hessian
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.PPs().size(),
               [table = info.contact_tabular().viewer().name("contact_tabular"),
                contact_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                PPs = info.PPs().viewer().name("PPs"),
                Gs  = info.PP_gradients().viewer().name("Gs"),
                Hs  = info.PP_hessians().viewer().name("Hs"),
                Ps  = info.positions().viewer().name("Ps"),
                prev_Ps = info.prev_positions().viewer().name("prev_Ps"),  // for friction calculation
                eps_v = info.eps_velocity(),
                d_hat = info.d_hat(),
                dt    = info.dt()] __device__(int i) mutable
               {
                   const auto& PP = PPs(i);

                   auto cid_L = contact_ids(PP[0]);
                   auto cid_R = contact_ids(PP[1]);

                   const auto& P0 = Ps(PP[0]);
                   const auto& P1 = Ps(PP[1]);

                   auto kappa = table(cid_L, cid_R).kappa * dt * dt;

                   // Use this to compute friction
                   auto friction_rate = table(cid_L, cid_R).mu;

                   sym::ipc_simplex_contact::PP_barrier_gradient_hessian(
                       Gs(i), Hs(i), kappa, d_hat * d_hat, P0, P1);
               });
}
}  // namespace uipc::backend::cuda