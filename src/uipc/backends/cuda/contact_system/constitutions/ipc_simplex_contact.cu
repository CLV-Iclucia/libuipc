#include "hip/hip_runtime.h"
#include <contact_system/constitutions/ipc_simplex_contact.h>
#include <muda/ext/geo/distance.h>

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(IPCSimplexContact);

void IPCSimplexContact::do_build(BuildInfo& info) {}


namespace ipc_contact
{
    namespace sym
    {
        using std::log;
#include <contact_system/constitutions/sym/ipc_contact.inl>
    }  // namespace sym


    __device__ Float PT_barrier_energy(Float          kappa,
                                       Float          squared_d_hat,
                                       const Vector3& P,
                                       const Vector3& T0,
                                       const Vector3& T1,
                                       const Vector3& T2)
    {
        using namespace muda::distance;
        Float D_hat = squared_d_hat;
        Float D     = 0.0;
        point_triangle_distance(P, T0, T1, T2, D);
        Float E = 0.0;
        sym::KappaBarrier(E, kappa, D, D_hat);
        return E;
    }

    __device__ void PT_barrier_gradient_hessian(Vector12&      G,
                                                Matrix12x12&   H,
                                                Float          kappa,
                                                Float          squared_d_hat,
                                                const Vector3& P,
                                                const Vector3& T0,
                                                const Vector3& T1,
                                                const Vector3& T2)
    {
        using namespace muda::distance;

        Float D_hat = squared_d_hat;
        Float D     = 0.0;
        point_triangle_distance(P, T0, T1, T2, D);

        Vector12 GradD;
        point_triangle_distance_gradient(P, T0, T1, T2, GradD);

        Float dBdD;
        sym::dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);

        //tex:
        //$$
        // G = \frac{\partial E}{\partial D} \frac{\partial D}{\partial x}
        //$$
        G = dBdD * GradD;

        Float ddBddD;
        sym::ddKappaBarrierddD(ddBddD, kappa, D, squared_d_hat);

        Matrix12x12 HessD;
        point_triangle_distance_hessian(P, T0, T1, T2, HessD);

        //tex:
        //$$
        // H = \frac{\partial^2 E}{\partial D^2} \frac{\partial D}{\partial x} \frac{\partial D}{\partial x}^T + \frac{\partial E}{\partial D} \frac{\partial^2 D}{\partial x^2}
        //$$
        H = ddBddD * GradD * GradD.transpose() + dBdD * HessD;
    }

    __device__ Float EE_barrier_energy(Float          kappa,
                                       Float          squared_d_hat,
                                       const Vector3& Ea0,
                                       const Vector3& Ea1,
                                       const Vector3& Eb0,
                                       const Vector3& Eb1)
    {
        using namespace muda::distance;
        Float D_hat = squared_d_hat;
        Float D     = 0.0;
        edge_edge_distance(Ea0, Ea1, Eb0, Eb1, D);
        Float E = 0.0;
        sym::KappaBarrier(E, kappa, D, D_hat);
        return E;
    }


    __device__ void EE_barrier_gradient_hessian(Vector12&      G,
                                                Matrix12x12&   H,
                                                Float          kappa,
                                                Float          squared_d_hat,
                                                const Vector3& Ea0,
                                                const Vector3& Ea1,
                                                const Vector3& Eb0,
                                                const Vector3& Eb1)
    {
        using namespace muda::distance;

        Float D_hat = squared_d_hat;
        Float D     = 0.0;
        edge_edge_distance(Ea0, Ea1, Eb0, Eb1, D);

        Vector12 GradD;
        edge_edge_distance_gradient(Ea0, Ea1, Eb0, Eb1, GradD);

        Float dBdD;
        sym::dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);

        //tex:
        //$$
        // G = \frac{\partial E}{\partial D} \frac{\partial D}{\partial x}
        //$$
        G = dBdD * GradD;

        Float ddBddD;
        sym::ddKappaBarrierddD(ddBddD, kappa, D, squared_d_hat);

        Matrix12x12 HessD;
        edge_edge_distance_hessian(Ea0, Ea1, Eb0, Eb1, HessD);

        //tex:
        //$$
        // H = \frac{\partial^2 E}{\partial D^2} \frac{\partial D}{\partial x} \frac{\partial D}{\partial x}^T + \frac{\partial E}{\partial D} \frac{\partial^2 D}{\partial x^2}
        //$$
        H = ddBddD * GradD * GradD.transpose() + dBdD * HessD;
    }

    __device__ Float PE_barrier_energy(Float          kappa,
                                       Float          squared_d_hat,
                                       const Vector3& P,
                                       const Vector3& E0,
                                       const Vector3& E1)
    {
        using namespace muda::distance;
        Float D_hat = squared_d_hat;
        Float D     = 0.0;
        point_edge_distance(P, E0, E1, D);
        Float E = 0.0;
        sym::KappaBarrier(E, kappa, D, D_hat);
        return E;
    }

    __device__ void PE_barrier_gradient_hessian(Vector9&       G,
                                                Matrix9x9&     H,
                                                Float          kappa,
                                                Float          squared_d_hat,
                                                const Vector3& P,
                                                const Vector3& E0,
                                                const Vector3& E1)
    {
        using namespace muda::distance;

        Float D_hat = squared_d_hat;
        Float D     = 0.0;
        point_edge_distance(P, E0, E1, D);

        Vector9 GradD;
        point_edge_distance_gradient(P, E0, E1, GradD);

        Float dBdD;
        sym::dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);

        //tex:
        //$$
        // G = \frac{\partial E}{\partial D} \frac{\partial D}{\partial x}
        //$$
        G = dBdD * GradD;

        Float ddBddD;
        sym::ddKappaBarrierddD(ddBddD, kappa, D, squared_d_hat);

        Matrix9x9 HessD;
        point_edge_distance_hessian(P, E0, E1, HessD);

        //tex:
        //$$
        // H = \frac{\partial^2 E}{\partial D^2} \frac{\partial D}{\partial x} \frac{\partial D}{\partial x}^T + \frac{\partial E}{\partial D} \frac{\partial^2 D}{\partial x^2}
        //$$
        H = ddBddD * GradD * GradD.transpose() + dBdD * HessD;
    }

    __device__ Float PP_barrier_energy(Float          kappa,
                                       Float          squared_d_hat,
                                       const Vector3& P0,
                                       const Vector3& P1)
    {
        using namespace muda::distance;
        Float D_hat = squared_d_hat;
        Float D     = 0.0;
        point_point_distance(P0, P1, D);
        Float E = 0.0;
        sym::KappaBarrier(E, kappa, D, D_hat);
        return E;
    }

    __device__ void PP_barrier_gradient_hessian(Vector6&       G,
                                                Matrix6x6&     H,
                                                Float          kappa,
                                                Float          squared_d_hat,
                                                const Vector3& P0,
                                                const Vector3& P1)
    {
        using namespace muda::distance;

        Float D_hat = squared_d_hat;
        Float D     = 0.0;
        point_point_distance(P0, P1, D);

        Vector6 GradD;
        point_point_distance_gradient(P0, P1, GradD);

        Float dBdD;
        sym::dKappaBarrierdD(dBdD, kappa, D, squared_d_hat);

        //tex:
        //$$
        // G = \frac{\partial E}{\partial D} \frac{\partial D}{\partial x}
        //$$
        G = dBdD * GradD;

        Float ddBddD;
        sym::ddKappaBarrierddD(ddBddD, kappa, D, squared_d_hat);

        Matrix6x6 HessD;
        point_point_distance_hessian(P0, P1, HessD);

        //tex:
        //$$
        // H = \frac{\partial^2 E}{\partial D^2} \frac{\partial D}{\partial x} \frac{\partial D}{\partial x}^T + \frac{\partial E}{\partial D} \frac{\partial^2 D}{\partial x^2}
        //$$
        H = ddBddD * GradD * GradD.transpose() + dBdD * HessD;
    }
}  // namespace ipc_contact


void IPCSimplexContact::do_compute_energy(EnergyInfo& info)
{
    using namespace muda;

    // Compute Point-Triangle energy
    ParallelFor()
        .kernel_name(__FUNCTION__ "-PT")
        .apply(PTs().size(),
               [PTs = PTs().viewer().name("PTs")] __device__(int i) {

               });
}

void IPCSimplexContact::do_assemble(ContactInfo& info) {}
}  // namespace uipc::backend::cuda