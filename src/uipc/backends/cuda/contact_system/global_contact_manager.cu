#include "hip/hip_runtime.h"
#include <contact_system/global_contact_manager.h>
#include <sim_engine.h>
namespace uipc::backend::cuda
{
template <>
class SimSystemCreator<GlobalContactManager>
{
  public:
    static U<GlobalContactManager> create(SimEngine& engine)
    {
        auto& info = engine.world().scene().info();

        return info["contact"]["enable"].get<bool>() ?
                   make_unique<GlobalContactManager>(engine) :
                   nullptr;
    }
};

REGISTER_SIM_SYSTEM(GlobalContactManager);

void GlobalContactManager::do_build()
{
    m_impl.global_vertex_manager = find<GlobalVertexManager>();
    const auto& info             = world().scene().info();
    m_impl.related_d_hat         = info["contact"]["d_hat"].get<Float>();
}

void GlobalContactManager::Impl::update_contact_parameters()
{
    AABB vert_aabb = global_vertex_manager->vertex_bounding_box();
    d_hat          = related_d_hat * vert_aabb.diagonal().norm();
}

void GlobalContactManager::update_contact_parameters()
{
    m_impl.update_contact_parameters();
}

Float GlobalContactManager::d_hat() const
{
    return m_impl.d_hat;
}
}  // namespace uipc::backend::cuda