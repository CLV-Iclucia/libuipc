#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <uipc/backends/module.h>
#include <uipc/common/log.h>
#include <muda/muda.h>
#include <kernel_cout.h>
#include <sim_engine_device_common.h>
#include <log_pattern_guard.h>
#include <sim_system_collection.h>

namespace uipc::backend::cuda
{
void say_hello_from_muda()
{
    using namespace muda;

    Launch()
        .apply([] __device__()
               { cout << "Hello from CudaEngine CUDA Kernel!\n"; })
        .wait();
}

SimEngine::SimEngine()
    : m_device_common(std::make_unique<DeviceCommon>())
{

    LogGuard guard;

    spdlog::info("Cuda Backend Init Success.");

    using namespace muda;

    if(!cout)
    {
        auto viewer_ptr         = device_logger_viewer_ptr();
        m_device_common->logger = std::make_unique<muda::Logger>(viewer_ptr);

        Debug::set_sync_callback(
            [this]
            {
                m_string_stream.str("");
                m_device_common->logger->retrieve(m_string_stream);
                if(m_string_stream.str().empty())
                    return;

                std::string str = m_string_stream.str();
                spdlog::info(R"([Kernel Console] 
-------------------------------------------------------------------------------
{}
-------------------------------------------------------------------------------)",
                             str);
            });
    }

    say_hello_from_muda();
}

SimEngine::~SimEngine()
{
    LogGuard guard;

    muda::wait_device();

    // remove the sync callback
    muda::Debug::set_sync_callback(nullptr);

    spdlog::info("Cuda Backend Shutdown Success.");
}
}  // namespace uipc::backend::cuda
