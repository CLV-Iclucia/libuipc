#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <uipc/backends/module.h>
#include <uipc/common/log.h>
#include <muda/muda.h>
#include <kernel_cout.h>
#include <sim_engine_device_common.h>
#include <log_pattern_guard.h>
#include <sim_system_collection.h>

namespace uipc::backend::cuda
{
void say_hello_from_muda()
{
    using namespace muda;

    Launch()
        .apply([] __device__()
               { cout << "CUDA Backend Kernel Console Init Success!\n"; })
        .wait();
}

SimEngine::SimEngine()
    : m_device_impl(make_unique<DeviceImpl>())
{
    LogGuard guard;

    spdlog::info("Cuda Backend Init Success.");

    using namespace muda;

    auto viewer_ptr       = device_logger_viewer_ptr();
    m_device_impl->logger = make_unique<muda::Logger>(viewer_ptr);

    Debug::set_sync_callback(
        [this]
        {
            m_string_stream.str("");
            m_device_impl->logger->retrieve(m_string_stream);
            if(m_string_stream.str().empty())
                return;

            spdlog::info(R"( 
-------------------------------------------------------------------------------
*                               Kernel  Console                               *
-------------------------------------------------------------------------------
{}
-------------------------------------------------------------------------------)",
                         m_string_stream.str());
        });

    say_hello_from_muda();

#ifndef NDEBUG
    // if in debug mode, sync all the time to check for errors
    muda::Debug::debug_sync_all(true);
#endif
}

SimEngine::~SimEngine()
{
    LogGuard guard;

    muda::wait_device();

    // remove the sync callback
    muda::Debug::set_sync_callback(nullptr);

    spdlog::info("Cuda Backend Shutdown Success.");
}

auto SimEngine::device_impl() noexcept -> DeviceImpl&
{
    return *m_device_impl;
}
}  // namespace uipc::backend::cuda
