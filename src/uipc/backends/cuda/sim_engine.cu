#include "hip/hip_runtime.h"
#include <uipc/backends/cuda/sim_engine.h>
#include <uipc/backends/module.h>
#include <uipc/common/log.h>
#include <muda/muda.h>
#include <uipc/backends/cuda/kernel_cout.h>
#include <uipc/backends/cuda/sim_engine_device_common.h>
#include <uipc/backends/cuda/log_pattern_guard.h>
#include <uipc/backends/cuda/sim_system_collection.h>

namespace uipc::backend::cuda
{
void say_hello_from_muda()
{
    using namespace muda;

    Launch()
        .apply([] __device__()
               { cout << "Hello from CudaEngine CUDA Kernel!\n"; })
        .wait();
}

SimEngine::SimEngine()
    : m_device_common(std::make_unique<DeviceCommon>())
{

    LogGuard guard;

    spdlog::info("Cuda Backend Init Success.");

    using namespace muda;

    if(!cout)
    {
        auto viewer_ptr         = device_logger_viewer_ptr();
        m_device_common->logger = std::make_unique<muda::Logger>(viewer_ptr);

        Debug::set_sync_callback(
            [this]
            {
                m_string_stream.str("");
                m_device_common->logger->retrieve(m_string_stream);
                if(m_string_stream.str().empty())
                    return;

                std::string str = m_string_stream.str();
                spdlog::info(R"([Kernel Console] 
-------------------------------------------------------------------------------
{}
-------------------------------------------------------------------------------)",
                             str);
            });
    }

    say_hello_from_muda();
}

auto SimEngine::device_common() noexcept -> DeviceCommon&
{
    return *m_device_common;
}

WorldVisitor& SimEngine::world() noexcept
{
    UIPC_ASSERT(m_world_visitor, "WorldVisitor is not initialized.");
    return *m_world_visitor;
}

SimEngine::~SimEngine()
{
    LogGuard guard;

    muda::wait_device();

    // remove the sync callback
    muda::Debug::set_sync_callback(nullptr);
    cout = {};

    spdlog::info("Cuda Backend Shutdown Success.");
}
}  // namespace uipc::backend::cuda
