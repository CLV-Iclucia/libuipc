#include <collision_detection/collision_classifier.h>
#include <sim_engine.h>
#include <muda/ext/geo/distance.h>
#include <muda/cub/device/device_select.h>

namespace uipc::backend::cuda
{
template <>
class SimSystemCreator<CollisionClassifier>
{
  public:
    static U<CollisionClassifier> create(SimEngine& engine)
    {
        auto& info = engine.world().scene().info();

        return info["contact"]["enable"].get<bool>() ?
                   make_unique<CollisionClassifier>(engine) :
                   nullptr;
    }
};

REGISTER_SIM_SYSTEM(CollisionClassifier);

void CollisionClassifier::do_build()
{
    auto global_collision_detector = find<GlobalCollisionDetector>();
    global_collision_detector->add_classifier(
        [this](GlobalCollisionDetector::ClassifyCandidateInfo& info)
        { m_impl.classify_candidates(info); });

    m_impl.global_vertex_manager  = find<GlobalVertexManager>();
    m_impl.global_contact_manager = find<GlobalContactManager>();
}

void CollisionClassifier::Impl::classify_candidates(GlobalCollisionDetector::ClassifyCandidateInfo& info)
{
    using namespace muda;
    auto d_hat         = global_contact_manager->d_hat();
    auto Ps            = global_vertex_manager->positions();
    auto candidate_PTs = info.candidate_PTs();
    auto candidate_EEs = info.candidate_EEs();

    temp_PTs.resize(candidate_PTs.size(), Vector4i::Ones() * -1);
    PTs.resize(candidate_PTs.size());
    temp_EEs.resize(candidate_EEs.size(), Vector4i::Ones() * -1);
    EEs.resize(candidate_EEs.size());

    // PE:
    SizeT max_PE_count = candidate_PTs.size() + candidate_EEs.size();
    temp_PEs.resize(max_PE_count, Vector3i::Ones() * -1);
    PEs.resize(max_PE_count);
    // PP:
    SizeT max_PP_count = candidate_PTs.size() + candidate_EEs.size();
    temp_PPs.resize(max_PP_count, Vector2i::Ones() * -1);
    PPs.resize(max_PP_count);  // reserve enough space for PTs


    auto PE_offset = 0;
    auto PE_count  = candidate_PTs.size();
    auto PP_offset = 0;
    auto PP_count  = candidate_PTs.size();

    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(candidate_PTs.size(),
               [Ps            = Ps.viewer().name("Ps"),
                candidate_PTs = candidate_PTs.viewer().name("candidate_PTs"),
                PTs           = temp_PTs.viewer().name("PTs"),
                PEs = temp_PEs.view(PE_offset, PE_count).viewer().name("PEs"),
                PPs = temp_PPs.view(PP_offset, PP_count).viewer().name("PPs")] __device__(int i) mutable
               {
                   Vector4i PT = candidate_PTs(i);
                   IndexT   P  = PT(0);
                   Vector3  T  = PT.tail<3>();


                   auto V  = Ps(PT(0));
                   auto F0 = Ps(PT(1));
                   auto F1 = Ps(PT(2));
                   auto F2 = Ps(PT(3));

                   auto dist_type = distance::point_triangle_distance_type(V, F0, F1, F2);

                   switch(dist_type)
                   {
                       case muda::distance::PointTriangleDistanceType::PP_PT0:
                           PPs(i) = Vector2i(P, T(0));
                           break;
                       case muda::distance::PointTriangleDistanceType::PP_PT1:
                           PPs(i) = Vector2i(P, T(1));
                           break;
                       case muda::distance::PointTriangleDistanceType::PP_PT2:
                           PPs(i) = Vector2i(P, T(2));
                           break;
                       case muda::distance::PointTriangleDistanceType::PE_PT0T1:
                           PEs(i) = Vector3i(P, T(0), T(1));
                           break;
                       case muda::distance::PointTriangleDistanceType::PE_PT1T2:
                           PEs(i) = Vector3i(P, T(1), T(2));
                           break;
                       case muda::distance::PointTriangleDistanceType::PE_PT2T0:
                           PEs(i) = Vector3i(P, T(2), T(0));
                           break;
                       case muda::distance::PointTriangleDistanceType::PT:
                           PTs(i) = PT;
                           break;
                       default:
                           break;
                   }
               });

    PE_offset = PE_count;
    PE_count  = candidate_EEs.size();
    PP_offset = PP_count;
    PP_count  = candidate_EEs.size();

    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(candidate_EEs.size(),
               [Ps            = Ps.viewer().name("Ps"),
                candidate_EEs = candidate_EEs.viewer().name("candidate_EEs"),
                EEs           = temp_EEs.viewer().name("EEs"),
                PEs = temp_PEs.view(PE_offset, PE_count).viewer().name("PEs"),
                PPs = temp_PPs.view(PP_offset, PP_count).viewer().name("PPs")] __device__(int i) mutable
               {
                   Vector4i EE = candidate_EEs(i);

                   IndexT Ea0 = EE(0);
                   IndexT Ea1 = EE(1);
                   IndexT Eb0 = EE(2);
                   IndexT Eb1 = EE(3);

                   Vector3 E0 = Ps(Ea0);
                   Vector3 E1 = Ps(Ea1);
                   Vector3 E2 = Ps(Eb0);
                   Vector3 E3 = Ps(Eb1);

                   auto dist_type = distance::edge_edge_distance_type(E0, E1, E2, E3);

                   switch(dist_type)
                   {
                       case muda::distance::EdgeEdgeDistanceType::PP_Ea0Eb0:
                           PPs(i) = Vector2i(Ea0, Eb0);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::PP_Ea0Eb1:
                           PPs(i) = Vector2i(Ea0, Eb1);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::PE_Ea0Eb0Eb1:
                           PEs(i) = Vector3i(Ea0, Eb0, Eb1);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::PP_Ea1Eb0:
                           PPs(i) = Vector2i(Ea1, Eb0);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::PP_Ea1Eb1:
                           PPs(i) = Vector2i(Ea1, Eb1);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::PE_Ea1Eb0Eb1:
                           PEs(i) = Vector3i(Ea1, Eb0, Eb1);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::PE_Eb0Ea0Ea1:
                           PEs(i) = Vector3i(Eb0, Ea0, Ea1);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::PE_Eb1Ea0Ea1:
                           PEs(i) = Vector3i(Eb1, Ea0, Ea1);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::EE:
                           EEs(i) = EE;
                           break;
                       default:
                           break;
                   }
               });

    DeviceSelect().If(temp_PTs.data(),
                      PTs.data(),
                      selected.data(),
                      temp_PTs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector4i& pt)
                      { return pt(0) != -1; });

    int h_selected = selected;
    PTs.resize(h_selected);

    DeviceSelect().If(temp_EEs.data(),
                      EEs.data(),
                      selected.data(),
                      temp_EEs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector4i& ee)
                      { return ee(0) != -1; });

    h_selected = selected;
    EEs.resize(h_selected);

    DeviceSelect().If(temp_PEs.data(),
                      PEs.data(),
                      selected.data(),
                      temp_PEs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector3i& pe)
                      { return pe(0) != -1; });

    h_selected = selected;
    PEs.resize(h_selected);

    DeviceSelect().If(temp_PPs.data(),
                      PPs.data(),
                      selected.data(),
                      temp_PPs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector2i& pp)
                      { return pp(0) != -1; });

    h_selected = selected;
    PPs.resize(h_selected);

    info.PTs(PTs);
    info.EEs(EEs);
    info.PEs(PEs);
    info.PPs(PPs);
}
}  // namespace uipc::backend::cuda
