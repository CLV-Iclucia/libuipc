#include "hip/hip_runtime.h"
#include <collision_detection/filters/lbvh_simplex_trajectory_filter.h>
#include <muda/cub/device/device_select.h>
#include <muda/ext/eigen/log_proxy.h>
#include <sim_engine.h>
#include <kernel_cout.h>
#include <utils/distance/distance_flagged.h>
#include <utils/distance.h>
#include <utils/codim_thickness.h>
#include <uipc/common/zip.h>

namespace uipc::backend::cuda
{
constexpr bool PrintDebugInfo = false;

REGISTER_SIM_SYSTEM(LBVHSimplexTrajectoryFilter);

void LBVHSimplexTrajectoryFilter::do_detect(DetectInfo& info)
{
    m_impl.detect(info);
}

void LBVHSimplexTrajectoryFilter::do_filter_active(FilterActiveInfo& info)
{
    m_impl.filter_active(info);
}

void LBVHSimplexTrajectoryFilter::do_filter_toi(FilterTOIInfo& info)
{
    m_impl.filter_toi(info);
}

void LBVHSimplexTrajectoryFilter::Impl::detect(DetectInfo& info)
{
    using namespace muda;

    auto alpha   = info.alpha();
    auto d_hat   = info.d_hat();
    auto Ps      = info.positions();
    auto dxs     = info.displacements();
    auto codimVs = info.codim_vertices();
    auto Vs      = info.surf_vertices();
    auto Es      = info.surf_edges();
    auto Fs      = info.surf_triangles();

    codim_point_aabbs.resize(codimVs.size());
    point_aabbs.resize(Vs.size());
    triangle_aabbs.resize(Fs.size());
    edge_aabbs.resize(Es.size());

    // build AABBs for codim vertices
    ParallelFor()
        .file_line(__FILE__, __LINE__)
        .apply(codimVs.size(),
               [codimVs     = codimVs.viewer().name("codimVs"),
                Ps          = Ps.viewer().name("Ps"),
                dxs         = dxs.viewer().name("dxs"),
                aabbs       = codim_point_aabbs.viewer().name("aabbs"),
                thicknesses = info.thicknesses().viewer().name("thicknesses"),
                alpha       = alpha,
                d_hat       = d_hat] __device__(int i) mutable
               {
                   auto vI = codimVs(i);

                   Float thickness = thicknesses(vI);

                   const auto& pos   = Ps(vI);
                   Vector3     pos_t = pos + dxs(vI) * alpha;

                   AABB aabb;
                   aabb.extend(pos).extend(pos_t);

                   Float expand = d_hat + thickness;

                   aabb.min().array() -= expand;
                   aabb.max().array() += expand;
                   aabbs(i) = aabb;
               });


    // build AABBs for surf vertices (including codim vertices)
    ParallelFor()
        .file_line(__FILE__, __LINE__)
        .apply(Vs.size(),
               [Vs          = Vs.viewer().name("V"),
                dxs         = dxs.viewer().name("dx"),
                Ps          = Ps.viewer().name("Ps"),
                aabbs       = point_aabbs.viewer().name("aabbs"),
                thicknesses = info.thicknesses().viewer().name("thicknesses"),
                alpha       = alpha,
                d_hat       = d_hat] __device__(int i) mutable
               {
                   auto vI = Vs(i);

                   Float thickness = thicknesses(vI);

                   const auto& pos   = Ps(vI);
                   Vector3     pos_t = pos + dxs(vI) * alpha;

                   AABB aabb;
                   aabb.extend(pos).extend(pos_t);

                   Float expand = d_hat + thickness;

                   aabb.min().array() -= expand;
                   aabb.max().array() += expand;
                   aabbs(i) = aabb;
               });

    // build AABBs for edges
    ParallelFor()
        .file_line(__FILE__, __LINE__)
        .apply(Es.size(),
               [Es          = Es.viewer().name("E"),
                Ps          = Ps.viewer().name("Ps"),
                aabbs       = edge_aabbs.viewer().name("aabbs"),
                dxs         = dxs.viewer().name("dx"),
                thicknesses = info.thicknesses().viewer().name("thicknesses"),
                alpha       = alpha,
                d_hat       = d_hat] __device__(int i) mutable
               {
                   auto eI = Es(i);

                   Float thickness =
                       edge_thickness(thicknesses(eI[0]), thicknesses(eI[1]));

                   const auto& pos0   = Ps(eI[0]);
                   const auto& pos1   = Ps(eI[1]);
                   Vector3     pos0_t = pos0 + dxs(eI[0]) * alpha;
                   Vector3     pos1_t = pos1 + dxs(eI[1]) * alpha;

                   Vector3 max = pos0_t;
                   Vector3 min = pos0_t;

                   AABB aabb;

                   aabb.extend(pos0).extend(pos1).extend(pos0_t).extend(pos1_t);

                   Float expand = d_hat + thickness;

                   aabb.min().array() -= expand;
                   aabb.max().array() += expand;
                   aabbs(i) = aabb;
               });

    // build AABBs for triangles
    ParallelFor()
        .file_line(__FILE__, __LINE__)
        .apply(Fs.size(),
               [Fs          = Fs.viewer().name("F"),
                Ps          = Ps.viewer().name("Ps"),
                aabbs       = triangle_aabbs.viewer().name("aabbs"),
                dxs         = dxs.viewer().name("dx"),
                thicknesses = info.thicknesses().viewer().name("thicknesses"),
                alpha       = alpha,
                d_hat       = d_hat] __device__(int i) mutable
               {
                   auto fI = Fs(i);

                   Float thickness = triangle_thickness(thicknesses(fI[0]),
                                                        thicknesses(fI[1]),
                                                        thicknesses(fI[2]));

                   const auto& pos0   = Ps(fI[0]);
                   const auto& pos1   = Ps(fI[1]);
                   const auto& pos2   = Ps(fI[2]);
                   Vector3     pos0_t = pos0 + dxs(fI[0]) * alpha;
                   Vector3     pos1_t = pos1 + dxs(fI[1]) * alpha;
                   Vector3     pos2_t = pos2 + dxs(fI[2]) * alpha;

                   AABB aabb;

                   aabb.extend(pos0)
                       .extend(pos1)
                       .extend(pos2)
                       .extend(pos0_t)
                       .extend(pos1_t)
                       .extend(pos2_t);

                   Float expand = d_hat + thickness;

                   aabb.min().array() -= expand;
                   aabb.max().array() += expand;
                   aabbs(i) = aabb;
               });

    // query CodimP and P
    {
        lbvh_PP.build(point_aabbs);
        muda::KernelLabel label{__FUNCTION__, __FILE__, __LINE__};
        candidate_PP_pairs = lbvh_PP.query(
            codim_point_aabbs,
            [codimVs     = codimVs.viewer().name("codimVs"),
             Vs          = Vs.viewer().name("Vs"),
             Ps          = Ps.viewer().name("Ps"),
             dxs         = dxs.viewer().name("dxs"),
             thicknesses = info.thicknesses().viewer().name("thicknesses"),
             d_hat       = d_hat,
             alpha       = alpha] __device__(IndexT i, IndexT j)
            {
                const auto& codimV = codimVs(i);
                const auto& V      = Vs(j);

                if(codimV >= V)  // avoid duplicate
                    return false;

                Vector3 P0  = Ps(codimV);
                Vector3 P1  = Ps(V);
                Vector3 dP0 = alpha * dxs(codimV);
                Vector3 dP1 = alpha * dxs(V);

                Float thickness = PP_thickness(thicknesses(codimV), thicknesses(V));

                Float expand = d_hat + thickness;

                if(!distance::point_point_ccd_broadphase(P0, P1, dP0, dP1, expand))
                    return false;

                return true;
            });
    }


    // query PE
    {
        lbvh_PE.build(edge_aabbs);
        muda::KernelLabel label{__FUNCTION__, __FILE__, __LINE__};
        candidate_PE_pairs = lbvh_PE.query(
            codim_point_aabbs,
            [codimVs     = codimVs.viewer().name("Vs"),
             Es          = Es.viewer().name("Es"),
             Ps          = Ps.viewer().name("Ps"),
             dxs         = dxs.viewer().name("dxs"),
             thicknesses = info.thicknesses().viewer().name("thicknesses"),
             d_hat       = d_hat,
             alpha       = alpha] __device__(IndexT i, IndexT j)
            {
                const auto& codimV = codimVs(i);
                const auto& E      = Es(j);

                MUDA_ASSERT(E[0] != codimV && E[1] != codimV,
                            "Edge (%d,%d) contains codim vertex (%d), why can it happen?",
                            E[0],
                            E[1],
                            codimV);

                Vector3 E0  = Ps(E[0]);
                Vector3 E1  = Ps(E[1]);
                Vector3 dE0 = alpha * dxs(E[0]);
                Vector3 dE1 = alpha * dxs(E[1]);

                Vector3 P  = Ps(codimV);
                Vector3 dP = alpha * dxs(codimV);

                Float thickness = PE_thickness(
                    thicknesses(codimV), thicknesses(E[0]), thicknesses(E[1]));

                Float expand = d_hat + thickness;

                if(!distance::point_edge_ccd_broadphase(P, E0, E1, dP, dE0, dE1, expand))
                    return false;

                return true;
            });
    }


    // query PT
    {
        lbvh_PT.build(triangle_aabbs);
        muda::KernelLabel label{__FUNCTION__, __FILE__, __LINE__};
        candidate_PT_pairs = lbvh_PT.query(
            point_aabbs,
            [Vs          = Vs.viewer().name("Vs"),
             Fs          = Fs.viewer().name("Fs"),
             Ps          = Ps.viewer().name("Ps"),
             dxs         = dxs.viewer().name("dxs"),
             thicknesses = info.thicknesses().viewer().name("thicknesses"),
             d_hat       = d_hat,
             alpha       = alpha] __device__(IndexT i, IndexT j)
            {
                // discard if the point is on the triangle
                auto V = Vs(i);
                auto F = Fs(j);

                if(F[0] == V || F[1] == V || F[2] == V)
                    return false;

                Vector3 P  = Ps(V);
                Vector3 dP = alpha * dxs(V);

                Vector3 F0 = Ps(F[0]);
                Vector3 F1 = Ps(F[1]);
                Vector3 F2 = Ps(F[2]);

                Vector3 dF0 = alpha * dxs(F[0]);
                Vector3 dF1 = alpha * dxs(F[1]);
                Vector3 dF2 = alpha * dxs(F[2]);

                Float thickness = triangle_thickness(
                    thicknesses(F[0]), thicknesses(F[1]), thicknesses(F[2]));

                Float expand = d_hat + thickness;

                if(!distance::point_triangle_ccd_broadphase(P, F0, F1, F2, dP, dF0, dF1, dF2, expand))
                    return false;

                return true;
            });
    }

    // query EE
    lbvh_EE.build(edge_aabbs);
    {
        muda::KernelLabel label{__FUNCTION__, __FILE__, __LINE__};
        candidate_EE_pairs = lbvh_EE.detect(
            [Es          = Es.viewer().name("Es"),
             Ps          = Ps.viewer().name("Ps"),
             dxs         = dxs.viewer().name("dxs"),
             thicknesses = info.thicknesses().viewer().name("thicknesses"),
             d_hat       = d_hat,
             alpha       = alpha] __device__(IndexT i, IndexT j)
            {
                // discard if the edges shared same vertex
                auto Ea = Es(i);
                auto Eb = Es(j);

                if(Ea[0] == Eb[0] || Ea[0] == Eb[1] || Ea[1] == Eb[0] || Ea[1] == Eb[1])
                    return false;

                Vector3 Ea0  = Ps(Ea[0]);
                Vector3 Ea1  = Ps(Ea[1]);
                Vector3 dEa0 = alpha * dxs(Ea[0]);
                Vector3 dEa1 = alpha * dxs(Ea[1]);

                Vector3 Eb0  = Ps(Eb[0]);
                Vector3 Eb1  = Ps(Eb[1]);
                Vector3 dEb0 = alpha * dxs(Eb[0]);
                Vector3 dEb1 = alpha * dxs(Eb[1]);

                Float thickness = EE_thickness(thicknesses(Ea[0]),
                                               thicknesses(Ea[1]),
                                               thicknesses(Eb[0]),
                                               thicknesses(Eb[1]));

                Float expand = d_hat + thickness;

                if(!distance::edge_edge_ccd_broadphase(
                       // position
                       Ea0,
                       Ea1,
                       Eb0,
                       Eb1,
                       // displacement
                       dEa0,
                       dEa1,
                       dEb0,
                       dEb1,
                       expand))
                    return false;

                return true;
            });
    }
}

void LBVHSimplexTrajectoryFilter::Impl::filter_active(FilterActiveInfo& info)
{
    using namespace muda;

    // we will filter-out the active pairs

    auto d_hat     = info.d_hat();
    auto positions = info.positions();

    SizeT N_PPs = candidate_PP_pairs.size();
    SizeT N_PEs = candidate_PE_pairs.size();
    SizeT N_PTs = candidate_PT_pairs.size();
    SizeT N_EEs = candidate_EE_pairs.size();

    // PT, EE, PT, PP can degenerate to PP
    temp_PPs.resize(N_PPs + N_PEs + N_PTs + N_EEs);
    // PT, EE, PT can degenerate to PE
    temp_PEs.resize(N_PEs + N_PTs + N_EEs);

    temp_PTs.resize(N_PTs);
    temp_EEs.resize(N_EEs);

    SizeT temp_PP_offset = 0;
    SizeT temp_PE_offset = 0;

    // PPs
    {
        auto PP_view = temp_PPs.view(temp_PP_offset, N_PPs);

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(candidate_PP_pairs.size(),
                   [positions = positions.viewer().name("positions"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    PP_pairs = candidate_PP_pairs.viewer().name("PP_pairs"),
                    temp_PPs = PP_view.viewer().name("temp_PPs"),
                    d_hat    = d_hat] __device__(int i) mutable
                   {
                       // default invalid
                       auto& PP = temp_PPs(i);
                       PP.setConstant(-1);

                       Vector2i indices = PP_pairs(i);

                       IndexT P0 = surf_vertices(indices(0));
                       IndexT P1 = surf_vertices(indices(1));


                       const auto& V0 = positions(P0);
                       const auto& V1 = positions(P1);

                       Float thickness = PP_thickness(thicknesses(P0), thicknesses(P1));

                       Vector2 range = D_range(thickness, d_hat);

                       Float D;
                       distance::point_point_distance2(V0, V1, D);


                       if(!is_active_D(range, D))
                           return;  // early return

                       PP = {P0, P1};
                   });

        temp_PP_offset += N_PPs;
    }
    // PEs
    {
        auto PP_view = temp_PPs.view(temp_PP_offset, N_PEs);
        auto PE_view = temp_PEs.view(temp_PE_offset, N_PEs);

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(candidate_PE_pairs.size(),
                   [positions = positions.viewer().name("positions"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    surf_edges = info.surf_edges().viewer().name("surf_edges"),
                    PE_pairs   = candidate_PE_pairs.viewer().name("PE_pairs"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    temp_PPs = PP_view.viewer().name("temp_PPs"),
                    temp_PEs = PE_view.viewer().name("temp_PEs"),
                    d_hat    = d_hat] __device__(int i) mutable
                   {
                       auto& PP = temp_PPs(i);
                       PP.setConstant(-1);
                       auto& PE = temp_PEs(i);
                       PE.setConstant(-1);

                       Vector2i indices = PE_pairs(i);
                       IndexT   V       = surf_vertices(indices(0));
                       Vector2i E       = surf_edges(indices(1));

                       Vector3i vIs  = {V, E(0), E(1)};
                       Vector3  Ps[] = {positions(vIs(0)),
                                        positions(vIs(1)),
                                        positions(vIs(2))};

                       Float thickness = PE_thickness(
                           thicknesses(V), thicknesses(E(0)), thicknesses(E(1)));

                       Vector3i flag =
                           distance::point_edge_distance_flag(Ps[0], Ps[1], Ps[2]);

                       Vector2 range = D_range(thickness, d_hat);

                       Float D;
                       distance::point_edge_distance2(flag, Ps[0], Ps[1], Ps[2], D);

                       if(!is_active_D(range, D))
                           return;  // early return

                       Vector3i offsets;
                       auto dim = distance::degenerate_point_edge(flag, offsets);

                       switch(dim)
                       {
                           case 2:  // PP
                           {
                               IndexT V0 = vIs(offsets(0));
                               IndexT V1 = vIs(offsets(1));
                               PP        = {V0, V1};
                           }
                           break;
                           case 3:  // PE
                           {
                               PE = vIs;
                           }
                           break;
                           default: {
                               MUDA_ERROR_WITH_LOCATION("unexpected degenerate case dim=%d", dim);
                           }
                           break;
                       }
                   });

        temp_PP_offset += N_PEs;
        temp_PE_offset += N_PEs;
    }
    // PTs
    {
        auto PP_view = temp_PPs.view(temp_PP_offset, N_PTs);
        auto PE_view = temp_PEs.view(temp_PE_offset, N_PTs);

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(candidate_PT_pairs.size(),
                   [positions = positions.viewer().name("Ps"),
                    PT_pairs  = candidate_PT_pairs.viewer().name("PT_pairs"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    surf_triangles = info.surf_triangles().viewer().name("surf_triangles"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    temp_PPs = PP_view.viewer().name("temp_PPs"),
                    temp_PEs = PE_view.viewer().name("temp_PEs"),
                    temp_PTs = temp_PTs.viewer().name("temp_PTs"),
                    d_hat    = d_hat] __device__(int i) mutable
                   {
                       auto& PP = temp_PPs(i);
                       PP.setConstant(-1);
                       auto& PE = temp_PEs(i);
                       PE.setConstant(-1);
                       auto& PT = temp_PTs(i);
                       PT.setConstant(-1);

                       Vector2i indices = PT_pairs(i);
                       IndexT   V       = surf_vertices(indices(0));
                       Vector3i F       = surf_triangles(indices(1));

                       Vector4i vIs  = {V, F(0), F(1), F(2)};
                       Vector3  Ps[] = {positions(vIs(0)),
                                        positions(vIs(1)),
                                        positions(vIs(2)),
                                        positions(vIs(3))};

                       Float thickness = PT_thickness(thicknesses(V),
                                                      thicknesses(F(0)),
                                                      thicknesses(F(1)),
                                                      thicknesses(F(2)));

                       Vector4i flag = distance::point_triangle_distance_flag(
                           Ps[0], Ps[1], Ps[2], Ps[3]);

                       Vector2 range = D_range(thickness, d_hat);

                       Float D;
                       distance::point_triangle_distance2(
                           flag, Ps[0], Ps[1], Ps[2], Ps[3], D);

                       if(!is_active_D(range, D))
                           return;  // early return

                       Vector4i offsets;
                       auto dim = distance::degenerate_point_triangle(flag, offsets);

                       switch(dim)
                       {
                           case 2:  // PP
                           {
                               IndexT V0 = vIs(offsets(0));
                               IndexT V1 = vIs(offsets(1));
                               PP        = {V0, V1};
                           }
                           break;
                           case 3:  // PE
                           {
                               IndexT V0 = vIs(offsets(0));
                               IndexT V1 = vIs(offsets(1));
                               IndexT V2 = vIs(offsets(2));
                               PE        = {V0, V1, V2};
                           }
                           break;
                           case 4:  // PT
                           {
                               PT = vIs;
                           }
                           break;
                           default: {
                               MUDA_ERROR_WITH_LOCATION("unexpected degenerate case dim=%d", dim);
                           }
                           break;
                       }
                   });

        temp_PP_offset += N_PTs;
        temp_PE_offset += N_PTs;
    }
    // EEs
    {
        auto PP_view = temp_PPs.view(temp_PP_offset, N_EEs);
        auto PE_view = temp_PEs.view(temp_PE_offset, N_EEs);


        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(
                candidate_EE_pairs.size(),
                [positions = positions.viewer().name("Ps"),
                 rest_positions = info.rest_positions().viewer().name("rest_positions"),
                 EE_pairs    = candidate_EE_pairs.viewer().name("EE_pairs"),
                 surf_edges  = info.surf_edges().viewer().name("surf_edges"),
                 thicknesses = info.thicknesses().viewer().name("thicknesses"),
                 temp_PPs    = PP_view.viewer().name("temp_PPs"),
                 temp_PEs    = PE_view.viewer().name("temp_PEs"),
                 temp_EEs    = temp_EEs.viewer().name("temp_EEs"),
                 d_hat       = d_hat] __device__(int i) mutable
                {
                    auto& PP = temp_PPs(i);
                    PP.setConstant(-1);
                    auto& PE = temp_PEs(i);
                    PE.setConstant(-1);
                    auto& EE = temp_EEs(i);
                    EE.setConstant(-1);

                    Vector2i indices = EE_pairs(i);
                    Vector2i E0      = surf_edges(indices(0));
                    Vector2i E1      = surf_edges(indices(1));

                    Vector4i vIs  = {E0(0), E0(1), E1(0), E1(1)};
                    Vector3  Ps[] = {positions(vIs(0)),
                                     positions(vIs(1)),
                                     positions(vIs(2)),
                                     positions(vIs(3))};

                    Float thickness = EE_thickness(thicknesses(E0(0)),
                                                   thicknesses(E0(1)),
                                                   thicknesses(E1(0)),
                                                   thicknesses(E1(1)));

                    Vector2 range = D_range(thickness, d_hat);

                    Vector4i flag =
                        distance::edge_edge_distance_flag(Ps[0], Ps[1], Ps[2], Ps[3]);

                    Float D;
                    distance::edge_edge_distance2(flag, Ps[0], Ps[1], Ps[2], Ps[3], D);

                    if(!is_active_D(range, D))
                        return;  // early return

                    Float eps_x;
                    distance::edge_edge_mollifier_threshold(rest_positions(vIs(0)),
                                                            rest_positions(vIs(1)),
                                                            rest_positions(vIs(2)),
                                                            rest_positions(vIs(3)),
                                                            eps_x);

                    if(distance::need_mollify(Ps[0], Ps[1], Ps[2], Ps[3], eps_x))
                    {
                        EE = vIs;
                        return;
                    }
                    else  // classify to EE/PE/PP
                    {
                        Vector4i offsets;
                        auto dim = distance::degenerate_edge_edge(flag, offsets);

                        switch(dim)
                        {
                            case 2:  // PP
                            {
                                IndexT V0 = vIs(offsets(0));
                                IndexT V1 = vIs(offsets(1));
                                PP        = {V0, V1};
                            }
                            break;
                            case 3:  // PE
                            {
                                IndexT V0 = vIs(offsets(0));
                                IndexT V1 = vIs(offsets(1));
                                IndexT V2 = vIs(offsets(2));
                                PE        = {V0, V1, V2};
                            }
                            break;
                            case 4:  // EE
                            {
                                EE = vIs;
                            }
                            break;
                            default: {
                                MUDA_ERROR_WITH_LOCATION("unexpected degenerate case dim=%d", dim);
                            }
                            break;
                        }
                    }
                });

        temp_PP_offset += N_EEs;
        temp_PE_offset += N_EEs;
    }

    UIPC_ASSERT(temp_PP_offset == temp_PPs.size(), "size mismatch");
    UIPC_ASSERT(temp_PE_offset == temp_PEs.size(), "size mismatch");

    {  // select the valid ones
        PPs.resize(temp_PPs.size());
        PEs.resize(temp_PEs.size());
        PTs.resize(temp_PTs.size());
        EEs.resize(temp_EEs.size());

        DeviceSelect().If(temp_PPs.data(),
                          PPs.data(),
                          selected_PP_count.data(),
                          temp_PPs.size(),
                          [] HIPCUB_RUNTIME_FUNCTION(const Vector2i& PP)
                          { return PP(0) != -1; });

        DeviceSelect().If(temp_PEs.data(),
                          PEs.data(),
                          selected_PE_count.data(),
                          temp_PEs.size(),
                          [] HIPCUB_RUNTIME_FUNCTION(const Vector3i& PE)
                          { return PE(0) != -1; });

        DeviceSelect().If(temp_PTs.data(),
                          PTs.data(),
                          selected_PT_count.data(),
                          temp_PTs.size(),
                          [] HIPCUB_RUNTIME_FUNCTION(const Vector4i& PT)
                          { return PT(0) != -1; });

        DeviceSelect().If(temp_EEs.data(),
                          EEs.data(),
                          selected_EE_count.data(),
                          temp_EEs.size(),
                          [] HIPCUB_RUNTIME_FUNCTION(const Vector4i& EE)
                          { return EE(0) != -1; });

        IndexT PP_count = selected_PP_count;
        IndexT PE_count = selected_PE_count;
        IndexT PT_count = selected_PT_count;
        IndexT EE_count = selected_EE_count;

        PPs.resize(PP_count);
        PEs.resize(PE_count);
        PTs.resize(PT_count);
        EEs.resize(EE_count);
    }

    info.PPs(PPs);
    info.PEs(PEs);
    info.PTs(PTs);
    info.EEs(EEs);

    if constexpr(PrintDebugInfo)
    {
        std::vector<Vector2i> PPs_host;
        std::vector<Float>    PP_thicknesses_host;

        std::vector<Vector3i> PEs_host;
        std::vector<Float>    PE_thicknesses_host;

        std::vector<Vector4i> PTs_host;
        std::vector<Float>    PT_thicknesses_host;

        std::vector<Vector4i> EEs_host;
        std::vector<Float>    EE_thicknesses_host;

        PPs.copy_to(PPs_host);
        PEs.copy_to(PEs_host);
        PTs.copy_to(PTs_host);
        EEs.copy_to(EEs_host);

        std::cout << "filter result:" << std::endl;

        for(auto&& [PP, thickness] : zip(PPs_host, PP_thicknesses_host))
        {
            std::cout << "PP: " << PP.transpose() << " thickness: " << thickness << "\n";
        }

        for(auto&& [PE, thickness] : zip(PEs_host, PE_thicknesses_host))
        {
            std::cout << "PE: " << PE.transpose() << " thickness: " << thickness << "\n";
        }

        for(auto&& [PT, thickness] : zip(PTs_host, PT_thicknesses_host))
        {
            std::cout << "PT: " << PT.transpose() << " thickness: " << thickness << "\n";
        }

        for(auto&& [EE, thickness] : zip(EEs_host, EE_thicknesses_host))
        {
            std::cout << "EE: " << EE.transpose() << " thickness: " << thickness << "\n";
        }

        std::cout << std::flush;
    }
}

void LBVHSimplexTrajectoryFilter::Impl::filter_toi(FilterTOIInfo& info)
{
    using namespace muda;

    auto toi_size = candidate_PP_pairs.size() + candidate_PE_pairs.size()
                    + candidate_PT_pairs.size() + candidate_EE_pairs.size();

    tois.resize(toi_size);

    auto offset  = 0;
    auto PP_tois = tois.view(offset, candidate_PP_pairs.size());
    offset += candidate_PP_pairs.size();
    auto PE_tois = tois.view(offset, candidate_PE_pairs.size());
    offset += candidate_PE_pairs.size();
    auto PT_tois = tois.view(offset, candidate_PT_pairs.size());
    offset += candidate_PT_pairs.size();
    auto EE_tois = tois.view(offset, candidate_EE_pairs.size());


    // TODO: Now hard code the minimum separation coefficient
    // gap = eta * (dist2_cur - thickness * thickness) / (dist_cur + thickness);
    constexpr Float eta = 0.1;

    // TODO: Now hard code the maximum iteration
    constexpr SizeT max_iter = 1000;

    // large enough toi (>1)
    constexpr Float large_enough_toi = 1.1;

    // PP
    {
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(candidate_PP_pairs.size(),
                   [PP_tois  = PP_tois.viewer().name("PP_tois"),
                    PP_pairs = candidate_PP_pairs.viewer().name("PP_pairs"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    positions = info.positions().viewer().name("Ps"),
                    dxs       = info.displacements().viewer().name("dxs"),
                    alpha     = info.alpha(),
                    d_hat     = info.d_hat(),
                    eta,
                    max_iter,
                    large_enough_toi] __device__(int i) mutable
                   {
                       auto   indices = PP_pairs(i);
                       IndexT V0      = surf_vertices(indices(0));
                       IndexT V1      = surf_vertices(indices(1));

                       Float thickness = PP_thickness(thicknesses(V0), thicknesses(V1));

                       Vector3 VP0  = positions(V0);
                       Vector3 VP1  = positions(V1);
                       Vector3 dVP0 = alpha * dxs(V0);
                       Vector3 dVP1 = alpha * dxs(V1);

                       Float toi = large_enough_toi;

                       bool faraway = !distance::point_point_ccd_broadphase(
                           VP0, VP1, dVP0, dVP1, d_hat + thickness);

                       if(faraway)
                       {
                           PP_tois(i) = toi;
                           return;
                       }

                       bool hit = distance::point_point_ccd(
                           VP0, VP1, dVP0, dVP1, eta, thickness, max_iter, toi);

                       if(!hit)
                           toi = large_enough_toi;

                       PP_tois(i) = toi;
                   });
    }

    // PE
    {
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(candidate_PE_pairs.size(),
                   [PE_tois  = PE_tois.viewer().name("PE_tois"),
                    PE_pairs = candidate_PE_pairs.viewer().name("PE_pairs"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    surf_edges = info.surf_edges().viewer().name("surf_edges"),
                    Ps         = info.positions().viewer().name("Ps"),
                    dxs        = info.displacements().viewer().name("dxs"),
                    alpha      = info.alpha(),
                    d_hat      = info.d_hat(),
                    eta,
                    max_iter,
                    large_enough_toi] __device__(int i) mutable
                   {
                       auto     indices   = PE_pairs(i);
                       IndexT   V         = surf_vertices(indices(0));
                       Vector2i E         = surf_edges(indices(1));
                       Float    thickness = PE_thickness(
                           thicknesses(V), thicknesses(E(0)), thicknesses(E(1)));

                       Vector3 VP  = Ps(V);
                       Vector3 dVP = alpha * dxs(V);

                       Vector3 EP0  = Ps(E[0]);
                       Vector3 EP1  = Ps(E[1]);
                       Vector3 dEP0 = alpha * dxs(E[0]);
                       Vector3 dEP1 = alpha * dxs(E[1]);

                       Float toi = large_enough_toi;

                       bool faraway = !distance::point_edge_ccd_broadphase(
                           VP, EP0, EP1, dVP, dEP0, dEP1, d_hat + thickness);

                       if(faraway)
                       {
                           PE_tois(i) = toi;
                           return;
                       }

                       bool hit = distance::point_edge_ccd(
                           VP, EP0, EP1, dVP, dEP0, dEP1, eta, thickness, max_iter, toi);

                       if(!hit)
                           toi = large_enough_toi;

                       PE_tois(i) = toi;
                   });
    }

    // PT
    {
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(candidate_PT_pairs.size(),
                   [PT_tois  = PT_tois.viewer().name("PT_tois"),
                    PT_pairs = candidate_PT_pairs.viewer().name("PT_pairs"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    surf_triangles = info.surf_triangles().viewer().name("surf_triangles"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    Ps  = info.positions().viewer().name("Ps"),
                    dxs = info.displacements().viewer().name("dxs"),

                    alpha = info.alpha(),
                    d_hat = info.d_hat(),
                    eta,

                    max_iter,
                    large_enough_toi] __device__(int i) mutable
                   {
                       auto     indices   = PT_pairs(i);
                       IndexT   V         = surf_vertices(indices(0));
                       Vector3i F         = surf_triangles(indices(1));
                       Float    thickness = PT_thickness(thicknesses(V),
                                                      thicknesses(F(0)),
                                                      thicknesses(F(1)),
                                                      thicknesses(F(2)));

                       Vector3 VP  = Ps(V);
                       Vector3 dVP = alpha * dxs(V);

                       Vector3 FP0 = Ps(F[0]);
                       Vector3 FP1 = Ps(F[1]);
                       Vector3 FP2 = Ps(F[2]);

                       Vector3 dFP0 = alpha * dxs(F[0]);
                       Vector3 dFP1 = alpha * dxs(F[1]);
                       Vector3 dFP2 = alpha * dxs(F[2]);

                       Float toi = large_enough_toi;


                       bool faraway = !distance::point_triangle_ccd_broadphase(
                           VP, FP0, FP1, FP2, dVP, dFP0, dFP1, dFP2, d_hat + thickness);

                       if(faraway)
                       {
                           PT_tois(i) = toi;
                           return;
                       }

                       bool hit = distance::point_triangle_ccd(
                           VP, FP0, FP1, FP2, dVP, dFP0, dFP1, dFP2, eta, thickness, max_iter, toi);

                       if(!hit)
                           toi = large_enough_toi;

                       PT_tois(i) = toi;
                   });
    }

    // EE
    {
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(candidate_EE_pairs.size(),
                   [EE_tois    = EE_tois.viewer().name("EE_tois"),
                    EE_pairs   = candidate_EE_pairs.viewer().name("EE_pairs"),
                    surf_edges = info.surf_edges().viewer().name("surf_edges"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    Ps    = info.positions().viewer().name("Ps"),
                    dxs   = info.displacements().viewer().name("dxs"),
                    alpha = info.alpha(),
                    d_hat = info.d_hat(),
                    eta,
                    max_iter,
                    large_enough_toi] __device__(int i) mutable
                   {
                       auto     indices   = EE_pairs(i);
                       Vector2i E0        = surf_edges(indices(0));
                       Vector2i E1        = surf_edges(indices(1));
                       Float    thickness = EE_thickness(thicknesses(E0(0)),
                                                      thicknesses(E0(1)),
                                                      thicknesses(E1(0)),
                                                      thicknesses(E1(1)));


                       Vector3 EP0  = Ps(E0[0]);
                       Vector3 EP1  = Ps(E0[1]);
                       Vector3 dEP0 = alpha * dxs(E0[0]);
                       Vector3 dEP1 = alpha * dxs(E0[1]);

                       Vector3 EP2  = Ps(E1[0]);
                       Vector3 EP3  = Ps(E1[1]);
                       Vector3 dEP2 = alpha * dxs(E1[0]);
                       Vector3 dEP3 = alpha * dxs(E1[1]);

                       Float toi = large_enough_toi;

                       bool faraway = !distance::edge_edge_ccd_broadphase(
                           // position
                           EP0,
                           EP1,
                           EP2,
                           EP3,
                           // displacement
                           dEP0,
                           dEP1,
                           dEP2,
                           dEP3,
                           d_hat + thickness);

                       if(faraway)
                       {
                           EE_tois(i) = toi;
                           return;
                       }

                       bool hit = distance::edge_edge_ccd(
                           // position
                           EP0,
                           EP1,
                           EP2,
                           EP3,
                           // displacement
                           dEP0,
                           dEP1,
                           dEP2,
                           dEP3,
                           eta,
                           thickness,
                           max_iter,
                           toi);

                       if(!hit)
                           toi = large_enough_toi;

                       EE_tois(i) = toi;
                   });
    }

    if(tois.size())
    {
        // get min toi
        DeviceReduce().Min(tois.data(), info.toi().data(), tois.size());
    }
    else
    {
        info.toi().fill(large_enough_toi);
    }
}
}  // namespace uipc::backend::cuda
