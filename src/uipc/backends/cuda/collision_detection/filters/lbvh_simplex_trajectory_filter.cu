#include "hip/hip_runtime.h"
#include <collision_detection/filters/lbvh_simplex_trajectory_filter.h>
#include <muda/cub/device/device_select.h>
#include <muda/ext/eigen/log_proxy.h>
#include <sim_engine.h>
#include <kernel_cout.h>
#include <utils/distance/distance_type.h>
#include <utils/distance.h>
#include <utils/codim_thickness.h>
#include <uipc/common/zip.h>

namespace uipc::backend::cuda
{
constexpr bool PrintDebugInfo = false;

REGISTER_SIM_SYSTEM(LBVHSimplexTrajectoryFilter);

void LBVHSimplexTrajectoryFilter::do_detect(DetectInfo& info)
{
    m_impl.detect(info);
}

void LBVHSimplexTrajectoryFilter::do_filter_active(FilterActiveInfo& info)
{
    m_impl.filter_active(info);
}

void LBVHSimplexTrajectoryFilter::do_filter_toi(FilterTOIInfo& info)
{
    m_impl.filter_toi(info);
}

void LBVHSimplexTrajectoryFilter::Impl::detect(DetectInfo& info)
{
    using namespace muda;

    auto alpha   = info.alpha();
    auto d_hat   = info.d_hat();
    auto Ps      = info.positions();
    auto dxs     = info.displacements();
    auto codimVs = info.codim_vertices();
    auto Vs      = info.surf_vertices();
    auto Es      = info.surf_edges();
    auto Fs      = info.surf_triangles();

    codim_point_aabbs.resize(codimVs.size());
    point_aabbs.resize(Vs.size());
    triangle_aabbs.resize(Fs.size());
    edge_aabbs.resize(Es.size());

    // build AABBs for codim vertices
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(codimVs.size(),
               [codimVs     = codimVs.viewer().name("codimVs"),
                Ps          = Ps.viewer().name("Ps"),
                dxs         = dxs.viewer().name("dxs"),
                aabbs       = codim_point_aabbs.viewer().name("aabbs"),
                thicknesses = info.thicknesses().viewer().name("thicknesses"),
                alpha       = alpha,
                d_hat       = d_hat] __device__(int i) mutable
               {
                   auto vI = codimVs(i);

                   Float thickness = thicknesses(vI);

                   const auto& pos   = Ps(vI);
                   Vector3     pos_t = pos + dxs(vI) * alpha;

                   AABB aabb;
                   aabb.extend(pos).extend(pos_t);

                   Float expand = d_hat + thickness;

                   aabb.min().array() -= expand;
                   aabb.max().array() += expand;
                   aabbs(i) = aabb;
               });


    // build AABBs for surf vertices (including codim vertices)
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(Vs.size(),
               [Vs          = Vs.viewer().name("V"),
                dxs         = dxs.viewer().name("dx"),
                Ps          = Ps.viewer().name("Ps"),
                aabbs       = point_aabbs.viewer().name("aabbs"),
                thicknesses = info.thicknesses().viewer().name("thicknesses"),
                alpha       = alpha,
                d_hat       = d_hat] __device__(int i) mutable
               {
                   auto vI = Vs(i);

                   Float thickness = thicknesses(vI);

                   const auto& pos   = Ps(vI);
                   Vector3     pos_t = pos + dxs(vI) * alpha;

                   AABB aabb;
                   aabb.extend(pos).extend(pos_t);

                   Float expand = d_hat + thickness;

                   aabb.min().array() -= expand;
                   aabb.max().array() += expand;
                   aabbs(i) = aabb;
               });

    // build AABBs for edges
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(Es.size(),
               [Es          = Es.viewer().name("E"),
                Ps          = Ps.viewer().name("Ps"),
                aabbs       = edge_aabbs.viewer().name("aabbs"),
                dxs         = dxs.viewer().name("dx"),
                thicknesses = info.thicknesses().viewer().name("thicknesses"),
                alpha       = alpha,
                d_hat       = d_hat] __device__(int i) mutable
               {
                   auto eI = Es(i);

                   Float thickness =
                       edge_thickness(thicknesses(eI[0]), thicknesses(eI[1]));

                   const auto& pos0   = Ps(eI[0]);
                   const auto& pos1   = Ps(eI[1]);
                   Vector3     pos0_t = pos0 + dxs(eI[0]) * alpha;
                   Vector3     pos1_t = pos1 + dxs(eI[1]) * alpha;

                   Vector3 max = pos0_t;
                   Vector3 min = pos0_t;

                   AABB aabb;

                   aabb.extend(pos0).extend(pos1).extend(pos0_t).extend(pos1_t);

                   Float expand = d_hat + thickness;

                   aabb.min().array() -= expand;
                   aabb.max().array() += expand;
                   aabbs(i) = aabb;
               });

    // build AABBs for triangles
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(Fs.size(),
               [Fs          = Fs.viewer().name("F"),
                Ps          = Ps.viewer().name("Ps"),
                aabbs       = triangle_aabbs.viewer().name("aabbs"),
                dxs         = dxs.viewer().name("dx"),
                thicknesses = info.thicknesses().viewer().name("thicknesses"),
                alpha       = alpha,
                d_hat       = d_hat] __device__(int i) mutable
               {
                   auto fI = Fs(i);

                   Float thickness = triangle_thickness(thicknesses(fI[0]),
                                                        thicknesses(fI[1]),
                                                        thicknesses(fI[2]));

                   const auto& pos0   = Ps(fI[0]);
                   const auto& pos1   = Ps(fI[1]);
                   const auto& pos2   = Ps(fI[2]);
                   Vector3     pos0_t = pos0 + dxs(fI[0]) * alpha;
                   Vector3     pos1_t = pos1 + dxs(fI[1]) * alpha;
                   Vector3     pos2_t = pos2 + dxs(fI[2]) * alpha;

                   AABB aabb;

                   aabb.extend(pos0)
                       .extend(pos1)
                       .extend(pos2)
                       .extend(pos0_t)
                       .extend(pos1_t)
                       .extend(pos2_t);

                   Float expand = d_hat + thickness;

                   aabb.min().array() -= expand;
                   aabb.max().array() += expand;
                   aabbs(i) = aabb;
               });

    // query CodimP and P
    lbvh_PP.build(point_aabbs);
    candidate_PP_pairs = lbvh_PP.query(
        codim_point_aabbs,
        [codimVs     = codimVs.viewer().name("codimVs"),
         Vs          = Vs.viewer().name("Vs"),
         Ps          = Ps.viewer().name("Ps"),
         dxs         = dxs.viewer().name("dxs"),
         thicknesses = info.thicknesses().viewer().name("thicknesses"),
         d_hat       = d_hat,
         alpha       = alpha] __device__(IndexT i, IndexT j)
        {
            const auto& codimV = codimVs(i);
            const auto& V      = Vs(j);

            Vector3 P0  = Ps(codimV);
            Vector3 P1  = Ps(V);
            Vector3 dP0 = alpha * dxs(codimV);
            Vector3 dP1 = alpha * dxs(V);

            Float thickness = PP_thickness(thicknesses(codimV), thicknesses(V));

            Float expand = d_hat + thickness;

            if(!distance::point_point_ccd_broadphase(P0, P1, dP0, dP1, expand))
                return false;

            return true;
        });

    // query PE
    lbvh_PE.build(edge_aabbs);
    candidate_PE_pairs = lbvh_PE.query(
        codim_point_aabbs,
        [codimVs     = codimVs.viewer().name("Vs"),
         Es          = Es.viewer().name("Es"),
         Ps          = Ps.viewer().name("Ps"),
         dxs         = dxs.viewer().name("dxs"),
         thicknesses = info.thicknesses().viewer().name("thicknesses"),
         d_hat       = d_hat,
         alpha       = alpha] __device__(IndexT i, IndexT j)
        {
            const auto& codimV = codimVs(i);
            const auto& E      = Es(j);

            Vector3 E0  = Ps(E[0]);
            Vector3 E1  = Ps(E[1]);
            Vector3 dE0 = alpha * dxs(E[0]);
            Vector3 dE1 = alpha * dxs(E[1]);

            Vector3 P  = Ps(codimV);
            Vector3 dP = alpha * dxs(codimV);

            Float thickness =
                PE_thickness(thicknesses(codimV), thicknesses(E[0]), thicknesses(E[1]));

            Float expand = d_hat + thickness;

            if(!distance::point_edge_ccd_broadphase(P, E0, E1, dP, dE0, dE1, expand))
                return false;

            return true;
        });


    // query PT
    lbvh_PT.build(triangle_aabbs);
    candidate_PT_pairs = lbvh_PT.query(
        point_aabbs,
        [Vs          = Vs.viewer().name("Vs"),
         Fs          = Fs.viewer().name("Fs"),
         Ps          = Ps.viewer().name("Ps"),
         dxs         = dxs.viewer().name("dxs"),
         thicknesses = info.thicknesses().viewer().name("thicknesses"),
         d_hat       = d_hat,
         alpha       = alpha] __device__(IndexT i, IndexT j)
        {
            // discard if the point is on the triangle
            auto V = Vs(i);
            auto F = Fs(j);

            if(F[0] == V || F[1] == V || F[2] == V)
                return false;

            Vector3 P  = Ps(V);
            Vector3 dP = alpha * dxs(V);

            Vector3 F0 = Ps(F[0]);
            Vector3 F1 = Ps(F[1]);
            Vector3 F2 = Ps(F[2]);

            Vector3 dF0 = alpha * dxs(F[0]);
            Vector3 dF1 = alpha * dxs(F[1]);
            Vector3 dF2 = alpha * dxs(F[2]);

            Float thickness = triangle_thickness(
                thicknesses(F[0]), thicknesses(F[1]), thicknesses(F[2]));

            Float expand = d_hat + thickness;

            if(!distance::point_triangle_ccd_broadphase(P, F0, F1, F2, dP, dF0, dF1, dF2, expand))
                return false;

            return true;
        });

    // query EE
    lbvh_EE.build(edge_aabbs);
    candidate_EE_pairs = lbvh_EE.detect(
        [Es          = Es.viewer().name("Es"),
         Ps          = Ps.viewer().name("Ps"),
         dxs         = dxs.viewer().name("dxs"),
         thicknesses = info.thicknesses().viewer().name("thicknesses"),
         d_hat       = d_hat,
         alpha       = alpha] __device__(IndexT i, IndexT j)
        {
            // discard if the edges shared same vertex
            auto Ea = Es(i);
            auto Eb = Es(j);

            if(Ea[0] == Eb[0] || Ea[0] == Eb[1] || Ea[1] == Eb[0] || Ea[1] == Eb[1])
                return false;

            Vector3 Ea0  = Ps(Ea[0]);
            Vector3 Ea1  = Ps(Ea[1]);
            Vector3 dEa0 = alpha * dxs(Ea[0]);
            Vector3 dEa1 = alpha * dxs(Ea[1]);

            Vector3 Eb0  = Ps(Eb[0]);
            Vector3 Eb1  = Ps(Eb[1]);
            Vector3 dEb0 = alpha * dxs(Eb[0]);
            Vector3 dEb1 = alpha * dxs(Eb[1]);

            Float thickness = EE_thickness(thicknesses(Ea[0]),
                                           thicknesses(Ea[1]),
                                           thicknesses(Eb[0]),
                                           thicknesses(Eb[1]));

            Float expand = d_hat + thickness;

            if(!distance::edge_edge_ccd_broadphase(
                   // position
                   Ea0,
                   Ea1,
                   Eb0,
                   Eb1,
                   // displacement
                   dEa0,
                   dEa1,
                   dEb0,
                   dEb1,
                   expand))
                return false;

            return true;
        });
}

void LBVHSimplexTrajectoryFilter::Impl::filter_active(FilterActiveInfo& info)
{
    using namespace muda;

    // we will filter-out the active pairs

    auto d_hat     = info.d_hat();
    auto positions = info.positions();

    auto get_total_count = [](muda::DeviceBuffer<IndexT>& offsets)
    {
        IndexT total_count = 0;
        offsets.view(offsets.size() - 1).copy_to(&total_count);
        return total_count;
    };

    // PPs
    {
        // +1 for total count
        PP_active_flags.resize(candidate_PP_pairs.size() + 1);
        PP_active_offsets.resize(candidate_PP_pairs.size() + 1);

        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(candidate_PP_pairs.size(),
                   [positions = positions.viewer().name("positions"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    PP_pairs = candidate_PP_pairs.viewer().name("PP_pairs"),
                    actives  = PP_active_flags.viewer().name("actives"),
                    d_hat    = d_hat] __device__(int i) mutable
                   {
                       // clear flag
                       actives(i)       = 0;
                       Vector2i indices = PP_pairs(i);

                       IndexT P0 = surf_vertices(indices(0));
                       IndexT P1 = surf_vertices(indices(1));

                       const auto& V0 = positions(P0);
                       const auto& V1 = positions(P1);

                       Float D;
                       distance::point_point_distance_unclassified(V0, V1, D);

                       Float thickness = PP_thickness(thicknesses(P0), thicknesses(P1));
                       Vector2 range = D_range(thickness, d_hat);

                       if(is_active_D(range, D))
                           actives(i) = 1;  // must use 1, we will scan the active pairs later
                   });

        // scan the active pairs
        DeviceScan().ExclusiveSum(PP_active_flags.data(),
                                  PP_active_offsets.data(),
                                  PP_active_offsets.size());

        IndexT total_count = get_total_count(PP_active_offsets);

        PPs.resize(total_count);

        // copy the active pairs
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(candidate_PP_pairs.size(),
                   [PP_active_flags = PP_active_flags.viewer().name("PP_active_flags"),
                    PP_active_offsets = PP_active_offsets.viewer().name("PP_active_offsets"),
                    PP_pairs = candidate_PP_pairs.viewer().name("PP_pairs"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    PPs = PPs.viewer().name("PPs")] __device__(int i) mutable
                   {
                       if(PP_active_flags(i))
                       {
                           auto     offset  = PP_active_offsets(i);
                           Vector2i surf_vI = PP_pairs(i);

                           IndexT P0 = surf_vertices(surf_vI(0));
                           IndexT P1 = surf_vertices(surf_vI(1));

                           PPs(offset) = {P0, P1};
                       }
                   });
    }
    // PEs
    {
        // +1 for total count
        PE_active_flags.resize(candidate_PE_pairs.size() + 1);
        PE_active_offsets.resize(candidate_PE_pairs.size() + 1);

        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(candidate_PE_pairs.size(),
                   [positions = positions.viewer().name("positions"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    surf_edges = info.surf_edges().viewer().name("surf_edges"),
                    PE_pairs   = candidate_PE_pairs.viewer().name("PE_pairs"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    actives = PE_active_flags.viewer().name("actives"),
                    d_hat   = d_hat] __device__(int i) mutable
                   {
                       // clear flag
                       actives(i) = 0;

                       Vector2i indices = PE_pairs(i);
                       IndexT   P       = surf_vertices(indices(0));
                       Vector2i E       = surf_edges(indices(1));

                       const auto& V  = positions(P);
                       const auto& E0 = positions(E(0));
                       const auto& E1 = positions(E(1));

                       Float D;
                       distance::point_edge_distance_unclassified(V, E0, E1, D);

                       Float thickness = PE_thickness(
                           thicknesses(P), thicknesses(E(0)), thicknesses(E(1)));

                       Vector2 range = D_range(thickness, d_hat);

                       if(is_active_D(range, D))
                           actives(i) = 1;  // must use 1, we will scan the active pairs later
                   });

        // scan the active pairs

        DeviceScan().ExclusiveSum(PE_active_flags.data(),
                                  PE_active_offsets.data(),
                                  PE_active_offsets.size());

        IndexT total_count = get_total_count(PE_active_offsets);

        PEs.resize(total_count);

        // copy the active pairs
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(candidate_PE_pairs.size(),
                   [PE_active_flags = PE_active_flags.viewer().name("PE_active_flags"),
                    PE_active_offsets = PE_active_offsets.viewer().name("PE_active_offsets"),
                    PE_pairs = candidate_PE_pairs.viewer().name("PE_pairs"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    surf_edges = info.surf_edges().viewer().name("surf_edges"),
                    PEs = PEs.viewer().name("PEs")] __device__(int i) mutable
                   {
                       if(PE_active_flags(i))
                       {
                           auto     offset       = PE_active_offsets(i);
                           Vector2i surf_indices = PE_pairs(i);

                           IndexT   P = surf_vertices(surf_indices(0));
                           Vector2i E = surf_edges(surf_indices(1));

                           PEs(offset) = {P, E(0), E(1)};
                       }
                   });
    }
    // PTs
    {
        // +1 for total count
        PT_active_flags.resize(candidate_PT_pairs.size() + 1);
        PT_active_offsets.resize(candidate_PT_pairs.size() + 1);

        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(candidate_PT_pairs.size(),
                   [Ps       = positions.viewer().name("Ps"),
                    PT_pairs = candidate_PT_pairs.viewer().name("PT_pairs"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    surf_triangles = info.surf_triangles().viewer().name("surf_triangles"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    actives = PT_active_flags.viewer().name("actives"),
                    d_hat   = d_hat] __device__(int i) mutable
                   {
                       // clear flag
                       actives(i) = 0;

                       Vector2i indices = PT_pairs(i);
                       IndexT   P       = surf_vertices(indices(0));
                       Vector3i T       = surf_triangles(indices(1));

                       const auto& V  = Ps(P);
                       const auto& F0 = Ps(T(0));
                       const auto& F1 = Ps(T(1));
                       const auto& F2 = Ps(T(2));


                       Float D;
                       distance::point_triangle_distance_unclassified(V, F0, F1, F2, D);

                       Float thickness = triangle_thickness(thicknesses(T(0)),
                                                            thicknesses(T(1)),
                                                            thicknesses(T(2)));

                       Vector2 range = D_range(thickness, d_hat);

                       if(is_active_D(range, D))
                           actives(i) = 1;  // must use 1, we will scan the active pairs later
                   });

        // scan the active pairs
        DeviceScan().ExclusiveSum(PT_active_flags.data(),
                                  PT_active_offsets.data(),
                                  PT_active_offsets.size());

        IndexT total_count = get_total_count(PT_active_offsets);

        PTs.resize(total_count);

        // copy the active pairs

        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(candidate_PT_pairs.size(),
                   [PT_active_flags = PT_active_flags.viewer().name("PT_active_flags"),
                    PT_active_offsets = PT_active_offsets.viewer().name("PT_active_offsets"),
                    PT_pairs = candidate_PT_pairs.viewer().name("PT_pairs"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    surf_triangles = info.surf_triangles().viewer().name("surf_triangles"),
                    PTs = PTs.viewer().name("PTs")] __device__(int i) mutable
                   {
                       if(PT_active_flags(i))
                       {
                           auto     offset       = PT_active_offsets(i);
                           Vector2i surf_indices = PT_pairs(i);

                           IndexT   P = surf_vertices(surf_indices(0));
                           Vector3i T = surf_triangles(surf_indices(1));

                           PTs(offset) = {P, T(0), T(1), T(2)};
                       }
                   });
    }
    // EEs
    {
        // +1 for total count
        EE_active_flags.resize(candidate_EE_pairs.size() + 1);
        EE_active_offsets.resize(candidate_EE_pairs.size() + 1);

        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(candidate_EE_pairs.size(),
                   [Ps         = positions.viewer().name("Ps"),
                    EE_pairs   = candidate_EE_pairs.viewer().name("EE_pairs"),
                    surf_edges = info.surf_edges().viewer().name("surf_edges"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    actives = EE_active_flags.viewer().name("actives"),
                    d_hat   = d_hat] __device__(int i) mutable
                   {
                       // clear flag
                       actives(i) = 0;

                       Vector2i indices = EE_pairs(i);
                       Vector2i E0      = surf_edges(indices(0));
                       Vector2i E1      = surf_edges(indices(1));

                       const auto& EP0 = Ps(E0(0));
                       const auto& EP1 = Ps(E0(1));
                       const auto& EP2 = Ps(E1(0));
                       const auto& EP3 = Ps(E1(1));

                       Float D;
                       distance::edge_edge_distance_unclassified(EP0, EP1, EP2, EP3, D);

                       Float thickness = EE_thickness(thicknesses(E0(0)),
                                                      thicknesses(E0(1)),
                                                      thicknesses(E1(0)),
                                                      thicknesses(E1(1)));

                       Vector2 range = D_range(thickness, d_hat);

                       if(is_active_D(range, D))
                           actives(i) = 1;  // must use 1, we will scan the active pairs later
                   });

        // scan the active pairs

        DeviceScan().ExclusiveSum(EE_active_flags.data(),
                                  EE_active_offsets.data(),
                                  EE_active_offsets.size());

        IndexT total_count = get_total_count(EE_active_offsets);

        EEs.resize(total_count);

        // copy the active pairs
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(candidate_EE_pairs.size(),
                   [EE_active_flags = EE_active_flags.viewer().name("EE_active_flags"),
                    EE_active_offsets = EE_active_offsets.viewer().name("EE_active_offsets"),
                    EE_pairs   = candidate_EE_pairs.viewer().name("EE_pairs"),
                    surf_edges = info.surf_edges().viewer().name("surf_edges"),
                    EEs = EEs.viewer().name("EEs")] __device__(int i) mutable
                   {
                       if(EE_active_flags(i))
                       {
                           auto     offset       = EE_active_offsets(i);
                           Vector2i surf_indices = EE_pairs(i);

                           Vector2i E0 = surf_edges(surf_indices(0));
                           Vector2i E1 = surf_edges(surf_indices(1));

                           EEs(offset) = {E0(0), E0(1), E1(0), E1(1)};
                       }
                   });
    }

    info.PPs(PPs);
    info.PEs(PEs);
    info.PTs(PTs);
    info.EEs(EEs);

    if constexpr(PrintDebugInfo)
    {
        std::vector<Vector2i> PPs_host;
        std::vector<Float>    PP_thicknesses_host;

        std::vector<Vector3i> PEs_host;
        std::vector<Float>    PE_thicknesses_host;

        std::vector<Vector4i> PTs_host;
        std::vector<Float>    PT_thicknesses_host;

        std::vector<Vector4i> EEs_host;
        std::vector<Float>    EE_thicknesses_host;

        PPs.copy_to(PPs_host);
        PEs.copy_to(PEs_host);
        PTs.copy_to(PTs_host);
        EEs.copy_to(EEs_host);

        std::cout << "filter result:" << std::endl;

        for(auto&& [PP, thickness] : zip(PPs_host, PP_thicknesses_host))
        {
            std::cout << "PP: " << PP.transpose() << " thickness: " << thickness << "\n";
        }

        for(auto&& [PE, thickness] : zip(PEs_host, PE_thicknesses_host))
        {
            std::cout << "PE: " << PE.transpose() << " thickness: " << thickness << "\n";
        }

        for(auto&& [PT, thickness] : zip(PTs_host, PT_thicknesses_host))
        {
            std::cout << "PT: " << PT.transpose() << " thickness: " << thickness << "\n";
        }

        for(auto&& [EE, thickness] : zip(EEs_host, EE_thicknesses_host))
        {
            std::cout << "EE: " << EE.transpose() << " thickness: " << thickness << "\n";
        }

        std::cout << std::flush;
    }
}

void LBVHSimplexTrajectoryFilter::Impl::filter_toi(FilterTOIInfo& info)
{
    using namespace muda;

    auto toi_size = candidate_PP_pairs.size() + candidate_PE_pairs.size()
                    + candidate_PT_pairs.size() + candidate_EE_pairs.size();

    tois.resize(toi_size);

    auto offset  = 0;
    auto PP_tois = tois.view(offset, candidate_PP_pairs.size());
    offset += candidate_PP_pairs.size();
    auto PE_tois = tois.view(offset, candidate_PE_pairs.size());
    offset += candidate_PE_pairs.size();
    auto PT_tois = tois.view(offset, candidate_PT_pairs.size());
    offset += candidate_PT_pairs.size();
    auto EE_tois = tois.view(offset, candidate_EE_pairs.size());


    // TODO: Now hard code the minimum separation coefficient
    // gap = eta * (dist2_cur - thickness * thickness) / (dist_cur + thickness);
    constexpr Float eta = 0.1;

    // TODO: Now hard code the maximum iteration
    constexpr SizeT max_iter = 1000;

    // large enough toi (>1)
    constexpr Float large_enough_toi = 1.1;

    // PP
    {
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(candidate_PP_pairs.size(),
                   [PP_tois  = PP_tois.viewer().name("PP_tois"),
                    PP_pairs = candidate_PP_pairs.viewer().name("PP_pairs"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    positions = info.positions().viewer().name("Ps"),
                    dxs       = info.displacements().viewer().name("dxs"),
                    alpha     = info.alpha(),
                    d_hat     = info.d_hat(),
                    eta,
                    max_iter,
                    large_enough_toi] __device__(int i) mutable
                   {
                       auto   indices = PP_pairs(i);
                       IndexT V0      = surf_vertices(indices(0));
                       IndexT V1      = surf_vertices(indices(1));

                       Float thickness = PP_thickness(thicknesses(V0), thicknesses(V1));

                       Vector3 VP0  = positions(V0);
                       Vector3 VP1  = positions(V1);
                       Vector3 dVP0 = alpha * dxs(V0);
                       Vector3 dVP1 = alpha * dxs(V1);

                       Float toi = large_enough_toi;

                       bool faraway =
                           !distance::point_point_ccd_broadphase(VP0, VP1, dVP0, dVP1, d_hat);

                       if(faraway)
                       {
                           PP_tois(i) = toi;
                           return;
                       }

                       bool hit = distance::point_point_ccd(
                           VP0, VP1, dVP0, dVP1, eta, thickness, max_iter, toi);

                       if(!hit)
                           toi = large_enough_toi;

                       PP_tois(i) = toi;
                   });
    }

    // PE
    {
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(candidate_PE_pairs.size(),
                   [PE_tois  = PE_tois.viewer().name("PE_tois"),
                    PE_pairs = candidate_PE_pairs.viewer().name("PE_pairs"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    surf_edges = info.surf_edges().viewer().name("surf_edges"),
                    Ps         = info.positions().viewer().name("Ps"),
                    dxs        = info.displacements().viewer().name("dxs"),
                    alpha      = info.alpha(),
                    d_hat      = info.d_hat(),
                    eta,
                    max_iter,
                    large_enough_toi] __device__(int i) mutable
                   {
                       auto     indices   = PE_pairs(i);
                       IndexT   V         = surf_vertices(indices(0));
                       Vector2i E         = surf_edges(indices(1));
                       Float    thickness = PE_thickness(
                           thicknesses(V), thicknesses(E(0)), thicknesses(E(1)));

                       Vector3 VP  = Ps(V);
                       Vector3 dVP = alpha * dxs(V);

                       Vector3 EP0  = Ps(E[0]);
                       Vector3 EP1  = Ps(E[1]);
                       Vector3 dEP0 = alpha * dxs(E[0]);
                       Vector3 dEP1 = alpha * dxs(E[1]);

                       Float toi = large_enough_toi;

                       bool faraway = !distance::point_edge_ccd_broadphase(
                           VP, EP0, EP1, dVP, dEP0, dEP1, d_hat);

                       if(faraway)
                       {
                           PE_tois(i) = toi;
                           return;
                       }

                       bool hit = distance::point_edge_ccd(
                           VP, EP0, EP1, dVP, dEP0, dEP1, eta, thickness, max_iter, toi);

                       if(!hit)
                           toi = large_enough_toi;

                       PE_tois(i) = toi;
                   });
    }

    // PT
    {
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(candidate_PT_pairs.size(),
                   [PT_tois  = PT_tois.viewer().name("PT_tois"),
                    PT_pairs = candidate_PT_pairs.viewer().name("PT_pairs"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    surf_triangles = info.surf_triangles().viewer().name("surf_triangles"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    Ps  = info.positions().viewer().name("Ps"),
                    dxs = info.displacements().viewer().name("dxs"),

                    alpha = info.alpha(),
                    d_hat = info.d_hat(),
                    eta,

                    max_iter,
                    large_enough_toi] __device__(int i) mutable
                   {
                       auto     indices   = PT_pairs(i);
                       IndexT   V         = surf_vertices(indices(0));
                       Vector3i F         = surf_triangles(indices(1));
                       Float    thickness = PT_thickness(thicknesses(V),
                                                      thicknesses(F(0)),
                                                      thicknesses(F(1)),
                                                      thicknesses(F(2)));

                       Vector3 VP  = Ps(V);
                       Vector3 dVP = alpha * dxs(V);

                       Vector3 FP0 = Ps(F[0]);
                       Vector3 FP1 = Ps(F[1]);
                       Vector3 FP2 = Ps(F[2]);

                       Vector3 dFP0 = alpha * dxs(F[0]);
                       Vector3 dFP1 = alpha * dxs(F[1]);
                       Vector3 dFP2 = alpha * dxs(F[2]);

                       Float toi = large_enough_toi;


                       bool faraway = !distance::point_triangle_ccd_broadphase(
                           VP, FP0, FP1, FP2, dVP, dFP0, dFP1, dFP2, d_hat);

                       if(faraway)
                       {
                           PT_tois(i) = toi;
                           return;
                       }

                       bool hit = distance::point_triangle_ccd(
                           VP, FP0, FP1, FP2, dVP, dFP0, dFP1, dFP2, eta, thickness, max_iter, toi);

                       if(!hit)
                           toi = large_enough_toi;

                       PT_tois(i) = toi;
                   });
    }

    // EE
    {
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(candidate_EE_pairs.size(),
                   [EE_tois    = EE_tois.viewer().name("EE_tois"),
                    EE_pairs   = candidate_EE_pairs.viewer().name("EE_pairs"),
                    surf_edges = info.surf_edges().viewer().name("surf_edges"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    Ps    = info.positions().viewer().name("Ps"),
                    dxs   = info.displacements().viewer().name("dxs"),
                    alpha = info.alpha(),
                    d_hat = info.d_hat(),
                    eta,
                    max_iter,
                    large_enough_toi] __device__(int i) mutable
                   {
                       auto     indices   = EE_pairs(i);
                       Vector2i E0        = surf_edges(indices(0));
                       Vector2i E1        = surf_edges(indices(1));
                       Float    thickness = EE_thickness(thicknesses(E0(0)),
                                                      thicknesses(E0(1)),
                                                      thicknesses(E1(0)),
                                                      thicknesses(E1(1)));


                       Vector3 EP0  = Ps(E0[0]);
                       Vector3 EP1  = Ps(E0[1]);
                       Vector3 dEP0 = alpha * dxs(E0[0]);
                       Vector3 dEP1 = alpha * dxs(E0[1]);

                       Vector3 EP2  = Ps(E1[0]);
                       Vector3 EP3  = Ps(E1[1]);
                       Vector3 dEP2 = alpha * dxs(E1[0]);
                       Vector3 dEP3 = alpha * dxs(E1[1]);

                       Float toi = large_enough_toi;

                       bool faraway = !distance::edge_edge_ccd_broadphase(
                           // position
                           EP0,
                           EP1,
                           EP2,
                           EP3,
                           // displacement
                           dEP0,
                           dEP1,
                           dEP2,
                           dEP3,
                           d_hat);

                       if(faraway)
                       {
                           EE_tois(i) = toi;
                           return;
                       }

                       bool hit = distance::edge_edge_ccd(
                           // position
                           EP0,
                           EP1,
                           EP2,
                           EP3,
                           // displacement
                           dEP0,
                           dEP1,
                           dEP2,
                           dEP3,
                           eta,
                           thickness,
                           max_iter,
                           toi);

                       if(!hit)
                           toi = large_enough_toi;

                       EE_tois(i) = toi;
                   });
    }

    if(tois.size())
    {
        // get min toi
        DeviceReduce().Min(tois.data(), info.toi().data(), tois.size());
    }
    else
    {
        info.toi().fill(large_enough_toi);
    }
}
}  // namespace uipc::backend::cuda
