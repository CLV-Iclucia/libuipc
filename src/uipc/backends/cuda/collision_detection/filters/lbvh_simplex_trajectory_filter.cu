#include "hip/hip_runtime.h"
#include <collision_detection/filters/lbvh_simplex_trajectory_filter.h>
#include <muda/cub/device/device_select.h>
#include <muda/ext/eigen/log_proxy.h>
#include <sim_engine.h>
#include <kernel_cout.h>
#include <muda/ext/geo/distance/distance_type.h>
#include <muda/ext/geo/distance.h>

namespace uipc::backend::cuda
{
constexpr bool PrintDebugInfo = false;

REGISTER_SIM_SYSTEM(LBVHSimplexTrajectoryFilter);

void LBVHSimplexTrajectoryFilter::do_detect(DetectInfo& info)
{
    m_impl.detect(info);
}

void LBVHSimplexTrajectoryFilter::do_filter_active(FilterActiveInfo& info)
{
    m_impl.filter_active(info);
}

void LBVHSimplexTrajectoryFilter::do_filter_toi(FilterTOIInfo& info)
{
    m_impl.filter_toi(info);
}

void LBVHSimplexTrajectoryFilter::Impl::detect(DetectInfo& info)
{
    using namespace muda;

    auto alpha = info.alpha();
    auto d_hat = info.d_hat();
    auto Ps    = info.positions();
    auto dxs   = info.displacements();
    auto Vs    = info.surf_vertices();
    auto Es    = info.surf_edges();
    auto Fs    = info.surf_triangles();

    point_aabbs.resize(Vs.size());
    triangle_aabbs.resize(Fs.size());
    edge_aabbs.resize(Es.size());

    if(alpha == 0.0f)
    {
        // build AABBs for points
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(Vs.size(),
                   [Vs    = Vs.viewer().name("V"),
                    Ps    = Ps.viewer().name("Ps"),
                    aabbs = point_aabbs.viewer().name("aabbs"),
                    d_hat = d_hat] __device__(int i) mutable
                   {
                       auto        vI  = Vs(i);
                       const auto& pos = Ps(vI);

                       AABB aabb;
                       aabb.extend(pos);

                       aabb.min().array() -= d_hat;
                       aabb.max().array() += d_hat;
                       aabbs(i) = aabb;
                   });

        // build AABBs for edges
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(Es.size(),
                   [Es    = Es.viewer().name("E"),
                    Ps    = Ps.viewer().name("Ps"),
                    aabbs = edge_aabbs.viewer().name("aabbs"),
                    d_hat = d_hat] __device__(int i) mutable
                   {
                       const auto& eI   = Es(i);
                       const auto& pos0 = Ps(eI[0]);
                       const auto& pos1 = Ps(eI[1]);

                       AABB aabb;
                       aabb.extend(pos0).extend(pos1);
                       aabb.min().array() -= d_hat;
                       aabb.max().array() += d_hat;
                       aabbs(i) = aabb;
                   });

        // build AABBs for triangles
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(Fs.size(),
                   [Fs    = Fs.viewer().name("F"),
                    Ps    = Ps.viewer().name("Ps"),
                    aabbs = triangle_aabbs.viewer().name("aabbs"),
                    d_hat = d_hat] __device__(int i) mutable
                   {
                       const auto& fI   = Fs(i);
                       const auto& pos0 = Ps(fI[0]);
                       const auto& pos1 = Ps(fI[1]);
                       const auto& pos2 = Ps(fI[2]);

                       AABB aabb;
                       aabb.extend(pos0).extend(pos1).extend(pos2);

                       aabb.min().array() -= d_hat;
                       aabb.max().array() += d_hat;
                       aabbs(i) = aabb;
                   });

        // query PT
        lbvh_PT.build(triangle_aabbs);
        auto PT_pairs = lbvh_PT.query(
            point_aabbs,
            [Vs    = Vs.viewer().name("Vs"),
             Fs    = Fs.viewer().name("Fs"),
             Ps    = Ps.viewer().name("Ps"),
             d_hat = d_hat] __device__(IndexT i, IndexT j)
            {
                // discard if the point is on the triangle
                auto        V = Vs(i);
                const auto& F = Fs(j);

                if(F[0] == V || F[1] == V || F[2] == V)
                    return false;


                Vector3 VP  = Ps(V);
                Vector3 FP0 = Ps(F[0]);
                Vector3 FP1 = Ps(F[1]);
                Vector3 FP2 = Ps(F[2]);
                Float   D;
                muda::distance::point_triangle_distance_unclassified(VP, FP0, FP1, FP2, D);

                if(D >= d_hat * d_hat)
                    return false;

                //cout << "PT: " << V << " " << F.transpose().eval()
                //     << " d: " << sqrt(D) << "d_hat: " << d_hat << "\n";

                return true;
            });

        // query EE
        lbvh_EE.build(edge_aabbs);
        auto EE_pairs = lbvh_EE.detect(
            [Es    = Es.viewer().name("Es"),
             Ps    = Ps.viewer().name("Ps"),
             d_hat = d_hat] __device__(IndexT i, IndexT j)
            {
                // discard if the edges shared same vertex
                const auto& E0 = Es(i);
                const auto& E1 = Es(j);

                if(E0[0] == E1[0] || E0[0] == E1[1] || E0[1] == E1[0] || E0[1] == E1[1])
                    return false;

                Vector3 EP0 = Ps(E0[0]);
                Vector3 EP1 = Ps(E0[1]);
                Vector3 EP2 = Ps(E1[0]);
                Vector3 EP3 = Ps(E1[1]);
                Float   D;
                muda::distance::edge_edge_distance_unclassified(EP0, EP1, EP2, EP3, D);

                if(D >= d_hat * d_hat)
                    return false;

                //cout << "EE: " << E0.transpose().eval() << " " << E1.transpose().eval()
                //     << " d: " << sqrt(D) << "d_hat: " << d_hat << "\n";

                return true;
            });

        candidate_PTs.resize(PT_pairs.size());
        candidate_EEs.resize(EE_pairs.size());

        // record the candidate pairs
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(PT_pairs.size(),
                   [PT_pairs = PT_pairs.viewer().name("PT_pairs"),
                    candidate_PTs = candidate_PTs.viewer().name("candidate_PTs"),
                    Fs = Fs.viewer().name("Fs"),
                    Vs = Vs.viewer().name("Vs")] __device__(int i) mutable
                   {
                       auto&       PT   = candidate_PTs(i);
                       const auto& pair = PT_pairs(i);
                       PT[0]            = Vs(pair[0]);
                       PT.segment<3>(1) = Fs(pair[1]);
                   });

        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(EE_pairs.size(),
                   [EE_pairs = EE_pairs.viewer().name("EE_pairs"),
                    candidate_EEs = candidate_EEs.viewer().name("candidate_EEs"),
                    Es = Es.viewer().name("Es")] __device__(int i) mutable
                   {
                       auto&       EE   = candidate_EEs(i);
                       const auto& pair = EE_pairs(i);
                       EE.segment<2>(0) = Es(pair[0]);
                       EE.segment<2>(2) = Es(pair[1]);
                   });
    }
    else  // alpha > 0
    {
        // build AABBs for points
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(Vs.size(),
                   [Vs    = Vs.viewer().name("V"),
                    dxs   = dxs.viewer().name("dx"),
                    Ps    = Ps.viewer().name("Ps"),
                    aabbs = point_aabbs.viewer().name("aabbs"),
                    alpha = alpha,
                    d_hat = d_hat] __device__(int i) mutable
                   {
                       auto        vI    = Vs(i);
                       const auto& pos   = Ps(vI);
                       Vector3     pos_t = pos + dxs(vI) * alpha;

                       AABB aabb;

                       aabb.extend(pos).extend(pos_t);

                       aabb.min().array() -= d_hat;
                       aabb.max().array() += d_hat;
                       aabbs(i) = aabb;
                   });

        // build AABBs for edges
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(Es.size(),
                   [Es    = Es.viewer().name("E"),
                    Ps    = Ps.viewer().name("Ps"),
                    aabbs = edge_aabbs.viewer().name("aabbs"),
                    dxs   = dxs.viewer().name("dx"),
                    alpha = alpha,
                    d_hat = d_hat] __device__(int i) mutable
                   {
                       auto        eI     = Es(i);
                       const auto& pos0   = Ps(eI[0]);
                       const auto& pos1   = Ps(eI[1]);
                       Vector3     pos0_t = pos0 + dxs(eI[0]) * alpha;
                       Vector3     pos1_t = pos1 + dxs(eI[1]) * alpha;

                       Vector3 max = pos0_t;
                       Vector3 min = pos0_t;

                       AABB aabb;

                       aabb.extend(pos0).extend(pos1).extend(pos0_t).extend(pos1_t);

                       aabb.min().array() -= d_hat;
                       aabb.max().array() += d_hat;
                       aabbs(i) = aabb;
                   });

        // build AABBs for triangles
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(Fs.size(),
                   [Fs    = Fs.viewer().name("F"),
                    Ps    = Ps.viewer().name("Ps"),
                    aabbs = triangle_aabbs.viewer().name("aabbs"),
                    dxs   = dxs.viewer().name("dx"),
                    alpha = alpha,
                    d_hat = d_hat] __device__(int i) mutable
                   {
                       auto        fI     = Fs(i);
                       const auto& pos0   = Ps(fI[0]);
                       const auto& pos1   = Ps(fI[1]);
                       const auto& pos2   = Ps(fI[2]);
                       Vector3     pos0_t = pos0 + dxs(fI[0]) * alpha;
                       Vector3     pos1_t = pos1 + dxs(fI[1]) * alpha;
                       Vector3     pos2_t = pos2 + dxs(fI[2]) * alpha;

                       AABB aabb;

                       aabb.extend(pos0)
                           .extend(pos1)
                           .extend(pos2)
                           .extend(pos0_t)
                           .extend(pos1_t)
                           .extend(pos2_t);

                       aabb.min().array() -= d_hat;
                       aabb.max().array() += d_hat;
                       aabbs(i) = aabb;
                   });

        // query PT
        lbvh_PT.build(triangle_aabbs);
        auto PT_pairs =
            lbvh_PT.query(point_aabbs,
                          [Vs    = Vs.viewer().name("Vs"),
                           Fs    = Fs.viewer().name("Fs"),
                           Ps    = Ps.viewer().name("Ps"),
                           dxs   = dxs.viewer().name("dxs"),
                           d_hat = d_hat,
                           alpha = alpha] __device__(IndexT i, IndexT j)
                          {
                              // discard if the point is on the triangle
                              auto V = Vs(i);
                              auto F = Fs(j);

                              if(F[0] == V || F[1] == V || F[2] == V)
                                  return false;

                              Vector3 VP  = Ps(V);
                              Vector3 dVP = alpha * dxs(V);

                              Vector3 FP0 = Ps(F[0]);
                              Vector3 FP1 = Ps(F[1]);
                              Vector3 FP2 = Ps(F[2]);

                              Vector3 dFP0 = alpha * dxs(F[0]);
                              Vector3 dFP1 = alpha * dxs(F[1]);
                              Vector3 dFP2 = alpha * dxs(F[2]);

                              if(!muda::distance::point_triangle_ccd_broadphase(
                                     VP, FP0, FP1, FP2, dVP, dFP0, dFP1, dFP2, d_hat))
                                  return false;


                              return true;
                          });

        // query EE
        lbvh_EE.build(edge_aabbs);
        auto EE_pairs = lbvh_EE.detect(
            [Es    = Es.viewer().name("Es"),
             Ps    = Ps.viewer().name("Ps"),
             dxs   = dxs.viewer().name("dxs"),
             d_hat = d_hat,
             alpha = alpha] __device__(IndexT i, IndexT j)
            {
                // discard if the edges shared same vertex
                auto E0 = Es(i);
                auto E1 = Es(j);

                if(E0[0] == E1[0] || E0[0] == E1[1] || E0[1] == E1[0] || E0[1] == E1[1])
                    return false;

                Vector3 EP0  = Ps(E0[0]);
                Vector3 EP1  = Ps(E0[1]);
                Vector3 dEP0 = alpha * dxs(E0[0]);
                Vector3 dEP1 = alpha * dxs(E0[1]);

                Vector3 EP2  = Ps(E1[0]);
                Vector3 EP3  = Ps(E1[1]);
                Vector3 dEP2 = alpha * dxs(E1[0]);
                Vector3 dEP3 = alpha * dxs(E1[1]);

                if(!muda::distance::edge_edge_ccd_broadphase(
                       // position
                       EP0,
                       EP1,
                       EP2,
                       EP3,
                       // displacement
                       dEP0,
                       dEP1,
                       dEP2,
                       dEP3,
                       d_hat))
                    return false;

                return true;
            });

        candidate_PTs.resize(PT_pairs.size());
        candidate_EEs.resize(EE_pairs.size());

        // record the candidate pairs
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(PT_pairs.size(),
                   [PT_pairs = PT_pairs.viewer().name("PT_pairs"),
                    candidate_PTs = candidate_PTs.viewer().name("candidate_PTs"),
                    Fs = Fs.viewer().name("Fs"),
                    Vs = Vs.viewer().name("Vs")] __device__(int i) mutable
                   {
                       auto& PT         = candidate_PTs(i);
                       auto  pair       = PT_pairs(i);
                       PT[0]            = Vs(pair[0]);
                       PT.segment<3>(1) = Fs(pair[1]);
                   });

        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(EE_pairs.size(),
                   [EE_pairs = EE_pairs.viewer().name("EE_pairs"),
                    candidate_EEs = candidate_EEs.viewer().name("candidate_EEs"),
                    Es = Es.viewer().name("Es")] __device__(int i) mutable
                   {
                       auto& EE         = candidate_EEs(i);
                       auto  pair       = EE_pairs(i);
                       EE.segment<2>(0) = Es(pair[0]);
                       EE.segment<2>(2) = Es(pair[1]);
                   });
    }

    if constexpr(PrintDebugInfo)
    {
        std::vector<Vector4i> candidate_PTs_host;
        std::vector<Vector4i> candidate_EEs_host;

        candidate_PTs.copy_to(candidate_PTs_host);
        candidate_EEs.copy_to(candidate_EEs_host);

        // print the candidate pairs
        for(auto& PT : candidate_PTs_host)
        {
            std::cout << "PT: " << PT.transpose() << std::endl;
        }

        for(auto& EE : candidate_EEs_host)
        {
            std::cout << "EE: " << EE.transpose() << std::endl;
        }
    }
}

void LBVHSimplexTrajectoryFilter::Impl::filter_active(FilterActiveInfo& info)
{
    using namespace muda;


    if constexpr(PrintDebugInfo)
    {
        std::vector<Vector4i> candidate_PTs_host;
        std::vector<Vector4i> candidate_EEs_host;

        candidate_PTs.copy_to(candidate_PTs_host);
        candidate_EEs.copy_to(candidate_EEs_host);

        // print the candidate pairs
        std::cout << "candidate pairs:" << std::endl;
        for(auto& PT : candidate_PTs_host)
        {
            std::cout << "PT: " << PT.transpose() << std::endl;
        }

        for(auto& EE : candidate_EEs_host)
        {
            std::cout << "EE: " << EE.transpose() << std::endl;
        }
    }

    auto d_hat = info.d_hat();
    auto Ps    = info.positions();

    // we will classify the PT EE pairs into PTs, EEs, PEs, PPs

    // PT: point-triangle, only 1 possible PT per candidate PT
    temp_PTs.resize(candidate_PTs.size());
    PTs.resize(candidate_PTs.size());

    // EE: edge-edge, only 1 possible EE per candidate EE
    temp_EEs.resize(candidate_EEs.size());
    EEs.resize(candidate_EEs.size());


    // PE: point-edge
    // 3 possible PE per candidate PT
    SizeT PT_to_PE_max_count = candidate_PTs.size() * 3;
    // 4 possible PE per candidate EE
    SizeT EE_to_PE_max_count = candidate_EEs.size() * 4;

    SizeT max_PE_count = PT_to_PE_max_count + EE_to_PE_max_count;
    temp_PEs.resize(max_PE_count);
    PEs.resize(max_PE_count);

    // PP: point-point
    // 3 possible PP per candidate PT
    SizeT PT_to_PP_max_count = candidate_PTs.size() * 3;
    // 4 possible PP per candidate EE
    SizeT EE_to_PP_max_count = candidate_EEs.size() * 4;

    SizeT max_PP_count = PT_to_PP_max_count + EE_to_PP_max_count;
    temp_PPs.resize(max_PP_count);
    PPs.resize(max_PP_count);


    // always use the squared distance to avoid numerical issue
    Float D_hat = d_hat * d_hat;


    // 1) PT-> PT, 3PE, 3PP
    auto PE_offset = 0;
    auto PE_count  = PT_to_PE_max_count;
    auto PP_offset = 0;
    auto PP_count  = PT_to_PP_max_count;

    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(candidate_PTs.size(),
               [Ps            = Ps.viewer().name("Ps"),
                candidate_PTs = candidate_PTs.viewer().name("candidate_PTs"),
                PTs           = temp_PTs.viewer().name("PTs"),
                PEs   = temp_PEs.view(PE_offset, PE_count).viewer().name("PEs"),
                PPs   = temp_PPs.view(PP_offset, PP_count).viewer().name("PPs"),
                D_hat = D_hat] __device__(int i) mutable
               {
                   auto I3 = i * 3;

                   // Invalidate all the candidates
                   {
                       PTs(i).array() = -1;

                       PEs(I3 + 0).array() = -1;
                       PEs(I3 + 1).array() = -1;
                       PEs(I3 + 2).array() = -1;

                       PPs(I3 + 0).array() = -1;
                       PPs(I3 + 1).array() = -1;
                       PPs(I3 + 2).array() = -1;
                   }

                   Vector4i PT = candidate_PTs(i);
                   IndexT   P  = PT(0);
                   Vector3i T  = PT.tail<3>();

                   const auto& V  = Ps(PT(0));
                   const auto& F0 = Ps(PT(1));
                   const auto& F1 = Ps(PT(2));
                   const auto& F2 = Ps(PT(3));

                   auto dist_type =
                       muda::distance::point_triangle_distance_type(V, F0, F1, F2);

                   if(dist_type == distance::PointTriangleDistanceType::PT)
                   {
                       if constexpr(PrintDebugInfo)
                       {
                           cout << "PT->PT:" << PT.transpose().eval() << "\n";
                       }

                       Float D;
                       distance::point_triangle_distance(V, F0, F1, F2, D);

                       if(D < D_hat)
                           PTs(i) = PT;

                       return;
                   }

                   // if not, then it can be PT->PE or PT->PP

                   // 3 possible PE
                   const Vector3i PE[3] = {
                       {P, T(0), T(1)}, {P, T(1), T(2)}, {P, T(2), T(0)}};

                   for(int j = 0; j < 3; ++j)
                   {
                       auto& pe = PE[j];
                       auto  E0 = Ps(pe(0));
                       auto  E1 = Ps(pe(1));
                       auto  E2 = Ps(pe(2));

                       auto dist_type = distance::point_edge_distance_type(V, E0, E1);

                       if(dist_type == distance::PointEdgeDistanceType::PE)
                       {
                           Float D;
                           distance::point_edge_distance(V, E0, E1, D);

                           if(D < D_hat)
                           {
                               if constexpr(PrintDebugInfo)
                               {
                                   cout << "PT->PE:" << PT.transpose().eval()
                                        << "->" << P << "," << T(j) << "\n";
                               }

                               PEs(I3 + j) = pe;
                           }
                       }
                   }

                   // 3 possible PP
                   const Vector2i PP[3] = {{P, T(0)}, {P, T(1)}, {P, T(2)}};

                   for(int j = 0; j < 3; ++j)
                   {
                       auto& pp = PP[j];
                       auto  P0 = Ps(pp(0));
                       auto  P1 = Ps(pp(1));

                       Float D;
                       distance::point_point_distance(P0, P1, D);

                       if(D < D_hat)
                       {
                           PPs(I3 + j) = pp;
                       }
                   }
               });

    // 2) EE-> PT, 4PE, 4PP
    PE_offset += PE_count;
    PE_count = EE_to_PE_max_count;
    PP_offset += PP_count;
    PP_count = EE_to_PP_max_count;

    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(candidate_EEs.size(),
               [Ps            = Ps.viewer().name("Ps"),
                candidate_EEs = candidate_EEs.viewer().name("candidate_EEs"),
                EEs           = temp_EEs.viewer().name("EEs"),
                PEs   = temp_PEs.view(PE_offset, PE_count).viewer().name("PEs"),
                PPs   = temp_PPs.view(PP_offset, PP_count).viewer().name("PPs"),
                D_hat = D_hat] __device__(int i) mutable
               {
                   auto I4 = i * 4;

                   // Invalidate all the candidates
                   {
                       EEs(i).array() = -1;

                       PEs(I4 + 0).array() = -1;
                       PEs(I4 + 1).array() = -1;
                       PEs(I4 + 2).array() = -1;
                       PEs(I4 + 3).array() = -1;

                       PPs(I4 + 0).array() = -1;
                       PPs(I4 + 1).array() = -1;
                       PPs(I4 + 2).array() = -1;
                       PPs(I4 + 3).array() = -1;
                   }


                   Vector4i EE = candidate_EEs(i);

                   IndexT Ea0 = EE(0);
                   IndexT Ea1 = EE(1);
                   IndexT Eb0 = EE(2);
                   IndexT Eb1 = EE(3);

                   Vector3 E0 = Ps(Ea0);
                   Vector3 E1 = Ps(Ea1);
                   Vector3 E2 = Ps(Eb0);
                   Vector3 E3 = Ps(Eb1);

                   auto dist_type = distance::edge_edge_distance_type(E0, E1, E2, E3);

                   if(dist_type == distance::EdgeEdgeDistanceType::EE)
                   {
                       if constexpr(PrintDebugInfo)
                       {
                           cout << "EE->EE:" << EE.transpose().eval() << "\n";
                       }


                       Float D;
                       distance::edge_edge_distance(E0, E1, E2, E3, D);

                       if(D < D_hat)
                           EEs(i) = EE;

                       return;
                   }

                   // if not, then it can be EE->PE or EE->PP

                   // 4 possible PE
                   const Vector3i PE[4] = {{Ea0, Eb0, Eb1},
                                           {Ea1, Eb0, Eb1},
                                           {Eb0, Ea0, Ea1},
                                           {Eb1, Ea0, Ea1}};

                   for(int j = 0; j < 4; ++j)
                   {
                       auto& pe = PE[j];
                       auto  E0 = Ps(pe(0));
                       auto  E1 = Ps(pe(1));
                       auto  E2 = Ps(pe(2));

                       auto dist_type = distance::point_edge_distance_type(E0, E1, E2);

                       if(dist_type == distance::PointEdgeDistanceType::PE)
                       {
                           Float D;
                           distance::point_edge_distance(E0, E1, E2, D);

                           if(D < D_hat)
                           {
                               if constexpr(PrintDebugInfo)
                               {
                                   cout << "EE->PE:" << EE.transpose().eval()
                                        << "->" << pe.transpose().eval() << "\n";
                               }

                               PEs(I4 + j) = pe;
                           }
                       }
                   }

                   // 4 possible PP
                   const Vector2i PP[4] = {{Ea0, Eb0}, {Ea0, Eb1}, {Ea1, Eb0}, {Ea1, Eb1}};

                   for(int j = 0; j < 4; ++j)
                   {
                       auto& pp = PP[j];
                       auto  P0 = Ps(pp(0));
                       auto  P1 = Ps(pp(1));

                       Float D;
                       distance::point_point_distance(P0, P1, D);

                       if(D < D_hat)
                       {
                           if constexpr(PrintDebugInfo)
                           {
                               cout << "EE->PP:" << EE.transpose().eval()
                                    << "->" << pp.transpose().eval() << "\n";
                           }

                           PPs(I4 + j) = pp;
                       }
                   }
               });

    DeviceSelect().If(temp_PTs.data(),
                      PTs.data(),
                      selected.data(),
                      temp_PTs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector4i& pt)
                      { return pt(0) != -1; });

    int h_selected = selected;
    PTs.resize(h_selected);

    DeviceSelect().If(temp_EEs.data(),
                      EEs.data(),
                      selected.data(),
                      temp_EEs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector4i& ee)
                      { return ee(0) != -1; });

    h_selected = selected;
    EEs.resize(h_selected);

    DeviceSelect().If(temp_PEs.data(),
                      PEs.data(),
                      selected.data(),
                      temp_PEs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector3i& pe)
                      { return pe(0) != -1; });

    h_selected = selected;
    PEs.resize(h_selected);

    DeviceSelect().If(temp_PPs.data(),
                      PPs.data(),
                      selected.data(),
                      temp_PPs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector2i& pp)
                      { return pp(0) != -1; });

    h_selected = selected;
    PPs.resize(h_selected);

    info.PTs(PTs);
    info.EEs(EEs);
    info.PEs(PEs);
    info.PPs(PPs);


    if constexpr(PrintDebugInfo)
    {
        std::vector<Vector4i> PTs_host;
        std::vector<Vector4i> EEs_host;
        std::vector<Vector3i> PEs_host;
        std::vector<Vector2i> PPs_host;

        PTs.copy_to(PTs_host);
        EEs.copy_to(EEs_host);
        PEs.copy_to(PEs_host);
        PPs.copy_to(PPs_host);

        std::cout << "classify result:" << std::endl;

        for(auto& PT : PTs_host)
        {
            std::cout << "PT: " << PT.transpose() << std::endl;
        }

        for(auto& EE : EEs_host)
        {
            std::cout << "EE: " << EE.transpose() << std::endl;
        }

        for(auto& PE : PEs_host)
        {
            std::cout << "PE: " << PE.transpose() << std::endl;
        }

        for(auto& PP : PPs_host)
        {
            std::cout << "PP: " << PP.transpose() << std::endl;
        }
    }
}

void LBVHSimplexTrajectoryFilter::Impl::filter_toi(FilterTOIInfo& info)
{
    using namespace muda;

    tois.resize(candidate_PTs.size() + candidate_EEs.size());

    auto PT_tois = tois.view(0, candidate_PTs.size());
    auto EE_tois = tois.view(candidate_PTs.size(), candidate_EEs.size());


    // TODO: Codimension IPC need thickness property, later we will add it
    constexpr Float thickness = 0.0;
    // TODO: Now hard code the minimum separation coefficient
    // gap = eta * (dist2_cur - thickness * thickness) / (dist_cur + thickness);
    constexpr Float eta = 0.1;

    // TODO: Now hard code the maximum iteration
    constexpr SizeT max_iter = 1000;

    // large enough toi (>1)
    constexpr Float large_enough_toi = 1.1;

    // PT
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(candidate_PTs.size(),
               [PT_tois       = PT_tois.viewer().name("PT_tois"),
                candidate_PTs = candidate_PTs.viewer().name("candidate_PTs"),
                Ps            = info.positions().viewer().name("Ps"),
                dxs           = info.displacements().viewer().name("dxs"),
                alpha         = info.alpha(),
                d_hat         = info.d_hat(),
                eta,
                thickness,
                max_iter,
                large_enough_toi] __device__(int i) mutable
               {
                   auto& PT = candidate_PTs(i);
                   auto  V  = PT[0];
                   auto  F  = PT.segment<3>(1);

                   Vector3 VP  = Ps(V);
                   Vector3 dVP = alpha * dxs(V);

                   Vector3 FP0 = Ps(F[0]);
                   Vector3 FP1 = Ps(F[1]);
                   Vector3 FP2 = Ps(F[2]);

                   Vector3 dFP0 = alpha * dxs(F[0]);
                   Vector3 dFP1 = alpha * dxs(F[1]);
                   Vector3 dFP2 = alpha * dxs(F[2]);

                   Float toi = large_enough_toi;


                   bool faraway = !muda::distance::point_triangle_ccd_broadphase(
                       VP, FP0, FP1, FP2, dVP, dFP0, dFP1, dFP2, d_hat);

                   if(faraway)
                   {
                       PT_tois(i) = toi;
                       return;
                   }

                   bool hit = muda::distance::point_triangle_ccd(
                       VP, FP0, FP1, FP2, dVP, dFP0, dFP1, dFP2, eta, thickness, max_iter, toi);

                   if(!hit)
                       toi = large_enough_toi;

                   PT_tois(i) = toi;
               });

    // EE
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(candidate_EEs.size(),
               [EE_tois       = EE_tois.viewer().name("EE_tois"),
                candidate_EEs = candidate_EEs.viewer().name("candidate_EEs"),
                Ps            = info.positions().viewer().name("Ps"),
                dxs           = info.displacements().viewer().name("dxs"),
                alpha         = info.alpha(),
                d_hat         = info.d_hat(),
                eta,
                thickness,
                max_iter,
                large_enough_toi] __device__(int i) mutable
               {
                   auto& EE = candidate_EEs(i);
                   auto  E0 = EE.segment<2>(0);
                   auto  E1 = EE.segment<2>(2);

                   Vector3 EP0  = Ps(E0[0]);
                   Vector3 EP1  = Ps(E0[1]);
                   Vector3 dEP0 = alpha * dxs(E0[0]);
                   Vector3 dEP1 = alpha * dxs(E0[1]);

                   Vector3 EP2  = Ps(E1[0]);
                   Vector3 EP3  = Ps(E1[1]);
                   Vector3 dEP2 = alpha * dxs(E1[0]);
                   Vector3 dEP3 = alpha * dxs(E1[1]);

                   Float toi = large_enough_toi;

                   bool faraway = !muda::distance::edge_edge_ccd_broadphase(
                       // position
                       EP0,
                       EP1,
                       EP2,
                       EP3,
                       // displacement
                       dEP0,
                       dEP1,
                       dEP2,
                       dEP3,
                       d_hat);

                   if(faraway)
                   {
                       EE_tois(i) = toi;
                       return;
                   }

                   bool hit = muda::distance::edge_edge_ccd(
                       // position
                       EP0,
                       EP1,
                       EP2,
                       EP3,
                       // displacement
                       dEP0,
                       dEP1,
                       dEP2,
                       dEP3,
                       eta,
                       thickness,
                       max_iter,
                       toi);

                   if(!hit)
                       toi = large_enough_toi;

                   EE_tois(i) = toi;
               });

    if(tois.size())
    {
        // get min toi
        DeviceReduce().Min(tois.data(), info.toi().data(), tois.size());
    }
    else
    {
        info.toi().fill(large_enough_toi);
    }
}
}  // namespace uipc::backend::cuda
