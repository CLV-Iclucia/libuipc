#include "hip/hip_runtime.h"
#include <collision_detection/filters/easy_vertex_half_plane_trajectory_filter.h>
#include <muda/cub/device/device_reduce.h>
#include <kernel_cout.h>

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(EasyVertexHalfPlaneTrajectoryFilter);

constexpr bool PrintDebugInfo = false;

void EasyVertexHalfPlaneTrajectoryFilter::do_detect(DetectInfo& info)
{
    // do nothing
}

void EasyVertexHalfPlaneTrajectoryFilter::do_filter_active(FilterActiveInfo& info)
{
    m_impl.filter_active(info);
}

void EasyVertexHalfPlaneTrajectoryFilter::do_filter_toi(FilterTOIInfo& info)
{
    m_impl.filter_toi(info);
}

void EasyVertexHalfPlaneTrajectoryFilter::Impl::filter_active(FilterActiveInfo& info)
{
    using namespace muda;

    auto query = [&]
    {
        num_collisions = 0;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(info.surf_vertices().size(),
                   [num = num_collisions.viewer().name("num_collisions"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    positions = info.positions().viewer().name("positions"),
                    half_plane_positions = info.plane_positions().viewer().name("plane_positions"),
                    half_plane_normals = info.plane_normals().viewer().name("plane_normals"),
                    d_hat     = info.d_hat(),
                    PHs       = PHs.viewer().name("PHs"),
                    max_count = PHs.size()] __device__(int i) mutable
                   {
                       IndexT  vI  = surf_vertices(i);
                       Vector3 pos = positions(vI);

                       for(int j = 0; j < half_plane_positions.total_size(); ++j)
                       {
                           Vector3 plane_pos    = half_plane_positions(j);
                           Vector3 plane_normal = half_plane_normals(j);
                           Vector3 diff         = pos - plane_pos;

                           Float dst = diff.dot(plane_normal);

                           MUDA_ASSERT(dst > 0.0f, "dst=%f, why?", dst);

                           if(dst < d_hat)
                           {
                               auto last = atomic_add(num.data(), 1);

                               if(last < max_count)
                               {
                                   PHs(last) = Vector2i{vI, j};
                               }
                           }
                       }
                   });
    };

    query();
    h_num_collisions = num_collisions;

    if(h_num_collisions > PHs.size())
    {
        PHs.resize(h_num_collisions * reserve_ratio);
        query();
    }

    info.PHs(PHs.view(0, h_num_collisions));

    if constexpr(PrintDebugInfo)
    {
        std::vector<Vector2i> phs(h_num_collisions);
        PHs.view(0, h_num_collisions).copy_to(phs.data());
        for(auto& ph : phs)
        {
            std::cout << "vI: " << ph[0] << ", pI: " << ph[1] << std::endl;
        }
    }
}

void EasyVertexHalfPlaneTrajectoryFilter::Impl::filter_toi(FilterTOIInfo& info)
{
    using namespace muda;

    info.toi().fill(1.1f);
    tois.resize(info.surf_vertices().size());

    // TODO: just hard code the slackness for now
    constexpr Float eta = 0.1;
    //constexpr Float slackness     = 0.8;
    //constexpr Float inv_slackness = 1.0 / slackness;

    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.surf_vertices().size(),
               [surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                positions = info.positions().viewer().name("positions"),
                displacements = info.displacements().viewer().name("displacements"),
                half_plane_positions = info.plane_positions().viewer().name("plane_positions"),
                half_plane_normals = info.plane_normals().viewer().name("plane_normals"),
                tois  = tois.viewer().name("tois"),
                alpha = info.alpha(),
                d_hat = info.d_hat(),
                eta] __device__(int i) mutable
               {
                   IndexT  vI  = surf_vertices(i);
                   Vector3 x   = positions(vI);
                   Vector3 dx  = displacements(vI) * alpha;
                   Vector3 x_t = x + dx;

                   Float min_toi = 1.1f;

                   for(int j = 0; j < half_plane_positions.total_size(); ++j)
                   {
                       Vector3 P = half_plane_positions(j);
                       Vector3 N = half_plane_normals(j);

                       Float t = N.dot(dx);
                       if(t >= 0)  // moving away from the plane, no collision
                           continue;

                       // t < 0, moving towards the plane

                       Vector3 diff = P - x;
                       Float t0 = N.dot(diff) * (1.0 - eta);  // gap should be larger than (eta * t0)

                       Float this_toi = t0 / t;

                       MUDA_ASSERT(this_toi > 0, "this_toi=%f, why?", this_toi);

                       min_toi = std::min(min_toi, this_toi);

                       if constexpr(PrintDebugInfo)
                       {
                           if(this_toi < 1.0)
                           {
                               cout << "vI: " << vI << ", pI: " << j
                                    << ", toi: " << this_toi << " d0: " << -t0 << "\n";
                           }
                       }
                   }

                   tois(i) = min_toi;
               });

    DeviceReduce().Min(tois.data(), info.toi().data(), info.surf_vertices().size());
}
}  // namespace uipc::backend::cuda
