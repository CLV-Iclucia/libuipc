#include "hip/hip_runtime.h"
#include <collision_detection/lbvh_simplex_dcd_filter.h>
#include <muda/cub/device/device_select.h>
#include <muda/ext/eigen/log_proxy.h>
#include <sim_engine.h>
#include <kernel_cout.h>
#include <muda/ext/geo/distance/distance_type.h>
#include <muda/ext/geo/distance.h>

namespace uipc::backend::cuda
{
template <>
class SimSystemCreator<LBVHSimplexDCDFilter>
{
  public:
    static U<LBVHSimplexDCDFilter> create(SimEngine& engine)
    {
        auto& info = engine.world().scene().info();

        return info["contact"]["enable"].get<bool>() ?
                   make_unique<LBVHSimplexDCDFilter>(engine) :
                   nullptr;
    }
};

REGISTER_SIM_SYSTEM(LBVHSimplexDCDFilter);

void LBVHSimplexDCDFilter::do_detect(SimplexDCDFilter::FilterInfo& info)
{
    m_impl.detect(info);
    m_impl.classify(info);
}

void LBVHSimplexDCDFilter::Impl::detect(SimplexDCDFilter::FilterInfo& info)
{
    using namespace muda;

    auto d_hat = info.d_hat();
    auto Ps    = info.positions();
    auto Vs    = info.surf_vertices();
    auto Es    = info.surf_edges();
    auto Fs    = info.surf_triangles();

    point_aabbs.resize(Vs.size());
    triangle_aabbs.resize(Fs.size());
    edge_aabbs.resize(Es.size());


    // build AABBs for points
    ParallelFor()
        .kernel_name(__FUNCTION__ "-points")
        .apply(Vs.size(),
               [Vs    = Vs.viewer().name("V"),
                Ps    = Ps.viewer().name("Ps"),
                aabbs = point_aabbs.viewer().name("aabbs"),
                d_hat = d_hat] __device__(int i) mutable
               {
                   auto        vI  = Vs(i);
                   const auto& pos = Ps(vI);

                   AABB aabb;
                   aabb.extend(pos);

                   aabb.min().array() -= d_hat;
                   aabb.max().array() += d_hat;
                   aabbs(i) = aabb;
               });

    // build AABBs for edges
    ParallelFor()
        .kernel_name(__FUNCTION__ "-edges")
        .apply(Es.size(),
               [Es    = Es.viewer().name("E"),
                Ps    = Ps.viewer().name("Ps"),
                aabbs = edge_aabbs.viewer().name("aabbs"),
                d_hat = d_hat] __device__(int i) mutable
               {
                   auto        eI   = Es(i);
                   const auto& pos0 = Ps(eI[0]);
                   const auto& pos1 = Ps(eI[1]);

                   AABB aabb;
                   aabb.extend(pos0).extend(pos1);
                   aabb.min().array() -= d_hat;
                   aabb.max().array() += d_hat;
                   aabbs(i) = aabb;
               });

    // build AABBs for triangles
    ParallelFor()
        .kernel_name(__FUNCTION__ "-triangles")
        .apply(Fs.size(),
               [Fs    = Fs.viewer().name("F"),
                Ps    = Ps.viewer().name("Ps"),
                aabbs = triangle_aabbs.viewer().name("aabbs"),
                d_hat = d_hat] __device__(int i) mutable
               {
                   auto        fI   = Fs(i);
                   const auto& pos0 = Ps(fI[0]);
                   const auto& pos1 = Ps(fI[1]);
                   const auto& pos2 = Ps(fI[2]);

                   AABB aabb;
                   aabb.extend(pos0).extend(pos1).extend(pos2);

                   aabb.min().array() -= d_hat;
                   aabb.max().array() += d_hat;
                   aabbs(i) = aabb;
               });

    // query PT
    lbvh_PT.build(triangle_aabbs);
    auto PT_pairs = lbvh_PT.query(point_aabbs,
                                  [Vs    = Vs.viewer().name("Vs"),
                                   Fs    = Fs.viewer().name("Fs"),
                                   Ps    = Ps.viewer().name("Ps"),
                                   d_hat = d_hat] __device__(IndexT i, IndexT j)
                                  {
                                      // discard if the point is on the triangle
                                      auto V = Vs(i);
                                      auto F = Fs(j);

                                      if(F[0] == V || F[1] == V || F[2] == V)
                                          return false;


                                      Vector3 VP  = Ps(V);
                                      Vector3 FP0 = Ps(F[0]);
                                      Vector3 FP1 = Ps(F[1]);
                                      Vector3 FP2 = Ps(F[2]);
                                      Float   d;
                                      muda::distance::point_triangle_distance_unclassified(
                                          VP, FP0, FP1, FP2, d);

                                      if(d > d_hat * d_hat)
                                          return false;

                                      return true;
                                  });

    // query EE
    lbvh_EE.build(edge_aabbs);
    auto EE_pairs = lbvh_EE.detect(
        [Es    = Es.viewer().name("Es"),
         Ps    = Ps.viewer().name("Ps"),
         d_hat = d_hat] __device__(IndexT i, IndexT j)
        {
            // discard if the edges shared same vertex
            auto E0 = Es(i);
            auto E1 = Es(j);

            if(E0[0] == E1[0] || E0[0] == E1[1] || E0[1] == E1[0] || E0[1] == E1[1])
                return false;

            Vector3 EP0 = Ps(E0[0]);
            Vector3 EP1 = Ps(E0[1]);
            Vector3 EP2 = Ps(E1[0]);
            Vector3 EP3 = Ps(E1[1]);
            Float   d;
            muda::distance::edge_edge_distance_unclassified(EP0, EP1, EP2, EP3, d);

            if(d > d_hat * d_hat)
                return false;

            return true;
        });

    candidate_PTs.resize(PT_pairs.size());
    candidate_EEs.resize(EE_pairs.size());

    // record the candidate pairs
    ParallelFor()
        .kernel_name(__FUNCTION__ "-record PT pairs")
        .apply(PT_pairs.size(),
               [PT_pairs      = PT_pairs.viewer().name("PT_pairs"),
                candidate_PTs = candidate_PTs.viewer().name("candidate_PTs"),
                Fs            = Fs.viewer().name("Fs"),
                Vs = Vs.viewer().name("Vs")] __device__(int i) mutable
               {
                   auto& PT         = candidate_PTs(i);
                   auto  pair       = PT_pairs(i);
                   PT[0]            = Vs(pair[0]);
                   PT.segment<3>(1) = Fs(pair[1]);
               });

    ParallelFor()
        .kernel_name(__FUNCTION__ "-record EE pairs")
        .apply(EE_pairs.size(),
               [EE_pairs      = EE_pairs.viewer().name("EE_pairs"),
                candidate_EEs = candidate_EEs.viewer().name("candidate_EEs"),
                Es = Es.viewer().name("Es")] __device__(int i) mutable
               {
                   auto& EE         = candidate_EEs(i);
                   auto  pair       = EE_pairs(i);
                   EE.segment<2>(0) = Es(pair[0]);
                   EE.segment<2>(2) = Es(pair[1]);
               });

    {
        std::vector<Vector4i> candidate_PTs_host;
        std::vector<Vector4i> candidate_EEs_host;

        candidate_PTs.copy_to(candidate_PTs_host);
        candidate_EEs.copy_to(candidate_EEs_host);

        // print the candidate pairs
        for(auto& PT : candidate_PTs_host)
        {
            std::cout << "PT: " << PT.transpose() << std::endl;
        }

        for(auto& EE : candidate_EEs_host)
        {
            std::cout << "EE: " << EE.transpose() << std::endl;
        }
    }
}

void LBVHSimplexDCDFilter::Impl::classify(SimplexDCDFilter::FilterInfo& info)
{
    using namespace muda;
    auto d_hat = info.d_hat();
    auto Ps    = info.positions();

    temp_PTs.resize(candidate_PTs.size(), Vector4i::Ones() * -1);
    PTs.resize(candidate_PTs.size());
    temp_EEs.resize(candidate_EEs.size(), Vector4i::Ones() * -1);
    EEs.resize(candidate_EEs.size());

    // PE:
    SizeT max_PE_count = candidate_PTs.size() + candidate_EEs.size();
    temp_PEs.resize(max_PE_count, Vector3i::Ones() * -1);
    PEs.resize(max_PE_count);
    // PP:
    SizeT max_PP_count = candidate_PTs.size() + candidate_EEs.size();
    temp_PPs.resize(max_PP_count, Vector2i::Ones() * -1);
    PPs.resize(max_PP_count);  // reserve enough space for PTs


    auto PE_offset = 0;
    auto PE_count  = candidate_PTs.size();
    auto PP_offset = 0;
    auto PP_count  = candidate_PTs.size();

    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(candidate_PTs.size(),
               [Ps            = Ps.viewer().name("Ps"),
                candidate_PTs = candidate_PTs.viewer().name("candidate_PTs"),
                PTs           = temp_PTs.viewer().name("PTs"),
                PEs = temp_PEs.view(PE_offset, PE_count).viewer().name("PEs"),
                PPs = temp_PPs.view(PP_offset, PP_count).viewer().name("PPs")] __device__(int i) mutable
               {
                   Vector4i PT = candidate_PTs(i);
                   IndexT   P  = PT(0);
                   Vector3i T  = PT.tail<3>();


                   auto V  = Ps(PT(0));
                   auto F0 = Ps(PT(1));
                   auto F1 = Ps(PT(2));
                   auto F2 = Ps(PT(3));

                   auto dist_type =
                       muda::distance::point_triangle_distance_type(V, F0, F1, F2);

                   switch(dist_type)
                   {
                       case muda::distance::PointTriangleDistanceType::PP_PT0:
                           PPs(i) = Vector2i(P, T(0));
                           break;
                       case muda::distance::PointTriangleDistanceType::PP_PT1:
                           PPs(i) = Vector2i(P, T(1));
                           break;
                       case muda::distance::PointTriangleDistanceType::PP_PT2:
                           PPs(i) = Vector2i(P, T(2));
                           break;
                       case muda::distance::PointTriangleDistanceType::PE_PT0T1:
                           PEs(i) = Vector3i(P, T(0), T(1));
                           break;
                       case muda::distance::PointTriangleDistanceType::PE_PT1T2:
                           PEs(i) = Vector3i(P, T(1), T(2));
                           break;
                       case muda::distance::PointTriangleDistanceType::PE_PT2T0:
                           PEs(i) = Vector3i(P, T(2), T(0));
                           break;
                       case muda::distance::PointTriangleDistanceType::PT:
                           PTs(i) = PT;
                           break;
                       default:
                           break;
                   }
               });

    PE_offset = PE_count;
    PE_count  = candidate_EEs.size();
    PP_offset = PP_count;
    PP_count  = candidate_EEs.size();

    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(candidate_EEs.size(),
               [Ps            = Ps.viewer().name("Ps"),
                candidate_EEs = candidate_EEs.viewer().name("candidate_EEs"),
                EEs           = temp_EEs.viewer().name("EEs"),
                PEs = temp_PEs.view(PE_offset, PE_count).viewer().name("PEs"),
                PPs = temp_PPs.view(PP_offset, PP_count).viewer().name("PPs")] __device__(int i) mutable
               {
                   Vector4i EE = candidate_EEs(i);

                   IndexT Ea0 = EE(0);
                   IndexT Ea1 = EE(1);
                   IndexT Eb0 = EE(2);
                   IndexT Eb1 = EE(3);

                   Vector3 E0 = Ps(Ea0);
                   Vector3 E1 = Ps(Ea1);
                   Vector3 E2 = Ps(Eb0);
                   Vector3 E3 = Ps(Eb1);

                   auto dist_type = distance::edge_edge_distance_type(E0, E1, E2, E3);

                   switch(dist_type)
                   {
                       case muda::distance::EdgeEdgeDistanceType::PP_Ea0Eb0:
                           PPs(i) = Vector2i(Ea0, Eb0);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::PP_Ea0Eb1:
                           PPs(i) = Vector2i(Ea0, Eb1);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::PE_Ea0Eb0Eb1:
                           PEs(i) = Vector3i(Ea0, Eb0, Eb1);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::PP_Ea1Eb0:
                           PPs(i) = Vector2i(Ea1, Eb0);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::PP_Ea1Eb1:
                           PPs(i) = Vector2i(Ea1, Eb1);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::PE_Ea1Eb0Eb1:
                           PEs(i) = Vector3i(Ea1, Eb0, Eb1);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::PE_Eb0Ea0Ea1:
                           PEs(i) = Vector3i(Eb0, Ea0, Ea1);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::PE_Eb1Ea0Ea1:
                           PEs(i) = Vector3i(Eb1, Ea0, Ea1);
                           break;
                       case muda::distance::EdgeEdgeDistanceType::EE:
                           EEs(i) = EE;
                           break;
                       default:
                           break;
                   }
               });

    DeviceSelect().If(temp_PTs.data(),
                      PTs.data(),
                      selected.data(),
                      temp_PTs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector4i& pt)
                      { return pt(0) != -1; });

    int h_selected = selected;
    PTs.resize(h_selected);

    DeviceSelect().If(temp_EEs.data(),
                      EEs.data(),
                      selected.data(),
                      temp_EEs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector4i& ee)
                      { return ee(0) != -1; });

    h_selected = selected;
    EEs.resize(h_selected);

    DeviceSelect().If(temp_PEs.data(),
                      PEs.data(),
                      selected.data(),
                      temp_PEs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector3i& pe)
                      { return pe(0) != -1; });

    h_selected = selected;
    PEs.resize(h_selected);

    DeviceSelect().If(temp_PPs.data(),
                      PPs.data(),
                      selected.data(),
                      temp_PPs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector2i& pp)
                      { return pp(0) != -1; });

    h_selected = selected;
    PPs.resize(h_selected);

    info.PTs(PTs);
    info.EEs(EEs);
    info.PEs(PEs);
    info.PPs(PPs);
}
}  // namespace uipc::backend::cuda
