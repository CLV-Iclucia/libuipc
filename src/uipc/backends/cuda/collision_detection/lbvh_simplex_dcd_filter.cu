#include "hip/hip_runtime.h"
#include <collision_detection/lbvh_simplex_dcd_filter.h>
#include <muda/cub/device/device_select.h>
#include <muda/ext/eigen/log_proxy.h>
#include <sim_engine.h>
#include <kernel_cout.h>
#include <muda/ext/geo/distance/distance_type.h>
#include <muda/ext/geo/distance.h>

namespace uipc::backend::cuda
{
constexpr bool PrintDebugInfo = false;

REGISTER_SIM_SYSTEM(LBVHSimplexDCDFilter);

void LBVHSimplexDCDFilter::do_detect(SimplexDCDFilter::FilterInfo& info)
{
    m_impl.detect(info);
    m_impl.classify(info);
}

void LBVHSimplexDCDFilter::Impl::detect(SimplexDCDFilter::FilterInfo& info)
{
    using namespace muda;

    auto d_hat = info.d_hat();
    auto Ps    = info.positions();
    auto Vs    = info.surf_vertices();
    auto Es    = info.surf_edges();
    auto Fs    = info.surf_triangles();

    point_aabbs.resize(Vs.size());
    triangle_aabbs.resize(Fs.size());
    edge_aabbs.resize(Es.size());


    // build AABBs for points
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(Vs.size(),
               [Vs    = Vs.viewer().name("V"),
                Ps    = Ps.viewer().name("Ps"),
                aabbs = point_aabbs.viewer().name("aabbs"),
                d_hat = d_hat] __device__(int i) mutable
               {
                   auto        vI  = Vs(i);
                   const auto& pos = Ps(vI);

                   AABB aabb;
                   aabb.extend(pos);

                   aabb.min().array() -= d_hat;
                   aabb.max().array() += d_hat;
                   aabbs(i) = aabb;
               });

    // build AABBs for edges
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(Es.size(),
               [Es    = Es.viewer().name("E"),
                Ps    = Ps.viewer().name("Ps"),
                aabbs = edge_aabbs.viewer().name("aabbs"),
                d_hat = d_hat] __device__(int i) mutable
               {
                   const auto& eI   = Es(i);
                   const auto& pos0 = Ps(eI[0]);
                   const auto& pos1 = Ps(eI[1]);

                   AABB aabb;
                   aabb.extend(pos0).extend(pos1);
                   aabb.min().array() -= d_hat;
                   aabb.max().array() += d_hat;
                   aabbs(i) = aabb;
               });

    // build AABBs for triangles
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(Fs.size(),
               [Fs    = Fs.viewer().name("F"),
                Ps    = Ps.viewer().name("Ps"),
                aabbs = triangle_aabbs.viewer().name("aabbs"),
                d_hat = d_hat] __device__(int i) mutable
               {
                   const auto& fI   = Fs(i);
                   const auto& pos0 = Ps(fI[0]);
                   const auto& pos1 = Ps(fI[1]);
                   const auto& pos2 = Ps(fI[2]);

                   AABB aabb;
                   aabb.extend(pos0).extend(pos1).extend(pos2);

                   aabb.min().array() -= d_hat;
                   aabb.max().array() += d_hat;
                   aabbs(i) = aabb;
               });

    // query PT
    lbvh_PT.build(triangle_aabbs);
    auto PT_pairs = lbvh_PT.query(
        point_aabbs,
        [Vs    = Vs.viewer().name("Vs"),
         Fs    = Fs.viewer().name("Fs"),
         Ps    = Ps.viewer().name("Ps"),
         d_hat = d_hat] __device__(IndexT i, IndexT j)
        {
            // discard if the point is on the triangle
            auto        V = Vs(i);
            const auto& F = Fs(j);

            if(F[0] == V || F[1] == V || F[2] == V)
                return false;


            Vector3 VP  = Ps(V);
            Vector3 FP0 = Ps(F[0]);
            Vector3 FP1 = Ps(F[1]);
            Vector3 FP2 = Ps(F[2]);
            Float   D;
            muda::distance::point_triangle_distance_unclassified(VP, FP0, FP1, FP2, D);

            if(D >= d_hat * d_hat)
                return false;

            //cout << "PT: " << V << " " << F.transpose().eval()
            //     << " d: " << sqrt(D) << "d_hat: " << d_hat << "\n";

            return true;
        });

    // query EE
    lbvh_EE.build(edge_aabbs);
    auto EE_pairs = lbvh_EE.detect(
        [Es    = Es.viewer().name("Es"),
         Ps    = Ps.viewer().name("Ps"),
         d_hat = d_hat] __device__(IndexT i, IndexT j)
        {
            // discard if the edges shared same vertex
            const auto& E0 = Es(i);
            const auto& E1 = Es(j);

            if(E0[0] == E1[0] || E0[0] == E1[1] || E0[1] == E1[0] || E0[1] == E1[1])
                return false;

            Vector3 EP0 = Ps(E0[0]);
            Vector3 EP1 = Ps(E0[1]);
            Vector3 EP2 = Ps(E1[0]);
            Vector3 EP3 = Ps(E1[1]);
            Float   D;
            muda::distance::edge_edge_distance_unclassified(EP0, EP1, EP2, EP3, D);

            if(D >= d_hat * d_hat)
                return false;

            //cout << "EE: " << E0.transpose().eval() << " " << E1.transpose().eval()
            //     << " d: " << sqrt(D) << "d_hat: " << d_hat << "\n";

            return true;
        });

    candidate_PTs.resize(PT_pairs.size());
    candidate_EEs.resize(EE_pairs.size());

    // record the candidate pairs
    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(PT_pairs.size(),
               [PT_pairs      = PT_pairs.viewer().name("PT_pairs"),
                candidate_PTs = candidate_PTs.viewer().name("candidate_PTs"),
                Fs            = Fs.viewer().name("Fs"),
                Vs = Vs.viewer().name("Vs")] __device__(int i) mutable
               {
                   auto&       PT   = candidate_PTs(i);
                   const auto& pair = PT_pairs(i);
                   PT[0]            = Vs(pair[0]);
                   PT.segment<3>(1) = Fs(pair[1]);
               });

    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(EE_pairs.size(),
               [EE_pairs      = EE_pairs.viewer().name("EE_pairs"),
                candidate_EEs = candidate_EEs.viewer().name("candidate_EEs"),
                Es = Es.viewer().name("Es")] __device__(int i) mutable
               {
                   auto&       EE   = candidate_EEs(i);
                   const auto& pair = EE_pairs(i);
                   EE.segment<2>(0) = Es(pair[0]);
                   EE.segment<2>(2) = Es(pair[1]);
               });

    if constexpr(PrintDebugInfo)
    {
        std::vector<Vector4i> candidate_PTs_host;
        std::vector<Vector4i> candidate_EEs_host;

        candidate_PTs.copy_to(candidate_PTs_host);
        candidate_EEs.copy_to(candidate_EEs_host);

        // print the candidate pairs
        for(auto& PT : candidate_PTs_host)
        {
            std::cout << "PT: " << PT.transpose() << std::endl;
        }

        for(auto& EE : candidate_EEs_host)
        {
            std::cout << "EE: " << EE.transpose() << std::endl;
        }
    }
}

void LBVHSimplexDCDFilter::Impl::classify(SimplexDCDFilter::FilterInfo& info)
{
    using namespace muda;


    if constexpr(PrintDebugInfo)
    {
        std::vector<Vector4i> candidate_PTs_host;
        std::vector<Vector4i> candidate_EEs_host;

        candidate_PTs.copy_to(candidate_PTs_host);
        candidate_EEs.copy_to(candidate_EEs_host);

        // print the candidate pairs
        std::cout << "candidate pairs:" << std::endl;
        for(auto& PT : candidate_PTs_host)
        {
            std::cout << "PT: " << PT.transpose() << std::endl;
        }

        for(auto& EE : candidate_EEs_host)
        {
            std::cout << "EE: " << EE.transpose() << std::endl;
        }
    }

    auto d_hat = info.d_hat();
    auto Ps    = info.positions();

    // we will classify the PT EE pairs into PTs, EEs, PEs, PPs

    // PT: point-triangle, only 1 possible PT per candidate PT
    temp_PTs.resize(candidate_PTs.size());
    temp_PTs.fill(Vector4i::Ones() * -1);  // fill -1 to indicate invalid PT
    PTs.resize(candidate_PTs.size());

    // EE: edge-edge, only 1 possible EE per candidate EE
    temp_EEs.resize(candidate_EEs.size());
    temp_EEs.fill(Vector4i::Ones() * -1);  // fill -1 to indicate invalid EE
    EEs.resize(candidate_EEs.size());


    // PE: point-edge
    // 3 possible PE per candidate PT
    SizeT PT_to_PE_max_count = candidate_PTs.size() * 3;
    // 4 possible PE per candidate EE
    SizeT EE_to_PE_max_count = candidate_EEs.size() * 4;

    SizeT max_PE_count = PT_to_PE_max_count + EE_to_PE_max_count;
    temp_PEs.resize(max_PE_count);
    temp_PEs.fill(Vector3i::Ones() * -1);  // fill -1 to indicate invalid PE
    PEs.resize(max_PE_count);

    // PP: point-point
    // 3 possible PP per candidate PT
    SizeT PT_to_PP_max_count = candidate_PTs.size() * 3;
    // 4 possible PP per candidate EE
    SizeT EE_to_PP_max_count = candidate_EEs.size() * 4;

    SizeT max_PP_count = PT_to_PP_max_count + EE_to_PP_max_count;
    temp_PPs.resize(max_PP_count);
    temp_PPs.fill(Vector2i::Ones() * -1);  // fill -1 to indicate invalid PP
    PPs.resize(max_PP_count);


    // always use the squared distance to avoid numerical issue
    Float D_hat = d_hat * d_hat;


    // 1) PT-> PT, 3PE, 3PP
    auto PE_offset = 0;
    auto PE_count  = PT_to_PE_max_count;
    auto PP_offset = 0;
    auto PP_count  = PT_to_PP_max_count;

    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(candidate_PTs.size(),
               [Ps            = Ps.viewer().name("Ps"),
                candidate_PTs = candidate_PTs.viewer().name("candidate_PTs"),
                PTs           = temp_PTs.viewer().name("PTs"),
                PEs   = temp_PEs.view(PE_offset, PE_count).viewer().name("PEs"),
                PPs   = temp_PPs.view(PP_offset, PP_count).viewer().name("PPs"),
                D_hat = D_hat] __device__(int i) mutable
               {
                   Vector4i PT = candidate_PTs(i);
                   IndexT   P  = PT(0);
                   Vector3i T  = PT.tail<3>();


                   const auto& V  = Ps(PT(0));
                   const auto& F0 = Ps(PT(1));
                   const auto& F1 = Ps(PT(2));
                   const auto& F2 = Ps(PT(3));

                   auto dist_type =
                       muda::distance::point_triangle_distance_type(V, F0, F1, F2);

                   if(dist_type == distance::PointTriangleDistanceType::PT)
                   {
                       if constexpr(PrintDebugInfo)
                       {
                           cout << "PT->PT:" << PT.transpose().eval() << "\n";
                       }

                       if constexpr(muda::RUNTIME_CHECK_ON)
                       {
                           Float D;
                           distance::point_triangle_distance(V, F0, F1, F2, D);
                           MUDA_ASSERT(D < D_hat,
                                       "[%d,%d,%d,%d] D(%f) < D_hat=(%f)",
                                       P,
                                       T(0),
                                       T(1),
                                       T(2),
                                       D,
                                       D_hat);
                       }

                       PTs(i) = PT;

                       return;
                   }

                   // if not, then it can be PT->PE or PT->PP

                   // 3 possible PE
                   Vector3i PE[3] = {{P, T(0), T(1)}, {P, T(1), T(2)}, {P, T(2), T(0)}};

                   for(int j = 0; j < 3; ++j)
                   {
                       auto& pe = PE[j];
                       auto  E0 = Ps(pe(0));
                       auto  E1 = Ps(pe(1));
                       auto  E2 = Ps(pe(2));

                       auto dist_type = distance::point_edge_distance_type(V, E0, E1);

                       if(dist_type == distance::PointEdgeDistanceType::PE)
                       {
                           Float D;
                           distance::point_edge_distance(V, E0, E1, D);

                           if(D < D_hat)
                           {
                               if constexpr(PrintDebugInfo)
                               {
                                   cout << "PT->PE:" << PT.transpose().eval()
                                        << "->" << P << "," << T(j) << "\n";
                               }

                               PEs(3 * i + j) = pe;
                           }
                       }
                   }

                   // 3 possible PP
                   Vector2i PP[3] = {{P, T(0)}, {P, T(1)}, {P, T(2)}};

                   for(int j = 0; j < 3; ++j)
                   {
                       auto& pp = PP[j];
                       auto  P0 = Ps(pp(0));
                       auto  P1 = Ps(pp(1));

                       Float D;
                       distance::point_point_distance(P0, P1, D);

                       if(D < D_hat)
                       {
                           PPs(3 * i + j) = pp;
                       }
                   }
               });

    // 2) EE-> PT, 4PE, 4PP
    PE_offset += PE_count;
    PE_count = EE_to_PE_max_count;
    PP_offset += PP_count;
    PP_count = EE_to_PP_max_count;

    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(candidate_EEs.size(),
               [Ps            = Ps.viewer().name("Ps"),
                candidate_EEs = candidate_EEs.viewer().name("candidate_EEs"),
                EEs           = temp_EEs.viewer().name("EEs"),
                PEs   = temp_PEs.view(PE_offset, PE_count).viewer().name("PEs"),
                PPs   = temp_PPs.view(PP_offset, PP_count).viewer().name("PPs"),
                D_hat = D_hat] __device__(int i) mutable
               {
                   Vector4i EE = candidate_EEs(i);

                   IndexT Ea0 = EE(0);
                   IndexT Ea1 = EE(1);
                   IndexT Eb0 = EE(2);
                   IndexT Eb1 = EE(3);

                   Vector3 E0 = Ps(Ea0);
                   Vector3 E1 = Ps(Ea1);
                   Vector3 E2 = Ps(Eb0);
                   Vector3 E3 = Ps(Eb1);

                   auto dist_type = distance::edge_edge_distance_type(E0, E1, E2, E3);

                   if(dist_type == distance::EdgeEdgeDistanceType::EE)
                   {
                       if constexpr(PrintDebugInfo)
                       {
                           cout << "EE->EE:" << EE.transpose().eval() << "\n";
                       }

                       if constexpr(muda::RUNTIME_CHECK_ON)
                       {
                           Float D;
                           distance::edge_edge_distance(E0, E1, E2, E3, D);
                           MUDA_ASSERT(
                               D < D_hat, "[%d,%d,%d,%d] D(%f) < D_hat=(%f)", Ea0, Ea1, Eb0, Eb1, D, D_hat);
                       }

                       EEs(i) = EE;

                       return;
                   }

                   // if not, then it can be EE->PE or EE->PP

                   // 4 possible PE
                   Vector3i PE[4] = {{Ea0, Eb0, Eb1},
                                     {Ea1, Eb0, Eb1},
                                     {Eb0, Ea0, Ea1},
                                     {Eb1, Ea0, Ea1}};

                   for(int j = 0; j < 4; ++j)
                   {
                       auto& pe = PE[j];
                       auto  E0 = Ps(pe(0));
                       auto  E1 = Ps(pe(1));
                       auto  E2 = Ps(pe(2));

                       auto dist_type = distance::point_edge_distance_type(E0, E1, E2);

                       if(dist_type == distance::PointEdgeDistanceType::PE)
                       {
                           Float D;
                           distance::point_edge_distance(E0, E1, E2, D);

                           if(D < D_hat)
                           {
                               if constexpr(PrintDebugInfo)
                               {
                                   cout << "EE->PE:" << EE.transpose().eval()
                                        << "->" << pe.transpose().eval() << "\n";
                               }

                               PEs(4 * i + j) = pe;
                           }
                       }
                   }

                   // 4 possible PP
                   Vector2i PP[4] = {{Ea0, Eb0}, {Ea0, Eb1}, {Ea1, Eb0}, {Ea1, Eb1}};

                   for(int j = 0; j < 4; ++j)
                   {
                       auto& pp = PP[j];
                       auto  P0 = Ps(pp(0));
                       auto  P1 = Ps(pp(1));

                       Float D;
                       distance::point_point_distance(P0, P1, D);

                       if(D < D_hat)
                       {

                           if constexpr(PrintDebugInfo)
                           {
                               cout << "EE->PP:" << EE.transpose().eval()
                                    << "->" << pp.transpose().eval() << "\n";
                           }

                           PPs(4 * i + j) = pp;
                       }
                   }
               });

    DeviceSelect().If(temp_PTs.data(),
                      PTs.data(),
                      selected.data(),
                      temp_PTs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector4i& pt)
                      { return pt(0) != -1; });

    int h_selected = selected;
    PTs.resize(h_selected);

    DeviceSelect().If(temp_EEs.data(),
                      EEs.data(),
                      selected.data(),
                      temp_EEs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector4i& ee)
                      { return ee(0) != -1; });

    h_selected = selected;
    EEs.resize(h_selected);

    DeviceSelect().If(temp_PEs.data(),
                      PEs.data(),
                      selected.data(),
                      temp_PEs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector3i& pe)
                      { return pe(0) != -1; });

    h_selected = selected;
    PEs.resize(h_selected);

    DeviceSelect().If(temp_PPs.data(),
                      PPs.data(),
                      selected.data(),
                      temp_PPs.size(),
                      [] HIPCUB_RUNTIME_FUNCTION(const Vector2i& pp)
                      { return pp(0) != -1; });

    h_selected = selected;
    PPs.resize(h_selected);

    info.PTs(PTs);
    info.EEs(EEs);
    info.PEs(PEs);
    info.PPs(PPs);


    if constexpr(PrintDebugInfo)
    {
        std::vector<Vector4i> PTs_host;
        std::vector<Vector4i> EEs_host;
        std::vector<Vector3i> PEs_host;
        std::vector<Vector2i> PPs_host;

        PTs.copy_to(PTs_host);
        EEs.copy_to(EEs_host);
        PEs.copy_to(PEs_host);
        PPs.copy_to(PPs_host);

        std::cout << "classify result:" << std::endl;

        for(auto& PT : PTs_host)
        {
            std::cout << "PT: " << PT.transpose() << std::endl;
        }

        for(auto& EE : EEs_host)
        {
            std::cout << "EE: " << EE.transpose() << std::endl;
        }

        for(auto& PE : PEs_host)
        {
            std::cout << "PE: " << PE.transpose() << std::endl;
        }

        for(auto& PP : PPs_host)
        {
            std::cout << "PP: " << PP.transpose() << std::endl;
        }
    }
}
}  // namespace uipc::backend::cuda
