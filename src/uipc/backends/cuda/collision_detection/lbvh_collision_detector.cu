#include "hip/hip_runtime.h"
#include <collision_detection/lbvh_collision_detector.h>
#include <collision_detection/global_collision_detector.h>
#include <muda/ext/geo/distance.h>
#include <sim_engine.h>
#include <kernel_cout.h>
#include <muda/ext/eigen/log_proxy.h>
namespace uipc::backend::cuda
{
template <>
class SimSystemCreator<LBVHCollisionDetector>
{
  public:
    static U<LBVHCollisionDetector> create(SimEngine& engine)
    {
        auto& info = engine.world().scene().info();

        return info["contact"]["enable"].get<bool>() ?
                   make_unique<LBVHCollisionDetector>(engine) :
                   nullptr;
    }
};

REGISTER_SIM_SYSTEM(LBVHCollisionDetector);

void LBVHCollisionDetector::do_build()
{
    m_impl.global_vertex_manager  = find<GlobalVertexManager>();
    m_impl.global_surface_manager = find<GlobalSurfaceManager>();
    m_impl.global_contact_manager = find<GlobalContactManager>();

    auto global_collision_detector = find<GlobalCollisionDetector>();

    // add the detector to the global collision detector
    global_collision_detector->add_detector(
        [this](GlobalCollisionDetector::DetectCandidateInfo& info)
        { m_impl.detect_candidates(info); });
}


void LBVHCollisionDetector::Impl::detect_candidates(GlobalCollisionDetector::DetectCandidateInfo& info)
{
    using namespace muda;
    auto alpha = info.alpha();
    auto d_hat = global_contact_manager->d_hat();
    auto Ps    = global_vertex_manager->positions();
    auto dxs   = global_vertex_manager->displacements();
    auto Vs    = global_surface_manager->surf_vertices();
    auto Es    = global_surface_manager->surf_edges();
    auto Fs    = global_surface_manager->surf_triangles();

    spdlog::info("d_hat:{}, alpha:{}", d_hat, alpha);

    point_aabbs.resize(Vs.size());
    triangle_aabbs.resize(Fs.size());
    edge_aabbs.resize(Es.size());

    // build AABBs for points
    ParallelFor()
        .kernel_name(__FUNCTION__ "-points")
        .apply(Vs.size(),
               [Vs    = Vs.viewer().name("V"),
                dxs   = dxs.viewer().name("dx"),
                Ps    = Ps.viewer().name("Ps"),
                aabbs = point_aabbs.viewer().name("aabbs"),
                alpha = alpha,
                d_hat = d_hat] __device__(int i) mutable
               {
                   auto        vI    = Vs(i);
                   const auto& pos   = Ps(vI);
                   Vector3     pos_t = pos + dxs(vI) * alpha;

                   Vector3 max = pos_t;
                   Vector3 min = pos_t;

                   max = max.cwiseMax(pos);
                   min = min.cwiseMin(pos);

                   AABB aabb{min, max};

                   aabb.min().array() -= d_hat;
                   aabb.max().array() += d_hat;
                   aabbs(i) = aabb;
               });

    // build AABBs for edges
    ParallelFor()
        .kernel_name(__FUNCTION__ "-edges")
        .apply(Es.size(),
               [Es    = Es.viewer().name("E"),
                Ps    = Ps.viewer().name("Ps"),
                aabbs = edge_aabbs.viewer().name("aabbs"),
                dxs   = dxs.viewer().name("dx"),
                alpha = alpha,
                d_hat = d_hat] __device__(int i) mutable
               {
                   auto        eI     = Es(i);
                   const auto& pos0   = Ps(eI[0]);
                   const auto& pos1   = Ps(eI[1]);
                   Vector3     pos0_t = pos0 + dxs(eI[0]) * alpha;
                   Vector3     pos1_t = pos1 + dxs(eI[1]) * alpha;

                   Vector3 max = pos0_t;
                   Vector3 min = pos0_t;

                   max = max.cwiseMax(pos1_t);
                   min = min.cwiseMin(pos1_t);

                   max = max.cwiseMax(pos0);
                   min = min.cwiseMin(pos0);

                   max = max.cwiseMax(pos1);
                   min = min.cwiseMin(pos1);

                   AABB aabb{min, max};

                   aabb.min().array() -= d_hat;
                   aabb.max().array() += d_hat;
                   aabbs(i) = aabb;
               });

    // build AABBs for triangles
    ParallelFor()
        .kernel_name(__FUNCTION__ "-triangles")
        .apply(Fs.size(),
               [Fs    = Fs.viewer().name("F"),
                Ps    = Ps.viewer().name("Ps"),
                aabbs = triangle_aabbs.viewer().name("aabbs"),
                dxs   = dxs.viewer().name("dx"),
                alpha = alpha,
                d_hat = d_hat] __device__(int i) mutable
               {
                   auto        fI     = Fs(i);
                   const auto& pos0   = Ps(fI[0]);
                   const auto& pos1   = Ps(fI[1]);
                   const auto& pos2   = Ps(fI[2]);
                   Vector3     pos0_t = pos0 + dxs(fI[0]) * alpha;
                   Vector3     pos1_t = pos1 + dxs(fI[1]) * alpha;
                   Vector3     pos2_t = pos2 + dxs(fI[2]) * alpha;

                   Vector3 max = pos0_t;
                   Vector3 min = pos0_t;

                   max = max.cwiseMax(pos1_t);
                   min = min.cwiseMin(pos1_t);

                   max = max.cwiseMax(pos2_t);
                   min = min.cwiseMin(pos2_t);

                   max = max.cwiseMax(pos0);
                   min = min.cwiseMin(pos0);

                   max = max.cwiseMax(pos1);
                   min = min.cwiseMin(pos1);

                   max = max.cwiseMax(pos2);
                   min = min.cwiseMin(pos2);

                   AABB aabb{min, max};

                   aabb.min().array() -= d_hat;
                   aabb.max().array() += d_hat;
                   aabbs(i) = aabb;
               });

    // query PT
    lbvh_PT.build(triangle_aabbs);
    auto PT_pairs = lbvh_PT.query(
        point_aabbs,
        [Vs    = Vs.viewer().name("Vs"),
         Fs    = Fs.viewer().name("Fs"),
         Ps    = Ps.viewer().name("Ps"),
         dxs   = dxs.viewer().name("dxs"),
         d_hat = d_hat,
         alpha = alpha] __device__(IndexT i, IndexT j)
        {
            // discard if the point is on the triangle
            auto V = Vs(i);
            auto F = Fs(j);

            if(F[0] == V || F[1] == V || F[2] == V)
            {
                return false;
            }

            if(alpha == 0.0)
            {
                Vector3 VP  = Ps(V);
                Vector3 FP0 = Ps(F[0]);
                Vector3 FP1 = Ps(F[1]);
                Vector3 FP2 = Ps(F[2]);
                Float   d;
                muda::distance::point_triangle_distance_unclassified(VP, FP0, FP1, FP2, d);
                if(d > d_hat * d_hat)
                    return false;
            }
            else
            {
                Vector3 VP  = Ps(V);
                Vector3 dVP = alpha * dxs(V);

                Vector3 FP0 = Ps(F[0]);
                Vector3 FP1 = Ps(F[1]);
                Vector3 FP2 = Ps(F[2]);

                Vector3 dFP0 = alpha * dxs(F[0]);
                Vector3 dFP1 = alpha * dxs(F[1]);
                Vector3 dFP2 = alpha * dxs(F[2]);

                if(!muda::distance::point_triangle_ccd_broadphase(
                       VP, FP0, FP1, FP2, dVP, dFP0, dFP1, dFP2, d_hat))
                    return false;
            }

            return true;
        });

    // query EE
    lbvh_EE.build(edge_aabbs);
    auto EE_pairs = lbvh_EE.detect(
        [Es    = Es.viewer().name("Es"),
         Ps    = Ps.viewer().name("Ps"),
         dxs   = dxs.viewer().name("dxs"),
         d_hat = d_hat,
         alpha = alpha] __device__(IndexT i, IndexT j)
        {
            // discard if the edges shared same vertex
            auto E0 = Es(i);
            auto E1 = Es(j);

            if(E0[0] == E1[0] || E0[0] == E1[1] || E0[1] == E1[0] || E0[1] == E1[1])
            {
                return false;
            }

            if(alpha == 0.0)
            {
                Vector3 EP0 = Ps(E0[0]);
                Vector3 EP1 = Ps(E0[1]);
                Vector3 EP2 = Ps(E1[0]);
                Vector3 EP3 = Ps(E1[1]);
                Float   d;
                muda::distance::edge_edge_distance_unclassified(EP0, EP1, EP2, EP3, d);

                if(d > d_hat * d_hat)
                    return false;
            }
            else
            {
                Vector3 EP0  = Ps(E0[0]);
                Vector3 EP1  = Ps(E0[1]);
                Vector3 dEP0 = alpha * dxs(E0[0]);
                Vector3 dEP1 = alpha * dxs(E0[1]);

                Vector3 EP2  = Ps(E1[0]);
                Vector3 EP3  = Ps(E1[1]);
                Vector3 dEP2 = alpha * dxs(E1[0]);
                Vector3 dEP3 = alpha * dxs(E1[1]);

                if(!muda::distance::edge_edge_ccd_broadphase(
                       EP0, EP1, dEP0, dEP1, EP2, EP3, dEP2, dEP3, d_hat))
                    return false;
            }

            return true;
        });

    candidate_PTs.resize(PT_pairs.size());
    candidate_EEs.resize(EE_pairs.size());

    // record the candidate pairs
    ParallelFor()
        .kernel_name(__FUNCTION__ "-record PT pairs")
        .apply(PT_pairs.size(),
               [PT_pairs      = PT_pairs.viewer().name("PT_pairs"),
                candidate_PTs = candidate_PTs.viewer().name("candidate_PTs"),
                Fs            = Fs.viewer().name("Fs"),
                Vs = Vs.viewer().name("Vs")] __device__(int i) mutable
               {
                   auto& PT         = candidate_PTs(i);
                   auto  pair       = PT_pairs(i);
                   PT[0]            = Vs(pair[0]);
                   PT.segment<3>(1) = Fs(pair[1]);
               });

    ParallelFor()
        .kernel_name(__FUNCTION__ "-record EE pairs")
        .apply(EE_pairs.size(),
               [EE_pairs      = EE_pairs.viewer().name("EE_pairs"),
                candidate_EEs = candidate_EEs.viewer().name("candidate_EEs"),
                Es = Es.viewer().name("Es")] __device__(int i) mutable
               {
                   auto& EE         = candidate_EEs(i);
                   auto  pair       = EE_pairs(i);
                   EE.segment<2>(0) = Es(pair[0]);
                   EE.segment<2>(2) = Es(pair[1]);
               });

    info.candidate_PTs(candidate_PTs);
    info.candidate_EEs(candidate_EEs);

    {
        std::vector<Vector4i> candidate_PTs_host;
        std::vector<Vector4i> candidate_EEs_host;

        candidate_PTs.copy_to(candidate_PTs_host);
        candidate_EEs.copy_to(candidate_EEs_host);

        // print the candidate pairs
        for(auto& PT : candidate_PTs_host)
        {
            std::cout << "PT: " << PT.transpose() << std::endl;
        }

        for(auto& EE : candidate_EEs_host)
        {
            std::cout << "EE: " << EE.transpose() << std::endl;
        }
    }
}
}  // namespace uipc::backend::cuda
