#include "hip/hip_runtime.h"
#include <collision_detection/simple_half_plane_ccd_filter.h>
#include <muda/cub/device/device_reduce.h>
#include <kernel_cout.h>

namespace uipc::backend::cuda
{
constexpr bool PrintDebugInfo = true;

REGISTER_SIM_SYSTEM(SimpleHalfPlaneCCDFilter);

void SimpleHalfPlaneCCDFilter::Impl::filter_toi(FilterInfo& info)
{
    using namespace muda;

    info.toi().fill(1.1f);
    tois.resize(info.surf_vertices().size());

    // TODO: just hard code the slackness for now
    constexpr Float slackness     = 0.8;
    constexpr Float inv_slackness = 1.0 / slackness;

    ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(info.surf_vertices().size(),
               [surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                positions = info.positions().viewer().name("positions"),
                displacements = info.displacements().viewer().name("displacements"),
                half_plane_positions = info.plane_positions().viewer().name("plane_positions"),
                half_plane_normals = info.plane_normals().viewer().name("plane_normals"),
                tois          = tois.viewer().name("tois"),
                alpha         = info.alpha(),
                d_hat         = info.d_hat(),
                slackness     = slackness,
                inv_slackness = inv_slackness] __device__(int i) mutable
               {
                   IndexT  vI       = surf_vertices(i);
                   Vector3 pos      = positions(vI);
                   Vector3 dx       = displacements(vI) * alpha;
                   Vector3 pos_next = pos + dx;

                   Float min_toi = 1.1f;

                   for(int j = 0; j < half_plane_positions.total_size(); ++j)
                   {
                       Vector3 plane_pos    = half_plane_positions(j);
                       Vector3 plane_normal = half_plane_normals(j);

                       Float t = plane_normal.dot(dx);
                       if(t >= 0)  // moving away from the plane, no collision
                           continue;

                       Vector3 diff = plane_pos - pos;
                       Float   t0   = plane_normal.dot(diff);

                       Float this_toi = t0 / t;

                       MUDA_ASSERT(this_toi > 0, "this_toi=%f, why?", this_toi);

                       if(this_toi <= inv_slackness)
                           this_toi *= slackness;

                       min_toi = std::min(min_toi, this_toi);

                       if constexpr(PrintDebugInfo)
                       {
                           if(this_toi < 1.0)
                           {
                               cout << "vI: " << vI << ", pI: " << j << ", toi: " << this_toi
                                    << "slackness: " << slackness << "\n";
                           }
                       }
                   }

                   tois(i) = min_toi;
               });

    DeviceReduce().Min(tois.data(), info.toi().data(), info.surf_vertices().size());
}

void SimpleHalfPlaneCCDFilter::do_filter_toi(FilterInfo& info)
{
    m_impl.filter_toi(info);
}
}  // namespace uipc::backend::cuda
