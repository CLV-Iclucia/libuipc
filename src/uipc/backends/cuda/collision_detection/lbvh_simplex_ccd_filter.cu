#include "hip/hip_runtime.h"
#include <collision_detection/lbvh_simplex_ccd_filter.h>
#include <collision_detection/global_dcd_filter.h>
#include <muda/ext/geo/distance.h>
#include <sim_engine.h>
#include <kernel_cout.h>
#include <muda/cub/device/device_reduce.h>

namespace uipc::backend::cuda
{
template <>
class SimSystemCreator<LBVHSimplexCCDFilter>
{
  public:
    static U<LBVHSimplexCCDFilter> create(SimEngine& engine)
    {
        auto& info = engine.world().scene().info();

        return info["contact"]["enable"].get<bool>() ?
                   make_unique<LBVHSimplexCCDFilter>(engine) :
                   nullptr;
    }
};

REGISTER_SIM_SYSTEM(LBVHSimplexCCDFilter);

void LBVHSimplexCCDFilter::Impl::broadphase_ccd(SimplexCCDFilter::FilterInfo& info)
{
    using namespace muda;
    auto alpha = info.alpha();
    auto d_hat = info.d_hat();
    auto Ps    = info.positions();
    auto dxs   = info.displacements();
    auto Vs    = info.surf_vertices();
    auto Es    = info.surf_edges();
    auto Fs    = info.surf_triangles();

    spdlog::info("d_hat:{}, alpha:{}", d_hat, alpha);

    point_aabbs.resize(Vs.size());
    triangle_aabbs.resize(Fs.size());
    edge_aabbs.resize(Es.size());

    // build AABBs for points
    ParallelFor()
        .kernel_name(__FUNCTION__ "-points")
        .apply(Vs.size(),
               [Vs    = Vs.viewer().name("V"),
                dxs   = dxs.viewer().name("dx"),
                Ps    = Ps.viewer().name("Ps"),
                aabbs = point_aabbs.viewer().name("aabbs"),
                alpha = alpha,
                d_hat = d_hat] __device__(int i) mutable
               {
                   auto        vI    = Vs(i);
                   const auto& pos   = Ps(vI);
                   Vector3     pos_t = pos + dxs(vI) * alpha;

                   AABB aabb;

                   aabb.extend(pos).extend(pos_t);

                   aabb.min().array() -= d_hat;
                   aabb.max().array() += d_hat;
                   aabbs(i) = aabb;
               });

    // build AABBs for edges
    ParallelFor()
        .kernel_name(__FUNCTION__ "-edges")
        .apply(Es.size(),
               [Es    = Es.viewer().name("E"),
                Ps    = Ps.viewer().name("Ps"),
                aabbs = edge_aabbs.viewer().name("aabbs"),
                dxs   = dxs.viewer().name("dx"),
                alpha = alpha,
                d_hat = d_hat] __device__(int i) mutable
               {
                   auto        eI     = Es(i);
                   const auto& pos0   = Ps(eI[0]);
                   const auto& pos1   = Ps(eI[1]);
                   Vector3     pos0_t = pos0 + dxs(eI[0]) * alpha;
                   Vector3     pos1_t = pos1 + dxs(eI[1]) * alpha;

                   Vector3 max = pos0_t;
                   Vector3 min = pos0_t;

                   AABB aabb;

                   aabb.extend(pos0).extend(pos1).extend(pos0_t).extend(pos1_t);

                   aabb.min().array() -= d_hat;
                   aabb.max().array() += d_hat;
                   aabbs(i) = aabb;
               });

    // build AABBs for triangles
    ParallelFor()
        .kernel_name(__FUNCTION__ "-triangles")
        .apply(Fs.size(),
               [Fs    = Fs.viewer().name("F"),
                Ps    = Ps.viewer().name("Ps"),
                aabbs = triangle_aabbs.viewer().name("aabbs"),
                dxs   = dxs.viewer().name("dx"),
                alpha = alpha,
                d_hat = d_hat] __device__(int i) mutable
               {
                   auto        fI     = Fs(i);
                   const auto& pos0   = Ps(fI[0]);
                   const auto& pos1   = Ps(fI[1]);
                   const auto& pos2   = Ps(fI[2]);
                   Vector3     pos0_t = pos0 + dxs(fI[0]) * alpha;
                   Vector3     pos1_t = pos1 + dxs(fI[1]) * alpha;
                   Vector3     pos2_t = pos2 + dxs(fI[2]) * alpha;

                   AABB aabb;

                   aabb.extend(pos0)
                       .extend(pos1)
                       .extend(pos2)
                       .extend(pos0_t)
                       .extend(pos1_t)
                       .extend(pos2_t);

                   aabb.min().array() -= d_hat;
                   aabb.max().array() += d_hat;
                   aabbs(i) = aabb;
               });

    // query PT
    lbvh_PT.build(triangle_aabbs);
    auto PT_pairs =
        lbvh_PT.query(point_aabbs,
                      [Vs    = Vs.viewer().name("Vs"),
                       Fs    = Fs.viewer().name("Fs"),
                       Ps    = Ps.viewer().name("Ps"),
                       dxs   = dxs.viewer().name("dxs"),
                       d_hat = d_hat,
                       alpha = alpha] __device__(IndexT i, IndexT j)
                      {
                          // discard if the point is on the triangle
                          auto V = Vs(i);
                          auto F = Fs(j);

                          if(F[0] == V || F[1] == V || F[2] == V)
                              return false;

                          Vector3 VP  = Ps(V);
                          Vector3 dVP = alpha * dxs(V);

                          Vector3 FP0 = Ps(F[0]);
                          Vector3 FP1 = Ps(F[1]);
                          Vector3 FP2 = Ps(F[2]);

                          Vector3 dFP0 = alpha * dxs(F[0]);
                          Vector3 dFP1 = alpha * dxs(F[1]);
                          Vector3 dFP2 = alpha * dxs(F[2]);

                          if(!muda::distance::point_triangle_ccd_broadphase(
                                 VP, FP0, FP1, FP2, dVP, dFP0, dFP1, dFP2, d_hat))
                              return false;


                          return true;
                      });

    // query EE
    lbvh_EE.build(edge_aabbs);
    auto EE_pairs = lbvh_EE.detect(
        [Es    = Es.viewer().name("Es"),
         Ps    = Ps.viewer().name("Ps"),
         dxs   = dxs.viewer().name("dxs"),
         d_hat = d_hat,
         alpha = alpha] __device__(IndexT i, IndexT j)
        {
            // discard if the edges shared same vertex
            auto E0 = Es(i);
            auto E1 = Es(j);

            if(E0[0] == E1[0] || E0[0] == E1[1] || E0[1] == E1[0] || E0[1] == E1[1])
                return false;

            Vector3 EP0  = Ps(E0[0]);
            Vector3 EP1  = Ps(E0[1]);
            Vector3 dEP0 = alpha * dxs(E0[0]);
            Vector3 dEP1 = alpha * dxs(E0[1]);

            Vector3 EP2  = Ps(E1[0]);
            Vector3 EP3  = Ps(E1[1]);
            Vector3 dEP2 = alpha * dxs(E1[0]);
            Vector3 dEP3 = alpha * dxs(E1[1]);

            if(!muda::distance::edge_edge_ccd_broadphase(
                   EP0, EP1, dEP0, dEP1, EP2, EP3, dEP2, dEP3, d_hat))
                return false;

            return true;
        });

    candidate_PTs.resize(PT_pairs.size());
    candidate_EEs.resize(EE_pairs.size());

    // record the candidate pairs
    ParallelFor()
        .kernel_name(__FUNCTION__ "-record PT pairs")
        .apply(PT_pairs.size(),
               [PT_pairs      = PT_pairs.viewer().name("PT_pairs"),
                candidate_PTs = candidate_PTs.viewer().name("candidate_PTs"),
                Fs            = Fs.viewer().name("Fs"),
                Vs = Vs.viewer().name("Vs")] __device__(int i) mutable
               {
                   auto& PT         = candidate_PTs(i);
                   auto  pair       = PT_pairs(i);
                   PT[0]            = Vs(pair[0]);
                   PT.segment<3>(1) = Fs(pair[1]);
               });

    ParallelFor()
        .kernel_name(__FUNCTION__ "-record EE pairs")
        .apply(EE_pairs.size(),
               [EE_pairs      = EE_pairs.viewer().name("EE_pairs"),
                candidate_EEs = candidate_EEs.viewer().name("candidate_EEs"),
                Es = Es.viewer().name("Es")] __device__(int i) mutable
               {
                   auto& EE         = candidate_EEs(i);
                   auto  pair       = EE_pairs(i);
                   EE.segment<2>(0) = Es(pair[0]);
                   EE.segment<2>(2) = Es(pair[1]);
               });

    {
        std::vector<Vector4i> candidate_PTs_host;
        std::vector<Vector4i> candidate_EEs_host;

        candidate_PTs.copy_to(candidate_PTs_host);
        candidate_EEs.copy_to(candidate_EEs_host);

        // print the candidate pairs
        for(auto& PT : candidate_PTs_host)
        {
            std::cout << "CCD-PT: " << PT.transpose() << std::endl;
        }

        for(auto& EE : candidate_EEs_host)
        {
            std::cout << "CCD-EE: " << EE.transpose() << std::endl;
        }
    }
}

void LBVHSimplexCCDFilter::Impl::narrowphase_ccd(SimplexCCDFilter::FilterInfo& info)
{
    using namespace muda;

    tois.resize(candidate_PTs.size() + candidate_EEs.size());

    auto PT_tois = tois.view(0, candidate_PTs.size());
    auto EE_tois = tois.view(candidate_PTs.size(), candidate_EEs.size());


    constexpr Float eta = 0.1;
    // TODO: Codimension IPC need thickness property
    constexpr Float thickness = 0.0;
    // TODO: Now hard code the max iteration
    constexpr SizeT max_iter = 1000;

    // PT
    ParallelFor()
        .kernel_name(__FUNCTION__ "-PT")
        .apply(candidate_PTs.size(),
               [PT_tois       = PT_tois.viewer().name("PT_tois"),
                candidate_PTs = candidate_PTs.viewer().name("candidate_PTs"),
                Ps            = info.positions().viewer().name("Ps"),
                dxs           = info.displacements().viewer().name("dxs"),
                alpha         = info.alpha(),
                d_hat         = info.d_hat(),
                eta,
                thickness,
                max_iter] __device__(int i) mutable
               {
                   auto& PT = candidate_PTs(i);
                   auto  V  = PT[0];
                   auto  F  = PT.segment<3>(1);

                   Vector3 VP  = Ps(V);
                   Vector3 dVP = alpha * dxs(V);

                   Vector3 FP0 = Ps(F[0]);
                   Vector3 FP1 = Ps(F[1]);
                   Vector3 FP2 = Ps(F[2]);

                   Vector3 dFP0 = alpha * dxs(F[0]);
                   Vector3 dFP1 = alpha * dxs(F[1]);
                   Vector3 dFP2 = alpha * dxs(F[2]);

                   Float toi = 1.1;  // large enough (>1)

                   bool hit = muda::distance::point_triangle_ccd(
                       VP, FP0, FP1, FP2, dVP, dFP0, dFP1, dFP2, eta, thickness, max_iter, toi);

                   if(!hit)
                       toi = 1.1;

                   PT_tois(i) = toi;
               });

    // EE
    ParallelFor()
        .kernel_name(__FUNCTION__ "-EE")
        .apply(candidate_EEs.size(),
               [EE_tois       = EE_tois.viewer().name("EE_tois"),
                candidate_EEs = candidate_EEs.viewer().name("candidate_EEs"),
                Ps            = info.positions().viewer().name("Ps"),
                dxs           = info.displacements().viewer().name("dxs"),
                alpha         = info.alpha(),
                d_hat         = info.d_hat(),
                eta,
                thickness,
                max_iter] __device__(int i) mutable
               {
                   auto& EE = candidate_EEs(i);
                   auto  E0 = EE.segment<2>(0);
                   auto  E1 = EE.segment<2>(2);

                   Vector3 EP0  = Ps(E0[0]);
                   Vector3 EP1  = Ps(E0[1]);
                   Vector3 dEP0 = alpha * dxs(E0[0]);
                   Vector3 dEP1 = alpha * dxs(E0[1]);

                   Vector3 EP2  = Ps(E1[0]);
                   Vector3 EP3  = Ps(E1[1]);
                   Vector3 dEP2 = alpha * dxs(E1[0]);
                   Vector3 dEP3 = alpha * dxs(E1[1]);

                   Float toi = 1.1;  // large enough (>1)

                   bool hit = muda::distance::edge_edge_ccd(
                       EP0, EP1, dEP0, dEP1, EP2, EP3, dEP2, dEP3, eta, thickness, max_iter, toi);

                   if(!hit)
                       toi = 1.1;

                   EE_tois(i) = toi;
               });

    // get min toi
    DeviceReduce().Min(tois.data(), info.toi().data(), tois.size());

    Float toi = -1;

    info.toi().copy_to(&toi);

    spdlog::info("toi:{}", toi);
}

void LBVHSimplexCCDFilter::do_filter_toi(SimplexCCDFilter::FilterInfo& info)
{
    m_impl.broadphase_ccd(info);
    m_impl.narrowphase_ccd(info);
}
}  // namespace uipc::backend::cuda
