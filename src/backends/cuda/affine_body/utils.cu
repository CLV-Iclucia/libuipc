#include <affine_body/utils.h>

namespace uipc::backend::cuda
{
MUDA_GENERIC Matrix3x3 q_to_A(const Vector12& q)
{
    Matrix3x3 A = Matrix3x3::Zero();
    A.row(0)    = q.segment<3>(3);
    A.row(1)    = q.segment<3>(6);
    A.row(2)    = q.segment<3>(9);
    return A;
}
MUDA_GENERIC Vector9 A_to_q(const Matrix3x3& A)
{
    Vector9 q       = Vector9::Zero();
    q.segment<3>(0) = A.row(0);
    q.segment<3>(3) = A.row(1);
    q.segment<3>(6) = A.row(2);
    return q;
}

MUDA_GENERIC Vector9 F_to_A(const Vector9& F)
{
    Vector9 A;
    A(0) = F(0);
    A(1) = F(3);
    A(2) = F(6);
    A(3) = F(1);
    A(4) = F(4);
    A(5) = F(7);
    A(6) = F(2);
    A(7) = F(5);
    A(8) = F(8);
    return A;
}

MUDA_GENERIC Matrix9x9 HF_to_HA(const Matrix9x9& HF)
{
    Matrix9x9 HA;
    HA(0, 0) = HF(0, 0);
    HA(0, 1) = HF(0, 3);
    HA(0, 2) = HF(0, 6);
    HA(0, 3) = HF(0, 1);
    HA(0, 4) = HF(0, 4);
    HA(0, 5) = HF(0, 7);
    HA(0, 6) = HF(0, 2);
    HA(0, 7) = HF(0, 5);
    HA(0, 8) = HF(0, 8);
    HA(1, 0) = HF(3, 0);
    HA(1, 1) = HF(3, 3);
    HA(1, 2) = HF(3, 6);
    HA(1, 3) = HF(3, 1);
    HA(1, 4) = HF(3, 4);
    HA(1, 5) = HF(3, 7);
    HA(1, 6) = HF(3, 2);
    HA(1, 7) = HF(3, 5);
    HA(1, 8) = HF(3, 8);
    HA(2, 0) = HF(6, 0);
    HA(2, 1) = HF(6, 3);
    HA(2, 2) = HF(6, 6);
    HA(2, 3) = HF(6, 1);
    HA(2, 4) = HF(6, 4);
    HA(2, 5) = HF(6, 7);
    HA(2, 6) = HF(6, 2);
    HA(2, 7) = HF(6, 5);
    HA(2, 8) = HF(6, 8);
    HA(3, 0) = HF(1, 0);
    HA(3, 1) = HF(1, 3);
    HA(3, 2) = HF(1, 6);
    HA(3, 3) = HF(1, 1);
    HA(3, 4) = HF(1, 4);
    HA(3, 5) = HF(1, 7);
    HA(3, 6) = HF(1, 2);
    HA(3, 7) = HF(1, 5);
    HA(3, 8) = HF(1, 8);
    HA(4, 0) = HF(4, 0);
    HA(4, 1) = HF(4, 3);
    HA(4, 2) = HF(4, 6);
    HA(4, 3) = HF(4, 1);
    HA(4, 4) = HF(4, 4);
    HA(4, 5) = HF(4, 7);
    HA(4, 6) = HF(4, 2);
    HA(4, 7) = HF(4, 5);
    HA(4, 8) = HF(4, 8);
    HA(5, 0) = HF(7, 0);
    HA(5, 1) = HF(7, 3);
    HA(5, 2) = HF(7, 6);
    HA(5, 3) = HF(7, 1);
    HA(5, 4) = HF(7, 4);
    HA(5, 5) = HF(7, 7);
    HA(5, 6) = HF(7, 2);
    HA(5, 7) = HF(7, 5);
    HA(5, 8) = HF(7, 8);
    HA(6, 0) = HF(2, 0);
    HA(6, 1) = HF(2, 3);
    HA(6, 2) = HF(2, 6);
    HA(6, 3) = HF(2, 1);
    HA(6, 4) = HF(2, 4);
    HA(6, 5) = HF(2, 7);
    HA(6, 6) = HF(2, 2);
    HA(6, 7) = HF(2, 5);
    HA(6, 8) = HF(2, 8);
    HA(7, 0) = HF(5, 0);
    HA(7, 1) = HF(5, 3);
    HA(7, 2) = HF(5, 6);
    HA(7, 3) = HF(5, 1);
    HA(7, 4) = HF(5, 4);
    HA(7, 5) = HF(5, 7);
    HA(7, 6) = HF(5, 2);
    HA(7, 7) = HF(5, 5);
    HA(7, 8) = HF(5, 8);
    HA(8, 0) = HF(8, 0);
    HA(8, 1) = HF(8, 3);
    HA(8, 2) = HF(8, 6);
    HA(8, 3) = HF(8, 1);
    HA(8, 4) = HF(8, 4);
    HA(8, 5) = HF(8, 7);
    HA(8, 6) = HF(8, 2);
    HA(8, 7) = HF(8, 5);
    HA(8, 8) = HF(8, 8);
    return HA;
}
}  // namespace uipc::backend::cuda
