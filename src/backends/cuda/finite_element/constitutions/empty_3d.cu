#include <finite_element/fem_3d_constitution.h>
#include <finite_element/constitutions/hookean_spring_1d_function.h>
#include <kernel_cout.h>
#include <muda/ext/eigen/log_proxy.h>
#include <Eigen/Dense>
#include <muda/ext/eigen/inverse.h>
#include <numbers>

namespace uipc::backend::cuda
{
class Empty3D final : public FEM3DConstitution
{
  public:
    // Constitution UID by libuipc specification
    static constexpr U64 ConstitutionUID = 0ull;

    using FEM3DConstitution::FEM3DConstitution;

    vector<Float>             h_kappas;
    muda::DeviceBuffer<Float> kappas;

    virtual U64 get_uid() const override { return ConstitutionUID; }

    virtual void do_report_extent(ReportExtentInfo& info)
    {
        info.energy_count(0);
        info.stencil_dim(4);
    }

    virtual void do_build(BuildInfo& info) override
    {
        // do nothing
    }

    virtual void do_init(FiniteElementMethod::FEM3DFilteredInfo& info) override
    {
        // do nothing
    }

    virtual void do_compute_energy(ComputeEnergyInfo& info) override
    {
        // do nothing
    }

    virtual void do_compute_gradient_hessian(ComputeGradientHessianInfo& info) override
    {
        // do nothing
    }
};

REGISTER_SIM_SYSTEM(Empty3D);
}  // namespace uipc::backend::cuda
