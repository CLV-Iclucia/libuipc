#include <finite_element/codim_0d_constitution.h>

namespace uipc::backend::cuda
{
class Particle0D final : public Codim0DConstitution
{
  public:
    // Constitution UID by libuipc specification
    static constexpr U64 ConstitutionUID = 13;

    using Codim0DConstitution::Codim0DConstitution;

    virtual U64 get_uid() const noexcept override { return ConstitutionUID; }

    virtual void do_build(BuildInfo& info) override
    {
        // Do nothing
    }

    virtual void do_init(FiniteElementMethod::Codim0DFilteredInfo& info) override
    {
        // Do nothing
    }
};

REGISTER_SIM_SYSTEM(Particle0D);
}  // namespace uipc::backend::cuda
