#include <contact_system/simplex_normal_contact.h>
#include <muda/ext/eigen/evd.h>
#include <muda/cub/device/device_merge_sort.h>
#include <utils/distance.h>
#include <utils/codim_thickness.h>

namespace uipc::backend::cuda
{
void SimplexNormalContact::do_build()
{
    m_impl.global_trajectory_filter = &require<GlobalTrajectoryFilter>();
    m_impl.global_contact_manager   = &require<GlobalContactManager>();
    m_impl.global_vertex_manager    = &require<GlobalVertexManager>();

    BuildInfo info;
    do_build(info);

    m_impl.global_contact_manager->add_reporter(this);
    m_impl.dt = world().scene().info()["dt"].get<Float>();

    on_init_scene(
        [this]
        {
            m_impl.simplex_trajectory_filter =
                m_impl.global_trajectory_filter->find<SimplexTrajectoryFilter>();
        });
}

void SimplexNormalContact::Impl::compute_energy(SimplexNormalContact* contact,
                                                GlobalContactManager::EnergyInfo& info)
{
    EnergyInfo this_info{this};

    auto filter = simplex_trajectory_filter;

    PT_count = filter->PTs().size();
    EE_count = filter->EEs().size();
    PE_count = filter->PEs().size();
    PP_count = filter->PPs().size();

    auto count_4 = (PT_count + EE_count);
    auto count_3 = PE_count;
    auto count_2 = PP_count;

    energies.resize(count_4 + count_3 + count_2);

    SizeT offset            = 0;
    this_info.m_PT_energies = energies.view(offset, PT_count);
    offset += PT_count;
    this_info.m_EE_energies = energies.view(offset, EE_count);
    offset += EE_count;
    this_info.m_PE_energies = energies.view(offset, PE_count);
    offset += PE_count;
    this_info.m_PP_energies = energies.view(offset, PP_count);


    contact->do_compute_energy(this_info);
    using namespace muda;

    // if(info.is_initial())
    //{
    //    DeviceMergeSort().SortKeys(energies.data(),
    //                               energies.size(),
    //                               [] HIPCUB_RUNTIME_FUNCTION(Float a, Float b)
    //                               { return a < b; });
    //}

    DeviceReduce().Sum(energies.data(), info.energy().data(), energies.size());
}

void SimplexNormalContact::do_compute_energy(GlobalContactManager::EnergyInfo& info)
{
    m_impl.compute_energy(this, info);
}

void SimplexNormalContact::do_report_extent(GlobalContactManager::ContactExtentInfo& info)
{
    auto& filter = m_impl.simplex_trajectory_filter;

    // m_impl.classify_constraints();
    //m_impl.PT_count = m_impl.PT_constraints.size();
    //m_impl.EE_count = m_impl.EE_constraints.size();
    //m_impl.PE_count = m_impl.PE_constraints.size();
    //m_impl.PP_count = m_impl.PP_constraints.size();

    m_impl.PT_count = filter->PTs().size();
    m_impl.EE_count = filter->EEs().size();
    m_impl.PE_count = filter->PEs().size();
    m_impl.PP_count = filter->PPs().size();


    auto count_4 = (m_impl.PT_count + m_impl.EE_count);
    auto count_3 = m_impl.PE_count;
    auto count_2 = m_impl.PP_count;

    // expand to hessian3x3 and graident3
    SizeT contact_gradient_count = 4 * count_4 + 3 * count_3 + 2 * count_2;
    SizeT contact_hessian_count = 4 * 4 * count_4 + 3 * 3 * count_3 + 2 * 2 * count_2;

    info.gradient_count(contact_gradient_count);
    info.hessian_count(contact_hessian_count);

    m_impl.loose_resize(m_impl.PT_EE_gradients, count_4);
    m_impl.loose_resize(m_impl.PT_EE_hessians, count_4);

    m_impl.loose_resize(m_impl.PE_gradients, count_3);
    m_impl.loose_resize(m_impl.PE_hessians, count_3);

    m_impl.loose_resize(m_impl.PP_gradients, count_2);
    m_impl.loose_resize(m_impl.PP_hessians, count_2);
}

void SimplexNormalContact::do_assemble(GlobalContactManager::ContactInfo& info)
{
    ContactInfo this_info{&m_impl};

    this_info.m_PT_gradients = m_impl.PT_EE_gradients.view(0, m_impl.PT_count);
    this_info.m_EE_gradients =
        m_impl.PT_EE_gradients.view(m_impl.PT_count, m_impl.EE_count);
    this_info.m_PE_gradients = m_impl.PE_gradients.view();
    this_info.m_PP_gradients = m_impl.PP_gradients.view();

    this_info.m_PT_hessians = m_impl.PT_EE_hessians.view(0, m_impl.PT_count);
    this_info.m_EE_hessians = m_impl.PT_EE_hessians.view(m_impl.PT_count, m_impl.EE_count);
    this_info.m_PE_hessians = m_impl.PE_hessians.view();
    this_info.m_PP_hessians = m_impl.PP_hessians.view();

    // let subclass to fill in the data
    do_assemble(this_info);

    // _assemble the data to the global contact manager
    m_impl.assemble(info);
}

muda::CBuffer2DView<ContactCoeff> SimplexNormalContact::BaseInfo::contact_tabular() const
{
    return m_impl->global_contact_manager->contact_tabular();
}

muda::CBufferView<Vector4i> SimplexNormalContact::BaseInfo::PTs() const
{
    // return m_impl->PT_constraints.view();
    return m_impl->simplex_trajectory_filter->PTs();
}

muda::CBufferView<Vector4i> SimplexNormalContact::BaseInfo::EEs() const
{
    return m_impl->simplex_trajectory_filter->EEs();
}

muda::CBufferView<Vector3i> SimplexNormalContact::BaseInfo::PEs() const
{
    return m_impl->simplex_trajectory_filter->PEs();
}

muda::CBufferView<Vector2i> SimplexNormalContact::BaseInfo::PPs() const
{
    return m_impl->simplex_trajectory_filter->PPs();
}

muda::CBufferView<Float> SimplexNormalContact::BaseInfo::thicknesses() const
{
    return m_impl->global_vertex_manager->thicknesses();
}

muda::CBufferView<Vector3> SimplexNormalContact::BaseInfo::positions() const
{
    return m_impl->global_vertex_manager->positions();
}

muda::CBufferView<Vector3> SimplexNormalContact::BaseInfo::prev_positions() const
{
    return m_impl->global_vertex_manager->prev_positions();
}

muda::CBufferView<Vector3> SimplexNormalContact::BaseInfo::rest_positions() const
{
    return m_impl->global_vertex_manager->rest_positions();
}

muda::CBufferView<IndexT> SimplexNormalContact::BaseInfo::contact_element_ids() const
{
    return m_impl->global_vertex_manager->contact_element_ids();
}

Float SimplexNormalContact::BaseInfo::d_hat() const
{
    return m_impl->global_contact_manager->d_hat();
}

Float SimplexNormalContact::BaseInfo::dt() const
{
    return m_impl->dt;
}

Float SimplexNormalContact::BaseInfo::eps_velocity() const
{
    return m_impl->global_contact_manager->eps_velocity();
}

namespace detail
{
    template <SizeT N>
    __inline__ __device__ void fill_contact_hessian(muda::TripletMatrixViewer<Float, 3>& H3x3,
                                                    int                      I,
                                                    const Vector<IndexT, N>& D,
                                                    const Matrix<Float, 3 * N, 3 * N>& H)
    {
        auto offset = N * N * I;
#pragma unroll
        for(int i = 0; i < N; ++i)
        {
            auto L = D(i);
#pragma unroll
            for(int j = 0; j < N; ++j)
            {
                auto R = D(j);
                H3x3(offset++).write(L, R, H.template block<3, 3>(3 * i, 3 * j));
            }
        }
    }

    template <SizeT N>
    __inline__ __device__ void fill_contact_gradient(muda::DoubletVectorViewer<Float, 3>& G3,
                                                     int                      I,
                                                     const Vector<IndexT, N>& D,
                                                     const Vector<Float, 3 * N>& G)
    {
        auto offset = N * I;
#pragma unroll
        for(int i = 0; i < N; ++i)
        {
            auto L = D(i);
            G3(offset++).write(L, G.segment<3>(3 * i));
        }
    }

    template <SizeT N>
    __inline__ __device__ void make_spd(Matrix<Float, N, N>& mat)
    {
        Vector<Float, N>    eigen_values;
        Matrix<Float, N, N> eigen_vectors;
        muda::eigen::template evd(mat, eigen_values, eigen_vectors);
#pragma unroll
        for(int i = 0; i < N; ++i)
        {
            auto& v = eigen_values(i);
            v       = v < 0.0 ? 0.0 : v;
        }
        mat = eigen_vectors * eigen_values.asDiagonal() * eigen_vectors.transpose();
    }
}  // namespace detail


void SimplexNormalContact::Impl::assemble(GlobalContactManager::ContactInfo& info)
{
    using namespace muda;

    auto H3x3 = info.hessian();
    auto G3   = info.gradient();
    auto PTs  = simplex_trajectory_filter->PTs();
    auto EEs  = simplex_trajectory_filter->EEs();
    auto PEs  = simplex_trajectory_filter->PEs();
    auto PPs  = simplex_trajectory_filter->PPs();

    auto PT_hessian  = PT_EE_hessians.view(0, PTs.size());
    auto PT_gradient = PT_EE_gradients.view(0, PTs.size());
    auto EE_hessian  = PT_EE_hessians.view(PTs.size(), EEs.size());
    auto EE_gradient = PT_EE_gradients.view(PTs.size(), EEs.size());

    SizeT H3x3_offset = 0;
    SizeT G3_offset   = 0;

    // PT
    {
        SizeT H3x3_count = PTs.size() * 16;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(PT_hessian.size(),
                   [PT_H12x12s = PT_hessian.cviewer().name("H12x12"),
                    PTs        = PTs.cviewer().name("PTs"),
                    H3x3 = H3x3.subview(H3x3_offset, H3x3_count).viewer().name("H3x3")] __device__(int I) mutable
                   {
                       Matrix12x12 H12x12 = PT_H12x12s(I);
                       Vector4i    D4     = PTs(I);
                       detail::make_spd<12>(H12x12);
                       detail::fill_contact_hessian<4>(H3x3, I, D4, H12x12);
                   });
        H3x3_offset += H3x3_count;

        SizeT G3_count = PT_gradient.size() * 4;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(PT_gradient.size(),
                   [PT_G12s = PT_gradient.cviewer().name("G12"),
                    PTs     = PTs.cviewer().name("PTs"),
                    G3 = G3.subview(G3_offset, G3_count).viewer().name("G3")] __device__(int I) mutable
                   {
                       const auto& G12 = PT_G12s(I);
                       Vector4i    D4  = PTs(I);
                       detail::fill_contact_gradient<4>(G3, I, D4, G12);
                   });
        G3_offset += G3_count;
    }

    // EE
    {
        SizeT H3x3_count = EEs.size() * 16;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(EE_hessian.size(),
                   [EE_H12x12s = EE_hessian.cviewer().name("H12x12"),
                    EEs        = EEs.cviewer().name("EEs"),
                    H3x3 = H3x3.subview(H3x3_offset, H3x3_count).viewer().name("H3x3")] __device__(int I) mutable
                   {
                       Matrix12x12 H12x12 = EE_H12x12s(I);
                       Vector4i    D4     = EEs(I);
                       detail::make_spd<12>(H12x12);
                       detail::fill_contact_hessian<4>(H3x3, I, D4, H12x12);
                   });

        H3x3_offset += H3x3_count;

        SizeT G3_count = EE_gradient.size() * 4;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(EE_gradient.size(),
                   [G12s = EE_gradient.cviewer().name("G12"),
                    EEs  = EEs.cviewer().name("EEs"),
                    G3 = G3.subview(G3_offset, G3_count).viewer().name("G3")] __device__(int I) mutable
                   {
                       const auto& G12 = G12s(I);
                       Vector4i    D4  = EEs(I);
                       detail::fill_contact_gradient<4>(G3, I, D4, G12);
                   });

        G3_offset += G3_count;
    }

    // PE
    {
        SizeT H3x3_count = PEs.size() * 9;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(PEs.size(),
                   [PE_H9x9s = PE_hessians.cviewer().name("H9x9"),
                    PEs      = PEs.cviewer().name("PEs"),
                    H3x3 = H3x3.subview(H3x3_offset, H3x3_count).viewer().name("H3x3")] __device__(int I) mutable
                   {
                       Matrix9x9 H9x9 = PE_H9x9s(I);
                       Vector3i  D3   = PEs(I);
                       detail::make_spd<9>(H9x9);
                       detail::fill_contact_hessian<3>(H3x3, I, D3, H9x9);
                   });

        H3x3_offset += H3x3_count;

        SizeT G3_count = PEs.size() * 3;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(PEs.size(),
                   [G9s = PE_gradients.cviewer().name("G9"),
                    PEs = PEs.cviewer().name("PEs"),
                    G3 = G3.subview(G3_offset, G3_count).viewer().name("G3")] __device__(int I) mutable
                   {
                       const auto& G9 = G9s(I);
                       Vector3i    D3 = PEs(I);
                       detail::fill_contact_gradient<3>(G3, I, D3, G9);
                   });

        G3_offset += G3_count;
    }


    // PP
    {
        SizeT H3x3_count = PPs.size() * 4;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(PPs.size(),
                   [PP_H6x6s = PP_hessians.cviewer().name("H6x6"),
                    PPs      = PPs.cviewer().name("PPs"),
                    H3x3 = H3x3.subview(H3x3_offset, H3x3_count).viewer().name("H3x3")] __device__(int I) mutable
                   {
                       Matrix6x6 H6x6 = PP_H6x6s(I);
                       Vector2i  D2   = PPs(I);
                       detail::make_spd<6>(H6x6);
                       detail::fill_contact_hessian<2>(H3x3, I, D2, H6x6);
                   });

        H3x3_offset += H3x3_count;

        SizeT G3_count = PPs.size() * 2;
        ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(PPs.size(),
                   [G6s = PP_gradients.cviewer().name("G6"),
                    PPs = PPs.cviewer().name("PPs"),
                    G3 = G3.subview(G3_offset, G3_count).viewer().name("G3")] __device__(int I) mutable
                   {
                       const auto& G6 = G6s(I);
                       Vector2i    D2 = PPs(I);
                       detail::fill_contact_gradient<2>(G3, I, D2, G6);
                   });

        G3_offset += G3_count;
    }

    UIPC_ASSERT(H3x3_offset == info.hessian().triplet_count(), "size mismatch");
    UIPC_ASSERT(G3_offset == info.gradient().doublet_count(), "size mismatch");
}

//void SimplexNormalContact::Impl::classify_constraints()
//{
//    using namespace muda;
//
//    auto& filter = simplex_trajectory_filter;
//
//    auto PPs = filter->PPs();
//    auto PEs = filter->PEs();
//    auto PTs = filter->PTs();
//    auto EEs = filter->EEs();
//
//    auto positions   = global_vertex_manager->positions();
//    auto thicknesses = global_vertex_manager->thicknesses();
//    auto d_hat       = global_contact_manager->d_hat();
//    auto D_hat       = d_hat * d_hat;
//
//    // PP, PE, PT, EE all can generate the PP constraint
//    SizeT possible_PP_constraint_count =
//        PPs.size() + PEs.size() + PTs.size() + EEs.size();
//    // PE, PT, EE can generate the PT constraint
//    SizeT possible_PE_constraint_count = PEs.size() + PTs.size() + EEs.size();
//    // only PT can generate the PT constraint
//    SizeT possible_PT_constraint_count = PTs.size();
//    // only EE can generate the EE constraint
//    SizeT possible_EE_constraint_count = EEs.size();
//
//    loose_resize(temp_PP_constraints, possible_PP_constraint_count);
//    loose_resize(temp_PE_constraints, possible_PE_constraint_count);
//    loose_resize(temp_PT_constraints, possible_PT_constraint_count);
//    loose_resize(temp_EE_constraints, possible_EE_constraint_count);
//
//    SizeT PP_offset = 0;
//    SizeT PE_offset = 0;
//
//    {  // PP in PPs
//
//        auto PP_constraint_view = temp_PP_constraints.view(PP_offset, PPs.size());
//        PP_offset += PPs.size();
//
//        ParallelFor()
//            .kernel_name(__FUNCTION__)
//            .apply(PPs.size(),
//                   [PPs         = PPs.cviewer().name("PPs"),
//                    positions   = positions.viewer().name("positions"),
//                    thicknesses = thicknesses.viewer().name("thicknesses"),
//                    PP_constraints = PP_constraint_view.viewer().name("PP_constraints"),
//                    D_hat = D_hat] __device__(int I) mutable
//                   {
//                       Vector2i PP = PPs(I);
//
//                       const auto& P0 = positions(PP[0]);
//                       const auto& P1 = positions(PP[1]);
//
//                       Float D;
//                       distance::point_point_distance_unclassified(P0, P1, D);
//
//                       Float thickness =
//                           PP_thickness(thicknesses(PP[0]), thicknesses(PP[1]));
//                       Vector2i range = D_range(thickness, D_hat);
//
//                       PP_constraints(I) =
//                           is_active_D(range, D) ?
//                               SimplexContactConstraint::PP(PP[0], PP[1]) :
//                               SimplexContactConstraint::None();
//                   });
//    }
//
//    {  // PP, PE in PEs
//
//        auto PE_constraint_view = temp_PE_constraints.view(PE_offset, PEs.size());
//        PE_offset += PEs.size();
//        auto PP_constraint_view = temp_PP_constraints.view(PP_offset, PEs.size());
//        PP_offset += PEs.size();
//
//        ParallelFor()
//            .kernel_name(__FUNCTION__)
//            .apply(PEs.size(),
//                   [PEs         = PEs.cviewer().name("PEs"),
//                    positions   = positions.viewer().name("positions"),
//                    thicknesses = thicknesses.viewer().name("thicknesses"),
//                    PE_constraints = PE_constraint_view.viewer().name("PE_constraints"),
//                    PP_constraints = PP_constraint_view.viewer().name("PP_constraints"),
//                    D_hat = D_hat] __device__(int I) mutable
//                   {
//                       Vector3i PE = PEs(I);
//
//                       const auto& P  = positions(PE[0]);
//                       const auto& E0 = positions(PE[1]);
//                       const auto& E1 = positions(PE[2]);
//
//                       auto type = distance::point_edge_distance_type(P, E0, E1);
//
//                       PE_constraints(I) = SimplexContactConstraint::None();
//                       PP_constraints(I) = SimplexContactConstraint::None();
//
//                       Float    thickness = PE_thickness(thicknesses(PE[0]),
//                                                      thicknesses(PE[1]),
//                                                      thicknesses(PE[2]));
//                       Vector2i range     = D_range(thickness, D_hat);
//
//                       switch(type)
//                       {
//                           case distance::PointEdgeDistanceType::PP_PE0: {
//                               Float D;
//                               distance::point_point_distance(P, E0, D);
//
//
//                               if(is_active_D(range, D))
//                               {
//                                   PP_constraints(I) = SimplexContactConstraint::PP_in_PE(
//                                       PE[0], PE[1], PE[2]);
//                               }
//                           }
//                           break;
//                           case distance::PointEdgeDistanceType::PP_PE1: {
//                               Float D;
//                               distance::point_point_distance(P, E1, D);
//
//                               if(is_active_D(range, D))
//                               {
//                                   PP_constraints(I) = SimplexContactConstraint::PP_in_PE(
//                                       PE[0], PE[2], PE[1]);
//                               }
//                           }
//                           break;
//                           case distance::PointEdgeDistanceType::PE: {
//                               Float D;
//                               distance::point_edge_distance(P, E0, E1, D);
//
//                               if(is_active_D(range, D))
//                               {
//                                   PE_constraints(I) = SimplexContactConstraint::PE(
//                                       PE[0], PE.segment<2>(1));
//                               }
//                           }
//                           break;
//                           default: {
//                               MUDA_ERROR_WITH_LOCATION("Invalid Type");
//                           }
//                           break;
//                       }
//                   });
//    }
//
//    {  // PT, PE, PP in PTs
//
//        auto PT_constraint_view = temp_PT_constraints.view(0, PTs.size());
//        auto PE_constraint_view = temp_PE_constraints.view(PE_offset, PTs.size());
//        PE_offset += PTs.size();
//        auto PP_constraint_view = temp_PP_constraints.view(PP_offset, PTs.size());
//        PP_offset += PTs.size();
//
//        ParallelFor()
//            .kernel_name(__FUNCTION__)
//            .apply(
//                PTs.size(),
//                [PTs         = PTs.cviewer().name("PTs"),
//                 positions   = positions.viewer().name("positions"),
//                 thicknesses = thicknesses.viewer().name("thicknesses"),
//                 PT_constraints = PT_constraint_view.viewer().name("PT_constraints"),
//                 PE_constraints = PE_constraint_view.viewer().name("PE_constraints"),
//                 PP_constraints = PP_constraint_view.viewer().name("PP_constraints"),
//                 D_hat = D_hat] __device__(int I) mutable
//                {
//                    Vector4i PT = PTs(I);
//
//                    const auto& P  = positions(PT[0]);
//                    const auto& T0 = positions(PT[1]);
//                    const auto& T1 = positions(PT[2]);
//                    const auto& T2 = positions(PT[3]);
//
//                    auto type = distance::point_triangle_distance_type(P, T0, T1, T2);
//
//                    PT_constraints(I) = SimplexContactConstraint::None();
//                    PE_constraints(I) = SimplexContactConstraint::None();
//                    PP_constraints(I) = SimplexContactConstraint::None();
//
//                    Float    thickness = PT_thickness(thicknesses(PT[0]),
//                                                   thicknesses(PT[1]),
//                                                   thicknesses(PT[2]),
//                                                   thicknesses(PT[3]));
//                    Vector2i range     = D_range(thickness, D_hat);
//
//                    switch(type)
//                    {
//                        case distance::PointTriangleDistanceType::PP_PT0: {
//                            Float D;
//                            distance::point_point_distance(P, T0, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PP_constraints(I) =
//                                    SimplexContactConstraint::PP_in_PT(PT[0],  // P
//                                                                       PT[1],  // P
//                                                                       {PT[2], PT[3]}  // inactive E
//                                    );
//                            }
//                        }
//                        break;
//                        case distance::PointTriangleDistanceType::PP_PT1: {
//                            Float D;
//                            distance::point_point_distance(P, T1, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PP_constraints(I) =
//                                    SimplexContactConstraint::PP_in_PT(PT[0],  // P
//                                                                       PT[2],  // P
//                                                                       {PT[1], PT[3]}  // inactive E
//                                    );
//                            }
//                        }
//                        break;
//                        case distance::PointTriangleDistanceType::PP_PT2: {
//                            Float D;
//                            distance::point_point_distance(P, T2, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PP_constraints(I) =
//                                    SimplexContactConstraint::PP_in_PT(PT[0],  // P
//                                                                       PT[3],  // P
//                                                                       {PT[1], PT[2]}  // inactive E
//                                    );
//                            }
//                        }
//                        break;
//                        case distance::PointTriangleDistanceType::PE_PT0T1: {
//                            Float D;
//                            distance::point_edge_distance(P, T0, T1, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PE_constraints(I) =
//                                    SimplexContactConstraint::PE_in_PT(PT[0],  // P
//                                                                       {PT[1], PT[2]},  // E
//                                                                       PT[3]  // inactive P
//                                    );
//                            }
//                        }
//                        break;
//                        case distance::PointTriangleDistanceType::PE_PT1T2: {
//                            Float D;
//                            distance::point_edge_distance(P, T1, T2, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PE_constraints(I) =
//                                    SimplexContactConstraint::PE_in_PT(PT[0],  // P
//                                                                       {PT[2], PT[3]},  // E
//                                                                       PT[1]  // inactive P
//                                    );
//                            }
//                        }
//                        break;
//                        case distance::PointTriangleDistanceType::PE_PT2T0: {
//                            Float D;
//                            distance::point_edge_distance(P, T2, T0, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PE_constraints(I) =
//                                    SimplexContactConstraint::PE_in_PT(PT[0],  // P
//                                                                       {PT[3], PT[1]},  // E
//                                                                       PT[2]  // inactive P
//                                    );
//                            }
//                        }
//                        break;
//                        case distance::PointTriangleDistanceType::PT: {
//                            Float D;
//                            distance::point_triangle_distance(P, T0, T1, T2, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PT_constraints(I) =
//                                    SimplexContactConstraint::PT(PT[0], PT.segment<3>(1));
//                            }
//                        }
//                        break;
//                        default: {
//                            MUDA_ERROR_WITH_LOCATION("Invalid Type");
//                        }
//                        break;
//                    }
//                });
//    }
//
//    {  // EE, PE, PP in EEs
//
//        auto EE_constraint_view = temp_EE_constraints.view(0, EEs.size());
//        auto PE_constraint_view = temp_PE_constraints.view(PE_offset, EEs.size());
//        PE_offset += EEs.size();
//        auto PP_constraint_view = temp_PP_constraints.view(PP_offset, EEs.size());
//        PP_offset += EEs.size();
//
//        ParallelFor()
//            .kernel_name(__FUNCTION__)
//            .apply(
//                EEs.size(),
//                [EEs         = EEs.cviewer().name("EEs"),
//                 positions   = positions.viewer().name("positions"),
//                 thicknesses = thicknesses.viewer().name("thicknesses"),
//                 EE_constraints = EE_constraint_view.viewer().name("EE_constraints"),
//                 PE_constraints = PE_constraint_view.viewer().name("PE_constraints"),
//                 PP_constraints = PP_constraint_view.viewer().name("PP_constraints"),
//                 D_hat = D_hat] __device__(int I) mutable
//                {
//                    Vector4i EE = EEs(I);
//
//                    const auto& Ea0 = positions(EE[0]);
//                    const auto& Ea1 = positions(EE[1]);
//                    const auto& Eb0 = positions(EE[2]);
//                    const auto& Eb1 = positions(EE[3]);
//
//                    auto type = distance::edge_edge_distance_type(Ea0, Ea1, Eb0, Eb1);
//
//                    EE_constraints(I) = SimplexContactConstraint::None();
//                    PE_constraints(I) = SimplexContactConstraint::None();
//                    PP_constraints(I) = SimplexContactConstraint::None();
//
//                    Float    thickness = EE_thickness(thicknesses(EE[0]),
//                                                   thicknesses(EE[1]),
//                                                   thicknesses(EE[2]),
//                                                   thicknesses(EE[3]));
//                    Vector2i range     = D_range(thickness, D_hat);
//
//                    switch(type)
//                    {
//                        case distance::EdgeEdgeDistanceType::PP_Ea0Eb0: {
//                            Float D;
//                            distance::point_point_distance(Ea0, Ea1, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PP_constraints(I) =
//                                    SimplexContactConstraint::PP_in_EE(EE[0],  // Pa
//                                                                       EE[2],  // Pb
//                                                                       EE[1],  // inactive Pa
//                                                                       EE[3]  // inactive Pb
//                                    );
//                            }
//                        }
//                        break;
//                        case distance::EdgeEdgeDistanceType::PP_Ea0Eb1: {
//                            Float D;
//                            distance::point_point_distance(Ea0, Ea1, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PP_constraints(I) =
//                                    SimplexContactConstraint::PP_in_EE(EE[0],  // Pa
//                                                                       EE[3],  // Pb
//                                                                       EE[1],  // inactive Pa
//                                                                       EE[2]  // inactive Pb
//                                    );
//                            }
//                        }
//                        break;
//                        case distance::EdgeEdgeDistanceType::PP_Ea1Eb0: {
//                            Float D;
//                            distance::point_point_distance(Ea0, Ea1, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PP_constraints(I) =
//                                    SimplexContactConstraint::PP_in_EE(EE[1],  // Pa
//                                                                       EE[2],  // Pb
//                                                                       EE[0],  // inactive Pa
//                                                                       EE[3]  // inactive Pb
//                                    );
//                            }
//                        }
//                        break;
//                        case distance::EdgeEdgeDistanceType::PP_Ea1Eb1: {
//                            Float D;
//                            distance::point_point_distance(Ea0, Ea1, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PP_constraints(I) =
//                                    SimplexContactConstraint::PP_in_EE(EE[1],  // Pa
//                                                                       EE[3],  // Pb
//                                                                       EE[0],  // inactive Pa
//                                                                       EE[2]  // inactive Pb
//                                    );
//                            }
//                        }
//                        break;
//                        case distance::EdgeEdgeDistanceType::PE_Ea0Eb0Eb1: {
//                            Float D;
//                            distance::point_edge_distance(Ea0, Ea1, Eb0, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PE_constraints(I) = SimplexContactConstraint::PE_in_EE(
//                                    EE[0],             // P
//                                    EE.segment<2>(2),  // E
//                                    EE[1]              // inactive P
//                                );
//                            }
//                        }
//                        break;
//                        case distance::EdgeEdgeDistanceType::PE_Ea1Eb0Eb1: {
//                            Float D;
//                            distance::point_edge_distance(Ea1, Ea0, Eb0, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PE_constraints(I) = SimplexContactConstraint::PE_in_EE(
//                                    EE[1],             // P
//                                    EE.segment<2>(2),  // E
//                                    EE[0]              // inactive P
//                                );
//                            }
//                        }
//                        case distance::EdgeEdgeDistanceType::PE_Eb0Ea0Ea1: {
//                            Float D;
//                            distance::point_edge_distance(Eb0, Eb1, Ea0, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PE_constraints(I) = SimplexContactConstraint::PE_in_EE(
//                                    EE[2],             // P
//                                    EE.segment<2>(0),  // E
//                                    EE[3]              // inactive P
//                                );
//                            }
//                        }
//                        break;
//                        case distance::EdgeEdgeDistanceType::PE_Eb1Ea0Ea1: {
//                            Float D;
//                            distance::point_edge_distance(Eb1, Eb0, Ea0, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                PE_constraints(I) = SimplexContactConstraint::PE_in_EE(
//                                    EE[3],             // P
//                                    EE.segment<2>(0),  // E
//                                    EE[2]              // inactive P
//                                );
//                            }
//                        }
//                        break;
//                        case distance::EdgeEdgeDistanceType::EE: {
//                            Float D;
//                            distance::edge_edge_distance(Ea0, Ea1, Eb0, Eb1, D);
//
//                            if(is_active_D(range, D))
//                            {
//                                EE_constraints(I) = SimplexContactConstraint::EE(
//                                    EE.segment<2>(0), EE.segment<2>(2));
//                            }
//                        }
//                    }
//                });
//    }
//
//    UIPC_ASSERT(PP_offset == temp_PP_constraints.size(), "size mismatch");
//    UIPC_ASSERT(PE_offset == temp_PE_constraints.size(), "size mismatch");
//
//    {  // PP constraints
//        DeviceSelect().If(temp_PP_constraints.data(),
//                          PP_constraints.data(),
//                          selected_count.data(),
//                          temp_PP_constraints.size(),
//                          [] HIPCUB_RUNTIME_FUNCTION(const SimplexContactConstraint& c)
//                          { return !c.is_none(); });
//
//        IndexT h_selected = selected_count;
//        PP_constraints.resize(h_selected);
//    }
//
//    {  // PE constraints
//        DeviceSelect().If(temp_PE_constraints.data(),
//                          PE_constraints.data(),
//                          selected_count.data(),
//                          temp_PE_constraints.size(),
//                          [] HIPCUB_RUNTIME_FUNCTION(const SimplexContactConstraint& c)
//                          { return !c.is_none(); });
//
//        IndexT h_selected = selected_count;
//
//        PE_constraints.resize(h_selected);
//    }
//
//    {  // PT constraints
//        DeviceSelect().If(temp_PT_constraints.data(),
//                          PT_constraints.data(),
//                          selected_count.data(),
//                          temp_PT_constraints.size(),
//                          [] HIPCUB_RUNTIME_FUNCTION(const SimplexContactConstraint& c)
//                          { return !c.is_none(); });
//
//        IndexT h_selected = selected_count;
//        PT_constraints.resize(h_selected);
//    }
//
//    {  // EE constraints
//        DeviceSelect().If(temp_EE_constraints.data(),
//                          EE_constraints.data(),
//                          selected_count.data(),
//                          temp_EE_constraints.size(),
//                          [] HIPCUB_RUNTIME_FUNCTION(const SimplexContactConstraint& c)
//                          { return !c.is_none(); });
//
//        IndexT h_selected = selected_count;
//        EE_constraints.resize(h_selected);
//    }
//}
}  // namespace uipc::backend::cuda
